#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>

#include "log_helper.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"

#define N 4096 
#define SWITCH_CHAR  '-'
#define GOLD_MATRIX_PATH "/home/carol/TestGPU/GenerateGoldMatrix/Double_GOLD_4096.matrix"
#define LOGFILE_MATRIXNAME "cudaMxM4096"

#define N_ERRORS_LOG 500
#define BLOCK_SIZE 32

#define ITERACTIONS 1

double *h_A;
double *h_B;
double *h_GOLD;
double *d_A;
double *d_B;
double *d_C;

unsigned int *d_errpos;
unsigned int *errpos;

FILE* file;
FILE* log_file;
FILE* timefile;

//void UpdateTimestamp(){
//	time_t timestamp = time(NULL);
//	char time_s[50];
//	sprintf(time_s, "%d", int(timestamp));
//
//	char string[100] = "echo ";
//	strcat(string, time_s);
//	strcat(string, " > /home/carol/TestGPU/timestamp.txt");
////	system(string);
//	}


void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);
    
    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("device: %d %s\n\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void ReadMatrixFromFile(){

	double time = mysecond();
	FILE *f_A, *f_B, *f_GOLD;

		
printf("open matrix...");
        f_A = fopen("/home/carol/TestGPU/GenerateGoldMatrix/Double_A_8192.matrix","rb");
        f_B = fopen("/home/carol/TestGPU/GenerateGoldMatrix/Double_B_8192.matrix","rb");
        f_GOLD = fopen(GOLD_MATRIX_PATH,"rb");
	if (!(f_A && f_B && f_GOLD)) { printf ("Error opening matrix.\n"); getchar(); exit(-1); }
printf("read...");
        fread(h_A,sizeof(double)*N*N, 1, f_A);
        fread(h_B,sizeof(double)*N*N, 1, f_B);
        fread(h_GOLD,sizeof(double)*N*N, 1, f_GOLD);
printf("ok in %f\n", mysecond() - time);


        fclose(f_A);
        fclose(f_B);
        fclose(f_GOLD);
}

__device__ int kerrors;

__global__ void GoldChkKernel (double *gk, double *ck, int n)
{
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                                      
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y; 
	if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
		atomicAdd(&kerrors, 1);

}

__global__ void MatrixMulKernel (double *d_A, double *d_B, double *d_C, int n)
{
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                                      
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y; 
	int k;
	
	d_C[ty*n + tx] = 0.0;
	for (k = 0;  k < n; k++)
	  d_C[ty*n + tx] += d_A[ty*n + k]*d_B[k*n + tx];

}

int main( int argc, char* argv[] )
{

	
	hipError_t malloc_mem1;
	hipError_t malloc_a;
	const char *erro_malloc;


	int ea=0; //wrong integers in the current loop
	int t_ea=0; //total number of wrong integers
	int old_ea = 0;

	double total_time = 0.0;

	int i, j, loop2;

	int kernel_errors=0;
	int zero = 0;


	double timeG;

	//////////BLOCK and GRID size///////////////////////
	int gridsize = N/BLOCK_SIZE < 1 ? 1 : N/BLOCK_SIZE;
	int blocksize = N/BLOCK_SIZE < 1 ? N : BLOCK_SIZE;
	dim3 dimBlock(blocksize,blocksize);
	dim3 dimGrid(gridsize,gridsize);
	////////////////////////////////////////////////////


	///////////////////////////////////////////////////////
	////////////////FILE NAME//////////////////////////////
//	time_t file_time;
//	struct tm *ptm;
//	char day[2], month[2], year[4], hour[2], second[2], minute[2];
//	char file_name[60];
//	char file_name_log[60];
//	
//	file_time = time(NULL);
//	ptm = gmtime(&file_time);
//
//	snprintf(day, sizeof(day + 1), "%d", ptm->tm_mday);
//	snprintf(month, sizeof(month + 1), "%d", ptm->tm_mon+1);
//	snprintf(year, sizeof(year + 1), "%d", ptm->tm_year+1900);
//	snprintf(hour, sizeof(hour + 1), "%d", ptm->tm_hour);
//	snprintf(minute, sizeof(minute + 1), "%d", ptm->tm_min);
//	snprintf(second, sizeof(second + 1), "%d", ptm->tm_sec);
//	strcpy(file_name,day);strcat(file_name,"_");
//	strcat(file_name,month);strcat(file_name,"_");
//	strcat(file_name,year);strcat(file_name,"_");
//	strcat(file_name,hour);strcat(file_name,"_");
//	strcat(file_name,minute);strcat(file_name,"_");
//	strcat(file_name,second);strcat(file_name,"_");
//	strcat(file_name,LOGFILE_MATRIXNAME);
//	strcpy(file_name_log, file_name);
//	
//	strcat(file_name,".txt");
//	strcat(file_name_log,"log.txt");
	///////////////////////////////////////////////////////
	///////////////////////////////////////////////////////
	char test_info[100];
	snprintf(test_info, 100, "size:%d",N);
	start_log_file(LOGFILE_MATRIXNAME, test_info);

	int size = N*N;

	h_A = ( double* ) malloc( size * sizeof( double ) );
	h_B = ( double* ) malloc( size * sizeof( double ) );
	h_GOLD = ( double* ) malloc( size * sizeof( double ) );

	ReadMatrixFromFile();


	kernel_errors=0;
	
	GetDevice();

	printf( "Cuda MxM Not optimized - %ix%i\n", N, N );


	for(loop2=0; loop2<ITERACTIONS; loop2++)
	{

	//	file = fopen(file_name, "a");	
	
		// ======> DEVICE MEMORY ALLOC
		malloc_a = hipMalloc( ( void** ) &d_A, size * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error a"); log_error_detail("error a"); end_log_file(); return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_B, size * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error b"); log_error_detail("error b"); end_log_file(); return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_C, size * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error c"); log_error_detail("error c"); end_log_file(); return 1;} //mem allocate failure
		// =======>


		malloc_mem1 = hipMemcpy( d_C, h_A, size * sizeof( double ), hipMemcpyHostToDevice ); // ZERA C
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error mem load c %s", erro_malloc); log_error_detail("error mem load c"); end_log_file(); return 1;}
	
		malloc_mem1 = hipMemcpy( d_A, h_A, size * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error mem load a %s", erro_malloc); log_error_detail("error mem load a"); end_log_file(); return 1;}

		malloc_mem1 = hipMemcpy( d_B, h_B, size * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error mem load b %s", erro_malloc); log_error_detail("error mem load b"); end_log_file(); return 1;}


		kernel_errors=0;
	
		//timeG = mysecond();
		start_iteration();
		MatrixMulKernel<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
		hipDeviceSynchronize();
		end_iteration();
		//timeG = mysecond() - timeG;

		//total_time += timeG;


		malloc_mem1 = hipMemcpy(d_A, h_GOLD, size * sizeof( double ), hipMemcpyHostToDevice );
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {printf("error mem load gold %s", erro_malloc); log_error_detail("error mem load gold"); end_log_file(); return 1;}
		// =======>

		hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));

		GoldChkKernel<<<dimGrid,dimBlock>>>(d_A, d_C, N);


		hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors), sizeof(unsigned int));
	

	//	///////////UPDATE FILE//////////////////////
	//	file_time = time(NULL);
	//	ptm = gmtime(&file_time);
	//	snprintf(hour, sizeof(hour + 1), "%d", ptm->tm_hour);
	//	snprintf(minute, sizeof(minute + 1), "%d", ptm->tm_min);
	//	snprintf(second, sizeof(second + 1), "%d", ptm->tm_sec);
	//	fprintf(file, "\n start time: %s/%s_%s:%s:%s", day,month,hour,minute,second);
	//	fclose(file);

	//	ea = 0;
	//	t_ea += kernel_errors;
 
	//	/////////////UPDATE TIMESTAMP///////////////////
	//	UpdateTimestamp(); // UNCOMENT THIS AFTER
		////////////////////////////////////////////////
		
		log_error_count(kernel_errors);

		if (kernel_errors!=0)
		{
			//file = fopen(file_name, "a");
			char error_detail[150];

			printf("\n kernel error: %d\n", kernel_errors);

			malloc_mem1 = hipMemcpy(h_A, d_C, size * sizeof( double ), hipMemcpyDeviceToHost);
			erro_malloc = hipGetErrorString(malloc_mem1);
			if(strcmp(erro_malloc, "no error") != 0)
				{printf("error mem load MEMDUMP %s", erro_malloc); fprintf(file, "error mem load MEMDUMP %s", erro_malloc); return 1;}

			for(i=0; (i<N) && (ea < N_ERRORS_LOG); i++)
			{
				for(j=0; (j<N) && (ea < N_ERRORS_LOG); j++)
				{
					if ((fabs((h_A[i+N*j]-h_GOLD[i+N*j])/h_A[i+N*j]) > 0.0000000001)||(fabs((h_A[i+N*j]-h_GOLD[i+N*j])/h_GOLD[i+N*j]) > 0.0000000001))
					{
						snprintf(error_detail, 150, "p: [%d, %d], r: %1.16e, e: %1.16e", i, j, h_A[i + N * j], h_GOLD[i + N * j]);
						log_error_detail(error_detail);
						//ea++;
						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, h_A[i + N * j], h_GOLD[i + N * j], ea);
										
					}
				}
			}

			///////////UPDATE LOG FILE//////////////////////
			//log_file = fopen(file_name_log, "a");
			//fprintf(log_file, "\ntest number: %d", loop2);
			//fprintf(log_file, "\ntime: %f", timeG);
			//fprintf(log_file, "\ntotal time: %f", total_time);
			//fprintf(log_file, "\nerrors: %d", kernel_errors);
			//fprintf(log_file, "\ntotal errors: %d", t_ea);
			//fclose(log_file);
			//fclose(file);

			ReadMatrixFromFile();
		}



		if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("\ntest number: %d", loop2);
			//printf("\ntotal time: %f", total_time);
			printf("\nerrors: %d", kernel_errors);
			//printf("\ntotal errors: %d\n", t_ea);
		//	if((kernel_errors != 0) && (kernel_errors == old_ea))
		//		{
		//			old_ea = 0;
		//			return 1;
		//		}
		//		
		//		old_ea = kernel_errors;
		}
		else
		{
			printf(".");
		}

		hipFree( d_A );
		hipFree( d_B );
		hipFree( d_C );
	}

	free(h_A);
	free(h_B);
	free(h_GOLD);

	end_log_file();

	return 0;
}
