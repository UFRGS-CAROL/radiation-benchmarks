#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>

#ifdef LOGS
#include "log_helper.h"
#endif

#include "hip/hip_runtime.h"
#include "hipblas.h"
//#include "cublas_v2.h"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define BLOCK_SIZE 32

int k=0; // N will be received on runtime
int iteractions=1; // iteractions will be received on runtime

using namespace std;

string gold_matrix_path, a_matrix_path, b_matrix_path;

double *A;
double *B;
double *d_A;
double *d_B;
double *d_C;

   int lda, ldb, ldc;

double *GOLD;


FILE* f_A;
FILE* f_B;
FILE* f_GOLD;

FILE* file;
FILE* log_file;
FILE* timefile;

void usage() {
    printf("Usage: cudaGemm <input_size> <A_MATRIX> <B_MATRIX> <GOLD_MATRIX>\n");
}

void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);
    
    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void ReadMatrixFromFile(){	
	
	int i;
	double time = mysecond();
	f_A = fopen(a_matrix_path.c_str(),"rb");
	f_B = fopen(b_matrix_path.c_str(),"rb");
	f_GOLD = fopen(gold_matrix_path.c_str(),"rb");
	if (!(f_A&&f_B&&f_GOLD))
	{
		printf ("Cant open matrices.\n");
#ifdef LOGS
		log_error_detail("Cant open matrices"); end_log_file(); 
#endif
		exit(-3);
	}
	for(i=0; i<k; i++)
	{
		fread (&A[ lda * i ], sizeof(double)*k, 1, f_A);
		fread (&B[ lda * i ], sizeof(double)*k, 1, f_B);
		fread (&GOLD[ lda * i ], sizeof(double)*k, 1, f_GOLD);
	}
	printf("Done reading matrices in %f\n", mysecond() - time);

	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);
}

__device__ int kerrors;

__global__ void GoldChkKernel (double *gk, double *ck, int n)//, int *kerrors)
{
	//ck[4] = 4.5;
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                                      
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y; 
	if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
		atomicAdd(&kerrors, 1);

}



int main( int argc, char* argv[] )
{

	
	hipError_t malloc_mem1;
	hipError_t malloc_a;
	const char *erro_malloc;

	int ea=0; //wrong integers in the current loop

	const double alpha = 1.0;
	const double beta = 1.0;

	char transa = 't', transb = 't';
	int i, j, loop2;

	int kernel_errors=0;
	int zero = 0;


	int sizea, sizeb, sizec;

	////////////////////////////////////////////////////
	////////////////////GET PARAM///////////////////////
	if (argc!=5) {
		usage();
		exit (-1);
	}

	k = atoi (argv[1]);
	if (((k%32)!=0)||(k<0)){
		printf ("Enter a valid input. (k=%i)\n", k);
		exit (-1);
	}

	a_matrix_path = argv[2];
	b_matrix_path = argv[3];
	gold_matrix_path = argv[4];

	//////////BLOCK and GRID size///////////////////////
	int gridsize = k/BLOCK_SIZE < 1 ? 1 : k/BLOCK_SIZE;
	int blocksize = k/BLOCK_SIZE < 1 ? k : BLOCK_SIZE;
	dim3 dimBlock(blocksize,blocksize);
	dim3 dimGrid(gridsize,gridsize);
	////////////////////////////////////////////////////

#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "size:%d", k);
	start_log_file("cudaGEMM", test_info);
#endif

	lda = max( 1, k + 16 );
	sizea = lda * k;
	ldb = max( 1, k + 16 );
	sizeb = ldb * k;
	ldc = max( 1, k + 16 );
	sizec = ldc * k;

	A = ( double* ) malloc( sizea * sizeof( double ) );
	B = ( double* ) malloc( sizeb * sizeof( double ) );

	GOLD = ( double* ) malloc( sizec * sizeof( double ) );

	kernel_errors=0;
	
	GetDevice();
	
	ReadMatrixFromFile();

	//A[72] = 7.2;

	printf( "cublasDGEMM\n" );

   
	for(loop2=0; loop2<iteractions; loop2++)
	{


		malloc_a = hipMalloc( ( void** ) &d_A, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error a"); end_log_file(); 
#endif
			return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_B, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error b"); end_log_file(); 
#endif
			return 1;} //mem allocate failure

		malloc_a = hipMalloc( ( void** ) &d_C, sizea * sizeof( double ) );
		erro_malloc = hipGetErrorString(malloc_a);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error c"); end_log_file(); 
#endif
			return 1;} //mem allocate failure


		malloc_mem1 = hipMemcpy( d_C, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // ZERA C
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error mem load c"); end_log_file(); 
#endif
			return 1;} //mem allocate failure
	
		malloc_mem1 = hipMemcpy( d_A, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error mem load b"); end_log_file(); 
#endif
			return 1;} //mem allocate failure

		malloc_mem1 = hipMemcpy( d_B, B, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error mem load b"); end_log_file(); 
#endif
			return 1;} //mem allocate failure

		kernel_errors=0;
		//cublasHandle_t blashandle;
		//cublasCreate(&blashandle);
	
		//printf("cublasDgemm... k=%d transa=%c transb=%c lda=%d ldb=%d ldc=%d\n", k, transa, transb, lda, ldb, ldc);
double time = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//cublasDgemm( blashandle, (cublasOperation_t)transa, (cublasOperation_t)transb,
		cublasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
			   k, k, k,
			   alpha,
			   d_A, lda,
			   d_B, ldb,
			   beta,
			   d_C, ldc );
		//printf("\nend\n");
		hipDeviceSynchronize();
#ifdef LOGS
		end_iteration();
#endif
time = mysecond() - time;

		malloc_mem1 = hipMemcpy(d_A, GOLD, sizea * sizeof( double ), hipMemcpyHostToDevice );
		erro_malloc = hipGetErrorString(malloc_mem1);
		if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
			log_error_detail("error mem load gold"); end_log_file(); 
#endif
			return 1;} //mem allocate failure

		hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));

		GoldChkKernel<<<dimGrid,dimBlock>>>(d_A, d_C, ldc);


		hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors), sizeof(unsigned int));
		
#ifdef LOGS
		log_error_count(kernel_errors);
#endif

		if (kernel_errors!=0)
		{

			printf("\n kernel error: %d\n", kernel_errors);

			malloc_mem1 = hipMemcpy(A, d_C, sizec * sizeof( double ), hipMemcpyDeviceToHost);
			erro_malloc = hipGetErrorString(malloc_mem1);
			if(strcmp(erro_malloc, "no error") != 0) {
#ifdef LOGS
				log_error_detail("error mem load c"); end_log_file(); 
#endif
			return 1;} //mem allocate failure
			char error_detail[150];

			for(i=0; (i<k) && (ea < 500); i++)
			{
				for(j=0; (j<k) && (ea < 500); j++)
				{
					if ((fabs((A[i+ldc*j]-GOLD[i+ldc*j])/A[i+ldc*j]) > 0.0000000001)||(fabs((A[i+ldc*j]-GOLD[i+ldc*j])/GOLD[i+ldc*j]) > 0.0000000001))
					{
						snprintf(error_detail, 150, "p: [%d, %d], r: %1.16e, e: %1.16e", i, j, A[i + ldc * j], GOLD[i + ldc * j]);
#ifdef LOGS
						log_error_detail(error_detail);
#endif
						//ea++;			
						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + ldc * j], GOLD[i + ldc * j], t_ea);
										
					}
				}
			}

				ReadMatrixFromFile();	
		}



		if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("\ntest number: %d", loop2);
			printf(" time: %f\n", time);
		}
		else
		{
			printf(".");
			fflush(stdout);
		}



		hipFree( d_A );
		hipFree( d_B );
		hipFree( d_C );
	}

	free( A );
	free( B );
#ifdef LOGS
	end_log_file();
#endif

	return 0;
}
