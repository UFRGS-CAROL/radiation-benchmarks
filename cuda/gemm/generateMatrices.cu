#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include <string>
#include <sys/time.h>

#include<hipblas.h>

#define MATRIX_PATH "./Double_"
#define INMATRIXSIZE 8192
using namespace std;

int k=0;
int lda, ldb, ldc;
int sizea, sizeb, sizec;	
double *A, *B, *GOLD;

string gold_matrix_path, a_matrix_path, b_matrix_path;

void generateInputMatrices()
{
	double temp;
	int i, j;
	FILE *f_A, *f_B;

	f_A = fopen(a_matrix_path.c_str(), "wb");
	f_B = fopen(b_matrix_path.c_str(), "wb");


	srand ( time(NULL) );

	for(i=0; i<INMATRIXSIZE; i++)
	{
		for(j=0; j<INMATRIXSIZE; j++){
			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.0004e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_A );
		

			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.4e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_B );
			
			
		}
	}

	fclose(f_A);
	fclose(f_B);

	return;
}

void ReadMatrixFromFile(){	
	
	int i;
	FILE *f_A, *f_B;

	f_A = fopen(a_matrix_path.c_str(),"rb");
	f_B = fopen(b_matrix_path.c_str(),"rb");
	if (!(f_A&&f_B))
	{
		printf("Error opening matrices A, B.\n");
		exit(-1);
	}
	for(i=0; i<k; i++)
	{
		fread (&A[ lda * i ], sizeof(double)*k, 1, f_A);
		fread (&B[ lda * i ], sizeof(double)*k, 1, f_B);
	}
printf("Done reading matrices\n");

	fclose(f_A);
	fclose(f_B);
}

void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("Get Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);
    
    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void generateGoldMatrix()
{
	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	char transa = 't', transb = 't';
	////////////////////////////////////////////////////
	
	////////////////////////////////////////////////////
	//////////DEVICE VARS///////////////////////////////	
	hipError_t cumalloc_err;
	const char *cumalloc_err_str;

	double *d_A;
	double *d_B;
	double *d_C;
	////////////////////////////////////////////////////

	A = ( double* ) malloc( sizea * sizeof( double ) );
	B = ( double* ) malloc( sizeb * sizeof( double ) );
	GOLD = ( double* ) malloc( sizec * sizeof( double ) );

	GetDevice();
	
	ReadMatrixFromFile();

	cumalloc_err = hipMalloc( ( void** ) &d_A, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);} //mem allocate failure

	cumalloc_err = hipMalloc( ( void** ) &d_B, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMalloc( ( void** ) &d_C, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}


	cumalloc_err = hipMemcpy( d_C, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // ZERA C
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_A, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_B, B, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	printf("cublasDgemm... k=%d transa=%c transb=%c lda=%d ldb=%d ldc=%d\n", k, transa, transb, lda, ldb, ldc);
	double time = mysecond();


	cublasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
			   k, k, k,
			   alpha,
			   d_A, lda,
			   d_B, ldb,
			   beta,
			   d_C, ldc );
	hipDeviceSynchronize();

	printf("\nend in %f\n", mysecond()-time);

	cumalloc_err = hipMemcpy(GOLD, d_C, sizec * sizeof( double ), hipMemcpyDeviceToHost);
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

	int i;
	FILE *f_GOLD;

	f_GOLD = fopen(gold_matrix_path.c_str(), "wb");

	//printf("-------------------------\n%.10f\n%.10f\n%.10f\n", GOLD[0], GOLD[1], GOLD[2]);

	for(i=0; i<k; i++)
	{
		fwrite( &GOLD[i * lda], sizeof(double)*k, 1, f_GOLD );
	}

	fclose(f_GOLD);

	return;
}

int main (int argc, char** argv)
{
	////////////////////////////////////////////////////
	////////////////////GET PARAM///////////////////////
	if (argc!=2) {
		printf ("Enter the required input. (1024/2048/4096/8192)\n");
		exit (-1);
	}
	k = atoi (argv[1]);
	if (((k%32)!=0)||(k<0)){
		printf ("Enter a valid input. (k=%i)\n", k);
		exit (-1);
	}
	string matrix_size_str(argv[1]);

	a_matrix_path = MATRIX_PATH;
	b_matrix_path = MATRIX_PATH;
	gold_matrix_path = MATRIX_PATH;
	a_matrix_path += "A_8192.matrix";
	b_matrix_path += "B_8192.matrix";
	gold_matrix_path += "GOLD_" + matrix_size_str + ".matrix";
	////////////////////////////////////////////////////

	lda = max( 1, k + 16 );
	sizea = lda * k;
	ldb = max( 1, k + 16 );
	sizeb = ldb * k;
	ldc = max( 1, k + 16 );
	sizec = ldc * k;
	
	FILE *test_file;
	test_file=fopen(a_matrix_path.c_str(), "rb");
	if (!test_file)
	{ 
		printf("Generating input matrices...\n");
		generateInputMatrices();
	}
	else
	{	printf("Input matrices already exist...\n");	}

	generateGoldMatrix();

	return 0;
}
