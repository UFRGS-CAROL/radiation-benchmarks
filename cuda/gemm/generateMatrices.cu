#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include <string>
#include <sys/time.h>

#include<hipblas.h>

#define MATRIX_PATH "./Double_"
#define INMATRIXSIZE 8192
using namespace std;

int k=0;
int lda, ldb, ldc;
int sizea, sizeb, sizec;	
double *A, *B, *GOLD;

string gold_matrix_path, a_matrix_path, b_matrix_path;

void usage() {
    printf("Usage: generateMatrices <input_size> <A_MATRIX> <B_MATRIX> <GOLD_MATRIX>\n");
}

void generateInputMatrices()
{
	double temp;
	int i, j;
	FILE *f_A, *f_B;

	f_A = fopen(a_matrix_path.c_str(), "wb");
	f_B = fopen(b_matrix_path.c_str(), "wb");


	srand ( time(NULL) );

	for(i=0; i<INMATRIXSIZE; i++)
	{
		for(j=0; j<INMATRIXSIZE; j++){
			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.0004e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_A );
		

			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.4e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_B );
			
			
		}
	}

	fclose(f_A);
	fclose(f_B);

	return;
}

void ReadMatrixFromFile(){	
	
	int i;
	FILE *f_A, *f_B;

	f_A = fopen(a_matrix_path.c_str(),"rb");
	f_B = fopen(b_matrix_path.c_str(),"rb");
	if (!(f_A&&f_B))
	{
		printf("Error opening matrices A, B.\n");
		exit(-1);
	}
	for(i=0; i<k; i++)
	{
		fread (&A[ lda * i ], sizeof(double)*k, 1, f_A);
		fread (&B[ lda * i ], sizeof(double)*k, 1, f_B);
	}
printf("Done reading matrices\n");

	fclose(f_A);
	fclose(f_B);
}

void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("Get Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);
    
    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void generateGoldMatrix()
{
	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	char transa = 't', transb = 't';
	////////////////////////////////////////////////////
	
	////////////////////////////////////////////////////
	//////////DEVICE VARS///////////////////////////////	
	hipError_t cumalloc_err;
	const char *cumalloc_err_str;

	double *d_A;
	double *d_B;
	double *d_C;
	////////////////////////////////////////////////////

	A = ( double* ) malloc( sizea * sizeof( double ) );
	B = ( double* ) malloc( sizeb * sizeof( double ) );
	GOLD = ( double* ) malloc( sizec * sizeof( double ) );

	GetDevice();
	
	ReadMatrixFromFile();

	cumalloc_err = hipMalloc( ( void** ) &d_A, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);} //mem allocate failure

	cumalloc_err = hipMalloc( ( void** ) &d_B, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMalloc( ( void** ) &d_C, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}


	cumalloc_err = hipMemcpy( d_C, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // ZERA C
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_A, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_B, B, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	printf("cublasDgemm... k=%d transa=%c transb=%c lda=%d ldb=%d ldc=%d\n", k, transa, transb, lda, ldb, ldc);
	double time = mysecond();


	cublasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
			   k, k, k,
			   alpha,
			   d_A, lda,
			   d_B, ldb,
			   beta,
			   d_C, ldc );
	hipDeviceSynchronize();

	time=mysecond()-time;

	/////////// PERF
    double flops = 2.0*(double)k*k*k;
    double gflops = flops / time;
    double outputpersec = (double)k*k/time;
    printf("kernel time: %lf\n",time);
    printf("SIZE:%d OUTPUT/S:%f FLOPS:%f\n",k, outputpersec, gflops);
	///////////

	cumalloc_err = hipMemcpy(GOLD, d_C, sizec * sizeof( double ), hipMemcpyDeviceToHost);
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

	int i;
	FILE *f_GOLD;

	f_GOLD = fopen(gold_matrix_path.c_str(), "wb");

	//printf("-------------------------\n%.10f\n%.10f\n%.10f\n", GOLD[0], GOLD[1], GOLD[2]);

	for(i=0; i<k; i++)
	{
		fwrite( &GOLD[i * lda], sizeof(double)*k, 1, f_GOLD );
	}

	fclose(f_GOLD);

	return;
}

int main (int argc, char** argv)
{
	////////////////////////////////////////////////////
	////////////////////GET PARAM///////////////////////
	if (argc!=5) {
		usage();
		exit (-1);
	}

	k = atoi (argv[1]);
	if (((k%32)!=0)||(k<0)){
		printf ("Enter a valid input. (k=%i)\n", k);
		exit (-1);
	}

	a_matrix_path = argv[2];
	b_matrix_path = argv[3];
	gold_matrix_path = argv[4];

	lda = max( 1, k + 16 );
	sizea = lda * k;
	ldb = max( 1, k + 16 );
	sizeb = ldb * k;
	ldc = max( 1, k + 16 );
	sizec = ldc * k;
	
	FILE *test_file;
	test_file=fopen(a_matrix_path.c_str(), "rb");
	if (!test_file)
	{ 
		printf("Generating input matrices...\n");
		generateInputMatrices();
	}
	else
	{	printf("Input matrices already exist...\n");	}

	generateGoldMatrix();

	return 0;
}
