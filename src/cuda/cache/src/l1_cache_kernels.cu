#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 26/01/2019
 *      Author: fernando
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstring>

#include "utils.h"
#include "Memory.h"
#include "L1Cache.h"

#define NUMBEROFELEMENTS 6144
#include "l1_move_function.h"

template<const uint32 SHARED_PER_SM>
__global__ void test_l1_cache_kernel(uint64 *in, uint64 *out, int64 *hits,
		int64 *miss, const int64 sleep_cycles) {

	__shared__ int64 l1_t_hit[SHARED_PER_SM];
	__shared__ int64 l1_t_miss[SHARED_PER_SM];

	const uint64 i = (blockIdx.x * blockDim.x + threadIdx.x) * NUMBEROFELEMENTS;

	uint64 rs[NUMBEROFELEMENTS]; //, rt[NUMBEROFELEMENTS];

	const int64 t1_miss = clock64();
	uint64 temp = 0;
	for(uint32 k = 0; k < NUMBEROFELEMENTS; k++){
		temp &= in[i + k];
	}
//	mov_cache_data(rs, in + i);
	l1_t_miss[threadIdx.x] = clock64() - t1_miss;

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	//last checking
	const register int64 t1_hit = clock64();
	mov_cache_data(rs, in + i);
	l1_t_hit[threadIdx.x] = clock64() - t1_hit;

	mov_cache_data(out + i, rs);
//	mov_cache_data(in + i, rs);
	in[i] = temp;

	//saving miss and hit
	miss[i] = l1_t_miss[threadIdx.x];
    hits[i] = l1_t_hit[threadIdx.x];
}

L1Cache::L1Cache(const Parameters& parameters) :
		Memory<uint64>(parameters) {
	uint32 v_size;
	switch (device) {
	case K20:
	case K40:
		v_size = MAX_KEPLER_L1_MEMORY / sizeof(uint64);
		break;
	case XAVIER:
	case TITANV:
		v_size = MAX_VOLTA_L1_MEMORY / sizeof(uint64);
		break;
	}

//	this->threads_per_block = dim3(v_size);
	this->threads_per_block = dim3(v_size / NUMBEROFELEMENTS);

	uint32 v_size_multiple_threads = v_size * parameters.number_of_sms;
//			* CACHE_LINE_SIZE_BY_INT32; // Each block with one thread using all l1 cache

	this->hit_vector_host.resize(v_size_multiple_threads);
	this->miss_vector_host.resize(v_size_multiple_threads);

	this->input_host_1.resize(v_size_multiple_threads);
	this->output_host_1.resize(v_size_multiple_threads);
}

void L1Cache::test(const uint64& mem) {
	//Set values to GPU
	std::fill(this->input_host_1.begin(), this->input_host_1.end(), mem);
	std::fill(this->hit_vector_host.begin(), this->hit_vector_host.end(), 0);
	std::fill(this->miss_vector_host.begin(), this->miss_vector_host.end(), 0);

	rad::DeviceVector<int64> hit_vector_device(this->hit_vector_host);
	rad::DeviceVector<int64> miss_vector_device(this->miss_vector_host);

	rad::DeviceVector<uint64> input_device_1(this->input_host_1);
	rad::DeviceVector<uint64> output_device_1(this->output_host_1);

	//This switch is only to set manually the cache line size
	//since it is hard to check it at runtime
	switch (device) {
	case K20:
	case K40: {
		// cache l1 has 65536 bytes
		//BUT, only 48kb are destined to L1 memory
		//so alloc 49152 bytes
		// cache line has 128 bytes
		//to force alloc maximum shared memory
//		constexpr uint32 v_size = MAX_KEPLER_L1_MEMORY / CACHE_LINE_SIZE;

		test_l1_cache_kernel<MAX_KEPLER_SHARED_MEMORY_TO_TEST_L1> <<<
				block_size, threads_per_block>>>(input_device_1.data(),
				output_device_1.data(), hit_vector_device.data(),
				miss_vector_device.data(), cycles);

		break;
	}
	case XAVIER:
	case TITANV: {
		// cache l1 has 128 Kbytes
		//BUT, only 98304 bytes are destined to L1 memory
		//so alloc 98304 bytes
		// cache line has 128 bytes
//		constexpr uint32 v_size = MAX_VOLTA_L1_MEMORY / CACHE_LINE_SIZE;

		test_l1_cache_kernel<MAX_VOLTA_SHARED_MEMORY_TO_TEST_L1> <<<
				block_size, threads_per_block>>>(input_device_1.data(),
				output_device_1.data(), hit_vector_device.data(),
				miss_vector_device.data(), cycles);
		break;
	}
	}

	cuda_check(hipPeekAtLastError());
	cuda_check(hipDeviceSynchronize());
	//Host arrays
	//Copy back to the host
	this->hit_vector_host = hit_vector_device.to_vector();
	this->miss_vector_host = miss_vector_device.to_vector();
	this->output_host_1 = output_device_1.to_vector();
}
