#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 26/01/2019
 *      Author: fernando
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <cstring>

#include "utils.h"
#include "Memory.h"
#include "L1Cache.h"

#define NUMBEROFELEMENTS 96
#include "l1_move_function.h"

template<const uint32 SHARED_PER_SM>
__global__ void test_l1_cache_kernel(uint64 *in, uint64 *out, int64 *hits,
		int64 *miss, const int64 sleep_cycles) {

	__shared__ int64 l1_t_hit[SHARED_PER_SM];
	__shared__ int64 l1_t_miss[SHARED_PER_SM];

//	printf("block idx %d block dim %d thread idx %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	const register uint64 i = (blockIdx.x * blockDim.x + threadIdx.x) * NUMBEROFELEMENTS;

	register uint64 rs[NUMBEROFELEMENTS]; //, rt[NUMBEROFELEMENTS];

	const int64 t1_miss = clock64();
	mov_cache_data(rs, in + i);
	const int64 t2_miss = clock64();

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	//last checking
	const register int64 t1_hit = clock64();
	mov_cache_data(rs, in + i);
	const register int64 t2_hit = clock64();

	mov_cache_data(out + i, rs);
//	mov_cache_data(in + i, rs);

//saving miss and hit
	l1_t_miss[threadIdx.x] = t2_miss - t1_miss;
	l1_t_hit[threadIdx.x] = t2_hit - t1_hit;
	miss[i] = l1_t_miss[threadIdx.x];
	hits[i] = l1_t_hit[threadIdx.x];

}

/*
 * l1_size size of the L1 cache
 * V_size = l1_size / sizeof(CacheLine)
 */
//template<const uint32 V_SIZE, const uint32 SHARED_PER_SM>
//__global__ void test_l1_cache_kernel(uint64 *in, uint64 *out, int64 *hits,
//		int64 *miss, const int64 sleep_cycles) {
//
//	__shared__ int64 l1_t_hit[SHARED_PER_SM];
//	__shared__ int64 l1_t_miss[SHARED_PER_SM];
//	const register uint64 i = blockIdx.x * V_SIZE + threadIdx.x;
//
//	if (threadIdx.x < V_SIZE && blockIdx.y == 0) {
//
//		const register uint64 index = i * CACHE_LINE_SIZE_BY_INT32;
//
//		register uint64 rs[CACHE_LINE_SIZE_BY_INT32];
//		register uint64 rt[CACHE_LINE_SIZE_BY_INT32];
//
//		const int64 t1_miss = clock64();
//		move_cache_line(rs, in + index);
//		const int64 t2_miss = clock64();
//
//		//wait for exposition to neutrons
//		sleep_cuda(sleep_cycles);
//
//		//last checking
//		const register int64 t1_hit = clock64();
//		move_cache_line(rt, in + index);
//		const register int64 t2_hit = clock64();
//
//		//triplication
//		move_cache_line(out + index, rt);
//		move_cache_line(in + index, rs);
//
////saving miss and hit
//		l1_t_miss[threadIdx.x] = t2_miss - t1_miss;
//		l1_t_hit[threadIdx.x] = t2_hit - t1_hit;
//		miss[i] = l1_t_miss[threadIdx.x];
//		hits[i] = l1_t_hit[threadIdx.x];
//	}
//
//}

L1Cache::L1Cache(const Parameters& parameters) :
		Memory<uint64>(parameters) {
	uint32 v_size;
	switch (device) {
	case K20:
	case K40:
		v_size = MAX_KEPLER_L1_MEMORY / sizeof(uint64);
		break;
	case XAVIER:
	case TITANV:
		v_size = MAX_VOLTA_L1_MEMORY / CACHE_LINE_SIZE;
		break;
	}

//	this->threads_per_block = dim3(v_size);
	this->threads_per_block = dim3(v_size / NUMBEROFELEMENTS);

	uint32 v_size_multiple_threads = v_size * parameters.number_of_sms;
//			* CACHE_LINE_SIZE_BY_INT32; // Each block with one thread using all l1 cache

	this->hit_vector_host.resize(v_size_multiple_threads);
	this->miss_vector_host.resize(v_size_multiple_threads);

	this->input_host_1.resize(v_size_multiple_threads);
	this->output_host_1.resize(v_size_multiple_threads);
}

void L1Cache::test(const uint64& mem) {
	//Set values to GPU
	std::fill(this->input_host_1.begin(), this->input_host_1.end(), mem);
	rad::DeviceVector<int64> hit_vector_device(this->hit_vector_host);
	rad::DeviceVector<int64> miss_vector_device(this->miss_vector_host);

	rad::DeviceVector<uint64> input_device_1(this->input_host_1);
	rad::DeviceVector<uint64> output_device_1(this->output_host_1);

	//This switch is only to set manually the cache line size
	//since it is hard to check it at runtime
	switch (device) {
	case K20:
	case K40: {
		// cache l1 has 65536 bytes
		//BUT, only 48kb are destined to L1 memory
		//so alloc 49152 bytes
		// cache line has 128 bytes
		//to force alloc maximum shared memory
//		constexpr uint32 v_size = MAX_KEPLER_L1_MEMORY / CACHE_LINE_SIZE;

		test_l1_cache_kernel<MAX_KEPLER_SHARED_MEMORY_TO_TEST_L1> <<<
				block_size, threads_per_block>>>(input_device_1.data(),
				output_device_1.data(), hit_vector_device.data(),
				miss_vector_device.data(), cycles);

		break;
	}
	case XAVIER:
	case TITANV: {
		// cache l1 has 128 Kbytes
		//BUT, only 98304 bytes are destined to L1 memory
		//so alloc 98304 bytes
		// cache line has 128 bytes
//		constexpr uint32 v_size = MAX_VOLTA_L1_MEMORY / CACHE_LINE_SIZE;

//		test_l1_cache_kernel<v_size, MAX_VOLTA_SHARED_MEMORY_TO_TEST_L1> <<<
//				block_size, threads_per_block>>>(input_device_1.data(),
//				output_device_1.data(), hit_vector_device.data(),
//				miss_vector_device.data(), cycles);
		break;
	}
	}

	cuda_check(hipPeekAtLastError());
	cuda_check(hipDeviceSynchronize());
	//Host arrays
	//Copy back to the host
	this->hit_vector_host = hit_vector_device.to_vector();
	this->miss_vector_host = miss_vector_device.to_vector();
	this->output_host_1 = output_device_1.to_vector();
}
