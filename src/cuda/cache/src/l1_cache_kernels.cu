#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 26/01/2019
 *      Author: fernando
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#include "utils.h"
#include "kernels.h"
#include "CacheLine.h"

__device__ uint64 l1_cache_err;

/*
 * l1_size size of the L1 cache
 * V_size = l1_size / sizeof(CacheLine)
 */
template<typename int_t, const uint32 V_SIZE,
		const uint32 LINE_SIZE>
__global__ void test_l1_cache_kernel(CacheLine<LINE_SIZE> *lines,
		int_t *l1_hit_array, int_t *l1_miss_array, std::int64_t sleep_cycles,
		byte t) {
	register uint32 tx = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int_t l1_t_hit[V_SIZE];
	__shared__ int_t l1_t_miss[V_SIZE];

	for (uint32 i = 0; i < V_SIZE; i++) {
		int_t t1 = clock();
		register auto r = lines[tx + i];
		int_t t2 = clock();
		l1_t_miss[i] = t2 - t1;
//		lines[tx + i] = r;
	}

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	for (uint32 i = 0; i < V_SIZE; i++) {
		//last checking
		int_t t1 = clock();
		register auto r = lines[tx + i];
		int_t t2 = clock();
		l1_t_hit[i] = t2 - t1;

		//bitwise operation
		if ((r ^ t) != 0)
			atomicAdd((unsigned long long*)&l1_cache_err, 1);

//		//saving the result
		l1_hit_array[tx + i] = l1_t_hit[i];
		l1_miss_array[tx + i] = l1_t_miss[i];
	}

}

std::vector<std::string> test_l1_cache(uint32 number_of_sms, Board device) {
	std::vector<std::string> errors;
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	const byte t_byte = 39;
	const uint32 l1_size = 64 * 1024; // cache l1 has 65536 bytes
	const uint32 cache_line_size = 128; // size in bytes
	const uint32 v_size = l1_size / cache_line_size; // 512 lines

	//device arrays
	int32 *l1_hit_array_device, *l1_miss_array_device;
	hipMalloc(&l1_hit_array_device, sizeof(int32) * v_size);
	hipMalloc(&l1_miss_array_device, sizeof(int32) * v_size);

	//Host arrays
	int32 *l1_hit_array_host = new int32[v_size];
	int32 *l1_miss_array_host = new int32[v_size];

	//Set each element of V array
	CacheLine<cache_line_size> *V_dev, *V_host;
	V_host = new CacheLine<cache_line_size> [v_size];
	for (int i = 0; i < v_size; i++) {
		V_host[i] = t_byte;
	}

	//copy to the gpu
	hipMalloc(&V_dev, sizeof(CacheLine<cache_line_size> ) * v_size);
	hipMemcpy(V_dev, V_host, sizeof(CacheLine<cache_line_size> ) * v_size,
			hipMemcpyDeviceToHost);

	test_l1_cache_kernel<int32, v_size, cache_line_size> <<<1, 1>>>(V_dev,
			l1_hit_array_device, l1_miss_array_device, 1000000000, t_byte);
	cuda_check(hipDeviceSynchronize());

	hipMemcpy(l1_hit_array_host, l1_hit_array_device,
			sizeof(int32) * v_size, hipMemcpyDeviceToHost);
	hipMemcpy(l1_miss_array_host, l1_miss_array_device,
			sizeof(int32) * v_size, hipMemcpyDeviceToHost);
	auto bad = 0;
	for (int i = 0; i < v_size; i++) {
		if ((l1_hit_array_host[i] - l1_miss_array_host[i]) > 0)
			bad++;
	}
	std::cout << "TOTAL BAD " << bad << std::endl;
	hipDeviceSetCacheConfig(hipFuncCachePreferNone);

	hipFree(l1_hit_array_device);
	hipFree(l1_miss_array_device);
	hipFree(V_dev);
	delete[] V_host;
	delete[] l1_hit_array_host;
	delete[] l1_miss_array_host;
	return errors;
}

std::vector<std::string> test_l1_cache(const Parameters& parameters){
	std::vector<std::string> errors;

	return errors;
}
