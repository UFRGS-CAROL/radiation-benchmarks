#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: 26/01/2019
 *      Author: fernando
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

#include "utils.h"
#include "kernels.h"
#include "CacheLine.h"

__device__ uint64 l1_cache_err;

/*
 * l1_size size of the L1 cache
 * V_size = l1_size / sizeof(CacheLine)
 */
template<typename int_t, const uint32 V_SIZE, const uint32 LINE_SIZE,
		const uint32 SHARED_PER_SM>
__global__ void test_l1_cache_kernel(CacheLine<LINE_SIZE> *lines,
		int_t *l1_hit_array, int_t *l1_miss_array, int64 sleep_cycles, byte t) {

	__shared__ int_t l1_t_hit[SHARED_PER_SM / 2];
	__shared__ int_t l1_t_miss[SHARED_PER_SM / 2];
	uint32 i = threadIdx.x;
	if (threadIdx.x < V_SIZE) {

		register CacheLine<LINE_SIZE> r;
		volatile int_t t1 = clock();
		r = lines[blockIdx.x * V_SIZE + i];
		volatile int_t t2 = clock();
		l1_t_miss[i] = t2 - t1;
		lines[blockIdx.x * V_SIZE + i] = r;

		//wait for exposition to neutrons
		sleep_cuda(sleep_cycles);

		//last checking
		t1 = clock();
		register CacheLine<LINE_SIZE> r2 = lines[blockIdx.x * V_SIZE + i];
		t2 = clock();
		l1_t_hit[i] = t2 - t1;
		lines[blockIdx.x * V_SIZE + i] = r2;

		if (r != t) {
			atomicAdd(&l1_cache_err, 1);
		}
	}

	__syncthreads();
	if (threadIdx.x < V_SIZE) {
		for (uint32 i = 0; i < V_SIZE; i++) {
			l1_miss_array[blockIdx.x * V_SIZE + i] = l1_t_miss[i];
			l1_hit_array[blockIdx.x * V_SIZE + i] = l1_t_hit[i];
		}
	}
}

template<const uint32 V_SIZE, const uint32 L1_LINE_SIZE,
		const uint32 SHARED_PER_SM>
Tuple test_l1_cache(const uint32 number_of_sms, const byte t_byte,
		const int64 cycles) {

	const uint32 v_size_multiple_threads = V_SIZE * number_of_sms; // Each block with one thread using all l1 cache

	//device arrays
	int32 *l1_hit_array_device, *l1_miss_array_device;
	cuda_check(
			hipMalloc(&l1_hit_array_device,
					sizeof(int32) * v_size_multiple_threads));
	cuda_check(
			hipMalloc(&l1_miss_array_device,
					sizeof(int32) * v_size_multiple_threads));

	//Set each element of V array
	CacheLine < L1_LINE_SIZE > *V_dev;
	std::vector<CacheLine<L1_LINE_SIZE> > V_host(v_size_multiple_threads);

	for (int i = 0; i < v_size_multiple_threads; i++) {
		V_host[i] = t_byte;
	}

	//copy to the GPU
	cuda_check(
			hipMalloc(&V_dev,
					sizeof(CacheLine<L1_LINE_SIZE> )
							* v_size_multiple_threads));

	cuda_check(
			hipMemcpy(V_dev, V_host.data(),
					sizeof(CacheLine<L1_LINE_SIZE> ) * v_size_multiple_threads,
					hipMemcpyHostToDevice));

	//Set to zero err_check
	uint64 l1_cache_err_host = 0;
	cuda_check(
			hipMemcpyToSymbol(HIP_SYMBOL(l1_cache_err), &l1_cache_err_host, sizeof(uint64),
					0));

	dim3 block_size(number_of_sms), threads_per_block(V_SIZE);

	test_l1_cache_kernel<int32, V_SIZE, L1_LINE_SIZE, SHARED_PER_SM> <<<
	block_size, threads_per_block>>>(V_dev, l1_hit_array_device,
			l1_miss_array_device, cycles, t_byte);
	cuda_check(hipDeviceSynchronize());

	//Host arrays
	//Copy back to the host
	std::vector<int32> l1_hit_array_host(v_size_multiple_threads),
			l1_miss_array_host(v_size_multiple_threads);
	cuda_check(
			hipMemcpy(l1_hit_array_host.data(), l1_hit_array_device,
					sizeof(int32) * v_size_multiple_threads,
					hipMemcpyDeviceToHost));
	cuda_check(
			hipMemcpy(l1_miss_array_host.data(), l1_miss_array_device,
					sizeof(int32) * v_size_multiple_threads,
					hipMemcpyDeviceToHost));
	cuda_check(
			hipMemcpy(V_host.data(), V_dev,
					sizeof(CacheLine<L1_LINE_SIZE> ) * v_size_multiple_threads,
					hipMemcpyDeviceToHost));

	cuda_check(
			hipMemcpyFromSymbol(&l1_cache_err_host, HIP_SYMBOL(l1_cache_err),
					sizeof(uint64), 0));

	cuda_check(hipFree(l1_hit_array_device));
	cuda_check(hipFree(l1_miss_array_device));
	cuda_check(hipFree(V_dev));

	Tuple t;

	t.cache_lines.assign((byte*) V_host.data(),
			(byte*) V_host.data()
					+ (sizeof(CacheLine<L1_LINE_SIZE> ) * V_host.size()));
	t.misses = std::move(l1_miss_array_host);

	t.hits = std::move(l1_hit_array_host);
	t.errors = l1_cache_err_host;

	return t;
}

Tuple test_l1_cache(const Parameters& parameters) {
	//This switch is only to set manually the cache line size
	//since it is hard to check it at runtime
	switch (parameters.device) {
	case K40: {
		// cache l1 has 65536 bytes
		//BUT, only 48kb are destined to L1 memory
		//so alloc 49152 bytes
		// cache line has 128 bytes
		//to force alloc maximum shared memory
		std::cout << "KEPLER\n";
		const uint32 max_l1_cache = 48 * 1024; //bytes
		const uint32 max_shared_mem = 8 * 1024;
		const uint32 cache_line_size = 128;
		const uint32 v_size = max_l1_cache / cache_line_size;
		return test_l1_cache<v_size, cache_line_size, max_shared_mem>(
				parameters.number_of_sms, parameters.t_byte,
				parameters.one_second_cycles);
//		break;
	}
	case TITANV: {
		// cache l1 has 128 Kbytes
		//BUT, only 98304 bytes are destined to L1 memory
		//so alloc 98304 bytes
		// cache line has 128 bytes
		const uint32 max_l1_cache = 96 * 1024; //bytes
		const uint32 max_shared_mem = 8 * 1024;
		const uint32 cache_line_size = 128;
		const uint32 v_size = max_l1_cache / cache_line_size;

		return test_l1_cache<v_size, cache_line_size, max_shared_mem>(
				parameters.number_of_sms, parameters.t_byte,
				parameters.one_second_cycles);
//		break;
	}
	}

	return Tuple();
}

