#include "hip/hip_runtime.h"
/*
 * shared_memory_kernels.cu
 *
 *  Created on: Jan 31, 2019
 *      Author: carol
 */

#include "Parameters.h"
#include "CacheLine.h"
#include "utils.h"
#include "SharedMemory.h"

__constant__ __device__
static uint64 volta_input[CACHE_LINE_SIZE_BY_INT64][2] =
		{ //teste
		{ 0xffffffffffffffff, 0x0000000000000000 }, { 0xffffffffffffffff,
				0x0000000000000000 },
				{ 0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 }, {
						0xffffffffffffffff, 0x0000000000000000 } };

template<const uint32 V_SIZE>
__global__ void test_shared_memory_kernel(uint64 *output1, uint64 *output2,
		uint64 *output3, const int64 sleep_cycles, const uint32 zero_or_one) {

	__shared__ uint64 V[V_SIZE * CACHE_LINE_SIZE_BY_INT64];
	const register uint64 index = (blockIdx.x * V_SIZE + threadIdx.x)
			* CACHE_LINE_SIZE_BY_INT64;

	move_cache_line(V + threadIdx.x, volta_input[zero_or_one]);

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	move_cache_line(output1 + index, V + threadIdx.x);
	move_cache_line(output2 + index, V + threadIdx.x);
	move_cache_line(output3 + index, V + threadIdx.x);

}

template<const uint32 V_SIZE>
__global__ void test_shared_memory_kernel(uint64 *input, uint64 *output,
		const int64 sleep_cycles) {

	__shared__ uint64 V[V_SIZE * CACHE_LINE_SIZE_BY_INT64];
	const register uint64 index = (blockIdx.x * V_SIZE + threadIdx.x)
			* CACHE_LINE_SIZE_BY_INT64;

	move_cache_line(V + threadIdx.x, input + index);

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	move_cache_line(output + index, V + threadIdx.x);
}

void SharedMemory::test(const uint64& mem) {
	//	//Set each element of V array

	std::fill(this->input_host_1.begin(), this->input_host_1.end(), mem);
	rad::DeviceVector<uint64> input_device_1 = this->input_host_1;
	rad::DeviceVector<uint64> output_device_1 = this->output_host_1;
	rad::DeviceVector<uint64> output_device_2;
	rad::DeviceVector<uint64> output_device_3;

	//Set the number of threads
	//These archs support two blocks per SM with 48KB of shared memory
	switch (this->device) {
	case K20:
	case K40: {
		constexpr uint32 v_size = MAX_KEPLER_SHARED_MEMORY / CACHE_LINE_SIZE;
		test_shared_memory_kernel<v_size> <<<block_size, threads_per_block>>>(
				input_device_1.data(), output_device_1.data(), cycles);
		break;
	}
	case XAVIER:
	case TITANV: {
		output_device_2 = this->output_host_1;
		output_device_3 = this->output_host_1;
		uint32 zero_or_one = (mem == 0);
		constexpr uint32 v_size = MAX_VOLTA_SHARED_MEMORY / CACHE_LINE_SIZE;
		test_shared_memory_kernel<v_size> <<<block_size, threads_per_block>>>(
				output_device_1.data(), output_device_2.data(),
				output_device_3.data(), cycles, zero_or_one);
		break;
	}
	}

	cuda_check(hipPeekAtLastError());
	cuda_check(hipDeviceSynchronize());

	//Host arrays
	//Copy back to the host
	this->output_host_1 = output_device_1.to_vector();
	this->output_host_2 = output_device_2.to_vector();
	this->output_host_3 = output_device_3.to_vector();
}

/**
 * Shared memory size is in bytes
 */
SharedMemory::SharedMemory(const Parameters& parameters) :
		Memory<uint64>(parameters) {
//This switch is only to set manually the cache line size
//since it is hard to check it at runtime
	uint32 v_size;
	switch (parameters.device) {
	case K20:
	case K40:
		v_size = MAX_KEPLER_SHARED_MEMORY / CACHE_LINE_SIZE;

		if (MAX_KEPLER_SHARED_MEMORY != parameters.shared_memory_size)
			error(
					"SHARED DEFAULT SIZE AND DRIVER OBTAINED VALUE DOES NOT MACH. REAL VALUE:"
							+ std::to_string(parameters.shared_memory_size));
		break;
	case XAVIER:
	case TITANV:
		v_size = MAX_VOLTA_SHARED_MEMORY / CACHE_LINE_SIZE;

		if (MAX_VOLTA_SHARED_MEMORY * 2 != parameters.shared_memory_size)
			error(
					"SHARED DEFAULT SIZE AND DRIVER OBTAINED VALUE DOES NOT MACH. REAL VALUE:"
							+ std::to_string(parameters.shared_memory_size));
		break;
	}

	this->threads_per_block = dim3(v_size);
	uint32 v_size_multiple_threads = v_size * parameters.number_of_sms
			* CACHE_LINE_SIZE_BY_INT64;
	this->input_host_1.resize(v_size_multiple_threads);
	this->output_host_1.resize(v_size_multiple_threads);
}

bool SharedMemory::call_checker(uint64& gold, Log& log, int64& hits,
		int64& misses, int64& false_hits) {
	uint64* out_ptr1 = (uint64*) (this->output_host_1.data());
	uint64* out_ptr2 = (uint64*) (this->output_host_2.data());
	uint64* out_ptr3 = (uint64*) (this->output_host_3.data());

	return this->check_output_errors(out_ptr1, out_ptr2, out_ptr3, gold, log,
			hits, misses, false_hits, this->output_host_1.size());
}
