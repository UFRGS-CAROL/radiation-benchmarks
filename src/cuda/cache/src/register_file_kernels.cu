#include "hip/hip_runtime.h"
/*
 * register_file_kernels.cu
 *
 *  Created on: Feb 2, 2019
 *      Author: carol
 */

#include <cstring>
#include <ctime>
#include <bitset>
#include "RegisterFile.h"
#include "Parameters.h"
#include "utils.h"
#include "register_kernel.h"
#include "register_kernel_volta.h"

RegisterFile::RegisterFile(const Parameters& parameters) :
		Memory<uint32>(parameters) {
	//Kepler and Volta have
	//256KB registers per SM
	//256 threads
	this->number_of_threads = parameters.registers_per_block / RF_SIZE;

	uint32 out_size = parameters.number_of_sms * parameters.registers_per_block;

	this->number_of_sms = parameters.number_of_sms;
	this->input_host_1 = std::vector<uint32>(RF_SIZE);
	this->output_host_1 = std::vector<uint32>(out_size);
}

void RegisterFile::test(const uint64& mem_) {
	uint32 mem = mem_;
	//Set values to GPU
	std::fill(this->input_host_1.begin(), this->input_host_1.end(), mem);
	rad::DeviceVector<uint32> input_device_1 = this->input_host_1;
	rad::DeviceVector<uint32> output_device_1 = this->output_host_1;
	rad::DeviceVector<uint32> output_device_2, output_device_3;
	switch (this->device) {
	case K20:
	case K40:
		test_register_file_kernel<<<number_of_sms, number_of_threads>>>(
				output_device_1.data(), input_device_1.data(), cycles);
		break;
	case TITANV:
	case XAVIER:
		output_device_2 = this->output_host_1;
		output_device_3 = this->output_host_1;
		uint32 zero_or_one = (mem_ == 0);
		test_register_file_kernel_volta<<<number_of_sms, number_of_threads>>>(
				output_device_1.data(), output_device_2.data(),
				output_device_3.data(), zero_or_one, this->cycles);
		break;
	};

    rad::checkFrameworkErrors(hipPeekAtLastError());
    rad::checkFrameworkErrors(hipDeviceSynchronize());
	this->output_host_1 = output_device_1.to_vector();
	this->output_host_2 = output_device_2.to_vector();
	this->output_host_3 = output_device_3.to_vector();

}

std::string RegisterFile::error_detail(uint64 i, uint64 e, uint64 r, int64 hits,
		int64 misses, int64 false_hits) {
	std::string error_detail = "";
	error_detail += " i:" + std::to_string(i);
	error_detail += " register:R" + std::to_string(i % 256);
	error_detail += " e:" + std::to_string(e);
	error_detail += " r:" + std::to_string(r);
	return error_detail;
}

bool RegisterFile::call_checker(uint64& gold, rad::Log& log, int64& hits,
		int64& misses, int64& false_hits, bool verbose) {
	uint32* out_ptr1 = (uint32*) this->output_host_1.data();
	uint32* out_ptr2 = (uint32*) this->output_host_2.data();
	uint32* out_ptr3 = (uint32*) this->output_host_3.data();

	uint32 gold_ = gold;

	return this->check_output_errors(out_ptr1, out_ptr2, out_ptr3, gold_, log,
			hits, misses, false_hits, this->output_host_1.size(), verbose);
}
