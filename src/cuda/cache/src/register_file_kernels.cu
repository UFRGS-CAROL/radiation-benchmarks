#include "hip/hip_runtime.h"
/*
 * register_file_kernels.cu
 *
 *  Created on: Feb 2, 2019
 *      Author: carol
 */

#include <cstring>
#include <ctime>
#include <bitset>

#include "RegisterFile.h"
#include "Parameters.h"
#include "utils.h"
#include "register_kernel.h"

RegisterFile::RegisterFile(const Parameters& parameters) :
		Memory<uint32>(parameters) {
	//Kepler and Volta have
	//256KB registers per SM
	//256 threads
	this->number_of_threads = parameters.registers_per_block / RF_SIZE;

	uint32 out_size = parameters.number_of_sms * number_of_threads * RF_SIZE;
	this->number_of_sms = parameters.number_of_sms;
	this->input_host_1 = std::vector<uint32>(RF_SIZE);
	this->input_host_2 = std::vector<uint32>(RF_SIZE);
	this->input_host_3 = std::vector<uint32>(RF_SIZE);

	this->output_host_1 = std::vector<uint32>(out_size);
	this->output_host_2 = std::vector<uint32>(out_size);
	this->output_host_3 = std::vector<uint32>(out_size);

}

void RegisterFile::test(byte t_byte) {
	uint32 reg_data;
	std::memset((uint32*) &reg_data, t_byte, sizeof(uint32));

	//Set values to GPU
	std::fill(this->input_host_1.begin(), this->input_host_1.end(), reg_data);
	std::fill(this->input_host_2.begin(), this->input_host_2.end(), reg_data);
	std::fill(this->input_host_3.begin(), this->input_host_3.end(), reg_data);

	rad::DeviceVector<uint32> input_device_1 = this->input_host_1;
	rad::DeviceVector<uint32> input_device_2 = this->input_host_2;
	rad::DeviceVector<uint32> input_device_3 = this->input_host_3;
	rad::DeviceVector<uint32> output_device_1 = this->output_host_1;
	rad::DeviceVector<uint32> output_device_2 = this->output_host_2;
	rad::DeviceVector<uint32> output_device_3 = this->output_host_3;

	double start = rad::mysecond();
	if (reg_data == 4294967295) {
		test_register_file_kernel_or<<<number_of_sms, number_of_threads>>>(
				output_device_1.data(), output_device_2.data(),
				output_device_3.data(), input_device_1.data(),
				input_device_2.data(), input_device_3.data(), reg_data, cycles);
	} else if (reg_data == 0) {
		test_register_file_kernel_and<<<number_of_sms, number_of_threads>>>(
				output_device_1.data(), output_device_2.data(),
				output_device_3.data(), input_device_1.data(),
				input_device_2.data(), input_device_3.data(), reg_data, cycles);
	}
	cuda_check(hipDeviceSynchronize());
	double end = rad::mysecond();

	this->output_host_1 = output_device_1.to_vector();
	this->output_host_2 = output_device_2.to_vector();
	this->output_host_3 = output_device_3.to_vector();

}

std::string RegisterFile::error_detail(uint32 i, uint32 e, uint32 r,
		uint64 hits, uint64 misses, uint64 false_hits) {
	std::string error_detail = "";
	error_detail += " i:" + std::to_string(i);
	error_detail += " register:R" + std::to_string(i % 256);
	error_detail += " e:" + std::to_string(e);
	error_detail += " r:" + std::to_string(r);
	return error_detail;
}

void RegisterFile::call_checker(const std::vector<uint32>& v1,
		const std::vector<uint32>& v2, const std::vector<uint32>& v3,
		byte valGold, Log& log, uint64 hits, uint64 misses, uint64 false_hits,
		bool verbose) {
	uint32 val_gold_tmp;
	std::memset((uint32*) &val_gold_tmp, valGold, sizeof(uint32));

	this->check_output_errors(v1.data(), v2.data(), v3.data(), val_gold_tmp,
			log, hits, misses, false_hits, verbose, v1.size());
}

