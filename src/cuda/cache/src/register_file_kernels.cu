/*
 * register_file_kernels.cu
 *
 *  Created on: Feb 2, 2019
 *      Author: carol
 */

#include "kernels.h"
#include "utils.h"
#include <cstring>

#include "register_kernel.h"

template<const uint32 RFSIZE>
Tuple test_register_file(const uint32 reg_data, const int64 cycles,
		dim3& block_size, dim3& threads_per_block) {

	//Allocate an array of the size of all register bank
	uint32 out_size = block_size.x * block_size.y * threads_per_block.x * RFSIZE;
	uint32 *output_dev1;
    uint32 *output_dev2;
    uint32 *output_dev3;

	//error variable
	uint64 register_file_errors_host1 = 0;
	uint64 register_file_errors_host2 = 0;
	uint64 register_file_errors_host3 = 0;

	cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(register_file_errors1), &register_file_errors_host1, sizeof(uint64), 0));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(register_file_errors2), &register_file_errors_host2, sizeof(uint64), 0));
    cuda_check(hipMemcpyToSymbol(HIP_SYMBOL(register_file_errors3), &register_file_errors_host3, sizeof(uint64), 0));

	//byte size
	const uint32 byte_size = sizeof(uint32) * out_size;

	//malloc on device
	cuda_check(hipMalloc(&output_dev1, byte_size));
	cuda_check(hipMalloc(&output_dev2, byte_size));
	cuda_check(hipMalloc(&output_dev3, byte_size));

	//malloc on host
	std::vector<uint32> output_host1(out_size, reg_data);
    std::vector<uint32> output_host2(out_size, reg_data);
    std::vector<uint32> output_host3(out_size, reg_data);

	cuda_check(hipMemcpy(output_dev1, output_host1.data(), byte_size, hipMemcpyHostToDevice));
	cuda_check(hipMemcpy(output_dev2, output_host2.data(), byte_size, hipMemcpyHostToDevice));
	cuda_check(hipMemcpy(output_dev3, output_host3.data(), byte_size, hipMemcpyHostToDevice));


	double start = Log::mysecond();
	test_register_file_kernel<<<block_size, threads_per_block>>>(output_dev1, output_dev2, output_dev3, reg_data, cycles);
	cuda_check(hipDeviceSynchronize());

	//Copy data back
	cuda_check(hipMemcpy(output_host1.data(), output_dev1, byte_size, hipMemcpyDeviceToHost));
    cuda_check(hipMemcpy(output_host2.data(), output_dev2, byte_size, hipMemcpyDeviceToHost));
    cuda_check(hipMemcpy(output_host3.data(), output_dev3, byte_size, hipMemcpyDeviceToHost));

	//Copy error var
	cuda_check(hipMemcpyFromSymbol(&register_file_errors_host1, HIP_SYMBOL(register_file_errors1), sizeof(uint64), 0));
    cuda_check(hipMemcpyFromSymbol(&register_file_errors_host2, HIP_SYMBOL(register_file_errors2), sizeof(uint64), 0));
    cuda_check(hipMemcpyFromSymbol(&register_file_errors_host3, HIP_SYMBOL(register_file_errors3), sizeof(uint64), 0));

	cuda_check(hipFree(output_dev1));
    cuda_check(hipFree(output_dev2));
    cuda_check(hipFree(output_dev3));

	Tuple t;
	t.register_file = std::move(output_host1);
	t.register_file2 = std::move(output_host2);
	t.register_file3 = std::move(output_host3);
	t.errors = register_file_errors_host1;
	t.errors2 = register_file_errors_host2;
	t.errors3 = register_file_errors_host3;

	return t;

}

Tuple test_register_file(const Parameters& parameters) {
	const uint32 rf_size = 256;
	//Kepler and Volta have
	//fucking 256KB registers per SM
	// so I have to allocate 4 blocks of
	// 256 threads
	dim3 block_size(parameters.number_of_sms, 4);
	dim3 threads_per_block(parameters.registers_per_block / rf_size);

	uint32 reg_data;
	std::memset(&reg_data, parameters.t_byte, sizeof(uint32));

	return test_register_file<rf_size>(reg_data, parameters.one_second_cycles,
			block_size, threads_per_block);
}
