#include "hip/hip_runtime.h"
/*
 * l2_cache_kernels.cu
 *
 *  Created on: Jan 31, 2019
 *      Author: carol
 */
#include <iostream>
#include <vector>
#include <hiprand.h>
#include <cstdlib>

#include "kernels.h"
#include "CacheLine.h"
#include "utils.h"

__device__ uint64 l2_cache_err;

template<typename int_t, const uint32 V_SIZE, const uint32 LINE_SIZE>
__global__ void test_l2_cache_kernel(CacheLine<LINE_SIZE> *lines,
		int_t *l2_hit_array, int_t *l2_miss_array, std::int64_t sleep_cycles,
		byte t) {
	uint32 i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < V_SIZE) {

		volatile int_t t1 = clock();
		CacheLine < LINE_SIZE > r = lines[i];
		volatile int_t t2 = clock();
		l2_miss_array[i] = t2 - t1;

		//wait for exposition to neutrons
		sleep_cuda(sleep_cycles);

		//last checking
		t1 = clock();
		CacheLine < LINE_SIZE > r2 = lines[i];
		t2 = clock();
		l2_hit_array[i] = t2 - t1;

		//bitwise operation
		if (r != t)
			atomicAdd((unsigned long long*) &l2_cache_err, 1);

		lines[i] = r2;
	}
	__syncthreads();
}

__global__ void clear_cache_kenel(float *random_array) {
	register uint32 tx = blockIdx.x * blockDim.x + threadIdx.x;
	random_array[tx] += random_array[tx] * 339 + 1 * (-random_array[tx]);
}

void clear_cache(uint32 n) {
	float *random_array_dev;
	/* Allocate n floats on device */
	cuda_check(hipMalloc((void ** )&random_array_dev, n * sizeof(float)));

	/* Create pseudo-random number generator */
	hiprandGenerator_t gen;

	(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

	/* Set seed */
	(hiprandSetPseudoRandomGeneratorSeed(gen, std::rand()));

	/* Generate n floats on device */
	(hiprandGenerateUniform(gen, random_array_dev, n));

	uint32 thread_number = std::ceil(float(n) / (BLOCK_SIZE * BLOCK_SIZE));
	uint32 block_number = std::ceil(n / float(thread_number));

	clear_cache_kenel<<<block_number, thread_number>>>(random_array_dev);
	cuda_check(hipDeviceSynchronize());

	(hiprandDestroyGenerator(gen));

	cuda_check(hipFree(random_array_dev));

}

template<const uint32 V_SIZE, const uint32 L2_LINE_SIZE>
Tuple test_l2_cache(const byte t_byte, const int64 cycles,
		const uint32 l2_size) {
	//device arrays
	int32 *l2_hit_array_device, *l2_miss_array_device;
	hipMalloc(&l2_hit_array_device, sizeof(int32) * V_SIZE);
	hipMalloc(&l2_miss_array_device, sizeof(int32) * V_SIZE);

	//Host arrays
	std::vector<int32> l2_hit_array_host(V_SIZE), l2_miss_array_host(V_SIZE);

	//Set each element of V array
	CacheLine < L2_LINE_SIZE > *V_dev;
	std::vector<CacheLine<L2_LINE_SIZE> > V_host(V_SIZE, t_byte);

	//copy to the gpu
	cuda_check(hipMalloc(&V_dev, sizeof(CacheLine<L2_LINE_SIZE> ) * V_SIZE));
	cuda_check(
			hipMemcpy(V_dev, V_host.data(),
					sizeof(CacheLine<L2_LINE_SIZE> ) * V_SIZE,
					hipMemcpyHostToDevice));

	//Clear the L2 Cache
	clear_cache(l2_size / sizeof(float));

	/**
	 * Split alongside the blocks
	 */
	dim3 block_size(V_SIZE / (BLOCK_SIZE * BLOCK_SIZE));
	dim3 threads_per_block(BLOCK_SIZE * BLOCK_SIZE);

	test_l2_cache_kernel<int32, V_SIZE, L2_LINE_SIZE> <<<block_size, threads_per_block>>>(V_dev,
			l2_hit_array_device, l2_miss_array_device, cycles, t_byte);
	cuda_check(hipDeviceSynchronize());

	cuda_check(
			hipMemcpy(l2_hit_array_host.data(), l2_hit_array_device,
					sizeof(int32) * V_SIZE, hipMemcpyDeviceToHost));
	cuda_check(
			hipMemcpy(l2_miss_array_host.data(), l2_miss_array_device,
					sizeof(int32) * V_SIZE, hipMemcpyDeviceToHost));

	cuda_check(
			hipMemcpy(V_host.data(), V_dev,
					sizeof(CacheLine<L2_LINE_SIZE> ) * V_SIZE,
					hipMemcpyDeviceToHost));

	//Set to zero err_check
	uint64 l2_cache_err_host = 0;
	cuda_check(
			hipMemcpyFromSymbol(&l2_cache_err_host, HIP_SYMBOL(l2_cache_err),
					sizeof(uint64), 0));

	cuda_check(hipFree(l2_hit_array_device));
	cuda_check(hipFree(l2_miss_array_device));
	cuda_check(hipFree(V_dev));

	Tuple t;

	t.cache_lines.assign((byte*) V_host.data(),
			(byte*) V_host.data()
					+ (sizeof(CacheLine<L2_LINE_SIZE> ) * V_host.size()));
	t.misses = std::move(l2_miss_array_host);

	t.hits = std::move(l2_hit_array_host);
	t.errors = l2_cache_err_host;

	return t;
}

Tuple test_l2_cache(const Parameters& parameters) {
	//This switch is only to set manually the cache line size
	//since it is hard to check it at runtime
	switch (parameters.device) {
	case K40: {
		const uint32 max_l2_cache = 1536 * 1024; //bytes
		if (max_l2_cache != parameters.l2_size)
			error(
					"L2 DEFAULT CACHE AND DRIVER OBTAINED VALUE DOES NOT MACH. REAL VALUE:"
							+ std::to_string(parameters.l2_size));

		const uint32 cache_line_size = 32;
		const uint32 v_size = max_l2_cache / cache_line_size;
		return test_l2_cache<v_size, cache_line_size>(parameters.t_byte,
				parameters.one_second_cycles, max_l2_cache);
//		break;
	}
	case TITANV: {
		const uint32 max_l2_cache = 6144 * 1024; //bytes
		if (max_l2_cache != parameters.l2_size)
			error(
					"L2 DEFAULT CACHE AND DRIVER OBTAINED VALUE DOES NOT MACH. REAL VALUE:"
							+ std::to_string(parameters.l2_size));

		const uint32 cache_line_size = 64;
		const uint32 v_size = max_l2_cache / cache_line_size;

		return test_l2_cache<v_size, cache_line_size>(parameters.t_byte,
				parameters.one_second_cycles, max_l2_cache);
//		break;
	}
	}

	return Tuple();
}
