#include "hip/hip_runtime.h"
/*
 * l2_cache_kernels.cu
 *
 *  Created on: Jan 31, 2019
 *      Author: carol
 */
#include <iostream>
#include <vector>
#include "kernels.h"
#include "CacheLine.h"
#include "utils.h"

__device__ uint64 l2_cache_err;

template<typename int_t, const uint32 V_SIZE, const uint32 LINE_SIZE>
__global__ void test_l2_cache_kernel(CacheLine<LINE_SIZE> *lines,
		int_t *l2_hit_array, int_t *l2_miss_array, std::int64_t sleep_cycles,
		byte t) {
	register uint32 tx = blockIdx.x * blockDim.x + threadIdx.x;

	for (uint32 i = 0; i < V_SIZE; i++) {
		int_t t1 = clock();
		volatile register auto r = lines[tx + i];
		int_t t2 = clock();
		l2_hit_array[tx + i] = t2 - t1;
	}

	//wait for exposition to neutrons
	sleep_cuda(sleep_cycles);

	for (uint32 i = 0; i < V_SIZE; i++) {
		//last checking
		int_t t1 = clock();
		volatile register auto r = lines[tx + i];
		int_t t2 = clock();
		l2_miss_array[tx + i] = t2 - t1;

		//bitwise operation
		if ((r ^ t) != 0)
			atomicAdd((unsigned long long*) &l2_cache_err, 1);
	}
}

void test_l2_cache(uint32 number_of_sms, Board device) {
	const byte t_byte = 39;
	const uint32 l2_size = 1536 * 1024; // cache l1 has 65536 bytes
	const uint32 cache_line_size = 128; // size in bytes
	const uint32 v_size = l2_size / cache_line_size; // 12288 lines

	//device arrays
	int32 *l2_hit_array_device, *l2_miss_array_device;
	hipMalloc(&l2_hit_array_device, sizeof(int32) * v_size);
	hipMalloc(&l2_miss_array_device, sizeof(int32) * v_size);

	//Host arrays
	std::vector<int32> l1_hit_array_host(v_size), l1_miss_array_host(v_size);

	//Set each element of V array
	CacheLine<cache_line_size> *V_dev;
	std::vector<CacheLine<cache_line_size> > V_host(v_size, t_byte);
//	V_host = new CacheLine<cache_line_size> [v_size];
//	for (int i = 0; i < v_size; i++) {
//		V_host[i] = t_byte;
//	}

	//copy to the gpu
	hipMalloc(&V_dev, sizeof(CacheLine<cache_line_size> ) * v_size);
	hipMemcpy(V_dev, V_host.data(),
			sizeof(CacheLine<cache_line_size> ) * v_size,
			hipMemcpyDeviceToHost);

	test_l2_cache_kernel<int32, v_size, cache_line_size> <<<1, 1>>>(V_dev,
			l2_hit_array_device, l2_miss_array_device, 1000000000, t_byte);
	cuda_check(hipDeviceSynchronize());

	hipMemcpy(l1_hit_array_host.data(), l2_hit_array_device,
			sizeof(int32) * v_size, hipMemcpyDeviceToHost);
	hipMemcpy(l1_miss_array_host.data(), l2_miss_array_device,
			sizeof(int32) * v_size, hipMemcpyDeviceToHost);

	uint64 bad = 0;
	for (int i = 0; i < v_size; i++) {
		if ((l1_hit_array_host[i] - l1_miss_array_host[i]) > 0)
			bad++;
	}
	std::cout << "TOTAL BAD " << bad << std::endl;

	hipFree(l2_hit_array_device);
	hipFree(l2_miss_array_device);
	hipFree(V_dev);
}

