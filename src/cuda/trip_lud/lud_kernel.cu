#include "hip/hip_runtime.h"
#include <stdio.h>

#include "lud_kernel.h"
#include "lud_half_kernel.h"

/**
 * For float and double precision
 * for half precision see lud_half_kernel.h
 */

template<typename real_t>
__global__ void lud_diagonal(real_t *m, int matrix_dim, int offset) {
	int i, j;
	__shared__ real_t shadow[BLOCK_SIZE][BLOCK_SIZE];

	int array_offset = offset * matrix_dim + offset;
	for (i = 0; i < BLOCK_SIZE; i++) {
		shadow[i][threadIdx.x] = m[array_offset + threadIdx.x];
		array_offset += matrix_dim;
	}
	__syncthreads();
	for (i = 0; i < BLOCK_SIZE - 1; i++) {

		if (threadIdx.x > i) {
			for (j = 0; j < i; j++)
				shadow[threadIdx.x][i] -= shadow[threadIdx.x][j] * shadow[j][i];
			shadow[threadIdx.x][i] /= shadow[i][i];
		}

		__syncthreads();
		if (threadIdx.x > i) {

			for (j = 0; j < i + 1; j++)
				shadow[i + 1][threadIdx.x] -= shadow[i + 1][j]
						* shadow[j][threadIdx.x];
		}
		__syncthreads();
	}

	/*
	 The first row is not modified, it
	 is no need to write it back to the
	 global memory

	 */
	array_offset = (offset + 1) * matrix_dim + offset;
	for (i = 1; i < BLOCK_SIZE; i++) {
		m[array_offset + threadIdx.x] = shadow[i][threadIdx.x];
		array_offset += matrix_dim;
	}
}

template<typename real_t>
__global__ void lud_perimeter(real_t *m, int matrix_dim, int offset) {
	__shared__ real_t dia[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ real_t peri_row[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ real_t peri_col[BLOCK_SIZE][BLOCK_SIZE];

	int i, j, array_offset;
	int idx;

	if (threadIdx.x < BLOCK_SIZE) {
		idx = threadIdx.x;

		array_offset = offset * matrix_dim + offset;
		for (i = 0; i < BLOCK_SIZE / 2; i++) {
			dia[i][idx] = m[array_offset + idx];
			array_offset += matrix_dim;
		}

		array_offset = offset * matrix_dim + offset;
		for (i = 0; i < BLOCK_SIZE; i++) {
			peri_row[i][idx] = m[array_offset + (blockIdx.x + 1) * BLOCK_SIZE
					+ idx];
			array_offset += matrix_dim;
		}

	} else {
		idx = threadIdx.x - BLOCK_SIZE;

		array_offset = (offset + BLOCK_SIZE / 2) * matrix_dim + offset;
		for (i = BLOCK_SIZE / 2; i < BLOCK_SIZE; i++) {
			dia[i][idx] = m[array_offset + idx];
			array_offset += matrix_dim;
		}

		array_offset = (offset + (blockIdx.x + 1) * BLOCK_SIZE) * matrix_dim
				+ offset;
		for (i = 0; i < BLOCK_SIZE; i++) {
			peri_col[i][idx] = m[array_offset + idx];
			array_offset += matrix_dim;
		}

	}
	__syncthreads();

	/* this version works ok on hardware, but not gpgpusim
	 **************************************************************
	 if (threadIdx.x < BLOCK_SIZE) { //peri-row
	 idx=threadIdx.x;
	 for(i=1; i < BLOCK_SIZE; i++){
	 for (j=0; j < i; j++)
	 peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
	 }


	 array_offset = (offset+1)*matrix_dim+offset;
	 for(i=1; i < BLOCK_SIZE; i++){
	 m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
	 array_offset += matrix_dim;
	 }
	 } else { //peri-col
	 idx=threadIdx.x - BLOCK_SIZE;
	 for(i=0; i < BLOCK_SIZE; i++){
	 for(j=0; j < i; j++)
	 peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
	 peri_col[idx][i] /= dia[i][i];
	 }

	 __syncthreads();

	 array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
	 for(i=0; i < BLOCK_SIZE; i++){
	 m[array_offset+idx] =  peri_col[i][idx];
	 array_offset += matrix_dim;
	 }
	 }
	 ***************************************************************/
	if (threadIdx.x < BLOCK_SIZE) { //peri-row
		idx = threadIdx.x;
		for (i = 1; i < BLOCK_SIZE; i++) {
			for (j = 0; j < i; j++)
				peri_row[i][idx] -= dia[i][j] * peri_row[j][idx];
		}
	} else { //peri-col
		idx = threadIdx.x - BLOCK_SIZE;
		for (i = 0; i < BLOCK_SIZE; i++) {
			for (j = 0; j < i; j++)
				peri_col[idx][i] -= peri_col[idx][j] * dia[j][i];
			peri_col[idx][i] /= dia[i][i];
		}
	}

	__syncthreads();

	if (threadIdx.x < BLOCK_SIZE) { //peri-row
		idx = threadIdx.x;
		array_offset = (offset + 1) * matrix_dim + offset;
		for (i = 1; i < BLOCK_SIZE; i++) {
			m[array_offset + (blockIdx.x + 1) * BLOCK_SIZE + idx] =
					peri_row[i][idx];
			array_offset += matrix_dim;
		}
	} else { //peri-col
		idx = threadIdx.x - BLOCK_SIZE;
		array_offset = (offset + (blockIdx.x + 1) * BLOCK_SIZE) * matrix_dim
				+ offset;
		for (i = 0; i < BLOCK_SIZE; i++) {
			m[array_offset + idx] = peri_col[i][idx];
			array_offset += matrix_dim;
		}
	}

}

template<typename real_t>
__global__ void lud_internal(real_t *m, int matrix_dim, int offset) {
	__shared__ real_t peri_row[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ real_t peri_col[BLOCK_SIZE][BLOCK_SIZE];

	int i;
	float sum;

	int global_row_id = offset + (blockIdx.y + 1) * BLOCK_SIZE;
	int global_col_id = offset + (blockIdx.x + 1) * BLOCK_SIZE;

	peri_row[threadIdx.y][threadIdx.x] = m[(offset + threadIdx.y) * matrix_dim
			+ global_col_id + threadIdx.x];
	peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id + threadIdx.y)
			* matrix_dim + offset + threadIdx.x];

	__syncthreads();

	sum = 0;
	for (i = 0; i < BLOCK_SIZE; i++)
		sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
	m[(global_row_id + threadIdx.y) * matrix_dim + global_col_id + threadIdx.x] -=
			sum;

}

template<typename real_t>
void lud_cuda(real_t *m, int matrix_dim) {
	int i = 0;
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	real_t *m_debug = (real_t*) malloc(
			matrix_dim * matrix_dim * sizeof(real_t));

	for (i = 0; i < matrix_dim - BLOCK_SIZE; i += BLOCK_SIZE) {
		lud_diagonal<real_t> <<<1, BLOCK_SIZE>>>(m, matrix_dim, i);

		lud_perimeter<real_t> <<<(matrix_dim - i) / BLOCK_SIZE - 1, BLOCK_SIZE * 2>>>(m,
				matrix_dim, i);

		dim3 dimGrid((matrix_dim - i) / BLOCK_SIZE - 1,
				(matrix_dim - i) / BLOCK_SIZE - 1);

		lud_internal<real_t> <<<dimGrid, dimBlock>>>(m, matrix_dim, i);
	}
	lud_diagonal<real_t> <<<1, BLOCK_SIZE>>>(m, matrix_dim, i);
	hipDeviceSynchronize();
}


void lud_cuda_float(float *m, int matrix_dim){
	lud_cuda<float>(m, matrix_dim);
}

void lud_cuda_double(double *m, int matrix_dim){
	lud_cuda<double>(m, matrix_dim);
}
//
//void lud_cuda_half(half *m, int matrix_dim){
//	lud_cuda<half>(m, matrix_dim);
//}
