
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>

#include "caffe/filler.hpp"
#include "caffe/layers/cudnn_conv_layer.hpp"
#include "caffe/net.hpp"
#include "caffe/solver.hpp"

namespace caffe {

template<typename Ftype, typename Btype>
void CuDNNConvolutionLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* weight = this->blobs_[0]->template gpu_data<Ftype>();
  shared_ptr<GPUMemory::Workspace>& ws = GPUMemory::workspace_[Caffe::current_device()];
  if (use_v7grouping()) {
    for (int i = 0; i < bottom.size(); ++i) {
      const Ftype *bottom_data = bottom[i]->gpu_data<Ftype>();
      Ftype *top_data = top[i]->mutable_gpu_data<Ftype>();
      // Forward through cuDNN in parallel over groups.
      CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(0),
          cudnn::dataType<Ftype>::one, fwd_bottom_descs_[i], bottom_data,
          fwd_filter_desc_, weight,
          fwd_conv_descs_[i], fwd_algo_[i], ws->data(), ws->size(),
          cudnn::dataType<Ftype>::zero, fwd_top_descs_[i], top_data));
      if (this->bias_term_) {
        const Ftype *bias_data = this->blobs_[1]->template gpu_data<Ftype>();
        CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(0),
            cudnn::dataType<Ftype>::one,
            fwd_bias_desc_, bias_data,
            cudnn::dataType<Ftype>::one,
            fwd_top_descs_[i], top_data));
      }
      CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));
    }  // end of for i
  } else {
    // "old" path
    for (int i = 0; i < bottom.size(); ++i) {
      const Ftype* bottom_data = bottom[i]->gpu_data<Ftype>();
      Ftype* top_data = top[i]->mutable_gpu_data<Ftype>();
      // Forward through cuDNN in parallel over groups.
      const size_t gsize = ws->size() / ws_groups();
      CHECK(is_even(gsize));
      for (int g = 0; g < groups(); ++g) {
        void* pspace = static_cast<unsigned char*>(ws->data()) + gsize * idxg(g);
        // Filters.
        CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(idxg(g)),
            cudnn::dataType<Ftype>::one, fwd_bottom_descs_[i], bottom_data + bottom_offset_ * g,
            fwd_filter_desc_, weight + this->weight_offset_ * g,
            fwd_conv_descs_[i], fwd_algo_[i], pspace, gsize,
            cudnn::dataType<Ftype>::zero, fwd_top_descs_[i], top_data + top_offset_ * g));
      }
      // NOLINT_NEXT_LINE(whitespace/operators)
      for (int ig = 0; ig < ws_groups(); ++ig) {
        CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(ig)));
      }

      if (this->bias_term_) {
        const Ftype* bias_data = this->blobs_[1]->template gpu_data<Ftype>();
        for (int g = 0; g < groups(); ++g) {
          CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(idxg(g)),
              cudnn::dataType<Ftype>::one,
              fwd_bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Ftype>::one,
              fwd_top_descs_[i], top_data + top_offset_ * g));
        }
        // Synchronize the work across groups, each of which went into its own stream
        // NOLINT_NEXT_LINE(whitespace/operators)
        for (int g = 0; g < ws_groups(); ++g) {
          CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(g)));
        }
      }
    }  // end of for i
  }

  ++fwd_count_;
}

template <typename Ftype, typename Btype>
void CuDNNConvolutionLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  propagate_down_ = propagate_down;
  shared_ptr<GPUMemory::Workspace>& ws = GPUMemory::workspace_[Caffe::current_device()];
  if (use_v7grouping()) {
    // compute dE/dB = sum_c(dE/dy)
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Btype *bias_diff = this->blobs_[1]->template mutable_gpu_diff<Btype>();
      for (int i = 0; i < top.size(); ++i) {
        Btype *top_diff = top[i]->mutable_gpu_diff<Btype>();
        // in parallel over groups
        CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(0),
            cudnn::dataType<Btype>::one, bwd_top_descs_[i], top_diff,
            cudnn::dataType<Btype>::one, bwd_bias_desc_, bias_diff));
        CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));
      }  // end of i
    }  // end of dB

    // compute dE/dW = dY * X
    if (this->param_propagate_down_[0]) {
      Btype *weight_diff = this->blobs_[0]->template mutable_gpu_diff<Btype>();
      for (int i = 0; i < top.size(); ++i) {
        Btype *top_diff = top[i]->mutable_gpu_diff<Btype>();
        const Btype *bottom_data = bottom[i]->gpu_data<Btype>();
        // Gradient w.r.t. weights.
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(Caffe::cudnn_handle(0),
            cudnn::dataType<Btype>::one, bwd_bottom_descs_[i], bottom_data,
            bwd_top_descs_[i], top_diff,
            bwd_conv_filter_descs_[i], bwd_filter_algo_[i], ws->data(), ws->size(),
            cudnn::dataType<Btype>::one, bwd_filter_desc_, weight_diff));
        CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));
      }  // end of i
    }

    // Backward propagate grad wrt bottom data dE/dX= dE/dY * W
    const Btype *weight = this->blobs_[0]->template gpu_data<Btype>();
    for (int i = 0; i < top.size(); ++i) {
      if (propagate_down[i]) {
        Btype *top_diff = top[i]->mutable_gpu_diff<Btype>();
        Btype *bottom_diff = bottom[i]->mutable_gpu_diff<Btype>();
        CUDNN_CHECK(cudnnConvolutionBackwardData(Caffe::cudnn_handle(0),
            cudnn::dataType<Btype>::one, bwd_filter_desc_, weight,
            bwd_top_descs_[i], top_diff,
            bwd_conv_data_descs_[i],
            bwd_data_algo_[i], ws->data(), ws->size(),
            cudnn::dataType<Btype>::zero, bwd_bottom_descs_[i], bottom_diff));
        CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));
      }  // end if propagate down
    }  // end for i
  } else {
    // "old" path
    const size_t gsize = ws->size() / ws_groups();
    // compute dE/dB = sum_c(dE/dy)
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Btype* bias_diff = this->blobs_[1]->template mutable_gpu_diff<Btype>();
      for (int i = 0; i < top.size(); ++i) {
        Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
        // in parallel over groups
        for (int g = 0; g < groups(); ++g) {
          CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(idxg(g)),
              cudnn::dataType<Btype>::one, bwd_top_descs_[i], top_diff + top_offset_ * g,
              cudnn::dataType<Btype>::one, bwd_bias_desc_, bias_diff + bias_offset_ * g));
        }  // end of groups
        // Synchronize the work across groups, each of which went into its own stream
        // NOLINT_NEXT_LINE(whitespace/operators)
        for (int g = 0; g < ws_groups(); ++g) {
          CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(g)));
        }
      }  // end of i
    }  // end of dB

    // compute dE/dW = dY * X
    if (this->param_propagate_down_[0]) {
      Btype* weight_diff = this->blobs_[0]->template mutable_gpu_diff<Btype>();
      for (int i = 0; i < top.size(); ++i) {
        Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
        const Btype* bottom_data = bottom[i]->gpu_data<Btype>();
        // Backward through cuDNN in parallel over groups and gradients.
        for (int g = 0; g < groups(); ++g) {
          unsigned char* pspace = static_cast<unsigned char*>(ws->data()) + gsize * idxg(g);
          // Gradient w.r.t. weights.
          CUDNN_CHECK(cudnnConvolutionBackwardFilter(Caffe::cudnn_handle(idxg(g)),
              cudnn::dataType<Btype>::one,
              bwd_bottom_descs_[i], bottom_data + bottom_offset_ * g,
              bwd_top_descs_[i], top_diff + top_offset_ * g,
              bwd_conv_filter_descs_[i], bwd_filter_algo_[i], pspace, gsize,
              cudnn::dataType<Btype>::one,
              bwd_filter_desc_, weight_diff + this->weight_offset_ * g));
        }  // end of groups
        // Synchronize the work across groups, each of which went into its own stream
        // NOLINT_NEXT_LINE(whitespace/operators)
        for (int g = 0; g < ws_groups(); ++g) {
          CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(g)));
        }
      }  // end of i
    }

    // Backward propagate grad wrt bottom data dE/dX= dE/dY * W
    const Btype* weight = this->blobs_[0]->template gpu_data<Btype>();
    for (int i = 0; i < top.size(); ++i) {
      if (propagate_down[i]) {
        // Backward in parallel over groups
        for (int g = 0; g < groups(); ++g) {
          Btype* top_diff = top[i]->mutable_gpu_diff<Btype>();
          Btype* bottom_diff = bottom[i]->mutable_gpu_diff<Btype>();
          unsigned char* pspace = static_cast<unsigned char*>(ws->data()) + gsize * idxg(g);
          CUDNN_CHECK(cudnnConvolutionBackwardData(Caffe::cudnn_handle(idxg(g)),
              cudnn::dataType<Btype>::one,
              bwd_filter_desc_, weight + this->weight_offset_ * g,
              bwd_top_descs_[i], top_diff + top_offset_ * g,
              bwd_conv_data_descs_[i],
              bwd_data_algo_[i], pspace, gsize,
              cudnn::dataType<Btype>::zero,
              bwd_bottom_descs_[i], bottom_diff + bottom_offset_ * g));
        }
        // Synchronize the work across groups.
        // NOLINT_NEXT_LINE(whitespace/operators)
        for (int g = 0; g < ws_groups(); ++g) {
          CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(g)));
        }
      }  // end if propagate down
    }  // end for i
  }

  ++bwd_count_;
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
