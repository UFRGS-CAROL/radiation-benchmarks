#include "hip/hip_runtime.h"
#include <vector>
#include <>

#include "caffe/layers/dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype>
__global__ void
DropoutForward(const int n, const Dtype* in, const unsigned int* mask, const unsigned int threshold,
    const float scale, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
      out[index] = in[index] * (mask[index] > threshold ? Dtype(1) : Dtype(0)) * scale;
  }
}

template<typename Ftype, typename Btype>
void
DropoutLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom, const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    hipStream_t stream = Caffe::thread_stream();
    unsigned int* mask = static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>
        (count, bottom_data, mask, uint_thres_, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
    CUDA_CHECK(hipStreamSynchronize(stream));
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template<typename Dtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff, const unsigned int* mask,
    const unsigned int threshold, const float scale, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (mask[index] > threshold ? Dtype(1) : Dtype(0)) * scale;
  }
}

template<typename Ftype, typename Btype>
void DropoutLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  if (propagate_down[0]) {
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
    if (this->phase_ == TRAIN) {  // Needed for TEST
      hipStream_t stream = Caffe::thread_stream();
      const unsigned int* mask = rand_vec_.gpu_data();
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>
          (count, top_diff, mask, uint_thres_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
      CUDA_CHECK(hipStreamSynchronize(stream));
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(DropoutLayer);

}  // namespace caffe
