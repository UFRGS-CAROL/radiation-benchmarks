#include "hip/hip_runtime.h"
#include <vector>
#include <>

#include "caffe/layers/concat_layer.hpp"
#include "caffe/util/half.cuh"

namespace caffe {

template <typename Dtype>
__global__ void Concat(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_concat_size = concat_size * bottom_concat_axis;
    const int concat_num = index / total_concat_size;
    const int concat_index = index % total_concat_size;
    const int top_index = concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
    }
  }
}

template <typename Ftype, typename Btype>
void ConcatLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
      const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;

  if (bottom.size() == 1) {
    return;
  }
  for (int i = 0; i < bottom.size(); ++i) {
    bottom_data = bottom[i]->gpu_data<Ftype>();
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int nthreads = bottom_concat_size * num_concats_;
    if (tp<Ftype>() == FLOAT16) {
      Concat<half>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<< CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream() >>> (
          nthreads, reinterpret_cast<const half*>(bottom_data), kForward, num_concats_,
          concat_input_size_, top_concat_axis, bottom_concat_axis, offset_concat_axis,
          reinterpret_cast<half*>(top_data));
    } else {
      Concat<Ftype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<< CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream() >>> (
          nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
              top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
    }
    CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
    offset_concat_axis += bottom_concat_axis;
  }
}

template <typename Ftype, typename Btype>
void ConcatLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
      const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  const Btype* top_diff = top[0]->gpu_diff<Btype>();
  Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
  if (bottom.size() == 1) {
    return;
  }
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int i = 0; i < bottom.size(); ++i) {
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      bottom_diff = bottom[i]->mutable_gpu_diff<Btype>();
      const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int nthreads = bottom_concat_size * num_concats_;
      Concat<Btype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff);
      CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(ConcatLayer);

}  // namespace caffe
