#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/prelu_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Ftype>
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const Ftype* in, Ftype* out, const Ftype* slope_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Btype>
__global__ void PReLUBackward(const int n, const int channels, const int dim,
    const Btype* in_diff, const Btype* in_data, Btype* out_diff,
    const Btype* slope_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * slope_data[c]);
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Btype>
__global__ void PReLUParamBackward(const int n,
    const int rows, const int rowPitch, const Btype* in_diff,
    const Btype* in_data, Btype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= 0);
    for ( int k = 1; k < rows; k++ ) {
        out_diff[index] += in_diff[index + k*rowPitch]
           * in_data[index + k*rowPitch] * (in_data[index + k*rowPitch] <= 0);
    }
  }
}

template <typename Ftype, typename Btype>
void PReLULayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Ftype* slope_data = this->blobs_[0]->template gpu_data<Ftype>();
  const int div_factor = channel_shared_ ? channels : 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy<Ftype>(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  PReLUForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
      count, channels, dim, bottom_data, top_data, slope_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Ftype, typename Btype>
void PReLULayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  const Btype* bottom_data = bottom[0]->gpu_data<Btype>();
  const Btype* top_diff = top[0]->gpu_diff<Btype>();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.template gpu_data<Btype>();
  }

  // Propagate to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.
  if (this->param_propagate_down_[0]) {
    Btype* slope_diff = this->blobs_[0]->template mutable_gpu_diff<Btype>();
    int cdim = channels * dim;

    // compute element-wise diff
    // NOLINT_NEXT_LINE(whitespace/operators)
    PReLUParamBackward<Btype><<<CAFFE_GET_BLOCKS(cdim),
      CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
      cdim, bottom[0]->num(), top[0]->offset(1), top_diff ,
      bottom_data ,
      backward_buff_.mutable_gpu_diff());
    CUDA_POST_KERNEL_CHECK;
    if (channel_shared_) {
      Btype dsum;
      caffe_gpu_dot(channels * dim, backward_buff_.gpu_diff(),
       multiplier_.gpu_data(), &dsum);
      caffe_gpu_add_scalar(this->blobs_[0]->count(), dsum, slope_diff);
    } else {
      caffe_gpu_gemv(CblasNoTrans, channels, dim, Btype(1.),
        backward_buff_.gpu_diff(), multiplier_.gpu_data(), Btype(1.),
        slope_diff);
    }
  }
  // Propagate to bottom
  if (propagate_down[0]) {
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
    const Btype* slope_data = this->blobs_[0]->template gpu_data<Btype>();
    int div_factor = channel_shared_ ? channels : 1;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PReLUBackward<Btype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data,
        div_factor);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(PReLULayer);

}  // namespace caffe
