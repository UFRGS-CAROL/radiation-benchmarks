#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/sigmoid_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-in[index]));
  }
}

template <typename Ftype, typename Btype>
void SigmoidLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (Dtype(1.) - sigmoid_x);
  }
}

template <typename Ftype, typename Btype>
void SigmoidLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (propagate_down[0]) {
    const Btype* top_data = top[0]->gpu_data<Btype>();
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
        count, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(SigmoidLayer);

}  // namespace caffe
