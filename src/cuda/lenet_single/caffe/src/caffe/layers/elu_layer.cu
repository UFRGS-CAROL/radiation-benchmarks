#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/elu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ELUForward(const int n, const Dtype* in, Dtype* out,
    float alpha, float lambda) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = lambda * in[index] > 0 ? in[index] :
        Dtype(alpha * (exp(in[index]) - 1.));
  }
}

template <typename Ftype, typename Btype>
void ELULayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const int count = bottom[0]->count();
  float alpha = this->layer_param_.elu_param().alpha();
  float lambda = this->layer_param_.elu_param().lambda();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ELUForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
      count, bottom_data, top_data, alpha, lambda);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ELUBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* in_data, Dtype* out_diff,
    float alpha, float lambda) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = lambda * in_data[index] > 0 ? in_diff[index] :
        Dtype(in_diff[index] * (out_data[index] + alpha));
  }
}

template <typename Ftype, typename Btype>
void ELULayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (propagate_down[0]) {
    const Btype* bottom_data = bottom[0]->gpu_data<Btype>();
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    const Btype* top_data = top[0]->gpu_data<Btype>();
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
    const int count = bottom[0]->count();
    float alpha = this->layer_param_.elu_param().alpha();
    float lambda = this->layer_param_.elu_param().lambda();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ELUBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, Caffe::thread_stream()>>>(
        count, top_diff, top_data, bottom_data, bottom_diff, alpha, lambda);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS_FB(ELULayer);


}  // namespace caffe
