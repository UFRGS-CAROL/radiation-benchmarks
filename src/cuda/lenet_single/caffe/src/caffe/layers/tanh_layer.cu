#include "hip/hip_runtime.h"
// TanH neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <vector>

#include "caffe/layers/tanh_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TanHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = tanh(in[index]);
  }
}

template <typename Ftype, typename Btype>
void TanHLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const int count = bottom[0]->count();
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TanHForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template <typename Dtype>
__global__ void TanHBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    float tanhx = out_data[index];
    out_diff[index] = in_diff[index] * (Dtype(1.) - tanhx * tanhx);
  }
}

template <typename Ftype, typename Btype>
void TanHLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (propagate_down[0]) {
    const Btype* top_data = top[0]->gpu_data<Btype>();
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
    const int count = bottom[0]->count();
    hipStream_t stream = Caffe::thread_stream();
    // NOLINT_NEXT_LINE(whitespace/operators)
    TanHBackward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
        count, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(TanHLayer);

}  // namespace caffe
