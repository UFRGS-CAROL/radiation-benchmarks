#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <>

#include "caffe/layers/permute_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PermuteKernel(const int nthreads,
    Dtype* bottom_data, const bool forward, const int* permute_order,
    const int* old_steps, const int* new_steps, const int num_axes,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp_idx = index;
    int old_idx = 0;
    for (int i = 0; i < num_axes; ++i) {
      int order = permute_order[i];
      old_idx += (temp_idx / new_steps[i]) * old_steps[order];
      temp_idx %= new_steps[i];
    }
    if (forward) {
      top_data[index] = bottom_data[old_idx];
    } else {
      bottom_data[old_idx] = top_data[index];
    }
  }
}

template <>
__global__ void PermuteKernel<float16>(const int nthreads, float16* bottom_data,
                                       const bool forward, const int* permute_order,
                                       const int* old_steps, const int* new_steps,
                                       const int num_axes, float16* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp_idx = index;
    int old_idx = 0;
    for (int i = 0; i < num_axes; ++i) {
      int order = permute_order[i];
      old_idx += (temp_idx / new_steps[i]) * old_steps[order];
      temp_idx %= new_steps[i];
    }
    if (forward) {
      top_data[index] = bottom_data[old_idx];
    } else {
      bottom_data[old_idx] = top_data[index];
    }
  }
}


template <typename Ftype, typename Btype>
void PermuteLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
      const vector<Blob*>& top) {
  if (need_permute_) {
    Dtype* bottom_data = const_cast<Dtype*>(bottom[0]->gpu_data<Dtype>());
    Dtype* top_data = top[0]->mutable_gpu_data<Dtype>();
    int count = top[0]->count();
    const int* permute_order = permute_order_.gpu_data();
    const int* new_steps = new_steps_.gpu_data();
    const int* old_steps = old_steps_.gpu_data();
    bool foward = true;
    hipStream_t stream = Caffe::thread_stream();
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteKernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
        count, bottom_data, foward, permute_order, old_steps, new_steps,
        num_axes_, top_data);
    CUDA_POST_KERNEL_CHECK;
    CUDA_CHECK(hipStreamSynchronize(stream));
  } else {
    // If there is no need to permute, we share data to save memory.
    top[0]->ShareData(*bottom[0]);
  }
}

template <typename Ftype, typename Btype>
void PermuteLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
      const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  if (need_permute_) {
    Dtype* top_diff = top[0]->mutable_gpu_diff<Dtype>();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff<Dtype>();
    const int count = bottom[0]->count();
    const int* permute_order = permute_order_.gpu_data();
    const int* new_steps = new_steps_.gpu_data();
    const int* old_steps = old_steps_.gpu_data();
    bool foward = false;
    hipStream_t stream = Caffe::thread_stream();
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
        count, bottom_diff, foward, permute_order, old_steps, new_steps,
        num_axes_, top_diff);
    CUDA_POST_KERNEL_CHECK;
    CUDA_CHECK(hipStreamSynchronize(stream));
  } else {
    // If there is no need to permute, we share diff to save memory.
    bottom[0]->ShareDiff(*top[0]);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(PermuteLayer);

}  // namespace caffe
