#include "hip/hip_runtime.h"
#include <algorithm>
#include <>

#include "caffe/layers/softmax_layer.hpp"

namespace caffe {

// TODO specialize these kernels for half
template <typename Dtype>
__global__ void kernel_channel_max(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype maxval = -max_dtype<Dtype>();
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_max, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] -= channel_max[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0.F;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_sum, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const int num, const int channels,
    const int spatial_dim, const Dtype* data_1, const Dtype* data_2,
    Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype dot = 0.F;
    for (int c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}

template <typename Ftype, typename Btype>
void SoftmaxLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  Ftype* scale_data = scale_.template mutable_gpu_data<Ftype>();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);
  hipStream_t stream = Caffe::thread_stream();
  // We need to subtract the max to avoid numerical issues, compute the exp,
  // and then normalize.
  // compute max
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      count, top_data, top_data);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template <typename Ftype, typename Btype>
void SoftmaxLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  const Btype* top_diff = top[0]->gpu_diff<Btype>();
  const Btype* top_data = top[0]->gpu_data<Btype>();
  Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();
  Btype* scale_data = scale_.template mutable_gpu_data<Btype>();
  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  hipStream_t stream = Caffe::thread_stream();
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_dot<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(outer_num_, channels, inner_num_,
      top_diff, top_data, scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS, 0, stream>>>(count, outer_num_, channels, inner_num_,
      scale_data, bottom_diff);
  CUDA_CHECK(hipStreamSynchronize(stream));
  // elementwise multiplication
  caffe_gpu_mul(top[0]->count(), bottom_diff, top_data, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(SoftmaxLayer);


}  // namespace caffe
