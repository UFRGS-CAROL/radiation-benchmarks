#include "hip/hip_runtime.h"
#include <vector>
#include <>

#include "caffe/filler.hpp"
#include "caffe/layers/embed_layer.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void EmbedForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int N, Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = abs(index * N + d);
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int N, Dtype* weight_diff) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(static_cast<double>(bottom_data[n]));
    const int weight_index = index * N + d;
    caffe_gpu_atomic_add(top_diff[top_index], weight_diff + weight_index);
  }
}

template <typename Ftype, typename Btype>
void EmbedLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  const Ftype* weight = this->blobs_[0]->template gpu_data<Ftype>();
  const int count = top[0]->count();
  hipStream_t stream = Caffe::thread_stream();
  EmbedForward  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      count, bottom_data, weight, N_, top_data);
  CUDA_CHECK(hipStreamSynchronize(stream));
  if (bias_term_) {
    caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, M_, N_, 1, Ftype(1),
        bias_multiplier_.template gpu_data<Ftype>(),
        this->blobs_[1]->template gpu_data<Ftype>(), Ftype(1), top_data);
  }
}

template <typename Ftype, typename Btype>
void EmbedLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  CHECK(!propagate_down[0]) << "Can't backpropagate to EmbedLayer input.";
  if (this->param_propagate_down_[0]) {
    const int top_count = top[0]->count();
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    const Btype* bottom_data = bottom[0]->gpu_data<Btype>();
    Btype* weight_diff = this->blobs_[0]->template mutable_gpu_diff<Btype>();
    hipStream_t stream = Caffe::thread_stream();
    EmbedBackward  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
        top_count, bottom_data, top_diff, N_, weight_diff);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Btype* top_diff = top[0]->gpu_diff<Btype>();
    Btype* bias_diff = this->blobs_[1]->template mutable_gpu_diff<Btype>();
    caffe_gpu_gemv(CblasTrans, M_, N_, Btype(1), top_diff,
        bias_multiplier_.template gpu_data<Btype>(), Btype(1), bias_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(EmbedLayer);

}  // namespace caffe
