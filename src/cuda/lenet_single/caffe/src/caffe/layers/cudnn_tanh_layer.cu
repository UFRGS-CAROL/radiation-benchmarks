
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_tanh_layer.hpp"

namespace caffe {

template <typename Ftype, typename Btype>
void CuDNNTanHLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data<Ftype>();
  Ftype* top_data = top[0]->mutable_gpu_data<Ftype>();
  CUDNN_CHECK(cudnnActivationForward(Caffe::cudnn_handle(0),
        activ_desc_,
        cudnn::dataType<Ftype>::one,
        fwd_bottom_desc_, bottom_data,
        cudnn::dataType<Ftype>::zero,
        fwd_top_desc_, top_data));
  CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream(0)));
}

template <typename Ftype, typename Btype>
void CuDNNTanHLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Btype* top_data = top[0]->gpu_data<Btype>();
  const Btype* top_diff = top[0]->gpu_diff<Btype>();
  const Btype* bottom_data = bottom[0]->gpu_data<Btype>();
  Btype* bottom_diff = bottom[0]->mutable_gpu_diff<Btype>();

  CUDNN_CHECK(cudnnActivationBackward(Caffe::cudnn_handle(0),
        activ_desc_,
        cudnn::dataType<Btype>::one,
        bwd_top_desc_, top_data, bwd_top_desc_, top_diff,
        bwd_bottom_desc_, bottom_data,
        cudnn::dataType<Btype>::zero,
        bwd_bottom_desc_, bottom_diff));
  CUDA_CHECK(cudaStreamSynchronize(Caffe::thread_stream()));
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(CuDNNTanHLayer);

}  // namespace caffe
#endif
