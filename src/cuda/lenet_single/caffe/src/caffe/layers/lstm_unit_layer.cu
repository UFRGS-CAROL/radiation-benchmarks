#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <vector>
#include <>

#include "caffe/layer.hpp"
#include "caffe/layers/lstm_layer.hpp"

namespace caffe {

template <typename Dtype>
__device__ Dtype sigmoid(const Dtype x) {
  return Dtype(1) / (Dtype(1) + exp(-x));
}

template <typename Dtype>
__device__ Dtype tanh(const Dtype x) {
  return Dtype(2) * sigmoid(Dtype(2) * x) - Dtype(1);
}

template <typename Dtype>
__global__ void LSTMActsForward(const int nthreads, const int dim,
                                const Dtype* X, Dtype* X_acts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int x_dim = 4 * dim;
    const int d = index % x_dim;
    if (d < 3 * dim) {
      X_acts[index] = sigmoid(X[index]);
    } else {
      X_acts[index] = tanh(X[index]);
    }
  }
}

template <typename Dtype>
__global__ void LSTMUnitForward(const int nthreads, const int dim,
    const Dtype* C_prev, const Dtype* X, const Dtype* cont,
    Dtype* C, Dtype* H) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / dim;
    const int d = index % dim;
    const Dtype* X_offset = X + 4 * dim * n;
    const Dtype i = X_offset[d];
    const Dtype f = X_offset[1 * dim + d];
    const Dtype o = X_offset[2 * dim + d];
    const Dtype g = X_offset[3 * dim + d];
    const Dtype c_prev = C_prev[index];
    const Dtype c = cont[n] * f * c_prev + i * g;
    C[index] = c;
    const Dtype tanh_c = tanh(c);
    H[index] = o * tanh_c;
  }
}

template<typename Ftype, typename Btype>
void LSTMUnitLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
                                              const vector<Blob*>& top) {
  const int count = top[1]->count();
  const Ftype* C_prev = bottom[0]->gpu_data<Ftype>();
  const Ftype* X = bottom[1]->gpu_data<Ftype>();
  const Ftype* cont = bottom[2]->gpu_data<Ftype>();
  Ftype* X_acts = X_acts_->mutable_gpu_data<Ftype>();
  Ftype* C = top[0]->mutable_gpu_data<Ftype>();
  Ftype* H = top[1]->mutable_gpu_data<Ftype>();
  const int X_count = bottom[1]->count();
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  LSTMActsForward<<<CAFFE_GET_BLOCKS(X_count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      X_count, hidden_dim_, X, X_acts);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
  // NOLINT_NEXT_LINE(whitespace/operators)
  LSTMUnitForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      count, hidden_dim_, C_prev, X_acts, cont, C, H);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template <typename Dtype>
__global__ void LSTMUnitBackward(const int nthreads, const int dim,
    const Dtype* C_prev, const Dtype* X, const Dtype* C, const Dtype* H,
    const Dtype* cont, const Dtype* C_diff, const Dtype* H_diff,
    Dtype* C_prev_diff, Dtype* X_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / dim;
    const int d = index % dim;
    const Dtype* X_offset = X + 4 * dim * n;
    const Dtype i = X_offset[d];
    const Dtype f = X_offset[1 * dim + d];
    const Dtype o = X_offset[2 * dim + d];
    const Dtype g = X_offset[3 * dim + d];
    const Dtype c_prev = C_prev[index];
    const Dtype c = C[index];
    const Dtype tanh_c = tanh(c);
    Dtype* c_prev_diff = C_prev_diff + index;
    Dtype* X_diff_offset = X_diff + 4 * dim * n;
    Dtype* i_diff = X_diff_offset + d;
    Dtype* f_diff = X_diff_offset + 1 * dim + d;
    Dtype* o_diff = X_diff_offset + 2 * dim + d;
    Dtype* g_diff = X_diff_offset + 3 * dim + d;
    const Dtype c_term_diff =
        C_diff[index] + H_diff[index] * o * (1 - tanh_c * tanh_c);
    const Dtype cont_n = cont[n];
    *c_prev_diff = cont_n * c_term_diff * f;
    *i_diff = c_term_diff * g;
    *f_diff = cont_n * c_term_diff * c_prev;
    *o_diff = H_diff[index] * tanh_c;
    *g_diff = c_term_diff * i;
  }
}

template <typename Dtype>
__global__ void LSTMActsBackward(const int nthreads, const int dim,
    const Dtype* X_acts, const Dtype* X_acts_diff, Dtype* X_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int x_dim = 4 * dim;
    const int d = index % x_dim;
    const Dtype X_act = X_acts[index];
    if (d < 3 * dim) {
      X_diff[index] = X_acts_diff[index] * X_act * (Dtype(1) - X_act);
    } else {
      X_diff[index] = X_acts_diff[index] * (Dtype(1) - X_act * X_act);
    }
  }
}

template<typename Ftype, typename Btype>
void LSTMUnitLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  CHECK(!propagate_down[2]) << "Cannot backpropagate to sequence indicators.";
  if (!propagate_down[0] && !propagate_down[1]) { return; }

  const int count = top[1]->count();
  const Btype* C_prev = bottom[0]->gpu_data<Btype>();
  const Btype* X_acts = X_acts_->gpu_data<Btype>();
  const Btype* cont = bottom[2]->gpu_data<Btype>();
  const Btype* C = top[0]->gpu_data<Btype>();
  const Btype* H = top[1]->gpu_data<Btype>();
  const Btype* C_diff = top[0]->gpu_diff<Btype>();
  const Btype* H_diff = top[1]->gpu_diff<Btype>();
  Btype* C_prev_diff = bottom[0]->mutable_gpu_diff<Btype>();
  Btype* X_acts_diff = X_acts_->mutable_gpu_diff<Btype>();

  hipStream_t stream = Caffe::thread_stream();
  LSTMUnitBackward  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(count, hidden_dim_,
      C_prev, X_acts, C, H, cont, C_diff, H_diff, C_prev_diff, X_acts_diff);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
  const int X_count = bottom[1]->count();
  Btype* X_diff = bottom[1]->mutable_gpu_diff<Btype>();
  LSTMActsBackward  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(X_count), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(
      X_count, hidden_dim_, X_acts, X_acts_diff, X_diff);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(LSTMUnitLayer);

}  // namespace caffe
