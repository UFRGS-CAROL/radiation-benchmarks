#include "hip/hip_runtime.h"
#include <string>
#include <>

#include "caffe/util/gpu_math_functions.cuh"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Gtype, typename Wtype, typename Htype>
__global__ void SGDRegUpdateAllAndClear(int N,
  Gtype* g, Wtype* w, Htype* h,
    float momentum, float local_rate, float local_decay, bool reg_L2,  bool clear_grads) {
  CUDA_KERNEL_LOOP(i, N) {
    Wtype reg = reg_L2 ? w[i] : Wtype((Wtype(0) < w[i]) - (w[i] < Wtype(0)));
    Wtype gr = Wtype(g[i]) + reg * local_decay;
    gr = h[i] = momentum * h[i] + local_rate * gr;
    w[i] -= gr;
    g[i] = clear_grads ? Gtype(0) : Gtype(gr);
  }
}

template<>
__global__ void SGDRegUpdateAllAndClear<half, half, half>(int N,
  half* g, half* w, half* h,
    float momentum, float local_rate, float local_decay, bool reg_L2,  bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float wf = __half2float(w[i]);
    float gf = __half2float(g[i]);
    float hf = __half2float(h[i]);

    float reg = reg_L2 ? wf : float((0.F < wf)-(wf < 0.F));
    gf += reg * local_decay;
    gf = hf = momentum * hf  + local_rate * gf;
    wf -= gf;

    h[i] = float2half_clip(hf);
    w[i] = float2half_clip(wf);
    g[i] = clear_grads ? hz : float2half_clip(gf);
  }
}

template<>
__global__ void SGDRegUpdateAllAndClear<float, float, half>(int N,
    float* g, float* w, half* h,
    float momentum, float local_rate, float local_decay, bool reg_L2,  bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float wf = w[i];
    float gf = g[i];
    float hf = __half2float(h[i]);

    float reg = reg_L2 ? wf : float((0.F < wf)-(wf < 0.F));
    gf += reg * local_decay;
    gf = hf = momentum * hf  + local_rate * gf;
    wf -= gf;

    h[i] = float2half_clip(hf);
    w[i] = wf;
    g[i] = clear_grads ? 0.F : gf;
  }
}

template<>
__global__ void SGDRegUpdateAllAndClear<half, float, float>(int N,
    half* g, float* w, float* h,
    float momentum, float local_rate, float local_decay, bool reg_L2, bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float reg = reg_L2 ? w[i] : (0.F < w[i]) - (w[i] < 0.F);
    float gr = __half2float(g[i]) + reg * local_decay;
    gr = h[i] = momentum * h[i] + local_rate * gr;
    w[i] -= gr;
    g[i] = clear_grads ? hz : float2half_clip(h[i]);
  }
}

template<typename Gtype, typename Wtype, typename Htype>
void sgd_reg_update_all_and_clear_gpu(int N,
  Gtype* g, Wtype* w, Htype* h,
  float momentum, float local_rate, const std::string& reg_type, float local_decay,
  void* handle,  bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));

  bool reg_L2 = (reg_type == "L2") || (reg_type == "L2_unitary");

  // NOLINT_NEXT_LINE(whitespace/operators)
  SGDRegUpdateAllAndClear<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>> (N,
    g, w, h,
    momentum, local_rate, local_decay, reg_L2,  clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template void sgd_reg_update_all_and_clear_gpu<float16, double, double>(
    int, float16*, double*, double*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<float, float, float>(
    int, float*, float*, float*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<float, double, double>(
    int, float*, double*, double*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<float, float16, float16>(
    int, float*, float16*, float16*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<double, float, float>(
    int, double*, float*, float*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<double, double, double>(
    int, double*, double*, double*,
  float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<double, float16, float16>(
    int, double*, float16*, float16*,
  float, float, const std::string&, float,  void*, bool);

template void sgd_reg_update_all_and_clear_gpu<float, float, float16>(
    int, float*, float*, float16*,
    float, float, const std::string&, float,  void*, bool);
template void sgd_reg_update_all_and_clear_gpu<float, float, double>(
    int, float*, float*, double*,
    float, float, const std::string&, float,  void*, bool);

template<>
void
sgd_reg_update_all_and_clear_gpu<float16, float16>(int N,
  float16* g, float16* w, float16* h,
  float momentum, float local_rate, const std::string& reg_type, float local_decay,
  void* handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  // NOLINT_NEXT_LINE(whitespace/operators)
  SGDRegUpdateAllAndClear<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>> (N,
      reinterpret_cast<half*>(g), reinterpret_cast<half*>(w), reinterpret_cast<half*>(h),
      momentum, local_rate, local_decay, reg_type == "L2",  clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void
sgd_reg_update_all_and_clear_gpu<float16, float>(int N,
    float16* g, float* w, float* h,
    float momentum,  float local_rate, const std::string& reg_type, float local_decay,
    void* handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  // NOLINT_NEXT_LINE(whitespace/operators)
  SGDRegUpdateAllAndClear<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>
      (N, reinterpret_cast<half*>(g), w, h, momentum, local_rate,
          local_decay, reg_type == "L2", clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

}  // namespace caffe
