#include "hip/hip_runtime.h"
#include <string>
#include <>

#include "caffe/util/gpu_math_functions.cuh"
#include "caffe/util/math_functions.hpp"

namespace caffe {

#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
template<typename Gtype, typename Wtype>
__global__ void RMSPropRegUpdateAllAndClear(int N,
    Gtype* g, Wtype *w, Wtype* h,
    float rms_decay, float delta, float local_rate, float local_decay, bool reg_L2,
    bool clear_grads) {
  CUDA_KERNEL_LOOP(i, N) {
    Wtype reg = reg_L2 ? w[i] : Wtype((Wtype(0) < w[i]) - (w[i] < Wtype(0)));
    Wtype gr = Wtype(g[i]) + reg * local_decay;
    Wtype hi = h[i] = rms_decay * h[i] + (1.F - rms_decay) * gr*gr;
    gr = local_rate * gr / (sqrt(hi) + delta);
    w[i] -= gr;
    g[i] = clear_grads ? Gtype(0) : Gtype(gr);
  }
}
#pragma clang diagnostic pop

template<>
__global__ void RMSPropRegUpdateAllAndClear<half, half>(int N,
    half* g, half* w, half* h,
    float rms_decay, float delta, float local_rate, float local_decay, bool reg_L2,
    bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float wf = __half2float(w[i]);
    float gf = __half2float(g[i]);
    float hf = __half2float(h[i]);

    float reg = reg_L2 ? wf : float((0.F < wf)-(wf < 0.F));
    gf += reg * local_decay;
    hf = rms_decay * hf + (1.F - rms_decay) * gf*gf;
    gf = local_rate * gf / (sqrt(hf) + delta);
    wf -= gf;

    h[i] = float2half_clip(hf);
    w[i] = float2half_clip(wf);
    g[i] = clear_grads ? hz : float2half_clip(gf);
  }
}


template<typename Gtype, typename Wtype>
void rmsprop_reg_update_and_clear_gpu(int N,
  Gtype* g, Wtype* w, Wtype* h,
  float rms_decay, float delta, float local_rate, const std::string& reg_type,
  float local_decay, void* handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  RMSPropRegUpdateAllAndClear  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N,
      g, w, h,
      rms_decay, delta, local_rate, local_decay, reg_type == "L2", clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void rmsprop_reg_update_and_clear_gpu<float16, float16>(int N,
  float16* g, float16* w, float16* h,
  float rms_decay, float delta, float local_rate, const std::string& reg_type,
  float local_decay, void* handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  RMSPropRegUpdateAllAndClear  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N,
        reinterpret_cast<half*>(g), reinterpret_cast<half*>(w), reinterpret_cast<half*>(h),
        rms_decay, delta, local_rate, local_decay, reg_type == "L2", clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template void rmsprop_reg_update_and_clear_gpu<float16, float>(int, float16*, float*, float*,
    float, float, float, const std::string&, float, void*, bool);
template void rmsprop_reg_update_and_clear_gpu<float16, double>(int, float16*, double*, double*,
    float, float, float, const std::string&, float, void*, bool);

template void rmsprop_reg_update_and_clear_gpu<float, float>(int, float*, float*, float*,
    float, float, float, const std::string&, float, void*, bool);
template void rmsprop_reg_update_and_clear_gpu<float, double>(int, float*, double*, double*,
    float, float, float, const std::string&, float, void*, bool);
template void rmsprop_reg_update_and_clear_gpu<float, float16>(int, float*, float16*, float16*,
    float, float, float, const std::string&, float, void*, bool);

template void rmsprop_reg_update_and_clear_gpu<double, float>(int, double*, float*, float*,
    float, float, float, const std::string&, float, void*, bool);
template void rmsprop_reg_update_and_clear_gpu<double, double>(int, double*, double*, double*,
    float, float, float, const std::string&, float, void*, bool);
template void rmsprop_reg_update_and_clear_gpu<double, float16>(int, double*, float16*, float16*,
    float, float, float, const std::string&, float, void*, bool);

}  // namespace caffe
