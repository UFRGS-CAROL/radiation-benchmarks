#include "hip/hip_runtime.h"
#include <string>

#include "caffe/util/gpu_math_functions.cuh"
#include "caffe/util/math_functions.hpp"

#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
namespace caffe {

template<typename Gtype, typename Wtype>
__global__ void AdamRegUpdateAllAndClear(int N,
  Gtype* g, Wtype *w, Wtype* m, Wtype* v,
    float beta1, float beta2, float eps_hat, float local_rate,  float local_decay,
    bool reg_L2,  bool clear_grads) {
  CUDA_KERNEL_LOOP(i, N) {
    Wtype reg = reg_L2 ? w[i] : Wtype((Wtype(0) < w[i]) - (w[i] < Wtype(0)));
    Wtype gr = Wtype(g[i]) + reg * local_decay;
    Wtype mi = m[i] = m[i] * beta1 + gr * (Wtype(1.) - beta1);
    Wtype vi = v[i] = v[i] * beta2 + gr * gr * (Wtype(1.) - beta2);
    gr = local_rate * mi / (sqrt(vi) + eps_hat);
    w[i] -= gr;
    g[i] = clear_grads ? Gtype(0) : Gtype(gr);
  }
}
#pragma clang diagnostic pop

template<>
__global__ void AdamRegUpdateAllAndClear<half, half>(int N,
  half* g, half *w, half* m, half* v,
    float beta1, float beta2, float eps_hat, float local_rate, float local_decay,
    bool reg_L2,  bool clear_grads) {
  half hz;
  CUDA_KERNEL_LOOP(i, N) {
    float wf = __half2float(w[i]);
    float gf = __half2float(g[i]);
    float mf = __half2float(m[i]);
    float vf = __half2float(v[i]);

    float reg = reg_L2 ? wf : float((0.F < wf)-(wf < 0.F));
    gf += reg * local_decay;
    mf = beta1 * mf + (1.F - beta1)*gf;
    vf = beta2 * vf + (1.F - beta2)*gf*gf;
    gf = local_rate * mf / sqrt(vf + eps_hat);
    wf -= gf;

    w[i] = float2half_clip(wf);
    m[i] = float2half_clip(mf);
    v[i] = float2half_clip(vf);
    g[i] = clear_grads ? hz : float2half_clip(gf);
  }
}

template<typename Gtype, typename Wtype>
void adam_reg_update_and_clear_gpu(int N,
  Gtype* g,  Wtype *w, Wtype* m, Wtype* v,
  float beta1,  float beta2, float eps_hat, float local_rate,
    const std::string& reg_type, float local_decay, void *handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  AdamRegUpdateAllAndClear  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N,
      g, w, m, v,
      beta1, beta2, eps_hat, local_rate, local_decay, reg_type == "L2",  clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void adam_reg_update_and_clear_gpu<float16, float16>(int N,
    float16 *g, float16 *w, float16 *m, float16 *v,
  float beta1,  float beta2, float eps_hat, float local_rate,
  const std::string& reg_type, float local_decay, void *handle, bool clear_grads) {
  hipblasHandle_t cublas_handle =
        handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  AdamRegUpdateAllAndClear  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N,
      reinterpret_cast<half*>(g), reinterpret_cast<half*>(w),
      reinterpret_cast<half*>(m), reinterpret_cast<half*>(v),
      beta1, beta2, eps_hat, local_rate, local_decay, reg_type == "L2",  clear_grads);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}


template void adam_reg_update_and_clear_gpu<float16, float>(int, float16*, float*, float*,
    float*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<float16, double>(int, float16*, double*, double*,
    double*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<float, float>(int, float*, float*, float*,
    float*, float, float, float, float, const std::string&, float,  void*, bool);
template void adam_reg_update_and_clear_gpu<float, double>(int, float*, double*, double*,
    double*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<float, float16>(int, float*, float16*, float16*,
    float16*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<double, float>(int, double*, float*, float*,
    float*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<double, double>(int, double*, double*, double*,
    double*, float, float, float, float, const std::string&, float, void*, bool);
template void adam_reg_update_and_clear_gpu<double, float16>(int, double*, float16*, float16*,
    float16*, float, float, float, float, const std::string&, float, void*, bool);
}  // namespace caffe
