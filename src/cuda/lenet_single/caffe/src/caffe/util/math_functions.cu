#include "hip/hip_runtime.h"
#include <algorithm>
#include <>

#include "caffe/util/half.cuh"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_math_functions.cuh"
#include "caffe/type.hpp"

namespace caffe {

template<>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(0), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(0), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_gemm<float16>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float16 alpha, const float16* A, const float16* B, const float16 beta,
    float16* C) {
  hipblasHandle_t handle = Caffe::cublas_handle(0);
  // Note that cublas follows fortran order.
  const int lda = (TransA == CblasNoTrans) ? K : M;
  const int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  if (Caffe::device_capability(Caffe::current_device()) >= 503) {
#if CUDA_VERSION >= 9000
    hipblasMath_t math_mode;
    CUBLAS_CHECK(hipblasGetMathMode(handle, &math_mode));
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
    const float alpha_fp32 = static_cast<float>(alpha);
    const float beta_fp32 = static_cast<float>(beta);
    CUBLAS_CHECK(hipblasGemmEx(handle, cuTransB, cuTransA,
        N, M, K, &alpha_fp32, B->gethp<half>(), HIP_R_16F, ldb,
        A->gethp<half>(), HIP_R_16F, lda, &beta_fp32, C->gethp<half>(),
        HIP_R_16F, N, HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
    CUBLAS_CHECK(hipblasSetMathMode(handle, math_mode));
#else
    CUBLAS_CHECK(hipblasHgemm(handle, cuTransB, cuTransA,
    N, M, K, alpha.gethp<half>(), B->gethp<half>(), ldb,
    A->gethp<half>(), lda, beta.gethp<half>(), C->gethp<half>(), N));
#endif
  } else {
    float alpha_fp32 = static_cast<float>(alpha);
    float beta_fp32 = static_cast<float>(beta);
    CUBLAS_CHECK(cublasSgemmEx(handle, cuTransB, cuTransA,
        N, M, K, &alpha_fp32, B->gethp<half>(), CAFFE_DATA_HALF, ldb,
        A->gethp<half>(), CAFFE_DATA_HALF, lda, &beta_fp32, C->gethp<half>(),
        CAFFE_DATA_HALF, N));
  }
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(0), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(0), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_gemv<float16>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float16 alpha, const float16* A, const float16* x,
    const float16 beta, float16* y) {
  hipblasHandle_t handle = Caffe::cublas_handle(0);
  hipblasOperation_t cuTransA = TransA == CblasNoTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  int m = cuTransA == HIPBLAS_OP_N ? N : M;
  int k = cuTransA == HIPBLAS_OP_N ? M : N;
  int LDA = cuTransA == HIPBLAS_OP_N ? m : k;
  int LDC = m;

  if (Caffe::device_capability(Caffe::current_device()) >= 503) {
#if CUDA_VERSION >= 9000
    hipblasMath_t math_mode;
    CUBLAS_CHECK(hipblasGetMathMode(handle, &math_mode));
    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
    const float alpha_fp32 = static_cast<float>(alpha);
    const float beta_fp32 = static_cast<float>(beta);
    CUBLAS_CHECK(hipblasGemmEx(handle, cuTransA, HIPBLAS_OP_N,
        m, 1, k, &alpha_fp32, A->gethp<half>(), HIP_R_16F, LDA,
        x->gethp<half>(), HIP_R_16F, k, &beta_fp32, y->gethp<half>(),
        HIP_R_16F, LDC, HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
    CUBLAS_CHECK(hipblasSetMathMode(handle, math_mode));
#else
    CUBLAS_CHECK(hipblasHgemm(handle, cuTransA, HIPBLAS_OP_N,
        m, 1, k, alpha.gethp<half>(), A->gethp<half>(), LDA,
        x->gethp<half>(), k, beta.gethp<half>(),
        y->gethp<half>(), LDC));
#endif
  } else {
    float alpha_fp32 = static_cast<float>(alpha);
    float beta_fp32 = static_cast<float>(beta);
    CUBLAS_CHECK(cublasSgemmEx(Caffe::cublas_handle(0), cuTransA, HIPBLAS_OP_N,
        m, 1, k, &alpha_fp32, A, CAFFE_DATA_HALF, LDA,
        x, CAFFE_DATA_HALF, k, &beta_fp32,
        y, CAFFE_DATA_HALF, LDC));
  }
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(0)));
}

template<>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y, void* handle) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  CUBLAS_CHECK(hipblasSaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y, void* handle) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  CUBLAS_CHECK(hipblasDaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<typename Dtype, typename Mtype>
__global__
void axpy_kernel(const int N, const Mtype alpha, const Dtype* x, Dtype* y) {
  for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N;
       idx += blockDim.x * gridDim.x) {
    y[idx] = alpha * (Mtype) x[idx] + (Mtype) y[idx];
  }
}

template<>
__global__
void axpy_kernel<half, half>(const int N, const half alpha, const half* x, half* y) {
#if __CUDA_ARCH__ >= 530
  CUDA_KERNEL_LOOP(idx, N) {
    y[idx] = __hfma(alpha, x[idx], y[idx]);
  }
#else
  CUDA_KERNEL_LOOP(idx, N) {
    y[idx] = float2half_clip(__half2float(y[idx]) + __half2float(alpha) * __half2float(x[idx]));
  }
#endif
}

template<>
void caffe_gpu_axpy<float16>(const int N, const float16 alpha, const float16* x, float16* y,
    void* handle) {
  hipblasHandle_t cublas_handle =
      handle == nullptr ? Caffe::cublas_handle(0) : reinterpret_cast<hipblasHandle_t>(handle);
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  half ha;
  ha.setx(alpha.getx());
  // NOLINT_NEXT_LINE(whitespace/operators)
  axpy_kernel<<<CAFFE_GET_BLOCKS_HALF(N), CAFFE_CUDA_NUM_THREADS_HALF, 0, stream>>>
      (N, ha, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y, int group) {
  if (X != Y) {
    hipStream_t stream = Caffe::thread_stream(group);
    CUDA_CHECK(hipMemcpyAsync(Y, X, N, hipMemcpyDefault, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

__global__
void scale_in_place_kernel(const int n, const half alpha, half* x) {
  CUDA_KERNEL_LOOP(idx, n) {
    x[idx] = hmul(alpha, x[idx]);
  }
}

template<>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X, hipblasHandle_t cublas_handle) {
  if (alpha == 1.F) { return; }
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  CUBLAS_CHECK(hipblasSscal(cublas_handle, N, &alpha, X, 1));
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
    hipblasHandle_t cublas_handle) {
  if (alpha == 1.0) { return; }
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  CUBLAS_CHECK(hipblasDscal(cublas_handle, N, &alpha, X, 1));
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_scal<float16>(const int n, const float16 alpha, float16* x,
    hipblasHandle_t cublas_handle) {
  if (alpha.getx() == 0x3c00U) { return; }
  hipStream_t stream;
  CUBLAS_CHECK(hipblasGetStream(cublas_handle, &stream));
  half ha;
  ha.setx(alpha.getx());
  // use cublasHscal when it will become available
  // NOLINT_NEXT_LINE(whitespace/operators)
  scale_in_place_kernel <<<CAFFE_GET_BLOCKS_HALF(n), CAFFE_CUDA_NUM_THREADS_HALF, 0, stream>>>
      (n, ha, reinterpret_cast<half*>(x));
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X) {
  caffe_gpu_scal(N, alpha, X, Caffe::cublas_handle(0));
}

template<>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X) {
  caffe_gpu_scal(N, alpha, X, Caffe::cublas_handle(0));
}

template<>
void caffe_gpu_scal<float16>(const int N, const float16 alpha, float16* X) {
  caffe_gpu_scal(N, alpha, X, Caffe::cublas_handle(0));
}

template<>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template<>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template<typename Dtype, typename Mtype>
__global__
void axpby_kernel(const int N, const Mtype alpha, const Dtype* X, const Mtype beta, Dtype* Y) {
  CUDA_KERNEL_LOOP(idx, N) {
    Y[idx] = alpha * X[idx] + beta * Y[idx];
  }
}

template<>
void caffe_gpu_axpby<float16>(const int N, const float16 alpha,
    const float16* X, const float16 beta, float16* Y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  axpby_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, alpha, X, beta, Y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_dot<float, float>(const int n, const float* x, const float* y, float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(0), n, x, 1, y, 1, out));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
}

template<>
void caffe_gpu_dot<double, double>(const int n, const double* x, const double* y, double* out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(0), n, x, 1, y, 1, out));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
}

template<>
void caffe_gpu_dot<double, float>(const int n, const double* x, const double* y, float* outf) {
  double out = 0.;
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(0), n, x, 1, y, 1, &out));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
  *outf = static_cast<float>(out);
}

template<typename Dtype, typename Mtype>
__global__
void gpu_dot_kernel(const int N, const Dtype* x, const Dtype* y, Mtype* out) {
  __shared__
  Mtype cache[CAFFE_CUDA_NUM_THREADS];
  const int tidx = threadIdx.x;
  cache[tidx] = 0.;
  __syncthreads();
  for (int i = tidx; i < N; i += blockDim.x) {
    cache[tidx] += static_cast<Mtype>(x[i]) * static_cast<Mtype>(y[i]);
  }
  __syncthreads();
  for (int s = CAFFE_CUDA_NUM_THREADS / 2; s > 0; s >>= 1) {
    if (tidx < s) cache[tidx] += cache[tidx + s];
    __syncthreads();
  }
  if (tidx == 0) *out = cache[tidx];
}

template<>
void
caffe_gpu_dot<float16, float16>(const int n, const float16* x, const float16* y, float16* out) {
  float fres;
  GPUMemory::Workspace ws(sizeof(float), Caffe::current_device());
  float* res = reinterpret_cast<float*>(ws.data());
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  gpu_dot_kernel<<<1, CAFFE_CUDA_NUM_THREADS, 0, stream>>>(n, x, y, res);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipMemcpyAsync(&fres, res, ws.size(), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));
  *out = static_cast<float16>(fres);
}

template<>
void caffe_gpu_dot<float16, float>(const int n, const float16* x, const float16* y, float* out) {
  GPUMemory::Workspace ws(sizeof(float), Caffe::current_device());
  float* res = reinterpret_cast<float*>(ws.data());
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  gpu_dot_kernel<<<1, CAFFE_CUDA_NUM_THREADS, 0, stream>>>(n, x, y, res);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipMemcpyAsync(out, res, ws.size(), hipMemcpyDeviceToHost, stream));
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_asum<float, float>(const int n, const float* x, float* y, int group) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(group), n, x, 1, y));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(group)));
}

template<>
void caffe_gpu_asum<float, double>(const int n, const float* x, double* y, int group) {
  float yf;
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(group), n, x, 1, &yf));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(group)));
  *y = yf;
}
template<>
void caffe_gpu_asum<double, double>(const int n, const double* x, double* y, int group) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(group), n, x, 1, y));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(group)));
}
template<>
void caffe_gpu_asum<double, float>(const int n, const double* x, float* y, int group) {
  double yd;
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(group), n, x, 1, &yd));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream(group)));
  *y = yd;
}

template<>
void caffe_gpu_scale<double>(const int n, const double alpha, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(0), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(0), n, &alpha, y, 1));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
}

template<>
void caffe_gpu_scale<float>(const int n, const float alpha, const float* x, float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(0), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(0), n, &alpha, y, 1));
  CUDA_CHECK(hipStreamSynchronize(Caffe::thread_stream()));
}

__global__
void scale_kernel(const int n, const half alpha, const half* x, half* y) {
  CUDA_KERNEL_LOOP(idx, n) {
    y[idx] = hmul(alpha, x[idx]);
  }
}

template<>
void caffe_gpu_scale<float16>(const int n, const float16 alpha, const float16* x, float16* y) {
  hipStream_t stream = Caffe::thread_stream();
  half ha;
  ha.setx(alpha.getx());
  // NOLINT_NEXT_LINE(whitespace/operators)
  scale_kernel <<<CAFFE_GET_BLOCKS_HALF(n), CAFFE_CUDA_NUM_THREADS_HALF, 0, stream>>>
      (n, ha, reinterpret_cast<const half*>(x), reinterpret_cast<half*>(y));
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<typename Dtype>
__global__ void set_kernel(const size_t n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template<typename Dtype>
void caffe_gpu_set(const size_t N, const Dtype alpha, Dtype* Y) {
  hipStream_t stream = Caffe::thread_stream();
  if (alpha == 0) {
    CUDA_CHECK(hipMemsetAsync(Y, 0, sizeof(Dtype) * N, stream));  // NOLINT(caffe/alt_fn)
  } else {
    // NOLINT_NEXT_LINE(whitespace/operators)
    set_kernel <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>> (N, alpha, Y);
    CUDA_POST_KERNEL_CHECK;
  }
  CUDA_CHECK_ARG2(hipStreamSynchronize(stream), stream, Caffe::current_device());
}

template void
caffe_gpu_set<int>(const size_t N, const int alpha, int* Y);
template void
caffe_gpu_set<float>(const size_t N, const float alpha, float* Y);
template void
caffe_gpu_set<double>(const size_t N, const double alpha, double* Y);
template void
caffe_gpu_set<float16>(const size_t N, const float16 alpha, float16* Y);

template<typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template<>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators
  add_scalar_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_add_scalar(const int N, const float16 alpha, float16* Y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, alpha, Y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a, const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template<>
__global__ void add_kernel<half>(const int n, const half* a, const half* b, half* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = hadd(a[index], b[index]);
  }
}

template<>
void caffe_gpu_add<float>(const int N, const float* a, const float* b, float* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_add<double>(const int N, const double* a, const double* b, double* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_add<float16>(const int N, const float16* a, const float16* b, float16* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<<<CAFFE_GET_BLOCKS_HALF(N), CAFFE_CUDA_NUM_THREADS_HALF, 0, stream>>>
      (N, reinterpret_cast<const half*>(a), reinterpret_cast<const half*>(b),
       reinterpret_cast<half*>(y));
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<typename Dtype>
__global__ void incr_kernel(const int n, const Dtype* a, Dtype* b) {
  CUDA_KERNEL_LOOP(index, n) {
    b[index] += a[index];
  }
}

template<>
__global__ void incr_kernel<half>(const int n, const half* a, half* b) {
  CUDA_KERNEL_LOOP(index, n) {
    b[index] = hadd(a[index], b[index]);
  }
}

template<>
void caffe_gpu_incr<float>(const int N, const float* a, float* b) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  incr_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_incr<double>(const int N, const double* a, double* b) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  incr_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_incr<float16>(const int N, const float16* a, float16* b) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  incr_kernel<<<CAFFE_GET_BLOCKS_HALF(N), CAFFE_CUDA_NUM_THREADS_HALF, 0, stream>>>
      (N, reinterpret_cast<const half*>(a), reinterpret_cast<half*>(b));
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template<>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b, float* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b, double* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

template<>
void caffe_gpu_sub<float16>(const int N, const float16* a, const float16* b, float16* y) {
  hipStream_t stream = Caffe::thread_stream();
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, stream>>>(N, a, b, y);
  CUDA_POST_KERNEL_CHECK;
  CUDA_CHECK(hipStreamSynchronize(stream));
}

}  // namespace caffe
