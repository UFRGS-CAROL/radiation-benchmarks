#include "hip/hip_runtime.h"
/*
 * HotspotExecute.cpp
 *
 *  Created on: 18/05/2019
 *      Author: fernando
 */

#include <hip/hip_fp16.h>


#include "HotspotExecute.h"
#include "kernels.h"

#ifdef LOGS
#include "log_helper.h"
#endif

#include "device_vector.h"


unsigned long long copy_errors() {
	unsigned long long errors_host = 0;
	//Copy errors first
	rad::checkFrameworkErrors(
			hipMemcpyFromSymbol((void*)&errors_host, errors,
					sizeof(unsigned long long), 0));

	unsigned long long temp = 0;
	//Reset the errors variable
	rad::checkFrameworkErrors(
				hipMemcpyToSymbol(HIP_SYMBOL(errors), (void*)&temp,
						sizeof(unsigned long long), 0));
	return errors_host;
}


HotspotExecute::HotspotExecute(Parameters& setup_parameters, Log& log) :
		setup_params(setup_parameters), log(log), flops(0) {
	if (this->setup_params.verbose)
		std::cout << "WG size of kernel = " << BLOCK_SIZE << " x " << BLOCK_SIZE
				<< std::endl;
}

template<typename full, typename incomplete>
int HotspotExecute::compute_tran_temp(rad::DeviceVector<full>& power_array,
		rad::DeviceVector<full>& temp_array_input,
		rad::DeviceVector<full>& temp_array_output,
		rad::DeviceVector<incomplete>& temp_array_output_incomplete, int col,
		int row, int sim_time, int num_iterations, int blockCols, int blockRows,
		int borderCols, int borderRows, hipStream_t stream) {
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(blockCols, blockRows);

	// Default values of hotpsot
	DefaultType t_chip(0.0005);
	DefaultType chip_height(0.016);
	DefaultType chip_width(0.016);
	DefaultType grid_height = chip_height / row;
	DefaultType grid_width = chip_width / col;
	DefaultType Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width
			* grid_height;
	DefaultType Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	DefaultType Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	DefaultType Rz = t_chip / (K_SI * grid_height * grid_width);
	DefaultType max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	DefaultType step = PRECISION / max_slope;

	//New precision converted types
	full time_elapsed = 0.001;
	full Cap_ = Cap;
	full Rx_ = Rx;
	full Ry_ = Ry;
	full Rz_ = Rz;
	full step_ = step;

	int src = 1, dst = 0;
	full* MatrixPower = power_array.data();
	full* MatrixTemp[2] = { temp_array_input.data(), temp_array_output.data() };
	incomplete* MatrixTempIncomplete = temp_array_output_incomplete.data();
	if (this->setup_params.redundancy == NONE) {
		for (int t = 0; t < sim_time; t += num_iterations) {
			std::swap(src, dst);
			calculate_temp<full> <<<dimGrid, dimBlock, 0, stream>>>(
					MIN(num_iterations, sim_time - t), MatrixPower,
					MatrixTemp[src], MatrixTemp[dst], col, row, borderCols,
					borderRows, Cap_, Rx_, Ry_, Rz_, step_, time_elapsed);
			this->flops += col * row * MIN(num_iterations, sim_time - t) * 15;
		}
	} else {
		for (int t = 0; t < sim_time; t += num_iterations) {
			std::swap(src, dst);
			calculate_temp<full, incomplete> <<<dimGrid, dimBlock, 0, stream>>>(
					MIN(num_iterations, sim_time - t), MatrixPower,
					MatrixTemp[src], MatrixTemp[dst], MatrixTempIncomplete, col,
					row, borderCols, borderRows, Cap_, Rx_, Ry_, Rz_, step_,
					time_elapsed);
			this->flops += col * row * MIN(num_iterations, sim_time - t) * 15;
		}
	}

//	hipStreamSynchronize(stream);
	return dst;
}

template<typename full, typename incomplete>
void HotspotExecute::generic_execute(int blockCols, int blockRows,
		int borderCols, int borderRows) {
	DataManagement<full, incomplete> hotspot_data(this->setup_params,
			this->log);
	hotspot_data.read_input();

	//====================================
	// Verbose in csv format
	if (this->setup_params.verbose == false) {
		std::cout << "output/s,iteration,time,output errors,relative errors"
				<< std::endl;
	}

	// ====================== MAIN BENCHMARK CYCLE ======================
	for (int loop = 0; loop < this->setup_params.setup_loops; loop++) {
		if (this->setup_params.verbose)
			std::cout << "======== Iteration #" << loop << " ========"
					<< std::endl;
		double global_time = this->log.mysecond();

		// ============ PREPARE ============
		double reload_time = this->log.mysecond();
		hotspot_data.reload();
		reload_time = this->log.mysecond();

		// ============ COMPUTE ============
		this->log.start_iteration_app();
		this->flops = 0;
		for (int streamIdx = 0; streamIdx < (this->setup_params.nstreams);
				streamIdx++) {
			rad::DeviceVector<full>& power_array_stream =
					hotspot_data.matrix_power_device[streamIdx];
			rad::DeviceVector<full>& temp_array_input_stream =
					hotspot_data.matrix_temperature_input_device[streamIdx];
			rad::DeviceVector<full>& temp_array_output_stream =
					hotspot_data.matrix_temperature_output_device[streamIdx];

			rad::DeviceVector<incomplete>& output_incomplete =
					hotspot_data.matrix_temperature_output_incomplete_device[streamIdx];

			hotspot_data.output_index[streamIdx] = compute_tran_temp<full,
					incomplete>(power_array_stream, temp_array_input_stream,
					temp_array_output_stream, output_incomplete, this->setup_params.grid_cols,
					this->setup_params.grid_rows, this->setup_params.sim_time,
					this->setup_params.pyramid_height, blockCols, blockRows,
					borderCols, borderRows, hotspot_data.streams[streamIdx]);
		}

		hotspot_data.sync();

		this->log.end_iteration_app();

		// ============ VALIDATE OUTPUT ============
		double copy_and_check_time = this->log.mysecond();

		hotspot_data.copy_from_gpu();
		hotspot_data.check_output_errors();

		auto dmr_errors = copy_errors();


		copy_and_check_time = this->log.mysecond();

		if (this->setup_params.verbose) {
			std::cout << "GPU prepare time: " << reload_time << "s"
					<< std::endl;

			// ============ MEASURE PERFORMANCE ============
			double outputpersec = double(
					((this->setup_params.grid_rows
							* this->setup_params.grid_rows
							* this->setup_params.nstreams)
							/ this->log.iteration_time()));
			std::cout << "Kernel time: " << this->log.iteration_time()
					<< std::endl;

			std::cout << "Performance - SIZE:" << this->setup_params.grid_rows
					<< " OUTPUT/S: " << outputpersec << " FLOPS: "
					<< flops / this->log.iteration_time() << " (GFLOPS: "
					<< flops / (this->log.iteration_time() * 1e9) << ")"
					<< std::endl;

			std::cout << "Gold check time: " << copy_and_check_time
					<< std::endl;

			//computing if the overall time is enough
			double iteration_time = this->log.mysecond() - global_time;

			std::cout << "Iteration time: " << iteration_time << " ("
					<< (this->log.iteration_time() / iteration_time) * 100.0
					<< "% Device)" << std::endl;
			std::cout << "Iteration errors " << this->log.error_count
					<< " DMR errors " << dmr_errors << std::endl;

			std::cout << "==============================" << std::endl;

		} else {
			//			std::cout << ".";
			// CSV format
			double outputpersec = double(
					((this->setup_params.grid_rows
							* this->setup_params.grid_rows
							* this->setup_params.nstreams)
							/ this->log.iteration_time()));
			std::cout << outputpersec << ",";
			std::cout << loop << ",";
			std::cout << this->log.iteration_time() << ",";
			std::cout << this->log.error_count << ",";
			std::cout << dmr_errors << std::endl;
		}

	}

	//this function already check if must generate a gold
	// or not
	hotspot_data.write_output();
}

HotspotExecute::~HotspotExecute() {
}

void HotspotExecute::run() {
	// ===============  pyramid parameters
	int borderCols = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int borderRows = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int smallBlockCol = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int smallBlockRow = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int blockCols = this->setup_params.grid_cols / smallBlockCol
			+ ((this->setup_params.grid_cols % smallBlockCol == 0) ? 0 : 1);
	int blockRows = this->setup_params.grid_rows / smallBlockRow
			+ ((this->setup_params.grid_rows % smallBlockRow == 0) ? 0 : 1);

	this->setup_params.size = (this->setup_params.grid_cols)
			* (this->setup_params.grid_rows);

	switch (this->setup_params.redundancy) {
	case NONE:
	case DMR:
		switch (this->setup_params.precision) {
		case HALF:

			generic_execute<half, half>(blockCols, blockRows, borderCols,
					borderRows);
			break;

		case SINGLE:
			generic_execute<float, float>(blockCols, blockRows, borderCols,
					borderRows);
			break;

		case DOUBLE:
			generic_execute<double, double>(blockCols, blockRows, borderCols,
					borderRows);
			break;

		}
		break;

	case DMRMIXED:
		switch (this->setup_params.precision) {
		case SINGLE:
			generic_execute<float, half>(blockCols, blockRows, borderCols,
					borderRows);
			break;

		case DOUBLE:
			generic_execute<double, float>(blockCols, blockRows, borderCols,
					borderRows);
			break;

		}
		break;

	}

}
