#include "hip/hip_runtime.h"
/*
 * HotspotExecute.cpp
 *
 *  Created on: 18/05/2019
 *      Author: fernando
 */

#include "HotspotExecute.h"
#include "none_kernels.h"

#include <hip/hip_fp16.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

HotspotExecute::HotspotExecute(Parameters& setup_parameters) {
	this->setup_params = setup_parameters;

	std::string test_info = std::string("streams:")
			+ std::to_string(this->setup_params.nstreams) + " precision:"
			+ this->setup_params.test_precision_description + " size:"
			+ std::to_string(this->setup_params.grid_rows) + +" pyramidHeight:"
			+ std::to_string(this->setup_params.pyramid_height) + " simTime:"
			+ std::to_string(this->setup_params.sim_time) + " redundancy:"
			+ this->setup_params.test_redundancy_description;
	std::string test_name = "cuda_hotspot_"
			+ this->setup_params.test_precision_description;

	this->log = Log(test_name, test_info, this->setup_params.generate);

	std::cout << "WG size of kernel = " << BLOCK_SIZE << " " << BLOCK_SIZE
			<< std::endl;
	std::cout << std::endl << test_name << std::endl << test_info << std::endl;

}

template<typename full>
int HotspotExecute::compute_tran_temp(DeviceVector<full>& power_array,
		DeviceVector<full>& temp_array_input,
		DeviceVector<full>& temp_array_output, int col, int row, int sim_time,
		int num_iterations, int blockCols, int blockRows, int borderCols,
		int borderRows, hipStream_t stream, double& flops) {

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(blockCols, blockRows);

	full t_chip(0.0005);
	full chip_height(0.016);
	full chip_width(0.016);

	full grid_height = chip_height / full(row);
	full grid_width = chip_width / full(col);

	full Cap = full(FACTOR_CHIP) * full(SPEC_HEAT_SI) * t_chip * grid_width
			* grid_height;
	full Rx = grid_width / (full(2.0) * full(K_SI) * t_chip * grid_height);
	full Ry = grid_height / (full(2.0) * full(K_SI) * t_chip * grid_width);
	full Rz = t_chip / (full(K_SI) * grid_height * grid_width);

	full max_slope = full(MAX_PD)
			/ (full(FACTOR_CHIP) * t_chip * full(SPEC_HEAT_SI));
	full step = full(PRECISION) / max_slope;
//	full t;
	full time_elapsed = 0.001;

	int src = 0, dst = 1;
	full* MatrixPower = power_array.data;
	full* MatrixTemp[2] = { temp_array_input.data, temp_array_output.data };

	for (int t = 0; t < sim_time; t += num_iterations) {
		calculate_temp<full> <<<dimGrid, dimBlock, 0, stream>>>(
				MIN(num_iterations, sim_time - t), MatrixPower, MatrixTemp[src],
				MatrixTemp[dst], col, row, borderCols, borderRows, Cap, Rx, Ry,
				Rz, step, time_elapsed);
		flops += col * row * MIN(num_iterations, sim_time - t) * 15;
		std::swap(src, dst);
	}
//	hipStreamSynchronize(stream);
	return dst;
}

template<typename full>
void HotspotExecute::generic_execute(int blockCols, int blockRows,
		int borderCols, int borderRows) {
	DataManagement<full> hotspot_data(this->setup_params);
	hotspot_data.readInput();

	// ====================== MAIN BENCHMARK CYCLE ======================
	for (int loop1 = 0; loop1 < (this->setup_params.setup_loops); loop1++) {
		if (this->setup_params.verbose)
			printf("======== Iteration #%06u ========\n", loop1);

		double globaltime = this->log.mysecond();
		// ============ PREPARE ============
		std::vector<int> ret(this->setup_params.nstreams);
		double timestamp = this->log.mysecond();
		hotspot_data.reload();
		if (this->setup_params.verbose)
			printf("GPU prepare time: %.4fs\n",
					this->log.mysecond() - timestamp);

		// ============ COMPUTE ============
		double kernel_time = this->log.mysecond();
		this->log.start_iteration_app();
		double flops = 0;
		for (int streamIdx = 0; streamIdx < (this->setup_params.nstreams);
				streamIdx++) {
			DeviceVector<full>& power_array_stream =
					hotspot_data.matrix_power_device[streamIdx];
			DeviceVector<full>& temp_array_input_stream =
					hotspot_data.matrix_temperature_input_device[streamIdx];
			DeviceVector<full>& temp_array_output_stream =
					hotspot_data.matrix_temperature_output_device[streamIdx];

			ret[streamIdx] = compute_tran_temp(power_array_stream,
					temp_array_input_stream, temp_array_output_stream,
					this->setup_params.grid_cols, this->setup_params.grid_rows,
					this->setup_params.sim_time,
					this->setup_params.pyramid_height, blockCols, blockRows,
					borderCols, borderRows, hotspot_data.streams[streamIdx],
					flops);
		}

		for (auto stream : hotspot_data.streams) {
			hipStreamSynchronize(stream);
		}
		this->log.end_iteration_app();
		kernel_time = this->log.mysecond() - kernel_time;
		// ============ MEASURE PERFORMANCE ============
		if (this->setup_params.verbose) {
			double outputpersec =
					(double) (((this->setup_params.grid_rows
							* this->setup_params.grid_rows
							* this->setup_params.nstreams) / kernel_time));
			std::cout << "Kernel time: " << kernel_time << std::endl;
			std::cout << "Performance - SIZE:" << this->setup_params.grid_rows
					<< " OUTPUT/S: " << outputpersec << " FLOPS: "
					<< flops / kernel_time << " (GFLOPS: "
					<< flops / (kernel_time * 1e9) << ")" << std::endl;
		}
		// ============ VALIDATE OUTPUT ============
		timestamp = this->log.mysecond();
		int kernel_errors = 0;

		hotspot_data.copy_from_gpu();

		if (this->setup_params.generate) {
			hotspot_data.writeOutput();
		} else {
			hotspot_data.check_output_errors();
		}

		if (this->setup_params.verbose)
			std::cout << "Gold check time: " << this->log.mysecond() - timestamp
					<< std::endl;

		if ((kernel_errors != 0) && !(this->setup_params.verbose))
			printf(".");

		double iteration_time = this->log.mysecond() - globaltime;
		if (this->setup_params.verbose)
			std::cout << "Iteration time: " << iteration_time << " ("
					<< (kernel_time / iteration_time) * 100.0 << "% Device)\n"
					<< std::endl;

		if (this->setup_params.verbose)
			std::cout << ("===================================\n");

	}
}

HotspotExecute::~HotspotExecute() {
}

void HotspotExecute::run() {
	// ===============  pyramid parameters
	int borderCols = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int borderRows = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int smallBlockCol = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int smallBlockRow = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int blockCols = this->setup_params.grid_cols / smallBlockCol
			+ ((this->setup_params.grid_cols % smallBlockCol == 0) ? 0 : 1);
	int blockRows = this->setup_params.grid_rows / smallBlockRow
			+ ((this->setup_params.grid_rows % smallBlockRow == 0) ? 0 : 1);

	this->setup_params.size = (this->setup_params.grid_cols)
			* (this->setup_params.grid_rows);

	switch (this->setup_params.precision) {
	case HALF:
		generic_execute<half>(blockCols, blockRows, borderCols, borderRows);

		break;

	case SINGLE:
		generic_execute<float>(blockCols, blockRows, borderCols, borderRows);

		break;

	case DOUBLE:
		generic_execute<double>(blockCols, blockRows, borderCols, borderRows);

		break;

	}

}
