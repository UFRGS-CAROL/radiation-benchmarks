#include "hip/hip_runtime.h"
/*
 * HotspotExecute.cpp
 *
 *  Created on: 18/05/2019
 *      Author: fernando
 */

#include "HotspotExecute.h"
#include "none_kernels.h"
#include "half.hpp"

#include <hip/hip_fp16.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

HotspotExecute::HotspotExecute(Parameters& setup_parameters, Log& log) :
		setup_params(setup_parameters), log(log) {
}

template<typename full>
int HotspotExecute::compute_tran_temp(DeviceVector<full>& power_array,
		DeviceVector<full>& temp_array_input,
		DeviceVector<full>& temp_array_output, int col, int row, int sim_time,
		int num_iterations, int blockCols, int blockRows, int borderCols,
		int borderRows, hipStream_t stream, double& flops) {
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(blockCols, blockRows);

	// Default values of hotpsot
	DefaultType t_chip(0.0005);
	DefaultType chip_height(0.016);
	DefaultType chip_width(0.016);
	DefaultType grid_height = chip_height / row;
	DefaultType grid_width = chip_width / col;
	DefaultType Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width
			* grid_height;
	DefaultType Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	DefaultType Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	DefaultType Rz = t_chip / (K_SI * grid_height * grid_width);
	DefaultType max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	DefaultType step = PRECISION / max_slope;

	//New precision converted types
	full time_elapsed = 0.001;
	full Cap_ = Cap;
	full Rx_ = Rx;
	full Ry_ = Ry;
	full Rz_ = Rz;
	full step_ = step;

	int src = 0, dst = 1;
	full* MatrixPower = power_array.data;
	full* MatrixTemp[2] = { temp_array_input.data, temp_array_output.data };

	std::cout << " output " << MatrixTemp[0] << " input " << MatrixTemp[1]
			<< std::endl;

	for (int t = 0; t < sim_time; t += num_iterations) {
		calculate_temp<full> <<<dimGrid, dimBlock, 0, stream>>>(
				MIN(num_iterations, sim_time - t), MatrixPower, MatrixTemp[src],
				MatrixTemp[dst], col, row, borderCols, borderRows, Cap_, Rx_,
				Ry_, Rz_, step_, time_elapsed);
		flops += col * row * MIN(num_iterations, sim_time - t) * 15;
		std::swap(src, dst);
	}
//	hipStreamSynchronize(stream);
	return dst;
}

template<typename full>
void HotspotExecute::generic_execute(int blockCols, int blockRows,
		int borderCols, int borderRows) {
	DataManagement<full> hotspot_data(this->setup_params);
	hotspot_data.read_input();

	// ====================== MAIN BENCHMARK CYCLE ======================
	for (int loop = 0; loop < this->setup_params.setup_loops; loop++) {
		if (this->setup_params.verbose)
			std::cout << "======== Iteration #" << loop << " ========"
					<< std::endl;
		double globaltime = this->log.mysecond();

		// ============ PREPARE ============
		std::vector<int> ret(this->setup_params.nstreams);
		double timestamp = this->log.mysecond();
		hotspot_data.reload();
		if (this->setup_params.verbose)
			std::cout << "GPU prepare time: "
					<< this->log.mysecond() - timestamp << "s" << std::endl;

		// ============ COMPUTE ============
		this->log.start_iteration_app();
		double flops = 0;
		for (int streamIdx = 0; streamIdx < (this->setup_params.nstreams);
				streamIdx++) {
			DeviceVector<full>& power_array_stream =
					hotspot_data.matrix_power_device[streamIdx];
			DeviceVector<full>& temp_array_input_stream =
					hotspot_data.matrix_temperature_input_device[streamIdx];
			DeviceVector<full>& temp_array_output_stream =
					hotspot_data.matrix_temperature_output_device[streamIdx];

//			ret[streamIdx] =
			compute_tran_temp<full>(power_array_stream, temp_array_input_stream,
					temp_array_output_stream, this->setup_params.grid_cols,
					this->setup_params.grid_rows, this->setup_params.sim_time,
					this->setup_params.pyramid_height, blockCols, blockRows,
					borderCols, borderRows, hotspot_data.streams[streamIdx],
					flops);
		}

		for (auto stream : hotspot_data.streams) {
			checkFrameworkErrors(hipStreamSynchronize(stream));
		}
		checkFrameworkErrors(hipDeviceSynchronize());
		checkFrameworkErrors(hipPeekAtLastError());

		this->log.end_iteration_app();
		// ============ MEASURE PERFORMANCE ============
		if (this->setup_params.verbose) {
			double outputpersec =
					(double) (((this->setup_params.grid_rows
							* this->setup_params.grid_rows
							* this->setup_params.nstreams)
							/ this->log.iteration_time()));
			std::cout << "Kernel time: " << this->log.iteration_time()
					<< std::endl;
			std::cout << "Performance - SIZE:" << this->setup_params.grid_rows
					<< " OUTPUT/S: " << outputpersec << " FLOPS: "
					<< flops / this->log.iteration_time() << " (GFLOPS: "
					<< flops / (this->log.iteration_time() * 1e9) << ")"
					<< std::endl;
		}
		// ============ VALIDATE OUTPUT ============
		timestamp = this->log.mysecond();
		int kernel_errors = 0;

		hotspot_data.copy_from_gpu();
		hotspot_data.check_output_errors();

		if (this->setup_params.verbose)
			std::cout << "Gold check time: " << this->log.mysecond() - timestamp
					<< std::endl;

		if ((kernel_errors != 0) && !(this->setup_params.verbose))
			std::cout << ".";

		if (this->setup_params.verbose) {
			//computing if the overall time is enough
			double iteration_time = this->log.mysecond() - globaltime;

			std::cout << "Iteration time: " << iteration_time << " ("
					<< (this->log.iteration_time() / iteration_time) * 100.0
					<< "% Device)\n" << std::endl;
		}
		if (this->setup_params.verbose)
			std::cout << ("==============================\n");

	}

	//this function already check if must generate a gold
	// or not
	hotspot_data.write_output();
}

HotspotExecute::~HotspotExecute() {
}

void HotspotExecute::run() {
	// ===============  pyramid parameters
	int borderCols = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int borderRows = (this->setup_params.pyramid_height) * EXPAND_RATE / 2;
	int smallBlockCol = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int smallBlockRow = BLOCK_SIZE
			- (this->setup_params.pyramid_height) * EXPAND_RATE;
	int blockCols = this->setup_params.grid_cols / smallBlockCol
			+ ((this->setup_params.grid_cols % smallBlockCol == 0) ? 0 : 1);
	int blockRows = this->setup_params.grid_rows / smallBlockRow
			+ ((this->setup_params.grid_rows % smallBlockRow == 0) ? 0 : 1);

	this->setup_params.size = (this->setup_params.grid_cols)
			* (this->setup_params.grid_rows);

	switch (this->setup_params.precision) {
	case HALF:

		generic_execute<half>(blockCols, blockRows, borderCols, borderRows);
		break;

	case SINGLE:
		generic_execute<float>(blockCols, blockRows, borderCols, borderRows);
		break;

	case DOUBLE:
		generic_execute<double>(blockCols, blockRows, borderCols, borderRows);
		break;

	}

}
