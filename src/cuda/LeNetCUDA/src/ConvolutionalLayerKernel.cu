#include "hip/hip_runtime.h"
/**
 * Here all kernels which were in kernels.ocl were translated
 */

#include "ConvolutionalLayerKernel.h"
#include "cudaUtil.h"
#include <cstdio>


__device__ float sigmod(float in) {
	return 1.0 / (1.0 + exp(-in));
}

__device__ inline int get_global_id(int index) {
	switch (index) {
	case 0:
		return blockIdx.x * blockDim.x + threadIdx.x;
	case 1:
		return blockIdx.y * blockDim.y + threadIdx.y;
	case 2:
		return blockIdx.z * blockDim.z + threadIdx.z;
	};
	return -1;
}

__global__ void forward_parallel(float* input_buf, float* weight_buf,
		float* b_buf, float* output_buf, int in_width, int in_height,
		int in_depth, int out_width, int out_height, int out_depth,
		int kernel_size) {

	if (get_global_id(0) > out_depth * out_width)
		return;
	if (get_global_id(1) > out_height)
		return;

	int out = get_global_id(0) / out_width;
	int w_index = get_global_id(0) % out_width;
	int h_index = get_global_id(1);

	float sum = 0;
	int size = kernel_size * kernel_size;

	for (unsigned int in = 0; in < in_depth; in++) {
		float weight_buf_sub[25]; // Set by brute force, NEED to be changed
		float input_buf_sub[25]; // Set by brute force, NEED to be changed
		// load input and weight for this sub area
		for (unsigned int y = 0; y < kernel_size; y++) {
			for (unsigned int x = 0; x < kernel_size; x++) {
				input_buf_sub[y * kernel_size + x] = input_buf[in
						* (in_width * in_height) + (h_index + y) * in_width + x
						+ w_index];
				weight_buf_sub[y * kernel_size + x] = weight_buf[in * out_depth
						* size + out * size + y * kernel_size + x];
			}
		}

		// compute the convolution
		for (unsigned int i = 0; i < size; i++) {
			sum += input_buf_sub[i] * weight_buf_sub[size - i - 1];
		}
	}

	unsigned int out_index = out * out_width * out_height + h_index * out_width
			+ w_index;
	unsigned int b_index = out_index;
	output_buf[out_index] = sigmod(sum + b_buf[b_index]);
}

void call_foward_parallel(float* input_buf, float* weight_buf, float* b_buf,
		float* output_buf, int in_width, int in_height, int in_depth,
		int out_width, int out_height, int out_depth, int kernel_size) {

	//PEDRO if these are the right dimentions
	long blocks_rows = ceil(float(out_height) / float(BLOCK_SIZE));
	long threads_rows = ceil(float(out_height) / float(blocks_rows));
	long blocks_cols = ceil(float(out_width) / float(BLOCK_SIZE));
	long threads_cols = ceil(float(out_width) / float(blocks_cols));

	dim3 blocks(blocks_rows, blocks_cols);
	dim3 threads(threads_rows, threads_cols);

	forward_parallel<<<blocks, threads>>>(input_buf,
			weight_buf, b_buf, output_buf, in_width, in_height, in_depth,
			out_width, out_height, out_depth, kernel_size);

	hipError_t ret = hipDeviceSynchronize();	
	CUDA_CHECK_RETURN(ret);
}
