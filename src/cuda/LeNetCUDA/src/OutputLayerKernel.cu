#include "hip/hip_runtime.h"
/*
 * OutputLayerKernel.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: carol
 */

#include "cudaUtil.h"
#include "OutputLayer.h"

__device__ float df_sigmod_gpu_output(float f_x) {
	return f_x * (1.0 - f_x);
}

__global__ void forward_output_layer_kernel(float *exp_y_vec, float *input_,
		float *reduce_output, float *output_, int in_depth_, int exp_y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > in_depth_)
		return;

	reduce_output[i] = 0.5 * (exp_y_vec[i] - input_[i])
			* (exp_y_vec[i] - input_[i]);

	//copy that was done in the host before
	output_[i] = input_[i];
}

void OutputLayer::call_forward_output_layer(float *exp_y_vec, float *input_,
		float *reduce_output, float *output_, int in_depth_, int exp_y) {
	dim3 blocks, threads;
	cuda_gridsize(&threads, &blocks, in_depth_);

	forward_output_layer_kernel<<<blocks, threads>>>(exp_y_vec, input_,
			reduce_output, output_, in_depth_, exp_y);

	CudaCheckError();
}

__global__ void backprop_output_layer_kernel(float *exp_y_vec, float *input_,
		float *g_, int in_depth_) {
	/* compute err terms of output layers */
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > in_depth_)
		return;
	g_[i] = ((exp_y_vec[i] - input_[i]) * df_sigmod_gpu_output(input_[i]));

}

void OutputLayer::call_backpropagation_output_layer(float *exp_y_vec, float *input_,
		float *g_, int in_depth_) {
	dim3 blocks, threads;
	cuda_gridsize(&threads, &blocks, in_depth_);
	backprop_output_layer_kernel<<<blocks, threads>>>(exp_y_vec, input_, g_,
			in_depth_);
	CudaCheckError();
}
