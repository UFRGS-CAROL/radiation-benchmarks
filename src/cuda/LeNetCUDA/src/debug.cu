#include "hip/hip_runtime.h"
/*
 * debug.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: carol
 *
 *      this file is only for debugging
 *      classes
 */
#include <iostream>
#include <vector>
#include <cstdio>

#include "DeviceVector.h"

// cpu implementation of dot product
float dot(const float* v1, const float* v2, int N) {
	float s = 0;
	for (int i = 0; i != N; ++i) {
		s += v1[i] * v2[i];
	}
	return s;
}

void print_matrix(float *m, size_t h, size_t w) {
	printf("matxix\n");
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			printf("%f ", m[i * w + j]);
		}
		printf("\n");
	}

}

void cuda_gridsize(dim3 *threads, dim3 *blocks, size_t x, size_t y, size_t z) {

	long blocks_x = ceil(float(x) / float(BLOCK_SIZE));
	long threads_x = ceil(float(x) / float(blocks_x));
	long blocks_y = ceil(float(y) / float(BLOCK_SIZE));
	long threads_y = ceil(float(y) / float(blocks_y));
	long blocks_z = ceil(float(z) / float(BLOCK_SIZE));
	long threads_z = ceil(float(z) / float(blocks_z));

	*blocks = dim3(blocks_x, blocks_y, blocks_z);
	*threads = dim3(threads_x, threads_y, threads_z);

	printf("b_x %d b_y %d b_z %d\nt_x %d t_y %d t_z %d\n", blocks->x, blocks->y,
			blocks->z, threads->x, threads->y, threads->z);

}

void test_dot_product() {
	const size_t ARRAY_SIZE = 3333000;	//1024 * 1024; //1Mi elements
//	const int BLOCKS = 64;	//512;
//	const int THREADS_PER_BLOCK = BLOCK_SIZE;//256; // total threads = 512 x 256 = 128ki threads;
	const size_t SIZE = ARRAY_SIZE * sizeof(float);
	float *dev_v1;
	float *dev_v2; // vector 2
	float* dev_out; // result array, final result is at position 0;
	hipMallocManaged(&dev_v1, SIZE);
	hipMallocManaged(&dev_v2, SIZE);

	// host storage
	std::vector<float> host_v1(ARRAY_SIZE);
	std::vector<float> host_v2(ARRAY_SIZE);

	dim3 blocks, threads;
	cuda_gridsize(&threads, &blocks, ARRAY_SIZE, 1, 1);

	hipMallocManaged(&dev_out, sizeof(float));


	hipFree(dev_v1);
	hipFree(dev_v2);
	hipFree(dev_out);
}

void forward_maxpool_layer_gpu() {
//
////
//	size_t out_width = 2;
//	size_t out_height = 2;
//	size_t out_depth = 1;
//	size_t in_height = 8;
//	size_t in_width = 8;
//	size_t bytes = sizeof(float);
//
//	float *input, *output, *max_loc;
//	hipMalloc(&input, bytes * in_height * in_width);
//	hipMalloc(&output, bytes * out_depth * out_height * out_width);
//	hipMalloc(&max_loc, bytes * in_height * in_width);
//
//	dim3 blocks, threads;
//	cuda_gridsize(&threads, &blocks, in_width, in_height, out_depth);
//
//	//fill first
//	fill<<<1, in_height * in_width>>>(input);
//
//	float host_input[in_height * in_width];
//	hipMemcpy(host_input, input, bytes * in_height * in_width,
//			hipMemcpyDeviceToHost);
//	print_matrix(host_input, in_height, in_width);
//
//	forward_maxpool_layer_kernel<<<blocks, threads>>>(input, max_loc, output,
//			out_width, out_height, out_depth, in_height, in_width);
//
//	float host_out[out_width * out_height * out_depth];
//
//	hipMemcpy(host_out, output, bytes * out_depth * out_height * out_width,
//			hipMemcpyDeviceToHost);
//
//	print_matrix(host_out, out_height, out_width);
//
//	hipError_t ret = hipDeviceSynchronize();
//	CUDA_CHECK_RETURN(ret);
//
//	hipFree(input);
//	hipFree(output);
//	hipFree(max_loc);
}


void test_device_vector(){
	const int siz = 1024;
	float t[siz];

	std::vector<float> t2(siz);

	for(int i = 0; i < siz; i++){
		t[i] = i * 12;
		t2[i] = 0;
	}

	DeviceVector<float> v;
	DeviceVector<float> v2(siz);
	DeviceVector<float> v3(t, siz);
	DeviceVector<float> v4;
	v4 = t2;
	for(int i = 0; i < siz; i++){
		std::cout << v2[i] << " " << v3[i] << " " << v4[i] << " " << std::endl;
	}

}

int main(int argc, char **argv) {

	test_device_vector();

	return 0;
}

