#include "hip/hip_runtime.h"
/*
 * debug.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: carol
 *
 *      this file is only for debugging
 *      classes
 */
#include <iostream>
#include <vector>
#include <cstdio>



// cpu implementation of dot product
float dot(const float* v1, const float* v2, int N) {
	float s = 0;
	for (int i = 0; i != N; ++i) {
		s += v1[i] * v2[i];
	}
	return s;
}

void print_matrix(float *m, size_t h, size_t w) {
	printf("matxix\n");
	for (int i = 0; i < h; i++) {
		for (int j = 0; j < w; j++) {
			printf("%f ", m[i * w + j]);
		}
		printf("\n");
	}

}

void cuda_gridsize(dim3 *threads, dim3 *blocks, size_t x, size_t y, size_t z) {

	long blocks_x = ceil(float(x) / float(BLOCK_SIZE));
	long threads_x = ceil(float(x) / float(blocks_x));
	long blocks_y = ceil(float(y) / float(BLOCK_SIZE));
	long threads_y = ceil(float(y) / float(blocks_y));
	long blocks_z = ceil(float(z) / float(BLOCK_SIZE));
	long threads_z = ceil(float(z) / float(blocks_z));

	*blocks = dim3(blocks_x, blocks_y, blocks_z);
	*threads = dim3(threads_x, threads_y, threads_z);

	printf("b_x %d b_y %d b_z %d\nt_x %d t_y %d t_z %d\n", blocks->x, blocks->y,
			blocks->z, threads->x, threads->y, threads->z);

}

void test_dot_product() {
	const size_t ARRAY_SIZE = 3333000;	//1024 * 1024; //1Mi elements
//	const int BLOCKS = 64;	//512;
//	const int THREADS_PER_BLOCK = BLOCK_SIZE;//256; // total threads = 512 x 256 = 128ki threads;
	const size_t SIZE = ARRAY_SIZE * sizeof(float);
	float *dev_v1;
	float *dev_v2; // vector 2
	float* dev_out; // result array, final result is at position 0;
	hipMallocManaged(&dev_v1, SIZE);
	hipMallocManaged(&dev_v2, SIZE);

	// host storage
	std::vector<float> host_v1(ARRAY_SIZE);
	std::vector<float> host_v2(ARRAY_SIZE);

	dim3 blocks, threads;
	cuda_gridsize(&threads, &blocks, ARRAY_SIZE, 1, 1);
//	const int BLOCKS = blocks.x;
	hipMallocManaged(&dev_out, sizeof(float));

	fill<<<blocks, threads>>>(dev_v1, 2, ARRAY_SIZE);
	hipMemcpy(host_v1.data(), dev_v1, SIZE, hipMemcpyDeviceToHost);

	// initialize vector 2 with kernel; much faster than using for loops on the cpu
	fill<<<blocks, threads>>>(dev_v2, 1, ARRAY_SIZE);
	hipMemcpy(host_v2.data(), dev_v2, SIZE, hipMemcpyDeviceToHost);

	full_dot<BLOCK_SIZE><<<blocks, threads>>>(dev_v1, dev_v2, dev_out, ARRAY_SIZE);
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	hipDeviceSynchronize();

	std::cout << "GPU: " << *dev_out << std::endl;
	std::cout << "CPU: " << dot(host_v1.data(), host_v2.data(), ARRAY_SIZE)
			<< std::endl;
	hipFree(dev_v1);
	hipFree(dev_v2);
	hipFree(dev_out);
}

void forward_maxpool_layer_gpu() {
//
////
//	size_t out_width = 2;
//	size_t out_height = 2;
//	size_t out_depth = 1;
//	size_t in_height = 8;
//	size_t in_width = 8;
//	size_t bytes = sizeof(float);
//
//	float *input, *output, *max_loc;
//	hipMalloc(&input, bytes * in_height * in_width);
//	hipMalloc(&output, bytes * out_depth * out_height * out_width);
//	hipMalloc(&max_loc, bytes * in_height * in_width);
//
//	dim3 blocks, threads;
//	cuda_gridsize(&threads, &blocks, in_width, in_height, out_depth);
//
//	//fill first
//	fill<<<1, in_height * in_width>>>(input);
//
//	float host_input[in_height * in_width];
//	hipMemcpy(host_input, input, bytes * in_height * in_width,
//			hipMemcpyDeviceToHost);
//	print_matrix(host_input, in_height, in_width);
//
//	forward_maxpool_layer_kernel<<<blocks, threads>>>(input, max_loc, output,
//			out_width, out_height, out_depth, in_height, in_width);
//
//	float host_out[out_width * out_height * out_depth];
//
//	hipMemcpy(host_out, output, bytes * out_depth * out_height * out_width,
//			hipMemcpyDeviceToHost);
//
//	print_matrix(host_out, out_height, out_width);
//
//	hipError_t ret = hipDeviceSynchronize();
//	CUDA_CHECK_RETURN(ret);
//
//	hipFree(input);
//	hipFree(output);
//	hipFree(max_loc);
}

int main(int argc, char **argv) {

//	std::string opt(argv[1]);
	test_dot_product();
//	if (opt == "maxpool") {
//		forward_maxpool_layer_gpu();
//	} else if (opt == "device_vector") {
//
//	}

	return 0;
}

