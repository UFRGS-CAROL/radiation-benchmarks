#include "hip/hip_runtime.h"
#include "cudaKernels.h"
#include "hip/hip_runtime.h"
#include "cudaUtil.h"
#include <hipblas.h>

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
		const int height, const int width, const int ksize, const int pad,
		const int stride, const int height_col, const int width_col,
		float *data_col) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (; index < n; index += blockDim.x * gridDim.x) {
		int w_out = index % width_col;
		int h_index = index / width_col;
		int h_out = h_index % height_col;
		int channel_in = h_index / height_col;
		int channel_out = channel_in * ksize * ksize;
		int h_in = h_out * stride - pad;
		int w_in = w_out * stride - pad;
		float* data_col_ptr = data_col;
		data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
		const float* data_im_ptr = data_im;
		data_im_ptr += (channel_in * height + h_in) * width + w_in;
		for (int i = 0; i < ksize; ++i) {
			for (int j = 0; j < ksize; ++j) {
				int h = h_in + i;
				int w = w_in + j;

				*data_col_ptr =
						(h >= 0 && w >= 0 && h < height && w < width) ?
								data_im_ptr[i * width + j] : 0;

				//*data_col_ptr = data_im_ptr[ii * width + jj];

				data_col_ptr += height_col * width_col;
			}
		}
	}
}

void im2col_ongpu(float *im, int channels, int height, int width, int ksize,
		int stride, int pad, float *data_col) {
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int num_kernels = channels * height_col * width_col;
	im2col_gpu_kernel<<<(num_kernels + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(num_kernels,
			im, height, width, ksize, pad, stride, height_col, width_col,
			data_col);
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void col2im_gpu_kernel(const int n, const float* data_col,
		const int height, const int width, const int ksize, const int pad,
		const int stride, const int height_col, const int width_col,
		float *data_im) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	for (; index < n; index += blockDim.x * gridDim.x) {
		float val = 0;
		int w = index % width + pad;
		int h = (index / width) % height + pad;
		int c = index / (width * height);
		// compute the start and end of the output
		int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
		int w_col_end = min(w / stride + 1, width_col);
		int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
		int h_col_end = min(h / stride + 1, height_col);
		// equivalent implementation
		int offset = (c * ksize * ksize + h * ksize + w) * height_col
				* width_col;
		int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
		int coeff_w_col = (1 - stride * height_col * width_col);
		for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
			for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
				val += data_col[offset + h_col * coeff_h_col
						+ w_col * coeff_w_col];
			}
		}
		data_im[index] += val;
	}
}

void col2im_ongpu(float *data_col, int channels, int height, int width,
		int ksize, int stride, int pad, float *data_im) {
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int num_kernels = channels * height * width;
	col2im_gpu_kernel<<<(num_kernels + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(num_kernels,
			data_col, height, width, ksize, pad, stride, height_col, width_col,
			data_im);
	CudaCheckError() ;
}

void gemm_ongpu(int TA, int TB, int M, int N, int K, float ALPHA, float *A_gpu,
		int lda, float *B_gpu, int ldb, float BETA, float *C_gpu, int ldc) {

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t status = hipblasSgemm(handle, (TB ? HIPBLAS_OP_T : HIPBLAS_OP_N),
			(TA ? HIPBLAS_OP_T : HIPBLAS_OP_N), N, M, K, &ALPHA, B_gpu, ldb,
			A_gpu, lda, &BETA, C_gpu, ldc);
	CudaCheckError() ;
	hipblasDestroy(handle);
}
