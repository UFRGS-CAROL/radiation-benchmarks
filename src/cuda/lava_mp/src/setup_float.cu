#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "setup_template.h"
#include "setup.h"
#include "KernelCaller.h"

#include "Parameters.h"
#include "Log.h"

/**
 * Define the threshold to use on
 * the comparison method
 */
//For 1 iteration
//00000000 01001100 00000000 00000000
#define THRESHOLD_1 5111808
#define ONE_BLOCK 1

//For 10 iterations
//3e-05
#define THRESHOLD_12 67108864
#define TWELVE_BLOCK 12

//For 100 iterations
//9e-05
#define THRESHOLD_96 5111808
#define NINETY_BLOCK 96

//For MAX_LAVA iterations
//4e-03
#define THRESHOLD_MAX 4194304
#define MAX_BLOCK NUMBER_PAR_PER_BOX

void setup_float(Parameters& parameters, Log& log) {

	switch (parameters.redundancy) {
//	case NONE: {
//		UnhardenedKernelCaller<float> kc;
//		setup_execution(parameters, log, kc);
//		break;
//	}
	/*
	case DMR: {
		switch (parameters.block_check) {
		case ONE_BLOCK: {
			//CASE FOR 1 Iteration-------------------
			DMRKernelCaller<ONE_BLOCK, double> kc;
			setup_execution(parameters, log, kc);

			break;
		}
			//---------------------------------------
		case TWELVE_BLOCK: {
			//CASE FOR 10 Iterations-----------------
			DMRKernelCaller<TWELVE_BLOCK, double> kc;
			setup_execution(parameters, log, kc);

			break;
		}
			//---------------------------------------

//		case NINETY_BLOCK: {
//			//CASE FOR 100 Iterations----------------
//			DMRKernelCaller<NINETY_BLOCK, double> kc;
//			setup_execution(parameters, log, kc);
//
//			break;
//		}
			//---------------------------------------

		case MAX_BLOCK: {
			//CASE FOR 100 Iterations----------------
			DMRKernelCaller<MAX_BLOCK, double> kc;
			setup_execution(parameters, log, kc);

			break;
		}
			//---------------------------------------

		default:
			error(
					std::to_string(parameters.block_check)
							+ " operation check block not supported");
		}
		break;
	}*/
	case DMRMIXED:
		switch (parameters.block_check) {
//		case ONE_BLOCK: {
//			//CASE FOR 1 Iteration-------------------
//			DMRMixedKernelCaller<ONE_BLOCK, double, float> kc(THRESHOLD_1);
//			setup_execution(parameters, log, kc);
//
//			break;
//		}
			//---------------------------------------
//		case TWELVE_BLOCK: {
//			//CASE FOR 10 Iterations-----------------
//			DMRMixedKernelCaller<TWELVE_BLOCK, THRESHOLD_12, float, double> kc;
//			setup_execution(parameters, log, kc);
//
//			break;
//		}
			//---------------------------------------
//
//		case NINETY_BLOCK: {
//			//CASE FOR 100 Iterations----------------
//			DMRMixedKernelCaller<NINETY_BLOCK, THRESHOLD_96, float, double> kc;
//			setup_execution(parameters, log, kc);
//
//			break;
//		}
			//---------------------------------------
//
//		case MAX_BLOCK: {
//			//CASE FOR 100 Iterations----------------
//			DMRMixedKernelCaller<MAX_BLOCK, THRESHOLD_MAX, float, double> kc;
//			setup_execution(parameters, log, kc);
//
//			break;
//		}
			//---------------------------------------

		default:
			error(
					std::to_string(parameters.block_check)
							+ " operation check block not supported");
		}
		break;
	}
}
