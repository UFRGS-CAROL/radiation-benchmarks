#include "hip/hip_runtime.h"
#define LIMIT -999
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <iostream>
#include <math.h>
#include "needle.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define GCHK_BLOCK_SIZE 32

// includes, kernels
#include "needle_kernel.cu"

//================== log include
#ifdef LOGS
#include "log_helper.h"
#endif
//====================================

///////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

#define N_ERRORS_LOG 500
#define ITERATIONS 1

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort =
		true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
				line);
		if (abort)
			exit(code);
	}
}
int blosum62[24][24] = { { 4, -1, -2, -2, 0, -1, -1, 0, -2, -1, -1, -1, -1, -2,
		-1, 1, 0, -3, -2, 0, -2, -1, 0, -4 }, { -1, 5, 0, -2, -3, 1, 0, -2, 0,
		-3, -2, 2, -1, -3, -2, -1, -1, -3, -2, -3, -1, 0, -1, -4 },
		{ -2, 0, 6, 1, -3, 0, 0, 0, 1, -3, -3, 0, -2, -3, -2, 1, 0, -4, -2, -3,
				3, 0, -1, -4 }, { -2, -2, 1, 6, -3, 0, 2, -1, -1, -3, -4, -1,
				-3, -3, -1, 0, -1, -4, -3, -3, 4, 1, -1, -4 }, { 0, -3, -3, -3,
				9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1,
				-3, -3, -2, -4 }, { -1, 1, 0, 0, -3, 5, 2, -2, 0, -3, -2, 1, 0,
				-3, -1, 0, -1, -2, -1, -2, 0, 3, -1, -4 },
		{ -1, 0, 0, 2, -4, 2, 5, -2, 0, -3, -3, 1, -2, -3, -1, 0, -1, -3, -2,
				-2, 1, 4, -1, -4 }, { 0, -2, 0, -1, -3, -2, -2, 6, -2, -4, -4,
				-2, -3, -3, -2, 0, -2, -2, -3, -3, -1, -2, -1, -4 }, { -2, 0, 1,
				-1, -3, 0, 0, -2, 8, -3, -3, -1, -2, -1, -2, -1, -2, -2, 2, -3,
				0, 0, -1, -4 }, { -1, -3, -3, -3, -1, -3, -3, -4, -3, 4, 2, -3,
				1, 0, -3, -2, -1, -3, -1, 3, -3, -3, -1, -4 }, { -1, -2, -3, -4,
				-1, -2, -3, -4, -3, 2, 4, -2, 2, 0, -3, -2, -1, -2, -1, 1, -4,
				-3, -1, -4 }, { -1, 2, 0, -1, -3, 1, 1, -2, -1, -3, -2, 5, -1,
				-3, -1, 0, -1, -3, -2, -2, 0, 1, -1, -4 }, { -1, -1, -2, -3, -1,
				0, -2, -3, -2, 1, 2, -1, 5, 0, -2, -1, -1, -1, -1, 1, -3, -1,
				-1, -4 }, { -2, -3, -3, -3, -2, -3, -3, -3, -1, 0, 0, -3, 0, 6,
				-4, -2, -2, 1, 3, -1, -3, -3, -1, -4 }, { -1, -2, -2, -1, -3,
				-1, -1, -2, -2, -3, -3, -1, -2, -4, 7, -1, -1, -4, -3, -2, -2,
				-1, -2, -4 }, { 1, -1, 1, 0, -1, 0, 0, 0, -1, -2, -2, 0, -1, -2,
				-1, 4, 1, -3, -2, -2, 0, 0, 0, -4 },
		{ 0, -1, 0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1, 1, 5, -2,
				-2, 0, -1, -1, 0, -4 }, { -3, -3, -4, -4, -2, -2, -3, -2, -2,
				-3, -2, -3, -1, 1, -4, -3, -2, 11, 2, -3, -4, -3, -2, -4 }, {
				-2, -2, -2, -3, -2, -1, -2, -3, 2, -1, -1, -2, -1, 3, -3, -2,
				-2, 2, 7, -1, -3, -2, -1, -4 }, { 0, -3, -3, -3, -1, -2, -2, -3,
				-3, 3, 1, -2, 1, -1, -2, -2, 0, -3, -1, 4, -3, -2, -1, -4 }, {
				-2, -1, 3, 4, -3, 0, 1, -1, 0, -3, -4, 0, -3, -3, -2, 0, -1, -4,
				-3, -3, 4, 1, -1, -4 }, { -1, 0, 0, 1, -3, 3, 4, -2, 0, -3, -3,
				1, -1, -3, -1, 0, -1, -3, -2, -2, 1, 4, -1, -4 }, { 0, -1, -1,
				-1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2, 0, 0, -2, -1,
				-1, -1, -1, -1, -4 }, { -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
				-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, 1 } };

double gettime() {
	struct timeval t;
	gettimeofday(&t, NULL);
	return t.tv_sec + t.tv_usec * 1e-6;
}

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void ReadArrayFromFile(int* input_itemsets, int* gold_itemsets, char** argv) {
	double time = mysecond();
	int n = atoi(argv[1]) + 1;

	std::string filenameinput(
			"/home/carol/TestGPU/GenerateNeedleArray/NeedleInput_");
	filenameinput += argv[1];
	filenameinput += ".array";
	std::string filenamegold(
			"/home/carol/TestGPU/GenerateNeedleArray/NeedleGOLD_");
	filenamegold += argv[1];
	filenamegold += ".array";

	std::cout << "open array...";

	FILE *f_a, *f_gold;
	f_a = fopen(filenameinput.c_str(), "rb");
	f_gold = fopen(filenamegold.c_str(), "rb");

	if ((f_a == NULL) || (f_gold == NULL)) {
		std::cout << "error.\n";
		exit(-3);
	}

	std::cout << "read...";
	fread(input_itemsets, sizeof(int) * n * n, 1, f_a);
	fread(gold_itemsets, sizeof(int) * n * n, 1, f_gold);
	fclose(f_a);
	fclose(f_gold);

	printf("ok in %f\n", mysecond() - time);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	printf("WG size of kernel = %d \n", BLOCK_SIZE);

	runTest(argc, argv);

	return EXIT_SUCCESS;
}

void UpdateTimestamp() {
	time_t timestamp = time(NULL);
	char time_s[50];
	sprintf(time_s, "%d", int(timestamp));

	char string[100] = "echo ";
	strcat(string, time_s);
	strcat(string, " > /home/carol/TestGPU/timestamp.txt");
	system(string);
}

void usage(int argc, char **argv) {
	fprintf(stderr, "Usage: %s <max_rows/max_cols> <penalty> \n", argv[0]);
	fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
	fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
	exit(1);
}

void runTest(int argc, char** argv) {
	int max_rows, max_cols, penalty;
	int *input_itemsets, *output_itemsets, *gold_itemsets, *referrence;
	int *matrix_cuda, *referrence_cuda;
	int size;
	int zero = 0;
	double timeG;

	// the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 3) {
		max_rows = atoi(argv[1]);
		max_cols = atoi(argv[1]);
		penalty = atoi(argv[2]);
	} else {
		usage(argc, argv);
	}

	int n = atoi(argv[1]) + 1;

	if (atoi(argv[1]) % 16 != 0) {
		fprintf(stderr, "The dimension values must be a multiple of 16\n");
		exit(1);
	}

	//////////BLOCK and GRID size for goldchk////////////
	int gchk_gridsize = n / GCHK_BLOCK_SIZE < 1 ? 1 : n / GCHK_BLOCK_SIZE;
	int gchk_blocksize = n / GCHK_BLOCK_SIZE < 1 ? n : GCHK_BLOCK_SIZE;
	dim3 gchk_dimBlock(gchk_blocksize, gchk_blocksize);
	dim3 gchk_dimGrid(gchk_gridsize, gchk_gridsize);
	////////////////////////////////////////////////////

	// Log files
	/*FILE* file;
	 FILE* log_file;
	 */
	//================== Init logs
#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "max_rows:%d max_cols:%d penalty:%d", max_rows, max_cols, penalty);
	start_log_file("cudaNW", test_info);
#endif
	//====================================
	int ea = 0; //wrong integers in the current loop
	int t_ea = 0; //total number of wrong integers
	int old_ea = 0;

	double total_time = 0.0;
	////////////////FILE NAME//////////////////////////////
	/*time_t file_time;
	 struct tm *ptm;
	 char day[2], month[2], year[4], hour[2], second[2], minute[2];
	 char file_name[60];
	 char file_name_log[60];

	 file_time = time(NULL);
	 ptm = gmtime(&file_time);

	 std::string logfile_matrixname("cudaNW");
	 logfile_matrixname += argv[1];

	 snprintf(day, sizeof(day + 1), "%d", ptm->tm_mday);
	 snprintf(month, sizeof(month + 1), "%d", ptm->tm_mon + 1);
	 snprintf(year, sizeof(year + 1), "%d", ptm->tm_year + 1900);
	 snprintf(hour, sizeof(hour + 1), "%d", ptm->tm_hour);
	 snprintf(minute, sizeof(minute + 1), "%d", ptm->tm_min);
	 snprintf(second, sizeof(second + 1), "%d", ptm->tm_sec);
	 strcpy(file_name, day);
	 strcat(file_name, "_");
	 strcat(file_name, month);
	 strcat(file_name, "_");
	 strcat(file_name, year);
	 strcat(file_name, "_");
	 strcat(file_name, hour);
	 strcat(file_name, "_");
	 strcat(file_name, minute);
	 strcat(file_name, "_");
	 strcat(file_name, second);
	 strcat(file_name, "_");
	 strcat(file_name, logfile_matrixname.c_str());
	 strcpy(file_name_log, file_name);

	 strcat(file_name, ".txt");
	 strcat(file_name_log, "log.txt");
	 *///////////////////////////////////////////////////////
	max_rows = max_rows + 1;
	max_cols = max_cols + 1;
	referrence = (int *) malloc(max_rows * max_cols * sizeof(int));
	input_itemsets = (int *) malloc(max_rows * max_cols * sizeof(int));
	output_itemsets = (int *) malloc(max_rows * max_cols * sizeof(int));
	gold_itemsets = (int *) malloc(max_rows * max_cols * sizeof(int));

	int *kerrors;
	kerrors = (int*) malloc(sizeof(int));

	if (!input_itemsets)
		fprintf(stderr, "error: can not allocate memory");

	printf("Start Needleman-Wunsch\n");

	ReadArrayFromFile(input_itemsets, gold_itemsets, argv);

	/*    	srand ( time(NULL) );
	 std::cout << "Original -  input : " << input_itemsets[1*max_cols+0] << "\treference : " << blosum62[input_itemsets[1*max_cols]][input_itemsets[1]] << "\n";
	 input_itemsets[1*max_cols+0] = rand () % 10 + 1;
	 std::cout << "Modified -  input : " << input_itemsets[1*max_cols+0] << "\treference : " << blosum62[input_itemsets[1*max_cols]][input_itemsets[1]] << "\n";
	 */
	/*for (int i = 0 ; i < max_cols; i++)
	 {
	 //getchar();
	 for (int j = 0 ; j < max_rows; j++)
	 std::cout << "[" << i << "][" << j << "] : " << input_itemsets[i*max_cols+j] << "\t";
	 }*/
	for (int i = 1; i < max_cols; i++) {
		for (int j = 1; j < max_rows; j++) {
			referrence[i * max_cols + j] =
					blosum62[input_itemsets[i * max_cols]][input_itemsets[j]];
		}
	}
	for (int i = 1; i < max_rows; i++)
		input_itemsets[i * max_cols] = -i * penalty;
	for (int j = 1; j < max_cols; j++)
		input_itemsets[j] = -j * penalty;

	size = max_cols * max_rows;

	for (int loop2 = 0; loop2 < ITERATIONS; loop2++) {
		//file = fopen(file_name, "a");
		//std::cout << "Allocating matrixes on GPU...";
		hipMalloc((void**) &referrence_cuda, sizeof(int) * size);
		hipMalloc((void**) &matrix_cuda, sizeof(int) * size);
		if ((referrence_cuda == NULL) || (matrix_cuda == NULL)) {
			std::cout << "error.\n";
			exit(-3);
		}
		//std::cout << "Done\n";
		//std::cout << "Sending matrixes to GPU...";

		timeG = mysecond();

		hipMemcpy(referrence_cuda, referrence, sizeof(int) * size,
				hipMemcpyHostToDevice);
		hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size,
				hipMemcpyHostToDevice);
		timeG = mysecond() - timeG;

		//std::cout << "Done in " << timeG << "s.\nRunning Needleman-Wunsch...";

		dim3 dimGrid;
		dim3 dimBlock(BLOCK_SIZE, 1);
		int block_width = (max_cols - 1) / BLOCK_SIZE;

		timeG = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//printf("Processing top-left matrix\n");
		//process top-left matrix
		for (int i = 1; i <= block_width; i++) {
			dimGrid.x = i;
			dimGrid.y = 1;
			needle_cuda_shared_1<<<dimGrid, dimBlock>>>(referrence_cuda,
					matrix_cuda, max_cols, penalty, i, block_width);
		}
		//printf("Processing bottom-right matrix\n");
		//process bottom-right matrix
		for (int i = block_width - 1; i >= 1; i--) {
			dimGrid.x = i;
			dimGrid.y = 1;
			needle_cuda_shared_2<<<dimGrid, dimBlock>>>(referrence_cuda,
					matrix_cuda, max_cols, penalty, i, block_width);
		}
#ifdef LOGS
		end_iteration();
#endif
		timeG = mysecond() - timeG;
		total_time += timeG;

		//std::cout << "Done in " << timeG << "s.\n";

		*kerrors = 0;
		// Check errors on GPU...
		//std::cout << "Sending gold matrix to GPU...";
		timeG = mysecond();
		hipMemcpy(referrence_cuda, gold_itemsets, sizeof(int) * size,
				hipMemcpyHostToDevice);
		// Using referrence just to avoid reallocation for gold
		hipMemcpyToSymbol(HIP_SYMBOL(gpukerrors), &zero, sizeof(int));
		timeG = mysecond() - timeG;
		//std::cout << "Done in " << timeG << "s.\nRunning GoldChk...";
		timeG = mysecond();

		GoldChkKernel<<<gchk_dimGrid, gchk_dimBlock>>>(referrence_cuda,
				matrix_cuda, n);

		timeG = mysecond() - timeG;
		//std::cout << "Done in " << timeG << "s.";
		gpuErrchk(hipPeekAtLastError());
		hipMemcpyFromSymbol(kerrors, HIP_SYMBOL(gpukerrors), sizeof(unsigned int));

		//std::cout << "Errors : " << *kerrors << "\n";

		///////////UPDATE FILE//////////////////////
		/*file_time = time(NULL);
		 ptm = gmtime(&file_time);
		 snprintf(hour, sizeof(hour + 1), "%d", ptm->tm_hour);
		 snprintf(minute, sizeof(minute + 1), "%d", ptm->tm_min);
		 snprintf(second, sizeof(second + 1), "%d", ptm->tm_sec);
		 fprintf(file, "\n start time: %s/%s_%s:%s:%s", day, month, hour, minute,
		 second);
		 fclose (file);
		 */

		ea = 0;

		/////////////UPDATE TIMESTAMP///////////////////
		//UpdateTimestamp();
		////////////////////////////////////////////////

		if (*kerrors > 0) {
			//file = fopen(file_name, "a");
			//std::cout <<  << *kerrors << "\n";
			std::stringstream sstm;
			sstm << "Error detected! kerrors = " << *kerrors;
#ifdef LOGS
			log_error_detail(sstm.str().c_str());
#endif
			hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size,
					hipMemcpyDeviceToHost);
			int host_errors =0;
			for (int i = 0; (i < n) && (ea < N_ERRORS_LOG); i++) {
				for (int j = 0; (j < n) && (ea < N_ERRORS_LOG); j++) {
					if (output_itemsets[i + n * j]
							!= gold_itemsets[i + n * j]) {
						ea++;
						char error_detail[200];
						sprintf(error_detail,
								"\n p: [%d, %d], r: %i, e: %i, error: %d\n", i,
								j, output_itemsets[i + n * j],
								gold_itemsets[i + n * j], ea);
#ifdef LOGS
						log_error_detail(error_detail);
						host_errors++;
#endif

					}
				}
			}
			t_ea += *kerrors;

			///////////UPDATE LOG FILE//////////////////////
			/*log_file = fopen(file_name_log, "a");
			fprintf(log_file, "\ntest number: %d", loop2);
			fprintf(log_file, "\ntime: %f", timeG);
			fprintf(log_file, "\ntotal time: %f", total_time);
			fprintf(log_file, "\nerrors: %d", *kerrors);
			fprintf(log_file, "\ntotal errors: %d", t_ea);
			fclose (log_file);
			fclose (file);
			*/
#ifdef LOGS
			log_error_count(host_errors);
#endif
		}

		if (*kerrors > 0 || (loop2 % 10 == 0)) {
			printf("\ntest number: %d", loop2);
			printf("\ntotal time: %f", total_time);
			printf("\nerrors: %d", *kerrors);
			printf("\ntotal errors: %d\n", t_ea);
			if ((*kerrors != 0) && (*kerrors == old_ea)) {
				old_ea = 0;
				return;
			}

			old_ea = *kerrors;
		} else {
			printf(".");
		}
		hipFree(referrence_cuda);
		hipFree(matrix_cuda);
	}
#ifdef LOGS
	end_log_file();
#endif
	free(referrence);
	free(input_itemsets);
	free(output_itemsets);

}

