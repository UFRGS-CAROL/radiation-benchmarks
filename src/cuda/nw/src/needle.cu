#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <vector>
#include <fstream>
#include <omp.h>
#include <numeric>

//helper kernels
#include "include/cuda_utils.h"
#include "include/device_vector.h"

// includes, kernels
#include "needle.h"
//================== log include
//#ifdef LOGS
//#include "log_helper.h"
//#endif
#include "include/generic_log.h"
//====================================

#define GCHK_BLOCK_SIZE 32
#define MAX_VALUE_NW 24
#define LIMIT -999
#define N_ERRORS_LOG 500
#define ITERATIONS 100
#define CHAR_CAST(x) (reinterpret_cast<char*>(x))
#define CONST_CAST(x) (const_cast<char*>(x))

int blosum62[24][24] = { { 4, -1, -2, -2, 0, -1, -1, 0, -2, -1, -1, -1, -1, -2,
		-1, 1, 0, -3, -2, 0, -2, -1, 0, -4 }, { -1, 5, 0, -2, -3, 1, 0, -2, 0,
		-3, -2, 2, -1, -3, -2, -1, -1, -3, -2, -3, -1, 0, -1, -4 },
		{ -2, 0, 6, 1, -3, 0, 0, 0, 1, -3, -3, 0, -2, -3, -2, 1, 0, -4, -2, -3,
				3, 0, -1, -4 }, { -2, -2, 1, 6, -3, 0, 2, -1, -1, -3, -4, -1,
				-3, -3, -1, 0, -1, -4, -3, -3, 4, 1, -1, -4 }, { 0, -3, -3, -3,
				9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1,
				-3, -3, -2, -4 }, { -1, 1, 0, 0, -3, 5, 2, -2, 0, -3, -2, 1, 0,
				-3, -1, 0, -1, -2, -1, -2, 0, 3, -1, -4 },
		{ -1, 0, 0, 2, -4, 2, 5, -2, 0, -3, -3, 1, -2, -3, -1, 0, -1, -3, -2,
				-2, 1, 4, -1, -4 }, { 0, -2, 0, -1, -3, -2, -2, 6, -2, -4, -4,
				-2, -3, -3, -2, 0, -2, -2, -3, -3, -1, -2, -1, -4 }, { -2, 0, 1,
				-1, -3, 0, 0, -2, 8, -3, -3, -1, -2, -1, -2, -1, -2, -2, 2, -3,
				0, 0, -1, -4 }, { -1, -3, -3, -3, -1, -3, -3, -4, -3, 4, 2, -3,
				1, 0, -3, -2, -1, -3, -1, 3, -3, -3, -1, -4 }, { -1, -2, -3, -4,
				-1, -2, -3, -4, -3, 2, 4, -2, 2, 0, -3, -2, -1, -2, -1, 1, -4,
				-3, -1, -4 }, { -1, 2, 0, -1, -3, 1, 1, -2, -1, -3, -2, 5, -1,
				-3, -1, 0, -1, -3, -2, -2, 0, 1, -1, -4 }, { -1, -1, -2, -3, -1,
				0, -2, -3, -2, 1, 2, -1, 5, 0, -2, -1, -1, -1, -1, 1, -3, -1,
				-1, -4 }, { -2, -3, -3, -3, -2, -3, -3, -3, -1, 0, 0, -3, 0, 6,
				-4, -2, -2, 1, 3, -1, -3, -3, -1, -4 }, { -1, -2, -2, -1, -3,
				-1, -1, -2, -2, -3, -3, -1, -2, -4, 7, -1, -1, -4, -3, -2, -2,
				-1, -2, -4 }, { 1, -1, 1, 0, -1, 0, 0, 0, -1, -2, -2, 0, -1, -2,
				-1, 4, 1, -3, -2, -2, 0, 0, 0, -4 },
		{ 0, -1, 0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1, 1, 5, -2,
				-2, 0, -1, -1, 0, -4 }, { -3, -3, -4, -4, -2, -2, -3, -2, -2,
				-3, -2, -3, -1, 1, -4, -3, -2, 11, 2, -3, -4, -3, -2, -4 }, {
				-2, -2, -2, -3, -2, -1, -2, -3, 2, -1, -1, -2, -1, 3, -3, -2,
				-2, 2, 7, -1, -3, -2, -1, -4 }, { 0, -3, -3, -3, -1, -2, -2, -3,
				-3, 3, 1, -2, 1, -1, -2, -2, 0, -3, -1, 4, -3, -2, -1, -4 }, {
				-2, -1, 3, 4, -3, 0, 1, -1, 0, -3, -4, 0, -3, -3, -2, 0, -1, -4,
				-3, -3, 4, 1, -1, -4 }, { -1, 0, 0, 1, -3, 3, 4, -2, 0, -3, -3,
				1, -1, -3, -1, 0, -1, -3, -2, -2, 1, 4, -1, -4 }, { 0, -1, -1,
				-1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2, 0, 0, -2, -1,
				-1, -1, -1, -1, -4 }, { -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
				-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, 1 } };

template<typename T>
bool read_from_file(std::string& path, std::vector<T>& array) {
	std::ifstream input(path, std::ios::binary);
	if (input.good()) {
		input.read(CHAR_CAST(array.data()), array.size() * sizeof(T));
		input.close();
		return true;
	}
	return false;
}

template<typename T>
bool write_to_file(std::string& path, std::vector<T>& array) {
	std::ofstream output(path, std::ios::binary);
	if (output.good()) {
		output.write(CHAR_CAST(array.data()), array.size() * sizeof(T));
		output.close();
		return true;
	}
	return false;
}

void GenerateInputFile(std::vector<int>& input_itemsets,
		std::string filenameinput) {
	std::cout << "Generating a random array  with size == "
			<< input_itemsets.size() << std::endl;
	for (auto& i : input_itemsets) {
		i = rand() % MAX_VALUE_NW; //24 is from blosum size
	}

	if (write_to_file(filenameinput, input_itemsets) == false) {
		std::cout << "error generating input.\n";
		exit(-3);
	}
}

void WriteGoldToFile(std::vector<int>& gold_array, std::string gold_name,
		int n) {
	if (write_to_file(gold_name, gold_array) == false) {
		std::cout << "error writing gold.\n";
		exit(-3);
	}
}

void ReadArrayFromFile(std::vector<int>& input_itemsets,
		std::vector<int>& gold_itemsets, std::string filenameinput,
		std::string filenamegold) {
	double time = rad::mysecond();
	std::cout << "open array...\n";

	if (read_from_file(filenameinput, input_itemsets) == false
			|| read_from_file(filenamegold, gold_itemsets) == false) {
		std::cout << "error.\n";
		exit(-3);
	}

	std::cout << "read...";
	printf("ok in %f\n", rad::mysecond() - time);
}

bool inline badass_memcmp(std::vector<int>& gold_vector,
		std::vector<int>& found_vector) {
//	uint32_t n = gold_vector.size();
//	uint32_t numthreads = 1;
//#pragma omp parallel
//	{
//		numthreads = omp_get_max_threads();
//	}
//	uint32_t chunk = ceil(float(n) / float(numthreads));
//	static std::vector<uint32_t> reduction_array(numthreads);
//
//#pragma omp parallel default(shared)
//	for (uint32_t i = 0; i < numthreads; i++) {
//		uint32_t slice = i * chunk;
//		reduction_array[i] = std::equal(
//				gold_vector.begin() + slice,
//				gold_vector.begin() + slice + chunk,
//				found_vector.begin() + slice);
//	}
//	uint32_t result = std::accumulate(reduction_array.begin(),
//			reduction_array.end(), 0);
//	return (result != numthreads);
//	return !std::equal(gold_vector.begin(), gold_vector.end(),
//			found_vector.begin());

	return !memcmp(gold_vector.data(), found_vector.data(), gold_vector.size());

}

void usage(int argc, char **argv) {
	fprintf(stderr,
			"Usage: %s <max_rows/max_cols> <penalty> <input_array> <gold_array> <iterations> "
					"<to generate gold 0 or 1>\n", argv[0]);
	fprintf(stderr, "\t<dimension>  - x and y dimensions\n");
	fprintf(stderr, "\t<penalty> - penalty(positive integer)\n");
	exit(1);
}

void runTest(int argc, char** argv) {
	int max_rows, max_cols, penalty;
	int iterations = 1;
	bool generate = false;
	std::string array_path, gold_path;

	// the lengths of the two sequences should be able to divided by 16.
	// And at current stage  max_rows needs to equal max_cols
	if (argc == 7) {
		max_rows = atoi(argv[1]);
		max_cols = atoi(argv[1]);
		penalty = atoi(argv[2]);
		array_path = std::string(argv[3]);
		gold_path = std::string(argv[4]);
		iterations = atoi(argv[5]);
		generate = atoi(argv[6]);
		if (generate)
			iterations = 1;
	} else {
		usage(argc, argv);
	}

	int n = atoi(argv[1]) + 1;

	if (atoi(argv[1]) % 16 != 0) {
		std::cerr << "The dimension values must be a multiple of 16\n";
		exit(1);
	}

	//////////BLOCK and GRID size for goldchk////////////
	int gchk_gridsize = n / GCHK_BLOCK_SIZE < 1 ? 1 : n / GCHK_BLOCK_SIZE;
	int gchk_blocksize = n / GCHK_BLOCK_SIZE < 1 ? n : GCHK_BLOCK_SIZE;
	dim3 gchk_dimBlock(gchk_blocksize, gchk_blocksize);
	dim3 gchk_dimGrid(gchk_gridsize, gchk_gridsize);
	////////////////////////////////////////////////////

	// Log files
	/*FILE* file;
	 FILE* log_file;
	 */
	//================== Init logs
//#ifdef LOGS
////"max_rows:%d max_cols:%d penalty:%d"
//	std::string test_info = "";
//	test_info += "max_rows:" + std::to_string(max_rows) + " ";
//	test_info += "max_cols:" + std::to_string(max_cols) + " ";
//	test_info += "penalty:" + std::to_string(penalty);
//	start_log_file(CONST_CAST("cudaNW"), CONST_CAST(test_info.c_str()));
//#endif
	std::string test_info = "", test_name = "cudaNW";
	test_info += "max_rows:" + std::to_string(max_rows) + " ";
	test_info += "max_cols:" + std::to_string(max_cols) + " ";
	test_info += "penalty:" + std::to_string(penalty);
	rad::Log log(test_name, test_info);
	std::cout << log << std::endl;
	//====================================
	KErrorsType ea = 0; //wrong integers in the current loop
	KErrorsType t_ea = 0; //total number of wrong integers
//	KErrorsType old_ea = 0;

	double total_time = 0.0;

	max_rows++;
	max_cols++;
	int size = max_cols * max_rows;

	std::vector<int> referrence(size);
	std::vector<int> input_itemsets(size);
	std::vector<int> output_itemsets(size);
	std::vector<int> gold_itemsets(size);

	rad::DeviceVector<int> referrence_cuda = referrence;
	rad::DeviceVector<int> matrix_cuda = input_itemsets;
	rad::DeviceVector<int> output_itemsets_cuda = output_itemsets;
	rad::DeviceVector<int> gold_itemsets_cuda = gold_itemsets;

	std::cout << "Starting Needleman-Wunsch" << std::endl;

	if (generate) {
		GenerateInputFile(input_itemsets, array_path);
	} else {
		ReadArrayFromFile(input_itemsets, gold_itemsets, array_path, gold_path);
	}

	for (int i = 1; i < max_cols; i++) {
		for (int j = 1; j < max_rows; j++) {
			referrence[i * max_cols + j] =
					blosum62[input_itemsets[i * max_cols]][input_itemsets[j]];
		}
	}
	for (int i = 1; i < max_rows; i++)
		input_itemsets[i * max_cols] = -i * penalty;
	for (int j = 1; j < max_cols; j++)
		input_itemsets[j] = -j * penalty;

	//Improve performance
	const rad::DeviceVector<int> save_input_itemsets_cuda = input_itemsets;

	for (int loop2 = 0; loop2 < iterations; loop2++) {
		auto mem_cpy_time = rad::mysecond();
		matrix_cuda = save_input_itemsets_cuda;
		mem_cpy_time = rad::mysecond() - mem_cpy_time;

		dim3 dimGrid;
		dim3 dimBlock(BLOCK_SIZE, 1);
		int block_width = (max_cols - 1) / BLOCK_SIZE;

		auto kernel_time = rad::mysecond();
//#ifdef LOGS
//		start_iteration();
//#endif
		log.start_iteration();
		//printf("Processing top-left matrix\n");
		//process top-left matrix
		for (int i = 1; i <= block_width; i++) {
			dimGrid.x = i;
			dimGrid.y = 1;
			needle_cuda_shared_1<<<dimGrid, dimBlock>>>(referrence_cuda.data(),
					matrix_cuda.data(), max_cols, penalty, i, block_width);
		}
		//printf("Processing bottom-right matrix\n");
		//process bottom-right matrix
		for (int i = block_width - 1; i >= 1; i--) {
			dimGrid.x = i;
			dimGrid.y = 1;
			needle_cuda_shared_2<<<dimGrid, dimBlock>>>(referrence_cuda.data(),
					matrix_cuda.data(), max_cols, penalty, i, block_width);
		}
		rad::checkFrameworkErrors(hipDeviceSynchronize());
		rad::checkFrameworkErrors(hipGetLastError());

//#ifdef LOGS
//		end_iteration();
//#endif
		log.end_iteration();
		kernel_time = rad::mysecond() - kernel_time;
		total_time += kernel_time;

		if (generate == false) {
			ea = 0;
			uint32_t host_errors = 0;

			auto copy_time = rad::mysecond();
			matrix_cuda.to_vector(output_itemsets);
			copy_time = rad::mysecond() - copy_time;

			auto cmp_time = rad::mysecond();
			auto is_equal = badass_memcmp(gold_itemsets, output_itemsets);
			cmp_time = rad::mysecond() - cmp_time;

			if (is_equal) {
#pragma omp parallel for default(shared)
				for (int i = 0; (i < n); i++) {
					for (int j = 0; (j < n); j++) {
						auto gold_ij = gold_itemsets[i * n + j];
						auto output_ij = output_itemsets[i * n + j];
						if (output_ij != gold_ij) {

							//p: [%d, %d], r: %i, e: %i, error: %d"
							std::string error_detail = "";
							error_detail += " p: [" + std::to_string(i) + ", "
									+ std::to_string(j) + "],";
							error_detail += " r: " + std::to_string(output_ij)
									+ ",";
							error_detail += " e: " + std::to_string(gold_ij);
//									+ ",";
//							error_detail += " error: " + std::to_string(ea);
#pragma omp critical
							{
								ea++;

//#ifdef LOGS
//								log_error_detail(CONST_CAST(error_detail.c_str()));
								host_errors++;
//#endif
								log.log_error_detail(error_detail);
							}

						}
					}
				}
				t_ea += host_errors;

//#ifdef LOGS
//				log_error_count(host_errors);
//#endif
				log.update_errors();
			}

			if (host_errors > 0 || (loop2 % 10 == 0)) {
				auto wasted_time = copy_time + cmp_time + mem_cpy_time;
				auto iteration_time = wasted_time + kernel_time;

				std::cout << "iteration: " << loop2;
				std::cout << " errors: " << host_errors;
				std::cout << " kernel time: " << kernel_time << "s.";
				std::cout << " matrix set time: " << mem_cpy_time << "s.";
				std::cout << " copy time: " << copy_time << "s.";
				std::cout << " compare time: " << cmp_time << "s.";
				std::cout << " ACC time: " << total_time << "s.";
				std::cout << " iteration time: " << iteration_time << "s.";
				std::cout << " wasted time: " << wasted_time << "s. ("
						<< (wasted_time / iteration_time) * 100 << "%)";
				std::cout << " total errors: " << t_ea << std::endl;
			} else {
				std::cout << "." << std::flush;
			}
		} else {
			output_itemsets = matrix_cuda.to_vector();
			WriteGoldToFile(output_itemsets, gold_path, max_rows);
		}

	}

//#ifdef LOGS
//	end_log_file();
//#endif
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

	std::cout << "WG size of kernel = " << BLOCK_SIZE << std::endl;

	runTest(argc, argv);

	return EXIT_SUCCESS;
}

