#include "hip/hip_runtime.h"
/* GEMM is a General Matrix Multiply - a subroutine in the Basic Linear Algebra Subprograms library*/

/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 16

/* ======================================================= */
/* CUDA implementation of dGEMM without using shared memory
/* ======================================================= */
__global__ void cuda_dgemm(int n, 
			   double alpha, 
			   const double *A, 
			   const double *B,
			   double beta, 
			   double *C) {

  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  
  //printf("row = %d col = %d  n= %d\n", row, col, n);
  if (row >= n || col >= n) return;
  
  double prod = 0;
  for (int k = 0; k < n; ++k){ 
    prod += B[row * n + k] * A[k * n + col];
    //printf(" %d %d %d %f %f\n",k,row,col,A[row*n+k],B[k*n+col]);
  }
  //printf("prod = %f\n", prod);
  C[row*n + col] = alpha * prod + beta * C[row*n+col]; 
}

/* ======================================================= */
/* CUDA implementation of dGEMM using shared memory
/* ======================================================= */
__global__ void cuda_dgemm_shmem(int n, 
			   double alpha, 
			   const double *B, 
			   const double *A,
			   double beta, 
			   double *C) {
  // Block index
  int block_col = blockIdx.x;
  int block_row = blockIdx.y;

  // Thread index
  int thread_col = threadIdx.x;
  int thread_row = threadIdx.y;

  //printf("row = %d col = %d  n= %d\n", block_col, block_row, n);
  //int row = blockDim.y * blockIdx.y + threadIdx.y;
  //int col = blockDim.x * blockIdx.x + threadIdx.x;
  
  int aBegin = n * blockDim.x * block_row;
  int aEnd = aBegin + n-1;
  int bBegin = blockDim.x * block_col;
  int bStep = n * blockDim.x;
  double Csub = 0;

  for (int a=aBegin, b=bBegin, istep=0;
       a <= aEnd; a+= blockDim.x, b+=bStep, ++istep){

    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

    if ((istep*blockDim.x+thread_col < n) && (block_row*blockDim.x+ thread_row < n))
      As[thread_row][thread_col] = A[a + n * thread_row + thread_col];
    else
      As[thread_row][thread_col] = 0;
      
    if ((block_col*blockDim.x+thread_col < n) && (istep*blockDim.x + thread_row < n))
      Bs[thread_row][thread_col] = B[b + n * thread_row + thread_col];
    else
      Bs[thread_row][thread_col] = 0;

    __syncthreads();

    // calculate the cell
    for (int k = 0; k < blockDim.x; ++k)
      Csub += As[thread_row][k] * Bs[k][thread_col];

    __syncthreads();
  }

  // Write the block sub-matrix to global memory;
  // each thread writes one element
  int c = n * blockDim.x * block_row + blockDim.x * block_col;
  if ((block_col*blockDim.x+thread_col < n) && (block_row*blockDim.x+ thread_row < n))
    C[c + n * thread_row + thread_col] = alpha * Csub + beta * C[c +n * thread_row + thread_col];

 }


/* ======================================================= */
/* Simple host implementation of a simple version of sgemm */
/* ======================================================= */
static void simple_dgemm(int n, double alpha, const double *A, const double *B,
                         double beta, double *C) {
  int i, j, k;
  for (i = 0; i < n; ++i) {
    for (j = 0; j < n; ++j){
      double prod = 0;
      for (k = 0; k < n; ++k){
    	  prod += A[k * n + i] * B[j * n + k];
      }
      C[j * n + i] = alpha * prod + beta * C[j * n + i];
    }
  }
}

/* ======================= */
/* dgemm from BLAS library */
/* ======================= */
extern "C"{
extern void dgemm_(char *, char * , 
		  int *, int *, int *,
		  double *, double *, int *,
		  double *, int *,
		   double *, double *, int *); };

/* ==== */
/* Main */
/* ==== */
int main(int argc, char **argv)
{
  hipblasStatus_t status;
  double *h_A, *h_B, *h_C, *h_C_blas, *h_C_simple, *h_C_0;
  double *d_A = 0; 
  double *d_B = 0;
  double *d_C = 0;
  double alpha = 1.0f;
  double beta = 0.0f;
  int n2, N;
  int i;
  double error_norm1, error_norm2;
  double ref_norm;
  double diff1, diff2;
  hipblasHandle_t handle;
  struct timeval tv1, tv2;


  /* get the size of the matrix from the command line */
  if (argc <2 ) N= 275;
  else N = atoi(argv[1]);
  //N=3;

  n2 = N * N;

  printf("\nRunning dgemm test for %d by %d matricies.\n", N, N);
  /* Initialize CUBLAS */
  status = hipblasCreate(&handle);
  
  /* Allocate host memory for the matrices */
  h_A = (double *)malloc(n2 * sizeof(double) );
  h_B = (double *)malloc(n2 * sizeof(double) );
  h_C = (double *)malloc(n2 * sizeof(double) );
  h_C_blas = (double *)malloc(n2 * sizeof(double) );
  h_C_simple = (double *)malloc(n2 * sizeof(double) );
  h_C_0 = (double *)malloc(n2 * sizeof(double) );

  /* Fill the matrices with test data */
  for (i = 0; i < n2; i++){
    h_A[i] = rand() / (double)RAND_MAX;
    h_B[i] = rand() / (double)RAND_MAX;
    h_C[i] = rand() / (double)RAND_MAX;
    h_C_blas[i] = h_C[i];
    h_C_simple[i] = h_C[i];
    h_C_0[i] = h_C[i];
    //printf("%f %f \n",h_A[i], h_B[i]);
  }

  printf("\tTesting simple C implementation of dgemm function.\n");
  gettimeofday(&tv1, NULL);
  /* Performs operation using plain C code */
  simple_dgemm(N, alpha, h_A, h_B, beta, h_C_simple);
  gettimeofday(&tv2, NULL);
  printf("\t\tdone...\n");
  printf("\t\tExecution time (in millisec): %.2f\n",
	 (double)(tv2.tv_usec-tv1.tv_usec)/1000 + 
	 (double)(tv2.tv_sec -tv1.tv_sec )*1000);


  printf("\tTesting dgemm function from BLAS library.\n");
  gettimeofday(&tv1, NULL);

  /* Performs operation using BLASS library */
  dgemm_("N","N", &N, &N, &N, &alpha, h_A, &N, h_B, &N, &beta, h_C_blas, &N);
  gettimeofday(&tv2, NULL);
  printf("\t\tdone...\n");
  printf("\t\tExecution time (in millisec): %.2f\n",
	 (double)(tv2.tv_usec-tv1.tv_usec)/1000 + 
	 (double)(tv2.tv_sec -tv1.tv_sec )*1000);


  printf("\tTesting dgemm function from cuBLAS library.\n");
  gettimeofday(&tv1, NULL);

  /* Allocate device memory for the matrices */
  hipMalloc((void **)&d_A, n2 * sizeof(d_A[0]));
  hipMalloc((void **)&d_B, n2 * sizeof(d_B[0]));
  hipMalloc((void **)&d_C, n2 * sizeof(d_C[0]));

  /* Initialize the device matrices with the host matrices */
  status = hipblasSetVector(n2, sizeof(h_A[0]), h_A, 1, d_A, 1);
  status = hipblasSetVector(n2, sizeof(h_B[0]), h_B, 1, d_B, 1);
  status = hipblasSetVector(n2, sizeof(h_C[0]), h_C, 1, d_C, 1);

  /* Performs operation using cublas */
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

  /* Read the result back */
  status = hipblasGetVector(n2, sizeof(h_C[0]), d_C, 1, h_C, 1);

  gettimeofday(&tv2, NULL);
  printf("\t\tdone...\n");
  printf("\t\tExecution time (in millisec): %.2f\n",
	 (double)(tv2.tv_usec-tv1.tv_usec)/1000 + 
	 (double)(tv2.tv_sec -tv1.tv_sec )*1000);

  printf("\n\tChecking results.\n");
  /* Check result against reference */
  error_norm1 = 0;
  error_norm2 = 0;
  ref_norm = 0;
  for (i = 0; i < n2; ++i){
    diff1 = h_C_simple[i] - h_C[i];
    diff2 = h_C_simple[i] - h_C_blas[i];
    error_norm1 += diff1 * diff1;
    error_norm2 += diff2 * diff2;
    ref_norm += h_C_simple[i] * h_C_simple[i];
  }

  error_norm1 = (double)sqrt((double)error_norm1);
  error_norm2 = (double)sqrt((double)error_norm2);
  ref_norm = (double)sqrt((double)ref_norm);

  if (fabs(ref_norm) < 1e-7)printf(" *** Error in Calculations! \n");
  if(error_norm1 / ref_norm < 1e-6f)printf("\t\tPassed cublas Dgemm vs. simple Dgemm comparison!\n");
  else printf("\t\tDid not pass cublas Dgemm vs. simple Dgemm comparison!\n");
  if(error_norm2 / ref_norm < 1e-6f)printf("\t\tPassed simple Dgemm vs. BLAS Dgemm comparison!\n");
  else printf("\t\tDid not pass simple Dgemm vs. BLAS Dgemm comparison!\n");

  /* free cuda memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  /* ============ CUDA implementation without shared memory =============== */
  printf("\tTesting CUDA dgemm function without using Shared memory.\n");
  gettimeofday(&tv1, NULL);

  /* Allocate device memory for the matrices */
  hipMalloc((void **)&d_A, n2 * sizeof(d_A[0]));
  hipMalloc((void **)&d_B, n2 * sizeof(d_B[0]));
  hipMalloc((void **)&d_C, n2 * sizeof(d_C[0]));

  /* copy A and B matrices to gpu */
  hipMemcpy(d_A, h_A,n2*sizeof(d_A[0]), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B,n2*sizeof(d_B[0]), hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C_0,n2*sizeof(d_C[0]), hipMemcpyHostToDevice);

  /* Kernel */
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(N/BLOCK_SIZE+1, N/BLOCK_SIZE+1);
  //printf(" beta=%f\n",beta);
  cuda_dgemm<<<dimGrid, dimBlock>>>(N, alpha, d_A, d_B, beta, d_C);
  /* wait until all threads finish their job */
  hipDeviceSynchronize();


  /* Read the result back */
  hipMemcpy(h_C, d_C,n2*sizeof(d_C[0]), hipMemcpyDeviceToHost);


  gettimeofday(&tv2, NULL);
  printf("\t\tdone...\n");
  printf("\t\tExecution time (in millisec): %.2f\n",
	 (double)(tv2.tv_usec-tv1.tv_usec)/1000 + 
	 (double)(tv2.tv_sec -tv1.tv_sec )*1000);

  printf("\n\tChecking results.\n");
  /* Check result against reference */
  error_norm1 = 0;
  for (i = 0; i < n2; ++i){
    //if (i<100)printf("%f %f\n",h_C[i], h_C_simple[i]);
    diff1 = h_C_simple[i] - h_C[i];
    error_norm1 += diff1 * diff1;
  }

  error_norm1 = (double)sqrt((double)error_norm1);

  //printf("%12.7f\n",error_norm1);
  if(error_norm1  < 1e-6f)printf("\t\tPassed cuda Dgemm without Shared Memoryvs. simple Dgemm comparison!\n");
  else printf("\t\tDid not pass cuda Dgemm (without Shared Memory) vs. simple Dgemm comparison!\n");



  /* free cuda memory */
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  /* ============ CUDA implementation using shared memory =============== */
  printf("\tTesting CUDA dgemm function without using Shared memory.\n");
  gettimeofday(&tv1, NULL);

  /* Allocate device memory for the matrices */
  hipMalloc((void **)&d_A, n2 * sizeof(d_A[0]));
  hipMalloc((void **)&d_B, n2 * sizeof(d_B[0]));
  hipMalloc((void **)&d_C, n2 * sizeof(d_C[0]));

  /* copy A and B matrices to gpu */
  hipMemcpy(d_A, h_A,n2*sizeof(d_A[0]), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B,n2*sizeof(d_B[0]), hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C_0,n2*sizeof(d_C[0]), hipMemcpyHostToDevice);

  /* Kernel */
  cuda_dgemm_shmem<<<dimGrid, dimBlock>>>(N, alpha, d_A, d_B, beta, d_C);
  /* wait until all threads finish their job */
  hipDeviceSynchronize();


  /* Read the result back */
  hipMemcpy(h_C, d_C,n2*sizeof(d_C[0]), hipMemcpyDeviceToHost);


  gettimeofday(&tv2, NULL);
  printf("\t\tdone...\n");
  printf("\t\tExecution time (in millisec): %.2f\n",
	 (double)(tv2.tv_usec-tv1.tv_usec)/1000 + 
	 (double)(tv2.tv_sec -tv1.tv_sec )*1000);

  printf("\n\tChecking results.\n");
  /* Check result against reference */
  error_norm1 = 0;
  for (i = 0; i < n2; ++i){
    //if (i<10)printf("%f %f %f\n",h_C[i], h_C_simple[i], h_C_0[i]);
    diff1 = h_C_simple[i] - h_C[i];
    error_norm1 += diff1 * diff1;
  }

  error_norm1 = (double)sqrt((double)error_norm1);
  //printf("%12.7f\n",error_norm1);
  if(error_norm1 < 1e-6f)printf("\t\tPassed cuda Dgemm using Shared Memoryvs. simple Dgemm comparison!\n");
  else printf("\t\tDid not pass cuda Dgemm (using Shared Memory) vs. simple Dgemm comparison!\n");


  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);


  /* Memory clean up */
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_simple);
  free(h_C_blas);

  /* Shutdown */
  status = hipblasDestroy(handle);

  return(0);
}
