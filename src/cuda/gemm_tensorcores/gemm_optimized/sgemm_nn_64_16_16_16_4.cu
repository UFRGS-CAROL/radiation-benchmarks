#include "hip/hip_runtime.h"
/*
 -- MAGMA (version 1.2.1) --
 Univ. of Tennessee, Knoxville
 Univ. of California, Berkeley
 Univ. of Colorado, Denver
 June 2012

 @precisions normal s

 */

// CUDA runtime
#include <hip/hip_runtime.h>
// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "cuda_utils.h"

__device__ __forceinline__ void fma__(const double a, const double b,
		float& c) {
	c = __fmaf_rn(__double2float_rn(a), __double2float_rn(b), c);
}

__device__ __forceinline__ void fma__(const float a, const float b,
		float& c) {
	c = __fma_rn(a, b, c);
}


__device__ __forceinline__ void fma__(const double a, const double b,
		double& c) {
	c = __fma_rn(a, b, c);
}

template<typename real_t>
__device__ void saxpy(real_t a, real_t *b, real_t *c) {
	c[0] += a * b[0];
	c[1] += a * b[1];
	c[2] += a * b[2];
	c[3] += a * b[3];
	c[4] += a * b[4];
	c[5] += a * b[5];
	c[6] += a * b[6];
	c[7] += a * b[7];
	c[8] += a * b[8];
	c[9] += a * b[9];
	c[10] += a * b[10];
	c[11] += a * b[11];
	c[12] += a * b[12];
	c[13] += a * b[13];
	c[14] += a * b[14];
	c[15] += a * b[15];
}

template<typename real_t, typename half_real_t>
__device__ void saxpy(real_t a, real_t *b, real_t *c, half_real_t *c_inc) {
	fma__(a, b[0], c[0]);
	fma__(a, b[1], c[1]);
	fma__(a, b[2], c[2]);
	fma__(a, b[3], c[3]);
	fma__(a, b[4], c[4]);
	fma__(a, b[5], c[5]);
	fma__(a, b[6], c[6]);
	fma__(a, b[7], c[7]);
	fma__(a, b[8], c[8]);
	fma__(a, b[9], c[9]);
	fma__(a, b[10], c[10]);
	fma__(a, b[11], c[11]);
	fma__(a, b[12], c[12]);
	fma__(a, b[13], c[13]);
	fma__(a, b[14], c[14]);
	fma__(a, b[15], c[15]);

	fma__(a, b[0], c_inc[0]);
	fma__(a, b[1], c_inc[1]);
	fma__(a, b[2], c_inc[2]);
	fma__(a, b[3], c_inc[3]);
	fma__(a, b[4], c_inc[4]);
	fma__(a, b[5], c_inc[5]);
	fma__(a, b[6], c_inc[6]);
	fma__(a, b[7], c_inc[7]);
	fma__(a, b[8], c_inc[8]);
	fma__(a, b[9], c_inc[9]);
	fma__(a, b[10], c_inc[10]);
	fma__(a, b[11], c_inc[11]);
	fma__(a, b[12], c_inc[12]);
	fma__(a, b[13], c_inc[13]);
	fma__(a, b[14], c_inc[14]);
	fma__(a, b[15], c_inc[15]);
}

template<typename real_t>
__global__ void sgemm_kernel(real_t *C, const real_t *A, const real_t *B, int m,
		int n, int k, int lda, int ldb, int ldc, real_t alpha, real_t beta) {
	/*  -- MAGMA (version 1.2.1) --
	 Purpose:
	 ========
	 This routine computes
	 C = alpha* A*B  + beta * C

	 B is put into shared memory
	 Parameters Used:
	 blk_M=64 blk_N=16 blk_K=16 nthd_x=16 nthd_y=4

	 This kernel is for matrices divisible by the corresponding
	 blocking sizes.
	 ===============================================================  */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;

	const int idt = ty * 16 + tx;

	B += tx + __mul24(iby + ty, ldb);
	A += ibx + idt;
	C += ibx + idt + __mul24(iby, ldc);

	const real_t *Bend = B + k;

	real_t Cb[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	m = 2 * lda;
	n = 3 * lda;

	do {
		//float Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
		real_t Ab[4] = { A[0], A[lda], A[m], A[n] };
		__shared__ real_t Bb[16][17];
		Bb[tx][ty + 0] = B[0];
		Bb[tx][ty + 4] = B[4 * ldb];
		Bb[tx][ty + 8] = B[8 * ldb];
		Bb[tx][ty + 12] = B[12 * ldb];

		__syncthreads();

		A += 4 * lda;
		saxpy(Ab[0], &Bb[0][0], Cb);
		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[1][0], Cb);
		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[2][0], Cb);
		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[3][0], Cb);
		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[4][0], Cb);
		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[5][0], Cb);
		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[6][0], Cb);
		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[7][0], Cb);
		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[8][0], Cb);
		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[9][0], Cb);
		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[10][0], Cb);
		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[11][0], Cb);
		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[12][0], Cb);
		saxpy(Ab[1], &Bb[13][0], Cb);
		saxpy(Ab[2], &Bb[14][0], Cb);
		saxpy(Ab[3], &Bb[15][0], Cb);

		B += 16;

		__syncthreads();
	} while (B < Bend);

#pragma unroll 16
	for (int i = 0; i < 16; i++, C += ldc) {
		C[0] = alpha * Cb[i] + beta * C[0];
	}
}

void sgemm(hipStream_t stream, float *C, const float *A, const float *B,
		int32_t m, int32_t n, int32_t k, int32_t lda, int32_t ldb, int32_t ldc,
		float alpha, float beta) {
	dim3 threads(16, 4);
	dim3 grid(m / 64, n / 16);

		// Allocate CUDA events that we'll use for timing
  	hipEvent_t start;
  	checkCudaErrors(hipEventCreate(&start));

  	hipEvent_t stop;
  	checkCudaErrors(hipEventCreate(&stop));	

  	// Record the start event
  	checkCudaErrors(hipEventRecord(start, NULL));

	sgemm_kernel<<<grid, threads, 0, stream>>>(C, A, B, m, n, k, lda, ldb, ldc,
			alpha, beta);

		// Record the stop event
  	checkCudaErrors(hipEventRecord(stop, NULL));

  	// Wait for the stop event to complete
  	checkCudaErrors(hipEventSynchronize(stop));

  	float msecTotal = 0.0f;
  	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  	
 	printf("%.3f ms \n", msecTotal);

	rad::checkFrameworkErrors(hipDeviceSynchronize());
	//end
}

void sgemm(hipStream_t stream, double *C, const double *A, const double *B,
		int32_t m, int32_t n, int32_t k, int32_t lda, int32_t ldb, int32_t ldc,
		double alpha, double beta) {
	dim3 threads(16, 4);
	dim3 grid(m / 64, n / 16);

	// Allocate CUDA events that we'll use for timing
  	hipEvent_t start;
  	checkCudaErrors(hipEventCreate(&start));

  	hipEvent_t stop;
  	checkCudaErrors(hipEventCreate(&stop));	

  	// Record the start event
  	checkCudaErrors(hipEventRecord(start, NULL));


	sgemm_kernel<<<grid, threads, 0, stream>>>(C, A, B, m, n, k, lda, ldb, ldc,
			alpha, beta);

	// Record the stop event
  	checkCudaErrors(hipEventRecord(stop, NULL));

  	// Wait for the stop event to complete
  	checkCudaErrors(hipEventSynchronize(stop));

  	float msecTotal = 0.0f;
  	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  	
 	printf("%.3f ms \n", msecTotal);
	rad::checkFrameworkErrors(hipDeviceSynchronize());
	//end
}


//DMR KERNEL
template<typename real_t, typename half_real_t>
__global__ void sgemm_kernel(half_real_t* C_inc, real_t *C, const real_t *A, const real_t *B, int m,
		int n, int k, int lda, int ldb, int ldc, real_t alpha, real_t beta) {

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y * 16;

	const int idt = ty * 16 + tx;

	B += tx + __mul24(iby + ty, ldb);
	A += ibx + idt;
	C += ibx + idt + __mul24(iby, ldc);

	C_inc += ibx + idt + __mul24(iby, ldc);

	const real_t *Bend = B + k;

	real_t Cb[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	half_real_t Cb_inc[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	m = 2 * lda;
	n = 3 * lda;

	do {
		//float Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
		real_t Ab[4] = { A[0], A[lda], A[m], A[n] };
		__shared__ real_t Bb[16][17];
		Bb[tx][ty + 0] = B[0];
		Bb[tx][ty + 4] = B[4 * ldb];
		Bb[tx][ty + 8] = B[8 * ldb];
		Bb[tx][ty + 12] = B[12 * ldb];

		__syncthreads();

		A += 4 * lda;
		saxpy(Ab[0], &Bb[0][0], Cb, Cb_inc);

		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[1][0], Cb, Cb_inc);

		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[2][0], Cb, Cb_inc);

		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[3][0], Cb, Cb_inc);
		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[4][0], Cb, Cb_inc);

		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[5][0], Cb, Cb_inc);

		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[6][0], Cb, Cb_inc);

		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[7][0], Cb, Cb_inc);

		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[8][0], Cb, Cb_inc);

		Ab[0] = A[0];
		saxpy(Ab[1], &Bb[9][0], Cb, Cb_inc);

		Ab[1] = A[lda];
		saxpy(Ab[2], &Bb[10][0], Cb, Cb_inc);

		Ab[2] = A[m];
		saxpy(Ab[3], &Bb[11][0], Cb, Cb_inc);

		Ab[3] = A[n];

		A += 4 * lda;
		saxpy(Ab[0], &Bb[12][0], Cb, Cb_inc);

		saxpy(Ab[1], &Bb[13][0], Cb, Cb_inc);

		saxpy(Ab[2], &Bb[14][0], Cb, Cb_inc);

		saxpy(Ab[3], &Bb[15][0], Cb, Cb_inc);

		B += 16;

		__syncthreads();
	} while (B < Bend);

	half_real_t alpha_inc = half_real_t(alpha);
	half_real_t beta_inc = half_real_t(beta);
#pragma unroll 16
	for (int i = 0; i < 16; i++, C += ldc, C_inc += ldc) {
		C[0] = alpha * Cb[i] + beta * C[0];
		C_inc[0] =  alpha_inc * Cb_inc[i] + beta_inc * C_inc[0];
	}
}


void sgemm_dmr(hipStream_t stream, double *C, float *C_inc, const double *A, const double *B,
		int32_t m, int32_t n, int32_t k, int32_t lda, int32_t ldb, int32_t ldc,
		double alpha, double beta) {
	dim3 threads(16, 4);
	dim3 grid(m / 64, n / 16);
			// Allocate CUDA events that we'll use for timing
  	hipEvent_t start;
  	checkCudaErrors(hipEventCreate(&start));

  	hipEvent_t stop;
  	checkCudaErrors(hipEventCreate(&stop));	

  	// Record the start event
  	checkCudaErrors(hipEventRecord(start, NULL));


	sgemm_kernel<<<grid, threads, 0, stream>>>(C_inc, C, A, B, m, n, k, lda, ldb, ldc,
			alpha, beta);

		// Record the stop event
  	checkCudaErrors(hipEventRecord(stop, NULL));

  	// Wait for the stop event to complete
  	checkCudaErrors(hipEventSynchronize(stop));

  	float msecTotal = 0.0f;
  	checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  	
 	printf("%.3f ms \n", msecTotal);
	rad::checkFrameworkErrors(hipDeviceSynchronize());
	//end
}
