#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_fp16.h>
#include "half.hpp"
#include <math.h>

typedef half_float::half half_h;

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(double *C, double *C1, double *A,double *A1,
    double *B,double *B1, int wA,
    int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;



  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
    volatile float Csub = 0;
    volatile double Csub1= 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];



    __shared__ double As1[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bs1[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    As1[ty][tx] = A1[a + wA * ty + tx];
    Bs1[ty][tx] = B1[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      
      Csub = fma_dmr(__double2float_rn(As[ty][k]), __double2float_rn(Bs[k][tx]), Csub);
      // Csub = fma_dmr(As[ty][k], Bs[k][tx],Csub);
      Csub1 = fma_dmr(As1[ty][k], Bs1[k][tx],Csub1);

      
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = (float)Csub;
  C1[c + wB * ty + tx] = Csub1;
}

// template <int BLOCK_SIZE> __global__ void MatrixMulCUDA_Half(half *C, half *C1, half *A,
//     half *B, int wA,
//     int wB) {
//   // Block index
//   int bx = blockIdx.x;
//   int by = blockIdx.y;

//   // Thread index
//   int tx = threadIdx.x;
//   int ty = threadIdx.y;

//   // Index of the first sub-matrix of A processed by the block
//   int aBegin = wA * BLOCK_SIZE * by;

//   // Index of the last sub-matrix of A processed by the block
//   int aEnd   = aBegin + wA - 1;



//   // Step size used to iterate through the sub-matrices of A
//   int aStep  = BLOCK_SIZE;

//   // Index of the first sub-matrix of B processed by the block
//   int bBegin = BLOCK_SIZE * bx;

//   // Step size used to iterate through the sub-matrices of B
//   int bStep  = BLOCK_SIZE * wB;

//   // Csub is used to store the element of the block sub-matrix
//   // that is computed by the thread
//   volatile half2 Csub = __float2half2_rn(0.0);

//  // half2 Csub1= __float2half2_rn(0.0);



//   // Loop over all the sub-matrices of A and B
//   // required to compute the block sub-matrix
//   for (int a = aBegin, b = bBegin;
//        a <= aEnd;
//        a += aStep, b += bStep) {
//     // Declaration of the shared memory array As used to
//     // store the sub-matrix of A
//     __shared__ half2 As[BLOCK_SIZE][BLOCK_SIZE];

//     // Declaration of the shared memory array Bs used to
//     // store the sub-matrix of B
//     __shared__ half2 Bs[BLOCK_SIZE][BLOCK_SIZE];

//     // Load the matrices from device memory
//     // to shared memory; each thread loads
//     // one element of each matrix
//     As[ty][tx] = __half2half2(A[a + wA * ty + tx]);
//     Bs[ty][tx] = __half2half2(B[b + wB/2 * ty + tx]);

//     // Synchronize to make sure the matrices are loaded
//     __syncthreads();

//     // Multiply the two matrices together;
//     // each thread computes one element
//     // of the block sub-matrix
// #pragma unroll

//     for (int k = 0; k < BLOCK_SIZE; ++k) {
//       Csub = __hfma2(As[ty][k], Bs[k][tx],Csub);
//      // Csub1 =__hfma2((As[ty][k]), (Bs[k][tx]),Csub1);

      
//     }

//     // Synchronize to make sure that the preceding
//     // computation is done before loading two new
//     // sub-matrices of A and B in the next iteration
//     __syncthreads();
//   }

//   // Write the block sub-matrix to device memory;
//   // each thread writes one element
//   int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
//   ((half2*)C)[c + wB/2 * ty + tx] = Csub;
//   //((half2*)C1)[c + wB/2 * ty + tx] = Csub1;
 
// }


void ConstantInit(double *data, int size, double val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(double) * size_A;
  double *h_A = reinterpret_cast<double *>(malloc(mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(double) * size_B;
  double *h_B = reinterpret_cast<double *>(malloc(mem_size_B));

  // Initialize host memory
  
  const double valA = 2.0f;
  const double valB = 2.0f;
  ConstantInit(h_A, size_A, valA);
  ConstantInit(h_B, size_B, valB);
  //printf("h_A = %f\n", h_A[0]);
  // Allocate device memory
  double *d_A, *d_A1,*d_B, *d_B1, *d_C, *d_C1;

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(double);
  double *h_C = reinterpret_cast<double *>(malloc(mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A1), mem_size_A));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B1), mem_size_B));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C1), mem_size_C));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A1, h_A, mem_size_A, hipMemcpyHostToDevice));

  checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B1, h_B, mem_size_B, hipMemcpyHostToDevice));

  


  // Setup execution parameters
  dim3 threads(block_size, block_size);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);




  // // HALF parameters 

  // dim3 threads(block_size/2.0, block_size);
  // dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);






  // Create and start timer
  printf("Computing result using CUDA Kernel...\n");


  MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_C1, d_A, d_A1, d_B, d_B1,
                                         dimsA.x, dimsB.x);
  //MatrixMulCUDA_Half<32> <<< grid, threads >>>(d_C,d_C1, d_A, d_B,
  //                                          dimsA.x, dimsB.x);

  

  printf("done\n");

  hipDeviceSynchronize();

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start;
  checkCudaErrors(hipEventCreate(&start));

  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&stop));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));

  // Execute the kernel
  int nIter = 10;

  for (int j = 0; j < nIter; j++) {
   
      MatrixMulCUDA<32> <<< grid, threads >>>(d_C, d_C1, d_A, d_A1, d_B, d_B1,
                                              dimsA.x, dimsB.x);
      // MatrixMulCUDA_Half<32> <<< grid, threads >>>(d_C,d_C1, d_A, d_B,
      //                                       dimsA.x, dimsB.x);
    
  }

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                             static_cast<double>(dimsA.y) *
                             static_cast<double>(dimsB.x);
  double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
                     (msecPerMatrixMul / 1000.0f);
  printf(
    "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops," \
    " WorkgroupSize= %u threads/block\n",
    gigaFlops,
    msecPerMatrixMul,
    flopsPerMatrixMul,
    threads.x * threads.y);

  // Copy result from device to host
  checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

  printf("Checking computed result for correctness: ");
  bool correct = true;

  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  //double eps = 1.e-6;  // machine zero

  // for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
  //   double abs_err = fabs(h_C[i] - (dimsA.x * valB));
  //   double dot_length = dimsA.x;
  //   double abs_val = fabs(h_C[i]);
  //   double rel_err = abs_err / abs_val / dot_length;

  //   if (rel_err > eps) {
  //     printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
  //            i, h_C[i], dimsA.x * valB, eps);
  //     correct = false;
  //   }
  // }

  // printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");
  //printf(" A= %f B= %f  C = %f \n", h_A[0], h_B[0], h_C[0]);
  // Clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipFree(d_C1));

  // printf("\nNOTE: The CUDA Samples are not meant for performance"\
  //        "measurements. Results may vary when GPU Boost is enabled.\n");

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}


/**
 * Program main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 32;

  dim3 dimsA(8192, 8192, 1);
  dim3 dimsB(8192, 8192, 1);

  dimsA.x = 8192;
  dimsA.y = 8192;

  dimsB.x = 8192;
  dimsB.y = 8192; 


  // dim3 dimsA(4096, 4096, 1);
  // dim3 dimsB(4096, 4096, 1);

  // dimsA.x = 4096;
  // dimsA.y = 4096;

  // dimsB.x = 4096;
  // dimsB.y = 4096; 


  // // width of Matrix A
  // if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
  //   dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  // }

  // // height of Matrix A
  // if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
  //   dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  // }

  // // width of Matrix B
  // if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
  //   dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  // }

  // // height of Matrix B
  // if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
  //   dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  // }

  // if (dimsA.x != dimsB.y) {
  //   printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
  //          dimsA.x, dimsB.y);
  //   exit(EXIT_FAILURE);
  // }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);

  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);

  exit(matrix_result);
}

__device__ __forceinline__ double fma_dmr(double a, double b, double acc) {
  return fma(a, b, acc);
}

__device__ __forceinline__ float fma_dmr(float a, float b, float acc) {
  return __fmaf_rn(a, b, acc);
}

__device__  __forceinline__ half fma_dmr(half a, half b, half acc) {
  return __hfma(a, b, acc);
}
