#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling approach.
 * It has been written for clarity of exposition to illustrate various CUDA programming
 * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>

#define BLOCK_SIZE 32
#define M_GLOBAL 8192 
#define N_GLOBAL 8192

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template<typename real_t>
__global__ void MatrixMulCUDA(real_t *C, real_t *A,
    real_t *B, int wA,
    int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  real_t Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ real_t As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ real_t Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}
template<typename real_t>
void ConstantInit(real_t *data, int size, real_t val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
template<typename real_t>
int MatrixMultiply(int argc, char **argv,
                   int block_size, const dim3 &dimsA,
                   const dim3 &dimsB) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(real_t) * size_A;
  real_t *h_A = reinterpret_cast<real_t *>(malloc(mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(real_t) * size_B;
  real_t *h_B = reinterpret_cast<real_t *>(malloc(mem_size_B));

  // Initialize host memory
  const real_t val = (real_t)1.0f;

  ConstantInit(h_A, size_A, val);
  ConstantInit(h_B, size_B, val);

  // Allocate device memory
  real_t *d_A, *d_B ,*d_C;
  

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(real_t);
  real_t *h_C = reinterpret_cast<real_t *>(malloc(mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));

  checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

  dim3 dim_grid, dim_block;
  
  uint32_t grid_rows = (M_GLOBAL + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint32_t grid_cols = (N_GLOBAL + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim_grid = dim3(grid_cols, grid_rows);
  dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);

 
  // Create and start timer
  printf("Computing result using CUDA Kernel...\n");

  // Performs warmup operation using matrixMul CUDA kernel

    MatrixMulCUDA<real_t> <<< dim_grid, dim_block >>>(d_C, d_A, d_B,
                                            dimsA.x, dimsB.x);
  

  printf("done\n");

  hipDeviceSynchronize();

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start;
  checkCudaErrors(hipEventCreate(&start));

  hipEvent_t stop;
  checkCudaErrors(hipEventCreate(&stop));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, NULL));


  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, NULL));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


  checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

  printf("Checking computed result for correctness: ");


  for (int i = 0; i < 10 ; i++) {
    
      printf("result = %f \n", (double)h_C[i]);      
    
  }



  // Clean up memory
  free(h_A);
  free(h_B);
  free(h_C);
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

    return EXIT_SUCCESS;

}

int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf("  Note: Outer matrix dimensions of A & B matrices" \
           " must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv);

  int block_size = 16;

  dim3 dimsA(M_GLOBAL,N_GLOBAL);
  dim3 dimsB(M_GLOBAL,N_GLOBAL);

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
         dimsB.x, dimsB.y);


  int matrix_result = MatrixMultiply<__half>(argc, argv, block_size, dimsA, dimsB);

  exit(matrix_result);
}
