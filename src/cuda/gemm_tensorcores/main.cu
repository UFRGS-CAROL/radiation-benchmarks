#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <random>
#include <fstream>      // std::ifstream
#include <sstream>      // std::stringstream

#include <iomanip>
#include <limits>

#ifdef OMP
#include <omp.h>
#endif

//#include "half.hpp"
#include "Log.h"
#include "GEMM.h"

#ifndef DEFAULT_INPUT_SIZE
#define DEFAULT_INPUT_SIZE 8192
#endif

#define GENERATOR_MAXABSVALUE 2.0
#define GENERATOR_MINABSVALUE 0

// THRESHOLDS
#define ZERO_HALF 6.5469
#define ZERO_FlOAT 0.0
#define ZERO_DOUBLE 0.0
#define ZERO_DMR 0.0

//typedef half_float::half host_half;
//typedef std::vector<host_half> half_vector;

template<class half_t, class real_t> void generate_matrices_files(
		std::vector<half_t>& a_host_vector, std::vector<half_t>& b_host_vector,
		std::vector<real_t>& c_host_vector, Log& log) {

	std::ofstream f_a(log.a_input_path, std::ios::out | std::ios::binary);
	std::ofstream f_b(log.b_input_path, std::ios::out | std::ios::binary);
	std::ofstream f_c(log.c_input_path, std::ios::out | std::ios::binary);

	if (f_a.is_open() && f_b.is_open() && f_c.is_open()) {
		std::random_device rd; //Will be used to obtain a seed for the random number engine
		std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
		std::uniform_real_distribution<double> dis(-GENERATOR_MAXABSVALUE,
		GENERATOR_MAXABSVALUE);

//		for (size_t i = 0; i < log.size_matrices; i++) {
//			for (size_t j = 0; j < log.size_matrices; j++) {
//				a_host_vector[i * log.size_matrices + j] = host_half(dis(gen));
//				b_host_vector[i * log.size_matrices + j] = host_half(dis(gen));
//				c_host_vector[i * log.size_matrices + j] = real_t(dis(gen));
//			}
//		}

		for (size_t i = 0; i < log.size_matrices; i++) {
			for (size_t j = 0; j < log.size_matrices; j++) {
				a_host_vector[i * log.size_matrices + j] = (half_t) 1.0;
				b_host_vector[i * log.size_matrices + j] = (half_t) 1.0;
				c_host_vector[i * log.size_matrices + j] = (real_t) 0.0;
			}
		}

		half_t zero(0.0);
		half_t nan_ = half_t(half_float::nan("0")); //half_t nan_ = half_t(half_float::nanh("0"));
		half_t inf_ = half_t(half_t(0x7C00));

		int numZeros = std::count(a_host_vector.begin(), a_host_vector.end(),
				zero);
		int numNans = std::count(a_host_vector.begin(), a_host_vector.end(),
				nan_);

		int numInfs = std::count(a_host_vector.begin(), a_host_vector.end(),
				inf_);
		std::cout << "Number of zeros/NaNs/INFs on matrix A: " << numZeros
				<< numNans << numInfs << std::endl;

//		std::cout << "entrou generate3" << std::endl;
		numZeros = std::count(b_host_vector.begin(), b_host_vector.end(), zero);
		numNans = std::count(b_host_vector.begin(), b_host_vector.end(), nan_);
		numInfs = std::count(b_host_vector.begin(), b_host_vector.end(), inf_);

		std::cout << "Number of zeros/NaNs/INFs on matrix B: " << numZeros
				<< numNans << numInfs << std::endl;

		numZeros = std::count(c_host_vector.begin(), c_host_vector.end(), zero);
		numNans = std::count(c_host_vector.begin(), c_host_vector.end(), nan_);
		numInfs = std::count(c_host_vector.begin(), c_host_vector.end(), inf_);

		std::cout << "Number of zeros/NaNs/INFs on matrix C: " << numZeros
				<< numNans << numInfs << std::endl;

		f_a.write(reinterpret_cast<char*>(a_host_vector.data()),
				a_host_vector.size() * sizeof(half_t));
		f_b.write(reinterpret_cast<char*>(b_host_vector.data()),
				b_host_vector.size() * sizeof(half_t));
		f_c.write(reinterpret_cast<char*>(c_host_vector.data()),
				c_host_vector.size() * sizeof(real_t));

		f_a.close();
		f_b.close();
		f_c.close();

	} else {
		throw std::runtime_error(
				"Some of the imput files could not be generated\n");
	}

}

template<class real_t>
void write_gold_to_file(std::string gold_path, std::vector<real_t>& gold) {
	std::ofstream f_gold(gold_path, std::ofstream::out | std::ofstream::binary);
	if (f_gold.is_open()) {
		f_gold.write(reinterpret_cast<char*>(gold.data()),
				sizeof(real_t) * gold.size());
		f_gold.close();
	} else {
		throw std::runtime_error("Could not write gold file\n");
	}
}

template<class real_t> int is_output_ok(std::vector<real_t>& d0,
		std::vector<real_t>& d1, std::vector<real_t>& d2,
		std::vector<real_t>& correct_vector) {

	int memory_errors = 0;
	for (size_t i = 0; i < d0.size(); i++) {
		real_t val_output0 = d0[i];
		real_t val_output1 = d1[i];
		real_t val_output2 = d2[i];
		real_t val_output = val_output0;

		if ((val_output0 != val_output1) || (val_output0 != val_output2)) {
			memory_errors++;

			if ((val_output0 != val_output1) && (val_output1 != val_output2)
					&& (val_output0 != val_output2)) {
				// All 3 values diverge
				memory_errors++;
			} else if (val_output1 == val_output2) {
				// Only value 0 diverge
				val_output = val_output1;
			} else if (val_output0 == val_output2) {
				// Only value 1 diverge
				val_output = val_output0;
			} else if (val_output0 == val_output1) {
				// Only value 2 diverge
				val_output = val_output0;
			}
		}
		correct_vector[i] = val_output;
	}
	return memory_errors;
}

template<class half_t, class real_t> void retrieve_matrices(
		std::vector<half_t>& a_host_vector, std::vector<half_t>& b_host_vector,
		std::vector<real_t>& c_host_vector,
		std::vector<real_t>& gold_host_vector, Log& log) {

	double start = log.mysecond();
	std::ifstream f_a(log.a_input_path, std::ios::in | std::ios::binary);
	std::ifstream f_b(log.b_input_path, std::ios::in | std::ios::binary);
	std::ifstream f_c(log.c_input_path, std::ios::in | std::ios::binary);
	std::ifstream f_gold(log.gold_inout_path,
			std::ifstream::in | std::ifstream::binary);

	if (f_a.is_open() && f_b.is_open() && f_c.is_open() && f_gold) {

		f_a.seekg(0, std::ios::beg);
		f_a.read(reinterpret_cast<char*>(a_host_vector.data()),
				sizeof(half_t) * a_host_vector.size());

		f_b.seekg(0, std::ios::beg);
		f_b.read(reinterpret_cast<char*>(b_host_vector.data()),
				sizeof(half_t) * b_host_vector.size());

		f_c.seekg(0, std::ios::beg);
		f_c.read(reinterpret_cast<char*>(c_host_vector.data()),
				sizeof(real_t) * c_host_vector.size());

		f_gold.seekg(0, std::ios::beg);
		f_gold.read(reinterpret_cast<char*>(gold_host_vector.data()),
				sizeof(real_t) * gold_host_vector.size());

		f_a.close();
		f_b.close();
		f_c.close();
		f_gold.close();
	} else {
		log.log_error("Could not retrieve the matrices");
		throw std::runtime_error("Could not retrieve the matrices\n");
	}

	std::cout << "Done with reading matrices " << log.mysecond() - start
			<< "s\n";
}

template<class real_t>
std::pair<int, int> compare_output_matrices(long long host_is_memory_bad,
		std::vector<real_t>& gold, std::vector<real_t>& c0,
		std::vector<real_t>& c1, std::vector<real_t>& c2, Log& log) {

	int host_errors = 0;
	int memory_errors = 0;

	// printf("r = %f \n", c0[2]);

	std::cout << "host_is_memory_bad: " << host_is_memory_bad << std::endl;

	if (host_is_memory_bad != 0) {
		std::string info_detail = "b: is_memory_bad: "
				+ std::to_string(host_is_memory_bad);
		if (log.verbose)
			std::cout << info_detail << std::endl;

		log.log_error(info_detail);
		memory_errors++;
	}

#ifdef OMP
#pragma omp parallel for shared(host_errors)
#endif
	for (size_t i = 0; i < gold.size(); i++) {
		register bool checkFlag = true;
		register real_t valGold = gold[i];
		register real_t valOutput0 = c0[i];
		register real_t valOutput1 = c1[i];
		register real_t valOutput2 = c2[i];
		register real_t valOutput = valOutput0;

		if ((valOutput0 != valOutput1) || (valOutput0 != valOutput2)) {
#ifdef OMP
#pragma omp critical
#endif
			{
				std::stringstream info_detail("");
				info_detail << "m: [" << int(floor(i / log.size_matrices))
						<< ", " << i % log.size_matrices << "], r0: "
						<< valOutput0 << ", r1: " << valOutput1 << ", r2: "
						<< valOutput2;

				if (log.verbose && (memory_errors < 10))
					std::cout << info_detail.str() << std::endl;

				log.log_info(info_detail.str());
				memory_errors++;
			}
			if ((valOutput0 != valOutput1) && (valOutput1 != valOutput2)
					&& (valOutput0 != valOutput2)) {
				// All 3 values diverge
				if (valOutput0 == valGold) {
					valOutput = valOutput0;
				} else if (valOutput1 == valGold) {
					valOutput = valOutput1;
				} else if (valOutput2 == valGold) {
					valOutput = valOutput2;
				} else {
					// NO VALUE MATCHES THE GOLD AND ALL 3 DIVERGE!
					checkFlag = false;
#ifdef OMP
#pragma omp critical
#endif
					{
						std::stringstream info_detail("");
						info_detail << "t: ["
								<< int(floor(i / log.size_matrices)) << ", "
								<< i % log.size_matrices << "], r0: "
								<< valOutput0 << ", r1: " << valOutput1
								<< ", r2: " << valOutput2 << ", e: " << valGold;

						if (log.verbose && (memory_errors < 10))
							std::cout << info_detail.str() << std::endl;

						log.log_info(std::string(info_detail.str()));

						memory_errors++;
					}
				}
			} else if (valOutput1 == valOutput2) {
				// Only value 0 diverge
				valOutput = valOutput1;
			} else if (valOutput0 == valOutput2) {
				// Only value 1 diverge
				valOutput = valOutput0;
			} else if (valOutput0 == valOutput1) {
				// Only value 2 diverge
				valOutput = valOutput0;
			}
		}
		// std::cout << "val gold: " << valGold << std::endl;
		if (valGold != valOutput) {
			if (checkFlag) {
#ifdef OMP
#pragma omp critical
#endif
				{
					// std::cout << "val out: " << valOutput << std::endl;

					std::stringstream error_detail("");
					error_detail << "p: [" << int(floor(i / log.size_matrices))
							<< ", " << i % log.size_matrices << "], r: "
							<< valOutput << ", e: " << valGold;

					if (log.verbose && (host_errors < 10))
						std::cout << error_detail.str() << std::endl;

					log.log_error(error_detail.str());
					host_errors++;
				}
			}
		}
	}

// printf("numErrors:%d", host_errors);

	log.update_info_count(memory_errors);
	log.update_error_count(host_errors);

	if (memory_errors != 0)
		std::cout << "M";
	if (host_errors != 0)
		std::cout << "#";

	std::pair<int, int> res(memory_errors, host_errors);
	return res;
}

template<class real_t>
bool cmp(const real_t lhs, const real_t rhs, Log& log) {
	const real_t diff = abs(lhs - rhs);
	const real_t zero;

	// std::cout << "d0= " << lhs << "d1 = " << rhs << std::endl;	
	// std::cout << "diff= " << diff << std::endl;

	if (log.use_tensor_cores) {
		zero = real_t(ZERO_HALF);
	} else {
		if (log.precision == "float")
			zero = real_t(ZERO_FlOAT);

		if (log.precision == "double")
			zero = real_t(ZERO_DOUBLE);

		if (log.precision == "DMR")
			zero = real_t(ZERO_DMR);
	}

	if (diff > zero) {
		return false;
	}
	return true;
}

template<class real_t>
std::pair<int, int> check_output_errors_dmr(std::vector<real_t>& gold,
		std::vector<real_t>& d0, std::vector<real_t>& d1, Log& log) {
	int host_errors = 0;

#ifdef OMP
#pragma omp parallel for shared(host_errors)
#endif
	for (size_t i = 0; i < gold.size(); i++) {
		real_t valGold = gold[i];
		real_t valOutput0 = d0[i];
		real_t valOutput1 = d1[i];

		if (valGold != valOutput1 || !cmp(valOutput0, valOutput1, log)) {

			std::stringstream error_detail("");
			error_detail << "p: [" << int(floor(i / log.size_matrices)) << ", "
					<< i % log.size_matrices << "], r: " << valOutput1
					<< ", e: " << valGold << " smaller_precision: "
					<< valOutput0;

			if (log.verbose && (host_errors < 10))
				std::cout << error_detail.str() << std::endl;

			log.log_error(error_detail.str());
			host_errors++;
		}
	}
	log.update_error_count(host_errors);
	if (host_errors != 0)
		std::cout << "#";

	std::pair<int, int> res(0, host_errors);
	return res;
}

template<class real_t>
std::pair<int, int> compare_output_matrices(std::vector<real_t>& gold,
		std::vector<real_t>& c0, Log& log) {
	int host_errors = 0;

#ifdef OMP
#pragma omp parallel for shared(host_errors)
#endif
	for (size_t i = 0; i < gold.size(); i++) {
		real_t valGold = gold[i];
		real_t valOutput = c0[i];
		if (valGold != valOutput) {
			std::stringstream error_detail("");
			error_detail << "p: [" << int(floor(i / log.size_matrices)) << ", "
					<< i % log.size_matrices << "], r: " << valOutput << ", e: "
					<< valGold;

			if (log.verbose && (host_errors < 10))
				std::cout << error_detail.str() << std::endl;

			log.log_error(error_detail.str());
			host_errors++;
		}
	}
	log.update_error_count(host_errors);
	if (host_errors != 0)
		std::cout << "#";

	std::pair<int, int> res(0, host_errors);
	return res;
}

template<class half_t, class real_t>
void call_mxm(Log& log_obj) {
	hipEvent_t start, stop;
	float elapsedTime;
	// Matrices A and B
	std::vector<half_t> host_matrix_a(
			log_obj.size_matrices * log_obj.size_matrices);
	std::vector<half_t> host_matrix_b(
			log_obj.size_matrices * log_obj.size_matrices);

// C matrix
	std::vector<real_t> host_matrix_c(
			log_obj.size_matrices * log_obj.size_matrices);
	std::vector<real_t> host_gold(
			log_obj.size_matrices * log_obj.size_matrices);
// D Matrix
	std::vector<real_t> host_matrix_d0(
			log_obj.size_matrices * log_obj.size_matrices);
	std::vector<real_t> host_matrix_d1(
			log_obj.size_matrices * log_obj.size_matrices);
	std::vector<real_t> host_matrix_d2(
			log_obj.size_matrices * log_obj.size_matrices);

	if (!log_obj.generate) {
		retrieve_matrices<half_t, real_t>(host_matrix_a, host_matrix_b,
				host_matrix_c, host_gold, log_obj);
	} else {
		generate_matrices_files<real_t, half_t>(host_matrix_a, host_matrix_b,
				host_matrix_c, log_obj);
	}

	//TODO
	//Set the correct version
	GEMMTYPE gemm_t = DMRGEMM;

	GEMM<half_t, real_t> mult_enviroment(host_matrix_a, host_matrix_b,
			host_matrix_c, log_obj.size_matrices, real_t(1.1f), real_t(1.2f));

	int tries = 0;
	hipEventCreate(&start);
	hipEventRecord(start, 0);
	// hipStream_t st;
	// hipStreamCreate(&st);	
	// assert(M_O > 512 && N_O > 512 && M_O % 64 == 0 && N_O % 16 == 0 && K_O % 16 == 0);
	for (int it = 0; it < log_obj.iterations; it++) {
		double start_computation = log_obj.mysecond();
		log_obj.start_iteration_app();
		mult_enviroment.gemm();
		log_obj.end_iteration_app();
		double end_computation = log_obj.mysecond();

		mult_enviroment.pull_array(host_matrix_d0, host_matrix_d1,
				host_matrix_d2);

		hipEventCreate(&stop);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsedTime, start, stop);
		printf("Elapsed time : %f ms\n", elapsedTime);

		//TODO check this
		if (log_obj.triplicated && log_obj.generate) {
			tries++;
			int has_errors = is_output_ok(host_matrix_d0, host_matrix_d1,
					host_matrix_d2, host_gold);
			if (has_errors != 0)
				it--;

			if (tries > 5)
				throw std::runtime_error(
						"More than 5 tries on matrix generate\n");
			std::cout << "Iteration: " << it << std::endl;
		}

		if (!log_obj.generate) {
			//fault test 
//			if(it == 2){
//			host_matrix_d0[2]= (real_t) 5.00;
//			}
			//
			std::pair<int, int> errors;
			double start, end;
			if (log_obj.triplicated) {
				start = log_obj.mysecond();
				errors = compare_output_matrices(
						mult_enviroment.get_memory_errors(), host_gold,
						host_matrix_d0, host_matrix_d1, host_matrix_d2,
						log_obj);
				end = log_obj.mysecond();
			} else {
				start = log_obj.mysecond();
				//errors = compare_output_matrices(host_gold, host_matrix_d0, log_obj);

				// errors = check_output_errors_dmr(host_gold, host_matrix_d0, host_matrix_d1,log_obj);
				end = log_obj.mysecond();
			}
			std::cout << "Iteration: " << it << " memory errors "
					<< errors.first << " radiation errors " << errors.second
					<< ". Time spent on computation "
					<< end_computation - start_computation
					<< "s. Time spent on comparing " << end - start << "s."
					<< std::endl;

			//If errors != 0 reload matrices to gpu
			if (errors.first != 0 || errors.second != 0) {
				mult_enviroment.push_arrays(host_matrix_a.data(),
						host_matrix_b.data(), host_matrix_c.data());
			}

		}

	}
	// hipStreamDestroy(st);
	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time : %f s\n", (elapsedTime / 1000));
	if (log_obj.generate) {
		if (log_obj.triplicated)
			write_gold_to_file<real_t>(log_obj.gold_inout_path, host_gold);
		else
			write_gold_to_file<real_t>(log_obj.gold_inout_path,
					host_matrix_d1);

	}

	// for(int z = 1;z < 15 ; z++) {
	// std::cout << "d0 = " << host_matrix_d0 [z] << " ||  d1 = " << host_matrix_d1 [z] << std::endl;
	// std::cout << "diff = " << (host_matrix_d0[0] - host_matrix_d1[0])<< std::endl;
	// }	

}

void usage(char **argv) {
	std::cout << "./" << argv[0]
			<< " --generate 0/1 --gold <gold file, DEFAULT=./gold.matrix > --size <matrix size, DEFAULT=8192> "
					"--iterations <how many iterations, optional> --input_a <input A, DEFAUL=./input_a.matrix> "
					"--input_b <input B, DEFAUL=./input_b.matrix> --input_c <input C, DEFAUL=./input_c.matrix>  --precision <float/double, DEFAULT=float>"
			<< std::endl;
}

int main(int argc, char** argv) {
	Log log_obj(argc, argv, DEFAULT_INPUT_SIZE);

	std::cout << "Generate: " << log_obj.generate << std::endl;
	std::cout << "A input path: " << log_obj.a_input_path << std::endl;
	std::cout << "B input path: " << log_obj.b_input_path << std::endl;
	std::cout << "C input path: " << log_obj.c_input_path << std::endl;
	std::cout << "Gold in/out path: " << log_obj.gold_inout_path << std::endl;
	std::cout << "Iterations: " << log_obj.iterations << std::endl;
	std::cout << "Matrix size: " << log_obj.size_matrices << std::endl;
	std::cout << "Precision: " << log_obj.precision << std::endl;
	std::cout << "Verbose: " << log_obj.verbose << std::endl;

// Alloc all memories on host
	if (log_obj.use_tensor_cores) {
		call_mxm<half, half>(log_obj);
	}

	// if (log_obj.precision == "float") {
	// 	call_mxm<float, float, half, float>(host_matrix_a, host_matrix_b, log_obj);
	// }
	// if (log_obj.precision == "double") {
	// 	call_mxm<double, double, half, double>(host_matrix_a, host_matrix_b, log_obj);
	// }
	// if (log_obj.precision == "DMR") {
	// 	call_mxm<double, double, half, float>(host_matrix_a, host_matrix_b, log_obj);
	// }

	std::cout << "Finished computation\n";
	return 0;
}
