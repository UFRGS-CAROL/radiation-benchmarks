#include "kernels.h"
#include <cassert>
#include <vector>
#include <iostream>

#define CHECK_BLOCK 16
#define THRESHOLD 1

int main(int argc, char **argv) {

	int m;
	int n;
	int k;
	m = n = k = 2048;

	real_t alpha = 1;
	real_t beta = 1;

	real_t* host_a = (real_t*)calloc(m * k, sizeof(real_t));
	real_t* host_b = (real_t*)calloc(k * n, sizeof(real_t));
	real_t* host_c = (real_t*)calloc(m * n, sizeof(real_t));
	half_t* host_c_half = (half_t*)calloc(m * n, sizeof(half_t));

	for (int i = 0; i < m * k; i++) host_a[i] = alpha;
	for (int i = 0; i < m * k; i++) host_b[i] = beta;

	real_t *device_a, *device_b, *device_c;
	half_t *device_c_half;
	hipMalloc((void**)&device_a, m * k * sizeof(real_t));
	hipMalloc((void**)&device_b, k * n * sizeof(real_t));
	hipMalloc((void**)&device_c, m * n * sizeof(real_t));
	hipMalloc((void**)&device_c_half, m * n * sizeof(half_t));

	hipMemcpy(device_a, host_a, m * k * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_b, host_b, k * n * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_c, host_c, m * n * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_c_half, host_c_half, m * n * sizeof(half_t), hipMemcpyHostToDevice);

	matrix_mult_dmr<THRESHOLD, CHECK_BLOCK>(device_a, device_b, m, n, k, device_c, device_c_half);

	hipMemcpy(host_c, device_c, m * n * sizeof(real_t), hipMemcpyDeviceToHost);
	hipMemcpy(host_c_half, device_c_half, m * n * sizeof(half_t), hipMemcpyDeviceToHost);

    std::cout << "FLOAT" << std::endl;
	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < 10; j++) {
			std::cout << host_c[i * m + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << "HALF" << std::endl;
	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < 10; j++) {
			std::cout << host_c_half[i * m + j] << " ";
		}
		std::cout << std::endl;
	}
    
	return 0;
}