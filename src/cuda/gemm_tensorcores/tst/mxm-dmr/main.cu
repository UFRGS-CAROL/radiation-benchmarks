#include "kernels.h"
#include <cassert>
#include <vector>
#include <iostream>

#define CHECK_BLOCK 16
#define THRESHOLD 1

typedef double real_t;
typedef float half_t;




int main(int argc, char **argv) {

	int m;
	int n;
	int k;
	m = n = k = 4096;

	real_t alpha = 1;
	real_t beta = 1;

	real_t* host_a = (real_t*)calloc(m * k, sizeof(real_t));
	real_t* host_b = (real_t*)calloc(k * n, sizeof(real_t));
	real_t* host_c = (real_t*)calloc(m * n, sizeof(real_t));
	real_t* host_d = (real_t*)calloc(m * n, sizeof(real_t));
	half_t* host_d_half = (half_t*)calloc(m * n, sizeof(half_t));

	for (int i = 0; i < m * k; i++) host_a[i] = alpha;
	for (int i = 0; i < m * k; i++) host_b[i] = beta;
	for (int i = 0; i < m * k; i++) host_c[i] = 0;	

	real_t *device_a, *device_b, *device_c, *device_d;
	half_t *device_d_half;
	hipMalloc((void**)&device_a, m * k * sizeof(real_t));
	hipMalloc((void**)&device_b, k * n * sizeof(real_t));
	hipMalloc((void**)&device_c, m * n * sizeof(real_t));
	hipMalloc((void**)&device_d, m * n * sizeof(real_t));
	hipMalloc((void**)&device_d_half, m * n * sizeof(half_t));

	hipMemcpy(device_a, host_a, m * k * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_b, host_b, k * n * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_c, host_c, m * n * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_d, host_d, m * n * sizeof(real_t), hipMemcpyHostToDevice);
	hipMemcpy(device_d_half, host_d_half, m * n * sizeof(half_t), hipMemcpyHostToDevice);

	matrix_mult_dmr<THRESHOLD, CHECK_BLOCK, real_t, half_t>(device_a, device_b, m, n, k, device_d, device_d_half, alpha, beta, device_c);


	
	hipMemcpy(host_d, device_d, m * n * sizeof(real_t), hipMemcpyDeviceToHost);
	hipMemcpy(host_d_half, device_d_half, m * n * sizeof(half_t), hipMemcpyDeviceToHost);

    std::cout << "FLOAT" << std::endl;
	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < 10; j++) {
			std::cout << host_d[i * m + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << "HALF" << std::endl;
	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < 10; j++) {
			std::cout << host_d_half[i * m + j] << " ";
		}
		std::cout << std::endl;
	}
    
	return 0;
}