#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 9.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
// matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the
//   A and B data from shared memory, thus reducing the number of data copies
//   from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
// additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
// warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory
//   contents to global memory, again avoiding redundant random global memory
//   accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
// utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define BLOCK_SIZE 32

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 8 two-byte
// "half" elements is chosen as the minimum possible shift because we must keep
// each row and column 128-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 8

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

__host__ void init_host_matrices(half *a, half *b, half *c) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      
      a[i * K_GLOBAL + j] = 1.0;
    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      
      b[i * K_GLOBAL + j] = 1.0;

    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      
      c[i * K_GLOBAL + j] = 1.0;

    }
  }
}

__global__ void matrix_mult_kernel_unhardened(	//Kernel without hardening
		half *A,  //A
		half *B,  //B
		half *C,  //C
		half alpha, half beta, int wA, int wB) {
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	half Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ half As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ half Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	const int index = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx + wB * ty + tx;
	C[index] = alpha * Csub + beta * C[index];
}


__global__ void compute_gemm(const half *A, const half *B, const half *C,
		half *D, half alpha, half beta) {
	extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

	// Warp and lane identification.
	const unsigned int warpId = threadIdx.x / WARP_SIZE;
	const unsigned int laneId = threadIdx.x % WARP_SIZE;

	// Offset in shared memory from which the B matrix is stored.
	const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

	// This pointer is used to access the C and D matrix tiles this warp computes.
	half *shmem_warp_tile_ptr = (half *) &shmem[0][0]
			+ (warpId / 2) * SHMEM_STRIDE * K * 2+
			(warpId % 2) * SHMEM_OFFSET;

	// This pointer is used to stream the C and D matrices block-wide tile to and
	// from shared memory.
	half *shmem_warp_stream_ptr = (half *) &shmem[0][0]
			+ warpId * SHMEM_STRIDE * K;

	// Adjust the beta scaler, as it'll be multiplied by alpha at the end of
	// each tile computation. Technically this is not generally correct (may
	// result in a loss of precision). Zero still needs to be specially handled
	// though.
	beta /= alpha;

	// Each CTA slides along the 128 x 128 tiles from the top left corner of the
	// matrix to the right and down, and selects the next tile to compute. Once
	// there's no such tile, all warps in this CTA exit.
	for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
		const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES)
				/ N_TILES) * (BLOCK_COL_TILES);
		const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES)
				% N_TILES;

		// Stop when there are no more D matrix tiles to compute in this CTA.
		if (block_tile_i >= M_TILES) {
			break;
		}

		// This warp's pointer to the C matrix data to copy memory from to shared
		// memory.
		const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE
				+ block_tile_j * N;
		const half *src_gmem_warp_stream_ptr = &C[gmem_idx];

		// Stream multiple C tiles to shared memory.
#pragma unroll
		for (int i = 0; i < K; i++) {
			typedef int4 copy_t;

			*((copy_t *) (shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
					*((copy_t *) (src_gmem_warp_stream_ptr
							+ GLOBAL_MEM_STRIDE * i) + laneId);
		}

		__syncthreads();

		// These fragments will accumulate the result of A and B matrix fragment
		// multiplications along the K_GLOBAL dimension.
		nvcuda::wmma::fragment < nvcuda::wmma::accumulator, M, N, K, half
				> c[WARP_COL_TILES][WARP_ROW_TILES];

		// Load the C matrix tiles into fragments from shared memory.
#pragma unroll
		for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
			for (int j = 0; j < WARP_ROW_TILES; j++) {
				const half *tile_ptr = shmem_warp_tile_ptr
						+ i * SHMEM_STRIDE * K + j * N;

				nvcuda::wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE,
				C_LAYOUT);
			}
		}

		__syncthreads();

		// Scale the C matrix.
#pragma unroll
		for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
			for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
				for (int t = 0; t < c[i][j].num_elements; t++) {
					c[i][j].x[t] *= beta;
				}
			}
		}

		// Select what warp copies what matrix to shared memory.
		// Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
		const half *warp_ptr =
				(warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
				M * K_GLOBAL * (warpId % 4) * 2) :
								(&B[block_tile_j * N * K_GLOBAL] +
								N * K_GLOBAL * (warpId % 4) * 2);

		// Go through the global K dimension by a fixed step at a time.
#pragma unroll
		for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
			// Copy slices of the A and B matrices to shared memory.
			// The first half of the warps in the CTA copy the A matrix, the rest copy
			// the B matrix.
			size_t shmem_idx =
					warpId < (WARPS_PER_BLOCK / 2) ?
							(M * (warpId % (WARPS_PER_BLOCK / 2)) * 2) :
							(N * (warpId % (WARPS_PER_BLOCK / 2)) * 2
									+ shmem_idx_b_off);

			// First half of the warp copies the first row / column of the matrix,
			// the second half of the warp copies the next.
			int4 *lane_ptr = (int4 *) (warp_ptr + tile_k * K
					+ (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL)
					+ (laneId % CHUNK_COPY_LINE_LANES);

			// Shift the second half of the warp to the next row / column in the
			// shared memory.
			shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
			for (int i = 0;
					i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
					i++) {
				// Copy 16 bytes dodasat once in each lane.
				*((int4 *) &shmem[shmem_idx][0]
						+ (laneId % CHUNK_COPY_LINE_LANES)) = *lane_ptr;

				// Advance the global memory pointer and the shared memory index.
				lane_ptr = (int4 *) ((half *) lane_ptr
						+ K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
				shmem_idx += CHUNK_COPY_LINES_PER_WARP;
			}

			__syncthreads();

			// Compute a grid of C matrix tiles in each warp.
#pragma unroll
			for (int k_step = 0; k_step < CHUNK_K; k_step++) {
				nvcuda::wmma::fragment < nvcuda::wmma::matrix_a, M, N, K, half, nvcuda::wmma::row_major
						> a[WARP_COL_TILES];
				nvcuda::wmma::fragment < nvcuda::wmma::matrix_b, M, N, K, half, nvcuda::wmma::col_major
						> b[WARP_ROW_TILES];

#pragma unroll
				for (int i = 0; i < WARP_COL_TILES; i++) {
					size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
					const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

					nvcuda::wmma::load_matrix_sync(a[i], tile_ptr,
					K * CHUNK_K + SKEW_HALF);

#pragma unroll
					for (int j = 0; j < WARP_ROW_TILES; j++) {
						if (i == 0) {
							// Load the B matrix fragment once, because it is going to be
							// reused against the other A matrix fragments.
							size_t shmem_idx_b = shmem_idx_b_off
									+ (WARP_ROW_TILES * N) * (warpId % 2)
									+ (j * N);
							const half *tile_ptr = &shmem[shmem_idx_b][k_step
									* K];

							nvcuda::wmma::load_matrix_sync(b[j], tile_ptr,
							K * CHUNK_K + SKEW_HALF);
						}

						nvcuda::wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
					}
				}
			}

			__syncthreads();
		}

		// Store the D fragments to shared memory.
#pragma unroll
		for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
			for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
				// Uniform, point-wise transformations of ALL fragment elements by ALL
				// threads in the warp are well-defined even though element indices
				// within fragment storage are not defined.
				for (int t = 0; t < c[i][j].num_elements; t++)
					c[i][j].x[t] *= alpha;

				half *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K
						+ j * N;

				nvcuda::wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE,
				C_LAYOUT);
			}
		}

		__syncthreads();

		// Now that shared memory contains all the D tiles, stream them to global
		// memory.
		half *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
		for (int i = 0; i < K; i++) {
			*((int4 *) (dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i)
					+ laneId) = *((int4 *) (shmem_warp_stream_ptr
					+ SHMEM_STRIDE * i) + laneId);
		}

		__syncthreads();
	}
}


int main(int argc, char **argv) {
  printf("Initializing...\n");

  int dev = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

  // Tensor cores require a GPU of Volta (SM7X) architecture or higher.
  if (deviceProp.major < 7) {
    printf(
        "cudaTensorCoreGemm requires SM 7.0 or higher to use Tensor "
        "Cores.  Exiting...\n");
    exit(EXIT_WAIVED);
  }

  printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
  printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
  printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

  half *A_h = NULL;
  half *B_h = NULL;
  half *C_h = NULL;

  half *result_hD = NULL;
  half *result_sw = NULL;


  A_h = (half *)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
  B_h = (half *)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
  C_h = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);

  result_hD = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
  result_sw = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);


  half *A = NULL;
  half *B = NULL;
  half *C = NULL;
  half *D = NULL;
  half *D_sw = NULL;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A),
                             sizeof(half) * M_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B),
                             sizeof(half) * N_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C),
                             sizeof(half) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D),
                             sizeof(half) * M_GLOBAL * N_GLOBAL));
   checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D_sw),
                             sizeof(half) * M_GLOBAL * N_GLOBAL));

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);
  assert(((unsigned long long)D) % 128 == 0);
  assert(((unsigned long long)D_sw) % 128 == 0);

  init_host_matrices(A_h, B_h, C_h);

  printf("Preparing data for GPU...\n");

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(half) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(D, 0, sizeof(half) * M_GLOBAL * N_GLOBAL));

  checkCudaErrors(hipMemset(D_sw, 0, sizeof(half) * M_GLOBAL * N_GLOBAL));

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
            (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(half))
  };

  printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

  const half alpha = 1.0f;
  const half beta = 1.0f;

  hipEvent_t start, stop;


  dim3 dim_grid, dim_block;
  
  uint32_t grid_rows = (M_GLOBAL + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint32_t grid_cols = (N_GLOBAL + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim_grid = dim3(grid_cols, grid_rows);
  dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);


  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));

  // If enough shared memory available on the GPU use high performant kernel
  // if (deviceProp.sharedMemPerMultiprocessor >= SHMEM_SZ) {
    printf("Computing... using high performance kernel compute_gemm \n");

    // checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(
    //     compute_gemm), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
    // checkKernelErrors(
    //     (compute_gemm<<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK,
    //                     SHMEM_SZ>>>(A, B, C, D, alpha, beta)));

    // checkCudaErrors(hipMemcpy(result_hD, D,
    //                            sizeof(half) * M_GLOBAL * N_GLOBAL,
    //                            hipMemcpyDeviceToHost));


	

  matrix_mult_kernel_unhardened<<<dim_grid, dim_block>>>(A, B, D_sw, alpha, beta, M_GLOBAL, N_GLOBAL);
  checkCudaErrors(hipMemcpy(result_sw, D_sw,
                           sizeof(half) * M_GLOBAL * N_GLOBAL,
                           hipMemcpyDeviceToHost));


  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));


  printf("Verifying correctness of the computations...\n");

  // memcpy(result_sw, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL);

  // matMultiplyOnHost(A_h, B_h, result_sw, alpha, beta, M_GLOBAL, K_GLOBAL,
  //                   K_GLOBAL, N_GLOBAL, M_GLOBAL, N_GLOBAL);


  for (int i = 0; i < 10; i++) {
   
      printf("mismatch i=%d result_hD=%f result_sw=%f\n", i, (double)result_hD[i],
             (double)result_sw[i]);
  }
  // for (int i = 0; i < N_GLOBAL * M_GLOBAL; i++) {
  //   if (fabs(result_hD[i] - result_sw[i]) > 0.1f)
  //     printf("mismatch i=%d result_hD=%f result_sw=%f\n", i, result_hD[i],
  //            result_sw[i]);
  // }
  free(result_hD);
  free(result_sw);


  float milliseconds = 0;

  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Time: %f ms\n", milliseconds);
  printf("TFLOPS: %.2f\n", static_cast<double>((static_cast<double>(M_GLOBAL) *
                                                N_GLOBAL * K_GLOBAL * 2) /
                                               (milliseconds / 1000.)) /
                               1e12);

  free(A_h);
  free(B_h);
  free(C_h);
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(D)));

  return 0;
}