#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>
#include <stdint.h>

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#include <mma.h>
using namespace nvcuda;

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT nvcuda::wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)


// Must be multiples of 16 for wmma code to work
#define MATRIX_M 4096 //16384
#define MATRIX_N 4096 //16384
#define MATRIX_K 4096 //16384

#define BLOCK_SIZE 32




// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__device__ __forceinline__ void axpy__(const double a, const double b, double &c) {
    c = __fma_rn(a, b, c);
}
__device__ __forceinline__ void axpy__(const float a, const float b, float &c) {
    //printf("A = %f   -- B =  %f\n", a, b);
    c = __fmaf_rn(a, b, c);
}
__device__ __forceinline__ void axpy__(const double a, const double b, float &c) {
    c = __fmaf_rn(__double2float_rn(a), __double2float_rn(b), c);
}
__device__ __forceinline__ void axpy__(const float a, const float b, __half &c) {
    c = __hfma(__float2half(a), __float2half(b), c);
}

__device__  __forceinline__ half axpy__(half a, half b, half acc) {
  return __hfma(a, b, acc);
}



__global__ void compute_gemm(const half *A, const half *B, const half *C,
    half *D, half alpha, half beta, int wA, int wB) {
  extern __shared__ half shmem[][CHUNK_K * K + SKEW_HALF];

  // Warp and lane identification.
  const unsigned int warpId = threadIdx.x / WARP_SIZE;
  const unsigned int laneId = threadIdx.x % WARP_SIZE;

  // Offset in shared memory from which the B matrix is stored.
  const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

  // This pointer is used to access the C and D matrix tiles this warp computes.
  half *shmem_warp_tile_ptr = (half *) &shmem[0][0]
      + (warpId / 2) * SHMEM_STRIDE * K * 2+
      (warpId % 2) * SHMEM_OFFSET;

  // This pointer is used to stream the C and D matrices block-wide tile to and
  // from shared memory.
  half *shmem_warp_stream_ptr = (half *) &shmem[0][0]
      + warpId * SHMEM_STRIDE * K;

  // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
  // each tile computation. Technically this is not generally correct (may
  // result in a loss of precision). Zero still needs to be specially handled
  // though.
  beta /= alpha;

  // Each CTA slides along the 128 x 128 tiles from the top left corner of the
  // matrix to the right and down, and selects the next tile to compute. Once
  // there's no such tile, all warps in this CTA exit.
  for (unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
    const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES)
        / N_TILES) * (BLOCK_COL_TILES);
    const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES)
        % N_TILES;

    // Stop when there are no more D matrix tiles to compute in this CTA.
    if (block_tile_i >= M_TILES) {
      break;
    }

    // This warp's pointer to the C matrix data to copy memory from to shared
    // memory.
    const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE
        + block_tile_j * N;
    const half *src_gmem_warp_stream_ptr = &C[gmem_idx];

    // Stream multiple C tiles to shared memory.
#pragma unroll
    for (int i = 0; i < K; i++) {
      typedef int4 copy_t;

      *((copy_t *) (shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
          *((copy_t *) (src_gmem_warp_stream_ptr
              + GLOBAL_MEM_STRIDE * i) + laneId);
    }

    __syncthreads();

    // These fragments will accumulate the result of A and B matrix fragment
    // multiplications along the K_GLOBAL dimension.
    nvcuda::wmma::fragment < nvcuda::wmma::accumulator, M, N, K, half
        > c[WARP_COL_TILES][WARP_ROW_TILES];

    // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
        const half *tile_ptr = shmem_warp_tile_ptr
            + i * SHMEM_STRIDE * K + j * N;

        nvcuda::wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE,
        C_LAYOUT);
      }
    }

    __syncthreads();

    // Scale the C matrix.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        for (int t = 0; t < c[i][j].num_elements; t++) {
          c[i][j].x[t] *= beta;
        }
      }
    }

    // Select what warp copies what matrix to shared memory.
    // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
    const half *warp_ptr =
        (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] +
        M * K_GLOBAL * (warpId % 4) * 2) :
                (&B[block_tile_j * N * K_GLOBAL] +
                N * K_GLOBAL * (warpId % 4) * 2);

    // Go through the global K dimension by a fixed step at a time.
#pragma unroll
    for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
      // Copy slices of the A and B matrices to shared memory.
      // The first half of the warps in the CTA copy the A matrix, the rest copy
      // the B matrix.
      size_t shmem_idx =
          warpId < (WARPS_PER_BLOCK / 2) ?
              (M * (warpId % (WARPS_PER_BLOCK / 2)) * 2) :
              (N * (warpId % (WARPS_PER_BLOCK / 2)) * 2
                  + shmem_idx_b_off);

      // First half of the warp copies the first row / column of the matrix,
      // the second half of the warp copies the next.
      int4 *lane_ptr = (int4 *) (warp_ptr + tile_k * K
          + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL)
          + (laneId % CHUNK_COPY_LINE_LANES);

      // Shift the second half of the warp to the next row / column in the
      // shared memory.
      shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
      for (int i = 0;
          i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2;
          i++) {
        // Copy 16 bytes at once in each lane.
        *((int4 *) &shmem[shmem_idx][0]
            + (laneId % CHUNK_COPY_LINE_LANES)) = *lane_ptr;

        // Advance the global memory pointer and the shared memory index.
        lane_ptr = (int4 *) ((half *) lane_ptr
            + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
        shmem_idx += CHUNK_COPY_LINES_PER_WARP;
      }

      __syncthreads();

      // Compute a grid of C matrix tiles in each warp.
#pragma unroll
      for (int k_step = 0; k_step < CHUNK_K; k_step++) {
        nvcuda::wmma::fragment < nvcuda::wmma::matrix_a, M, N, K, half, nvcuda::wmma::row_major
            > a[WARP_COL_TILES];
        nvcuda::wmma::fragment < nvcuda::wmma::matrix_b, M, N, K, half, nvcuda::wmma::col_major
            > b[WARP_ROW_TILES];

#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
          size_t shmem_idx_a = (warpId / 2) * M * 2 + (i * M);
          const half *tile_ptr = &shmem[shmem_idx_a][k_step * K];

          nvcuda::wmma::load_matrix_sync(a[i], tile_ptr,
          K * CHUNK_K + SKEW_HALF);

#pragma unroll
          for (int j = 0; j < WARP_ROW_TILES; j++) {
            if (i == 0) {
              // Load the B matrix fragment once, because it is going to be
              // reused against the other A matrix fragments.
              size_t shmem_idx_b = shmem_idx_b_off
                  + (WARP_ROW_TILES * N) * (warpId % 2)
                  + (j * N);
              const half *tile_ptr = &shmem[shmem_idx_b][k_step
                  * K];

              nvcuda::wmma::load_matrix_sync(b[j], tile_ptr,
              K * CHUNK_K + SKEW_HALF);
            }

            nvcuda::wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
          }
        }
      }

      __syncthreads();
    }

    // Store the D fragments to shared memory.
#pragma unroll
    for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
      for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
        // Uniform, point-wise transformations of ALL fragment elements by ALL
        // threads in the warp are well-defined even though element indices
        // within fragment storage are not defined.
        for (int t = 0; t < c[i][j].num_elements; t++)
          c[i][j].x[t] *= alpha;

        half *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K
            + j * N;

        nvcuda::wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE,
        C_LAYOUT);
      }
    }

    __syncthreads();

    // Now that shared memory contains all the D tiles, stream them to global
    // memory.
    half *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
    for (int i = 0; i < K; i++) {
      *((int4 *) (dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i)
          + laneId) = *((int4 *) (shmem_warp_stream_ptr
          + SHMEM_STRIDE * i) + laneId);
    }

    __syncthreads();
  }
}

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.

__global__ void wmma_example(half *a, half *b, half *c, int M, int N, int K, half alpha, half beta) {
   // Leading dimensions. Packed with no transpositions.
   int lda = M;
   int ldb = K;
   int ldc = M;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < K; i += WMMA_K) {
      int aRow = warpM * WMMA_M;
      int aCol = i;

      int bRow = i;
      int bCol = warpN * WMMA_N;

      // Bounds checking
      if (aRow < M && aCol < K && bRow < K && bCol < N) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < M && cCol < N) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);


      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16. 
//  3) Neither A nor B are transposed.
// Note: This is NOT a high performance example but is for demonstration purposes only
//       For a high performance code please use the GEMM provided in cuBLAS.


// __global__ void wmma_example_dmr(half *a, half *b, float *c, float *d_sw, float *d_wmma, int M, int N, int K, float alpha, float beta) {

//   // Leading dimensions. Packed with no transpositions.
//   int lda = M;
//   int ldb = K;
//   int ldc = M;
 

//   // Tile using a 2D grid
//   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
//   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

//   // Declare the fragments
//   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
//   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
//   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
//   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
      
//   int row = blockIdx.x * blockDim.x + threadIdx.x;
//   int col = blockIdx.y * blockDim.y + threadIdx.y;

        
//     if (row < M && col < N) {
//       register float acc_real_t = 0.0;
         

     
//       for (int i = 0; i < K; i++) {
//         axpy__((float)a[row * M + i], (float)b[col * N + i], acc_real_t);
//       }   
       

     

//       d_sw[row * M + col] = acc_real_t;
        
//     }
    
    
    
    
//   wmma::fill_fragment(acc_frag, 0.0f);

//     // Loop over k
//   for (int i = 0; i < K; i += WMMA_K) {
//     int aRow = warpM * WMMA_M;
//     int aCol = i;

//     int bRow = i;
//     int bCol = warpN * WMMA_N;

    
//     // Bounds checking
//     if (aRow < M && aCol < K && bRow < K && bCol < N) {
//          // Load the inputs
//       wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
//       wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

//          // Perform the matrix multiplication
//       wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

//     }
// }

//    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
// int cRow = warpM * WMMA_M;
// int cCol = warpN * WMMA_N;

// if (cRow < M && cCol < N) {
//   wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);


//   for(int i=0; i < c_frag.num_elements; i++) {
   
//     c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
//     if (row < M && col < N) {
      
//       register float acc_real_t = 0.0;

//       //for (int internal = i; internal < WMMA_N; internal++) {
//       //  axpy__((float)a[row * M + internal], (float)b[col * N + internal], acc_real_t);    
//       for (int i = 0; i < K; i++) {
//         acc_real_t += (float)a[row * M + i] * (float)b[col * N + i];
//       }   
      
//       d_sw[row * M + col] = acc_real_t * alpha + beta * c[row * M + col];
//     }      
//   }

//       // Store the output
//   wmma::store_matrix_sync(d_wmma + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
// }
   

// }



__global__ void matrix_mult(half *A, half *B, int wA,
    int wB, half *C, half alpha, half beta) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  half Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ half As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ half Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  //int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  const int index = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx + wB * ty + tx;
  half half_val = alpha * Csub + beta * C[index];
  C[index] = Csub;
}



__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}



int main(int argc, char* argv[]) {
  float *a_fp32;
  float *b_fp32;
  half *a_fp16;
  half *b_fp16;

  half *c;
  half *c_wmma;
  half *c_cublas;
  half *d_wmma;
  half *d_sw;  
  half *d_host_cublas;
  half *d_host_wmma;
  half *d_host_sw;
  

  
  hiprandGenerator_t gen;
  hipblasHandle_t cublasHandle;
  
  hipEvent_t startWMMA;
  hipEvent_t stopWMMA;
  hipEvent_t startMXM;
  hipEvent_t stopMXM;

  hipEvent_t startcublas;
  hipEvent_t stopcublas;
   
  cudaErrCheck(hipEventCreate(&startWMMA));
  cudaErrCheck(hipEventCreate(&stopWMMA));

  cudaErrCheck(hipEventCreate(&startMXM));
  cudaErrCheck(hipEventCreate(&stopMXM));
  
  cudaErrCheck(hipEventCreate(&startcublas));
  cudaErrCheck(hipEventCreate(&stopcublas));
   
   
  cublasErrCheck(hipblasCreate(&cublasHandle));
   
  // Use tensor cores
  cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
  
  cudaErrCheck(hipMalloc((void**)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));
  cudaErrCheck(hipMalloc((void**)&b_fp32, MATRIX_K * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMalloc((void**)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&b_fp16, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&c, MATRIX_M * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&d_sw, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void**)&d_wmma, MATRIX_K * MATRIX_N * sizeof(half)));

  d_host_cublas = (half*)malloc(MATRIX_M * MATRIX_N * sizeof(half));
  d_host_wmma = (half*)malloc(MATRIX_M * MATRIX_N * sizeof(half));
  d_host_sw = (half*)malloc(MATRIX_M * MATRIX_N * sizeof(half));

  
   
   curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

   curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
   curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

  
   
   // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (a_fp16, a_fp32, MATRIX_M * MATRIX_K);
   convertFp32ToFp16 <<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (b_fp16, b_fp32, MATRIX_K * MATRIX_N);

   //curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));
   
   curandErrCheck(hiprandDestroyGenerator(gen));
   


  //cudaErrCheck(hipMemset(a_fp16, 6462.8195679, MATRIX_M * MATRIX_N * sizeof(half)));
  //cudaErrCheck(hipMemset(b_fp16, 6462.8195679, MATRIX_M * MATRIX_N * sizeof(half)));

  cudaErrCheck(hipMemset(c_cublas, 0.0f, MATRIX_M * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMemset(c_wmma, 0.0f, MATRIX_M * MATRIX_N * sizeof(half)));
  
  cudaErrCheck(hipMemset(d_sw, 0.0f, MATRIX_M * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMemset(d_wmma, 0.0f, MATRIX_M * MATRIX_N * sizeof(half)));



 
  half alpha = 1.0f;
  half beta = 1.0f;


   
   

  // WMMA TENSOR //
  dim3 gridDim;
  dim3 blockDim;

  dim3 dim_grid, dim_block;
  
  uint32_t grid_rows = (MATRIX_M + BLOCK_SIZE - 1) / BLOCK_SIZE;
  uint32_t grid_cols = (MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim_grid = dim3(grid_cols, grid_rows);
  dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
 
  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
   
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

   
  //printf("Running with wmma thread dimensions...\n");
  //cudaErrCheck(hipEventRecord(startWMMA));
  //wmma_example <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  
  //cudaErrCheck(hipEventRecord(stopWMMA));

   

   // MXM DIMENSIONS
   
  //blockDim.x = WMMA_M; //128;
  //blockDim.y = WMMA_N;
  //blockDim.x = 128;
  //blockDim.y = 4;

  //printf("Running  mxm with MXM thread dimensions...\n");

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF)
            * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N
            * (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(half))
  };

 
  hipStream_t stream1, stream2;
  cudaErrCheck(hipStreamCreate(&stream1)); 
  cudaErrCheck(hipStreamCreate(&stream2));
  int ite = 10; 
  //printf("Running  dmr with MXM thread dimensions...\n");
  cudaErrCheck(hipEventRecord(startMXM));


   for (int i = 0; i < ite; ++i)
   {
     // ---- MXM SW ----//
    matrix_mult<<< dim_grid, dim_block,0, stream1 >>> (a_fp16, b_fp16, MATRIX_M, MATRIX_N, d_sw, alpha, beta);

    //wmma_example <<< gridDim, blockDim,0, stream2 >>> (a_fp16, b_fp16, d_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
    compute_gemm <<<deviceProp.multiProcessorCount, THREADS_PER_BLOCK, SHMEM_SZ,
      stream2>>> (A, B, C, d_wmma, alpha, beta, M_GLOBAL,  M_GLOBAL);

    cudaErrCheck(hipDeviceSynchronize());
   }


   
   // ---- DMR --- //
  //printf("Running  dmr with tensor thread dimensions...\n");
  
  //wmma_example_dmr <<< gridDim, blockDim >>> (a_fp16, b_fp16, c_wmma, d_sw, d_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  cudaErrCheck(hipEventRecord(stopMXM));
 
   


  /* 
  // Now using cuBLAS
  printf("Running with cuBLAS...\n");
  cudaErrCheck(hipEventRecord(startcublas));
  cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                a_fp16, HIP_R_16F, MATRIX_M,
                b_fp16, HIP_R_16F, MATRIX_K,
                &beta, 
                c_cublas, HIP_R_32F, MATRIX_M,
                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
  cudaErrCheck(hipEventRecord(stopcublas));
  */ 


  // Error checking
  printf("\nChecking results...\n");
  //cudaErrCheck(hipMemcpy(d_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(d_host_sw, d_sw, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(d_host_wmma, d_wmma, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyDeviceToHost));



    for (int i = 0; i <  20; i++) {      
    float v1 = d_host_wmma[i];
    float v2 = d_host_sw[i];
    //float v3 = d_host_cublas[i]; 
    //half v4 = v2/v1;     
    printf("TENSOR = %f  | ------  MXM = %f  ----- | \n", v1, v2);

  }
   
  
  //float wmmaTime;
  //float cublasTime;
  float mxmTime;
  //cudaErrCheck(hipEventSynchronize(stopWMMA));
  //cudaErrCheck(hipEventSynchronize(stopcublas));
  cudaErrCheck(hipEventSynchronize(stopMXM));

  //cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
  //cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
  cudaErrCheck(hipEventElapsedTime(&mxmTime, startMXM, stopMXM));
  //printf("wmma took %fms\n", wmmaTime);
  //printf("cublas took %fms\n", cublasTime);
  printf("mxm took %fms\n", mxmTime);
  
     
   
 
 
 cudaErrCheck(hipEventDestroy(startWMMA));
 cudaErrCheck(hipEventDestroy(stopWMMA));

 cudaErrCheck(hipEventDestroy(startcublas));             
 cudaErrCheck(hipEventDestroy(stopcublas));
 
 cudaErrCheck(hipFree(a_fp32));
 cudaErrCheck(hipFree(b_fp32));
 cudaErrCheck(hipFree(a_fp16));
 cudaErrCheck(hipFree(b_fp16));
 cudaErrCheck(hipFree(d_wmma));
 cudaErrCheck(hipFree(d_sw));

 cudaErrCheck(hipFree(c));
 cudaErrCheck(hipFree(c_cublas));
 cudaErrCheck(hipFree(c_wmma));

 free(d_host_cublas);
 free(d_host_wmma);
 free(d_host_sw);

 cudaErrCheck(hipDeviceReset());
 return 0;
}
