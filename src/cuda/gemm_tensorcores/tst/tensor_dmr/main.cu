#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 9.

// In this program, the compute_gemm kernel computes the result of a matrix
// multiplication and addition: D = alpha * A * B + beta * C. The dimensions of
// both C and D matrices are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x
// K_GLOBAL (row-major), the B matrix is K_GLOBAL x N_GLOBAL (column-major). In
// that kernel, each CTA computes one 128 x 128 tile of the resulting matrix per
// iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes
// eight 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array. Warps
// compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and
// accumulating the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments
//   from shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B
// matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the
//   A and B data from shared memory, thus reducing the number of data copies
//   from global memory.
// - The portions of the A and B matrices are stored in shared memory with an
// additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_HALF macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each
// warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory
//   contents to global memory, again avoiding redundant random global memory
//   accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register
// utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU configuration.

#define WARP_SIZE 32
#define BLOCK_SIZE 32

// MMA matrix tile dimensions.

#define M 32
#define N 32
#define K 32

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 256
#define N_TILES 256
#define K_TILES 256

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B
// matrix in shared memory to minimize possible bank conflicts. Before
// performing the nvcuda::wmma::mma_sync operation, the warp must load the
// matrix data using the nvcuda::wmma::load_matrix_sync operation. Although the
// memory access pattern is not specified for that function, each lane in the
// warp can read one or multiple matrix elements from different matrix rows or
// columns. For shared memory, such access can result in bank conflicts if
// different rows / columns of the matrix map to the same bank. By shifting each
// row and column by a few bytes, we make sure that they map to different banks,
// thus reducing the number of possible bank conflicts. The number of 8 two-byte
// "half" elements is chosen as the minimum possible shift because we must keep
// each row and column 128-bit aligned, as required by
// nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 8



#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

__host__ void init_host_matrices(half *a, half *b, float *c) {
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      a[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      b[i * K_GLOBAL + j] = (half)(rand() % 3);
    }
  }

  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = 0;
  }
}

__device__ __forceinline__ void axpy__(const double a, const double b, double &c) {
    c = __fma_rn(a, b, c);
}
__device__ __forceinline__ void axpy__(const float a, const float b, float &c) {
    //printf("A = %f   -- B =  %f\n", a, b);
    c = __fmaf_rn(a, b, c);
}
__device__ __forceinline__ void axpy__(const double a, const double b, float &c) {
    c = __fmaf_rn(__double2float_rn(a), __double2float_rn(b), c);
}
__device__ __forceinline__ void axpy__(const float a, const float b, __half &c) {
    c = __hfma(__float2half(a), __float2half(b), c);
}

__device__  __forceinline__ half axpy__(half a, half b, half acc) {
  return __hfma(a, b, acc);
}


// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, half *d_sw, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = m_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = m_ld * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + m_ld - 1;



  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * n_ld;

  volatile half Csub = 0;
  
  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int A = aBegin, B = bBegin; A <= aEnd;  A += aStep, B += bStep) {
    

      __shared__ half As[BLOCK_SIZE][BLOCK_SIZE];

      __shared__ half Bs[BLOCK_SIZE][BLOCK_SIZE];

      As[ty][tx] = a[A + m_ld * ty + tx];
      Bs[ty][tx] = b[B + n_ld * ty + tx];

        // Synchronize to make sure the matrices are loaded
      __syncthreads();

  #pragma unroll

      for (int k = 0; k < BLOCK_SIZE; ++k) {
        
          Csub = axpy__(As[ty][k], Bs[k][tx],Csub);
      }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
      __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
  int c_p = n_ld * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  d_sw[c_p + n_ld * ty + tx] = Csub;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;

    int bCol = i;
    int bRow = warpN * WMMA_N;

    // Bounds checking
    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bCol + bRow * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                           wmma::mem_row_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                            wmma::mem_row_major);
  }
}

__host__ void matMultiplyOnHost(half *A, half *B, float *C, float alpha,
                                float beta, int numARows, int numAColumns,
                                int numBRows, int numBColumns, int numCRows,
                                int numCColumns) {
  for (int i = 0; i < numCRows; i++) {
    for (int j = 0; j < numCColumns; j++) {
      float temp = 0.0;

      for (int k = 0; k < numAColumns; k++) {
        temp += (float)A[i * numAColumns + k] * (float)B[j * numBRows + k];
      }

      C[i * numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
    }
  }
}

int main(int argc, char **argv) {
  printf("Initializing...\n");

  int dev = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

  // Tensor cores require a GPU of Volta (SM7X) architecture or higher.
  if (deviceProp.major < 7) {
    printf(
        "cudaTensorCoreGemm requires SM 7.0 or higher to use Tensor "
        "Cores.  Exiting...\n");
    exit(EXIT_WAIVED);
  }

  printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
  printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
  printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

  half *A_h = NULL;
  half *B_h = NULL;
  float *C_h = NULL;

  float *result_hD = NULL;
  half *result_hD_sw = NULL;
  float *result_host = NULL;



  A_h = (half *)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
  B_h = (half *)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
  C_h = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);


  result_hD = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  result_hD_sw = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
  result_host = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);


  half *A = NULL;
  half *B = NULL;
  float *C = NULL;
  float *D = NULL;
  half *D_sw = NULL;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A),
                             sizeof(half) * M_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B),
                             sizeof(half) * N_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C),
                             sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D),
                             sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D),
                             sizeof(float) * M_GLOBAL * N_GLOBAL));
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D_sw),
                             sizeof(half) * M_GLOBAL * N_GLOBAL));

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);
  assert(((unsigned long long)D) % 128 == 0);
  assert(((unsigned long long)D_sw) % 128 == 0);

  init_host_matrices(A_h, B_h, C_h);

  printf("Preparing data for GPU...\n");

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMemset(D_sw, 0, sizeof(half) * M_GLOBAL * N_GLOBAL));

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
            (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float))
  };

  printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

  const float alpha = 1.1f;
  const float beta = 1.2f;

  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));


  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 128x4 means we have 16 warps and a block computes a 64x64 output tile
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x = (M_GLOBAL + (WMMA_M * blockDim.x / 32 - 1)) /
              (WMMA_M * blockDim.x / 32);
  gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  printf("Computing... using simple_wmma_gemm kernel\n");
  simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, D_sw, M_GLOBAL, N_GLOBAL,
                                          K_GLOBAL, alpha, beta);

  checkCudaErrors(hipMemcpy(result_hD, D,
                             sizeof(float) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(result_hD_sw, D,
                             sizeof(float) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyDeviceToHost));

  

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));

#if CPU_DEBUG
  printf("Verifying correctness of the computations...\n");

  memcpy(result_host, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL);

  matMultiplyOnHost(A_h, B_h, result_host, alpha, beta, M_GLOBAL, K_GLOBAL,
                    K_GLOBAL, N_GLOBAL, M_GLOBAL, N_GLOBAL);

  for (int i = 0; i < N_GLOBAL * M_GLOBAL; i++) {
    if (fabs(result_hD[i] - result_host[i]) > 0.1f)
      printf("mismatch i=%d result_hD=%f result_host=%f\n", i, result_hD[i],
             result_host[i]);
  }
  printf("result_hD = &d \n",result_hD[0]);
  free(result_hD);
  free(result_host);
#endif

  float milliseconds = 0;

  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Time: %f ms\n", milliseconds);
  printf("TFLOPS: %.2f\n", static_cast<double>((static_cast<double>(M_GLOBAL) *
                                                N_GLOBAL * K_GLOBAL * 2) /
                                               (milliseconds / 1000.)) /
                               1e12);

  free(A_h);
  free(B_h);
  free(C_h);
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(D)));

  return 0;
}