#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

//extern "C" {
#include "crop_layer.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include "image.h"
//}

__device__ real_t3 make_real_t3(real_t_device x, real_t_device y, real_t_device z) {
	real_t3 mem;
	mem.x = x;
	mem.y = y;
	mem.z = z;
	return mem;
}

__device__ real_t_device get_pixel_kernel(real_t_device *image, int w, int h, int x, int y,
		int c) {
	if (x < 0 || x >= w || y < 0 || y >= h)
		return 0;
	return image[x + w * (y + c * h)];
}

__device__ real_t3 rgb_to_hsv_kernel(real_t3 rgb) {
	real_t_device r = rgb.x;
	real_t_device g = rgb.y;
	real_t_device b = rgb.z;

	real_t_device h, s, v;
	real_t_device max = (r > g) ? ((r > b) ? r : b) : ((g > b) ? g : b);
	real_t_device min = (r < g) ? ((r < b) ? r : b) : ((g < b) ? g : b);
	real_t_device delta = max - min;
	v = max;
	if (max == real_t_device(0)) {
		s = 0;
		h = -1;
	} else {
		s = delta / max;
		if (r == max) {
			h = (g - b) / delta;
		} else if (g == max) {
			h = real_t_device(2) + (b - r) / delta;
		} else {
			h = real_t_device(4) + (r - g) / delta;
		}
		if (h < real_t_device(0))
			h += 6;
	}
	return make_real_t3(h, s, v);
}

__device__ real_t3 hsv_to_rgb_kernel(real_t3 hsv) {
	real_t_device h = hsv.x;
	real_t_device s = hsv.y;
	real_t_device v = hsv.z;

	real_t_device r, g, b;
	real_t_device f, p, q, t;

	if (s == real_t_device(0)) {
		r = g = b = v;
	} else {
		int index = (int) floor_real(h);
		f = h - real_t_device(index);
		p = v * (real_t_device(1) - s);
		q = v * (real_t_device(1) - s * f);
		t = v * (real_t_device(1) - s * (real_t_device(1) - f));
		if (index == 0) {
			r = v;
			g = t;
			b = p;
		} else if (index == 1) {
			r = q;
			g = v;
			b = p;
		} else if (index == 2) {
			r = p;
			g = v;
			b = t;
		} else if (index == 3) {
			r = p;
			g = q;
			b = v;
		} else if (index == 4) {
			r = t;
			g = p;
			b = v;
		} else {
			r = v;
			g = p;
			b = q;
		}
	}
	r = (r < real_t_device(0)) ? real_t_device(0) : ((r > real_t_device(1)) ? real_t_device(1) : r);
	g = (g < real_t_device(0)) ? real_t_device(0) : ((g > real_t_device(1)) ? real_t_device(1) : g);
	b = (b < real_t_device(0)) ? real_t_device(0) : ((b > real_t_device(1)) ? real_t_device(1) : b);
	return make_real_t3(r, g, b);
}

__device__ real_t_device bilinear_interpolate_kernel(real_t_device *image, int w, int h,
		real_t_device x, real_t_device y, int c) {
	int ix = (int) floor_real(x);
	int iy = (int) floor_real(y);

	real_t_device dx = x - real_t_device(ix);
	real_t_device dy = y - real_t_device(iy);

	real_t_device val = (real_t_device(1) - dy) * (real_t_device(1) - dx) * get_pixel_kernel(image, w, h, ix, iy, c)
			+ dy * (real_t_device(1) - dx) * get_pixel_kernel(image, w, h, ix, iy + 1, c)
			+ (real_t_device(1) - dy) * dx * get_pixel_kernel(image, w, h, ix + 1, iy, c)
			+ dy * dx * get_pixel_kernel(image, w, h, ix + 1, iy + 1, c);
	return val;
}

__global__ void levels_image_kernel(real_t_device *image, real_t_device *rand, int batch,
		int w, int h, int train, real_t_device saturation, real_t_device exposure,
		real_t_device translate, real_t_device scale, real_t_device shift) {
	int size = batch * w * h;
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= size)
		return;
	int x = id % w;
	id /= w;
	int y = id % h;
	id /= h;
	real_t_device rshift = rand[0];
	real_t_device gshift = rand[1];
	real_t_device bshift = rand[2];
	real_t_device r0 = rand[8 * id + 0];
	real_t_device r1 = rand[8 * id + 1];
	real_t_device r2 = rand[8 * id + 2];
	real_t_device r3 = rand[8 * id + 3];

	saturation = r0 * (saturation - real_t_device(1)) + real_t_device(1);
	saturation = (r1 > real_t_device(.5f)) ? real_t_device(1.f) / saturation : saturation;
	exposure = r2 * (exposure - real_t_device(1)) + real_t_device(1);
	exposure = (r3 > real_t_device(.5f)) ? real_t_device(1.f) / exposure : exposure;

	size_t offset = id * h * w * 3;
	image += offset;
	real_t_device r = image[x + w * (y + h * 0)];
	real_t_device g = image[x + w * (y + h * 1)];
	real_t_device b = image[x + w * (y + h * 2)];
	real_t3 rgb = make_real_t3(r, g, b);
	if (train) {
		real_t3 hsv = rgb_to_hsv_kernel(rgb);
		hsv.y *= saturation;
		hsv.z *= exposure;
		rgb = hsv_to_rgb_kernel(hsv);
	} else {
		shift = 0;
	}
	image[x + w * (y + h * 0)] = rgb.x * scale + translate
			+ (rshift - real_t_device(.5f)) * shift;
	image[x + w * (y + h * 1)] = rgb.y * scale + translate
			+ (gshift - real_t_device(.5f)) * shift;
	image[x + w * (y + h * 2)] = rgb.z * scale + translate
			+ (bshift - real_t_device(.5f)) * shift;
}

__global__ void forward_crop_layer_kernel(real_t_device *input, real_t_device *rand, int size,
		int c, int h, int w, int crop_height, int crop_width, int train,
		int flip, real_t_device angle, real_t_device *output) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= size)
		return;

	real_t_device cx = w / 2.f;
	real_t_device cy = h / 2.f;

	int count = id;
	int j = id % crop_width;
	id /= crop_width;
	int i = id % crop_height;
	id /= crop_height;
	int k = id % c;
	id /= c;
	int b = id;

	real_t_device r4 = rand[8 * b + 4];
	real_t_device r5 = rand[8 * b + 5];
	real_t_device r6 = rand[8 * b + 6];
	real_t_device r7 = rand[8 * b + 7];

	real_t_device dw = real_t_device(w - crop_width) * r4;
	real_t_device dh = real_t_device(h - crop_height) * r5;
	flip = (flip && (r6 > real_t_device(.5f)));
	angle = real_t_device(2) * angle * r7 - angle;
	if (!train) {
		dw = (w - crop_width) / 2.f;
		dh = (h - crop_height) / 2.f;
		flip = 0;
		angle = 0;
	}

	input += w * h * c * b;

	real_t_device x = (flip) ? real_t_device(w) - dw - real_t_device(j) - real_t_device(1) : real_t_device(j) + dw;
	real_t_device y = real_t_device(i) + dh;

	real_t_device rx = cos_real(angle) * (x - cx) - sin_real(angle) * (y - cy) + cx;
	real_t_device ry = sin_real(angle) * (x - cx) + cos_real(angle) * (y - cy) + cy;

	output[count] = bilinear_interpolate_kernel(input, w, h, rx, ry, k);
}

//extern "C"
void forward_crop_layer_gpu(crop_layer layer, network net) {
	cuda_random(layer.rand_gpu, layer.batch * 8);

	real_t radians = real_t(layer.angle * 3.14159265f / 180.f);

	real_t scale = real_t(2);
	real_t translate = real_t(-1);
	if (layer.noadjust) {
		scale = real_t(1);
		translate = real_t(0);
	}

	int size = layer.batch * layer.w * layer.h;

	levels_image_kernel<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu,
			layer.rand_gpu, layer.batch, layer.w, layer.h, net.train,
			CAST(layer.saturation), CAST(layer.exposure), CAST(translate), CAST(scale), CAST(layer.shift));
	check_error(hipPeekAtLastError());

	size = layer.batch * layer.c * layer.out_w * layer.out_h;

	forward_crop_layer_kernel<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu,
			layer.rand_gpu, size, layer.c, layer.h, layer.w, layer.out_h,
			layer.out_w, net.train, layer.flip, CAST(radians), layer.output_gpu);
	check_error(hipPeekAtLastError());

	/*
	 cuda_pull_array(layer.output_gpu, layer.output, size);
	 image im = real_t_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 0*(size/layer.batch));
	 image im2 = real_t_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 1*(size/layer.batch));
	 image im3 = real_t_to_image(layer.crop_width, layer.crop_height, layer.c, layer.output + 2*(size/layer.batch));

	 translate_image(im, -translate);
	 scale_image(im, 1/scale);
	 translate_image(im2, -translate);
	 scale_image(im2, 1/scale);
	 translate_image(im3, -translate);
	 scale_image(im3, 1/scale);
	 
	 show_image(im, "cropped");
	 show_image(im2, "cropped2");
	 show_image(im3, "cropped3");
	 cvWaitKey(0);
	 */
}

