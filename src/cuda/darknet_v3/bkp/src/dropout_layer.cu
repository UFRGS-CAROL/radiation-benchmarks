#include "dropout_layer.h"
#include "utils.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

dropout_layer make_dropout_layer(int batch, int inputs, real_t probability) {
	dropout_layer l; // = { 0 };
	l.type = DROPOUT;
	l.probability = probability;
	l.inputs = inputs;
	l.outputs = inputs;
	l.batch = batch;
	l.rand = (real_t*) calloc(inputs * batch, sizeof(real_t));
	l.scale = 1. / (1. - probability);
	l.forward = forward_dropout_layer;
	l.backward = backward_dropout_layer;
#ifdef GPU
	l.forward_gpu = forward_dropout_layer_gpu;
	l.backward_gpu = backward_dropout_layer_gpu;
	l.rand_gpu = cuda_make_array(l.rand, inputs * batch);
#endif
	fprintf(stderr, "dropout       p = %.2f               %4d  ->  %4d\n",
			probability, inputs, inputs);
	return l;
}

void resize_dropout_layer(dropout_layer *l, int inputs) {
	l->rand = (real_t*) realloc(l->rand, l->inputs * l->batch * sizeof(real_t));
#ifdef GPU
	cuda_free(l->rand_gpu);

	l->rand_gpu = cuda_make_array(l->rand, inputs * l->batch);
#endif
}

void forward_dropout_layer(dropout_layer l, network net) {
	int i;
	if (!net.train)
		return;
	for (i = 0; i < l.batch * l.inputs; ++i) {
		real_t r = rand_uniform(real_t(0), real_t(1));
		l.rand[i] = r;
		if (r < l.probability)
			net.input[i] = 0;
		else
			net.input[i] *= l.scale;
	}
}

void backward_dropout_layer(dropout_layer l, network net) {
	int i;
	if (!net.delta)
		return;
	for (i = 0; i < l.batch * l.inputs; ++i) {
		real_t r = l.rand[i];
		if (r < l.probability)
			net.delta[i] = 0;
		else
			net.delta[i] *= l.scale;
	}
}

