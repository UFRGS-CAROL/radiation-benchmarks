#include "hip/hip_runtime.h"
/*
 * DetectionGold.cu
 *
 *  Created on: 28/09/2018
 *      Author: fernando
 */

#include "DetectionGold.h"
#include <iterator>
#include "helpful.h"
#include <sstream>
#include <ctime>

void DetectionGold::write_gold_header() {
	//	0       1           2              3              4            5            6        7
	//	thresh; hier_tresh; img_list_size; img_list_path; config_file; config_data; model;weights;
	std::string gold_header = std::to_string(this->thresh) + ";";
	gold_header += std::to_string(this->hier_thresh) + ";";
	gold_header += std::to_string(this->plist_size) + ";";
	gold_header += this->img_list_path + ";";
	gold_header += this->config_file + ";";
	gold_header += this->cfg_data + ";";
	gold_header += this->model + ";";
	gold_header += this->weights + ";";

	std::ofstream gold(this->gold_inout, std::ofstream::trunc);
	if (gold.is_open()) {
		gold << gold_header << std::endl;
		gold.close();
	} else {
		std::cout << "ERROR ON OPENING GOLD OUTPUT FILE\n";
		exit(-1);
	}
}

DetectionGold::DetectionGold(int argc, char **argv, real_t thresh,
		real_t hier_thresh, char *img_list_path, char *config_file,
		char *config_data, char *model, char *weights) {
	char *def;
	this->gold_inout = std::string(find_char_arg(argc, argv, "-gold", def));
	this->generate = find_int_arg(argc, argv, "-generate", 0);
	this->network_name = "darknet_v3_";
#if REAL_TYPE == HALF
	this->network_name += "half";
#elif REAL_TYPE == FLOAT
	this->network_name += "single";
#elif REAL_TYPE == DOUBLE
	this->network_name += "double";
#endif

	this->iterations = find_int_arg(argc, argv, "-iterations", 1);
	this->tensor_core_mode = find_int_arg(argc, argv, "-tensor_cores", 0);
	this->stream_mr = find_int_arg(argc, argv, "-smx_redundancy", 0);
	this->thresh = thresh;
	this->hier_thresh = hier_thresh;

	std::cout << this->generate << " " << this->iterations << " "
			<< this->gold_inout << "\n";

	if (!this->generate) {

		//		Log(std::string gold, int save_layer, int abft, int iterations,
		//				std::string app, unsigned char use_tensor_core_mode)
		this->app_logging = new Log(this->gold_inout, 0, 0, this->iterations,
				this->network_name, this->tensor_core_mode);

		//	detection gold;
		std::string line;
		std::ifstream gold_file(this->gold_inout, std::ifstream::in);
		if (gold_file.is_open()) {
			getline(gold_file, line);
		} else {
			std::cout << "ERROR ON OPENING GOLD FILE\n";
			exit(-1);
		}

		std::vector < std::string > split_ret = split(line, ';');
		//	0       1           2              3              4            5            6      7
		//	thresh; hier_tresh; img_list_size; img_list_path; config_file; config_data; model;weights;
		this->thresh = std::stof(split_ret[0]);
		this->hier_thresh = std::stof(split_ret[1]);
		this->plist_size = std::stoi(split_ret[2]);
		this->img_list_path = split_ret[3];
		this->config_file = split_ret[4];
		this->cfg_data = split_ret[5];
		this->model = split_ret[6];
		this->weights = split_ret[7];

		//allocate detector
		this->load_gold_hash(gold_file);
		std::cout << "Size of vector outside "
				<< this->gold_hash_var[this->gold_img_names[0]].size() << "\n";
		gold_file.close();

	} else {
		this->img_list_path = std::string(img_list_path);

		//reading the img list path content
		std::ifstream tmp_img_file(this->img_list_path);
		std::copy(std::istream_iterator < std::string > (tmp_img_file),
				std::istream_iterator<std::string>(),
				std::back_inserter(this->gold_img_names));

		this->plist_size = this->gold_img_names.size();
		this->config_file = std::string(config_file);
		this->cfg_data = std::string(config_data);
		this->model = std::string(model);
		this->weights = std::string(weights);

		this->write_gold_header();
	}

	//check if iterations is bigger than img_list_size
	if (this->iterations < this->plist_size) {
		this->iterations = this->plist_size;
	}

}

bool operator!=(const box& a, const box& b) {
	return (a.h != b.h || a.w != b.w || a.x != b.x || a.y != a.y);
}

void DetectionGold::cmp(detection* found_dets, int nboxes, int img_index,
		int classes) {
	std::string img = this->gold_img_names[img_index];

	std::vector<Detection> gold_dets = this->gold_hash_var[img];

	int min_nboxes = gold_dets.size();
	int error_count = 0;

	std::cout << "size of gold_dets " << gold_dets.size() << "\n";

	if (min_nboxes != nboxes) {
		std::string error_detail = "img: " + img + " nboxes_e: "
				+ std::to_string(min_nboxes) + " nboxes_r: "
				+ std::to_string(nboxes);
		this->app_logging->log_error_info(error_detail);
		min_nboxes = std::min(nboxes, min_nboxes);
		error_count++;
	}

	for (int nb = 0; nb < min_nboxes; nb++) {
		Detection g_det = gold_dets[nb];
		detection f_det = found_dets[nb];

		box g_box = g_det.bbox;
		box f_box = f_det.bbox;

		real_t g_objectness = g_det.objectness;
		real_t f_objectness = f_det.objectness;

		int g_sort_class = g_det.sort_class;
		int f_sort_class = f_det.sort_class;

		if ((g_box != f_box) || (g_objectness != f_objectness)
				|| (g_sort_class != f_sort_class)) {
			std::ostringstream error_info("");
			error_info.precision(6);

			error_info << "img: " << img << " detection: " << nb << " x_e: "
					<< g_box.x << " x_r: " << f_box.x << " y_e: " << g_box.y
					<< " y_r: " << f_box.y << " h_e: " << g_box.h << " h_r: "
					<< f_box.h << " w_e: " << g_box.w << " w_r: " << f_box.w
					<< " objectness_e: " << g_objectness << " objectness_r: "
					<< f_objectness << " sort_class_e: " << g_sort_class
					<< " sort_class_r: " << f_sort_class;

			this->app_logging->log_error_info(error_info.str());
			error_count++;
		}

		for (int cl = 0; cl < classes; ++cl) {
			real_t g_prob = g_det.prob[cl];
			real_t f_prob = f_det.prob[cl];

			if (g_prob != f_prob) {
				std::ostringstream error_info("");
				error_info.precision(6);

				error_info << "img: " << img << " detection: " << nb
						<< " class: " << cl << " prob_e: " << g_prob
						<< " prob_r: " << f_prob;
				this->app_logging->log_error_info(error_info.str());
				error_count++;
			}
		}
	}

	this->app_logging->update_error_count(error_count);
}

void DetectionGold::run(detection *dets, int nboxes, int img_index,
		int classes) {
	// To generate function
	//std::string img, detection* dets, int nboxes, int classes, int l_coord
	if (this->generate) {

		std::ofstream gold_file(this->gold_inout, std::ofstream::app);
		if (!gold_file.is_open()) {
			std::cerr << "ERROR ON OPENING GOLD FILE\n";
			exit(-1);
		}
		this->gen(dets, nboxes, img_index, gold_file, classes);
		gold_file.close();
	} else {
		// To compare function
		//detection is allways nboxes size
		std::time_t start = std::time(nullptr);

		this->cmp(dets, nboxes, img_index, classes);

		std::cout << "Seconds to compare: "
				<< std::difftime(std::time(nullptr), start) << " s.\n";

	}
}

void DetectionGold::gen(detection *dets, int nboxes, int img_index,
		std::ofstream& gold_file, int classes) {
	//first write the image string name
	std::string img = this->gold_img_names[img_index];

	gold_file << img << ";" << nboxes << ";" << classes << ";" << std::endl;

	for (int i = 0; i < nboxes; ++i) {

		box b = dets[i].bbox;

		gold_file << dets[i].objectness << ";" << dets[i].sort_class << ";"
				<< b.x << ";" << b.y << ";" << b.w << ";" << b.h << ";"
				<< std::endl;

		for (int cl = 0; cl < classes; ++cl) {
			real_t prob = dets[i].prob[cl];
			if (prob != 0)
				gold_file << prob << ";" << cl << ";" << std::endl;
		}
		//just to end the box info
		gold_file << "--;" << std::endl;
	}

}

void DetectionGold::load_gold_hash(std::ifstream& gold_file) {
//allocate detector
	this->gold_img_names = std::vector < std::string > (this->plist_size);
	std::string line;

	for (int i = 0; i < this->plist_size && getline(gold_file, line); i++) {
		//	gold_file << img << ";" << nboxes << ";" << classes << ";" << std::endl;
		std::vector < std::string > splited_line = split(line, ';');
		// Set each img_name path
		this->gold_img_names[i] = splited_line[0];

		// Probarray creation
		int nboxes = std::stoi(splited_line[1]);
		int classes = std::stoi(splited_line[2]);
		//
		std::vector<Detection> detections(nboxes);

		for (int bb = 0; bb < nboxes; ++bb) {

			// Getting bb box
			box b;
			getline(gold_file, line);
			splited_line = split(line, ';');

			real_t objectness = std::stof(splited_line[0]);
			int sort_class = std::stoi(splited_line[1]);
			b.x = std::stof(splited_line[2]);
			b.y = std::stof(splited_line[3]);
			b.w = std::stof(splited_line[4]);
			b.h = std::stof(splited_line[5]);

			// Getting the probabilities
			std::vector < real_t > probs(classes, 0.0);

			while (getline(gold_file, line)) {
				splited_line = split(line, ';');

				if (splited_line[0] == "--")
					break;
				real_t prob = std::stof(splited_line[0]);
				int class_ = std::stoi(splited_line[1]);

				probs[class_] = prob;

			}

			detections[bb] = Detection(nboxes, sort_class, objectness, probs,
					b);

		}

		this->gold_hash_var.put(this->gold_img_names[i], detections);
	}

}

DetectionGold::~DetectionGold() {
	if (this->app_logging) {
		delete this->app_logging;
	}
}

void DetectionGold::start_iteration() {
	if (!this->generate)
		this->app_logging->start_iteration_app();
}

void DetectionGold::end_iteration() {
	if (!this->generate)
		this->app_logging->end_iteration_app();
}
