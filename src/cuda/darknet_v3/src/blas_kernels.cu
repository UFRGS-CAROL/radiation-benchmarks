#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <assert.h>

//extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
//}

__global__ void scale_bias_kernel(real_t_device *output, real_t_device *biases,
		int n, int size) {
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int filter = blockIdx.y;
	int batch = blockIdx.z;

	if (offset < size)
		output[(batch * n + filter) * size + offset] *= biases[filter];
}

void scale_bias_gpu(real_t_device *output, real_t_device *biases, int batch,
		int n, int size) {
	dim3 dimGrid((size - 1) / BLOCK + 1, n, batch);
	dim3 dimBlock(BLOCK, 1, 1);

	scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
	check_error(hipPeekAtLastError());
}

__global__ void backward_scale_kernel(real_t_device *x_norm,
		real_t_device *delta, int batch, int n, int size,
		real_t_device *scale_updates) {
	__shared__ real_t_device part[BLOCK];
	int i, b;
	int filter = blockIdx.x;
	int p = threadIdx.x;
	real_t_device sum = 0;
	for (b = 0; b < batch; ++b) {
		for (i = 0; i < size; i += BLOCK) {
			int index = p + i + size * (filter + n * b);
			sum += (p + i < size) ?
					delta[index] * x_norm[index] : real_t_device(0);
		}
	}
	part[p] = sum;
	__syncthreads();
	if (p == 0) {
		for (i = 0; i < BLOCK; ++i)
			scale_updates[filter] += part[i];
	}
}

void backward_scale_gpu(real_t_device *x_norm, real_t_device *delta, int batch,
		int n, int size, real_t_device *scale_updates) {
	backward_scale_kernel<<<n, BLOCK>>>(x_norm, delta, batch, n, size,
			scale_updates);
	check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(real_t_device *output, real_t_device *biases,
		int batch, int n, int size) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= n * size * batch)
		return;
	int i = index % size;
	index /= size;
	int j = index % n;
	index /= n;
	int k = index;

	output[(k * n + j) * size + i] += biases[j];
}

void add_bias_gpu(real_t_device *output, real_t_device *biases, int batch,
		int n, int size) {
	int num = n * size * batch;

	add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n,
			size);
	check_error(hipPeekAtLastError());
}

__global__ void backward_bias_conn_kernel(real_t_device *bias_updates,
		real_t_device *delta, int batch, int n) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= n)
		return;
	int b;
	real_t_device sum = 0;
	for (b = 0; b < batch; ++b) {
		int i = b * n + index;
		sum += delta[i];
	}
	bias_updates[index] += sum;
}

__global__ void backward_bias_kernel(real_t_device *bias_updates,
		real_t_device *delta, int batch, int n, int size) {
	__shared__ real_t_device part[BLOCK];
	int i, b;
	int filter = blockIdx.x;
	int p = threadIdx.x;
	real_t_device sum = 0;
	for (b = 0; b < batch; ++b) {
		for (i = 0; i < size; i += BLOCK) {
			int index = p + i + size * (filter + n * b);
			sum += (p + i < size) ? delta[index] : real_t_device(0);
		}
	}
	part[p] = sum;
	__syncthreads();
	if (p == 0) {
		for (i = 0; i < BLOCK; ++i)
			bias_updates[filter] += part[i];
	}
}

void backward_bias_gpu(real_t_device *bias_updates, real_t_device *delta,
		int batch, int n, int size) {
	if (size == 1) {
		backward_bias_conn_kernel<<<cuda_gridsize(n), BLOCK>>>(bias_updates,
				delta, batch, n);
	} else {
		backward_bias_kernel<<<n, BLOCK>>>(bias_updates, delta, batch, n, size);
	}
	check_error(hipPeekAtLastError());
}

/*
 __global__ void dot_kernel(real_t_device *output, real_t_device scale, int batch, int n, int size, real_t_device *delta)
 {
 int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
 int f1 = index / n;
 int f2 = index % n;
 if (f2 <= f1) return;
 
 real_t_device sum = 0;
 real_t_device norm1 = 0;
 real_t_device norm2 = 0;
 int b, i;
 for(b = 0; b <  batch; ++b){
 for(i = 0; i < size; ++i){
 int i1 = b * size * n + f1 * size + i;
 int i2 = b * size * n + f2 * size + i;
 sum += output[i1] * output[i2];
 norm1 += output[i1] * output[i1];
 norm2 += output[i2] * output[i2];
 }
 }
 norm1 = sqrt(norm1);
 norm2 = sqrt(norm2);
 real_t_device norm = norm1 * norm2;
 sum = sum / norm;
 for(b = 0; b <  batch; ++b){
 for(i = 0; i < size; ++i){
 int i1 = b * size * n + f1 * size + i;
 int i2 = b * size * n + f2 * size + i;
 delta[i1] += - scale * sum * output[i2] / norm;
 delta[i2] += - scale * sum * output[i1] / norm;
 }
 }
 }

 void dot_error_gpu(layer l)
 {
 dot_kernel<<<cuda_gridsize(l.n*l.n), BLOCK>>>(l.output_gpu, l.dot, l.batch, l.n, l.out_w * l.out_h, l.delta_gpu);
 check_error(hipPeekAtLastError());
 }
 */

__global__ void adam_kernel(int N, real_t_device *x, real_t_device *m,
		real_t_device *v, real_t_device B1, real_t_device B2,
		real_t_device rate, real_t_device eps, int t) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;

	real_t_device mhat = m[index] / (real_t_device(1.f) - pow_real(B1, t));
	real_t_device vhat = v[index] / (real_t_device(1.f) - pow_real(B2, t));

	x[index] = x[index] + rate * mhat / (sqrt_real(vhat) + eps);
}

//extern "C"
void adam_gpu(int n, real_t_device *x, real_t_device *m, real_t_device *v,
		real_t_device B1, real_t_device B2, real_t_device rate,
		real_t_device eps, int t) {
	adam_kernel<<<cuda_gridsize(n), BLOCK>>>(n, x, m, v, B1, B2, rate, eps, t);
	check_error(hipPeekAtLastError());
}

//extern "C"
void adam_update_gpu(real_t_device *w, real_t_device *d, real_t_device *m,
		real_t_device *v, real_t_device B1, real_t_device B2, real_t_device eps,
		real_t_device decay, real_t_device rate, int n, int batch, int t) {
	scal_gpu(n, B1, m, 1);
	scal_gpu(n, B2, v, 1);
	axpy_gpu(n, -real_t(decay) * batch, w, 1, d, 1);

	axpy_gpu(n, (1 - real_t(B1)), d, 1, m, 1);
	mul_gpu(n, d, 1, d, 1);
	axpy_gpu(n, (1 - real_t(B2)), d, 1, v, 1);

	adam_gpu(n, w, m, v, B1, B2, rate, eps, t);
	fill_gpu(n, 0, d, 1);
}

__global__ void normalize_kernel(int N, real_t_device *x, real_t_device *mean,
		real_t_device *variance, int batch, int filters, int spatial) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int f = (index / spatial) % filters;

	x[index] = (x[index] - mean[f])
			/ (sqrt_real(variance[f] + real_t_device(.00001f)));
}

__global__ void normalize_delta_kernel(int N, real_t_device *x,
		real_t_device *mean, real_t_device *variance, real_t_device *mean_delta,
		real_t_device *variance_delta, int batch, int filters, int spatial,
		real_t_device *delta) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int f = (index / spatial) % filters;

	delta[index] = delta[index] * real_t_device(1.f)
			/ (sqrt_real(variance[f] + real_t_device(.00001f)))
			+ variance_delta[f] * real_t_device(2.f) * (x[index] - mean[f])
					/ real_t_device(spatial * batch)
			+ mean_delta[f] / real_t_device(spatial * batch);
}

//extern "C"
void normalize_delta_gpu(real_t_device *x, real_t_device *mean,
		real_t_device *variance, real_t_device *mean_delta,
		real_t_device *variance_delta, int batch, int filters, int spatial,
		real_t_device *delta) {
	size_t N = batch * filters * spatial;
	normalize_delta_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance,
			mean_delta, variance_delta, batch, filters, spatial, delta);
	check_error(hipPeekAtLastError());
}

__global__ void variance_delta_kernel(real_t_device *x, real_t_device *delta,
		real_t_device *mean, real_t_device *variance, int batch, int filters,
		int spatial, real_t_device *variance_delta) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	variance_delta[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			variance_delta[i] += delta[index] * (x[index] - mean[i]);
		}
	}
	variance_delta[i] *= real_t_device(-.5f)
			* pow_real(variance[i] + real_t_device(.00001f), (-3.f / 2.f));
}

__global__ void accumulate_kernel(real_t_device *x, int n, int groups,
		real_t_device *sum) {
	int k;
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= groups)
		return;
	sum[i] = 0;
	for (k = 0; k < n; ++k) {
		sum[i] += x[k * groups + i];
	}
}

__global__ void fast_mean_delta_kernel(real_t_device *delta,
		real_t_device *variance, int batch, int filters, int spatial,
		real_t_device *mean_delta) {
	const int threads = BLOCK;
	__shared__ real_t_device local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;
			local[id] += (i + id < spatial) ? delta[index] : real_t_device(0);
		}
	}

	__syncthreads();

	if (id == 0) {
		mean_delta[filter] = 0;
		for (i = 0; i < threads; ++i) {
			mean_delta[filter] += local[i];
		}
		mean_delta[filter] *= (real_t_device(-1.f)
				/ sqrt_real(variance[filter] + real_t_device(.00001f)));
	}
}

__global__ void fast_variance_delta_kernel(real_t_device *x,
		real_t_device *delta, real_t_device *mean, real_t_device *variance,
		int batch, int filters, int spatial, real_t_device *variance_delta) {
	const int threads = BLOCK;
	__shared__ real_t_device local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;

			local[id] +=
					(i + id < spatial) ?
							delta[index] * (x[index] - mean[filter]) :
							real_t_device(0);
		}
	}

	__syncthreads();

	if (id == 0) {
		variance_delta[filter] = 0;
		for (i = 0; i < threads; ++i) {
			variance_delta[filter] += local[i];
		}
		variance_delta[filter] *= real_t_device(-.5f)
				* pow_real(variance[filter] + real_t_device(.00001f),
						(-3.f / 2.f));
	}
}

__global__ void mean_delta_kernel(real_t_device *delta, real_t_device *variance,
		int batch, int filters, int spatial, real_t_device *mean_delta) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	mean_delta[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			mean_delta[i] += delta[index];
		}
	}
	mean_delta[i] *= (real_t_device(-1.f)
			/ sqrt_real(variance[i] + real_t_device(.00001f)));
}

//extern "C"
void mean_delta_gpu(real_t_device *delta, real_t_device *variance, int batch,
		int filters, int spatial, real_t_device *mean_delta) {
	mean_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(delta, variance, batch,
			filters, spatial, mean_delta);
	check_error(hipPeekAtLastError());
}

//extern "C"
void fast_mean_delta_gpu(real_t_device *delta, real_t_device *variance,
		int batch, int filters, int spatial, real_t_device *mean_delta) {
	fast_mean_delta_kernel<<<filters, BLOCK>>>(delta, variance, batch, filters,
			spatial, mean_delta);
	check_error(hipPeekAtLastError());
}

//extern "C"
void fast_variance_delta_gpu(real_t_device *x, real_t_device *delta,
		real_t_device *mean, real_t_device *variance, int batch, int filters,
		int spatial, real_t_device *variance_delta) {
	fast_variance_delta_kernel<<<filters, BLOCK>>>(x, delta, mean, variance,
			batch, filters, spatial, variance_delta);
	check_error(hipPeekAtLastError());
}

__global__ void mean_kernel(real_t_device *x, int batch, int filters,
		int spatial, real_t_device *mean) {
	real_t_device scale = 1.f / (batch * spatial);
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	mean[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			mean[i] += x[index];
		}
	}
	mean[i] *= scale;
}

__global__ void variance_kernel(real_t_device *x, real_t_device *mean,
		int batch, int filters, int spatial, real_t_device *variance) {
	real_t_device scale = 1.f / (batch * spatial - 1);
	int j, k;
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	variance[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			variance[i] += powf((x[index] - mean[i]), 2);
		}
	}
	variance[i] *= scale;
}

__global__ void reorg_kernel(int N, real_t_device *x, int w, int h, int c,
		int batch, int stride, int forward, real_t_device *out) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int in_index = i;
	int in_w = i % w;
	i = i / w;
	int in_h = i % h;
	i = i / h;
	int in_c = i % c;
	i = i / c;
	int b = i % batch;

	int out_c = c / (stride * stride);

	int c2 = in_c % out_c;
	int offset = in_c / out_c;
	int w2 = in_w * stride + offset % stride;
	int h2 = in_h * stride + offset / stride;
	//printf("%d\n", offset);
	int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

	// printf("%d %d %d\n", w2, h2, c2);
	//printf("%d %d\n", in_index, out_index);
	//if(out_index >= N || out_index < 0) printf("bad bad bad \n");

	if (forward)
		out[out_index] = x[in_index];
	else
		out[in_index] = x[out_index];
	//if(forward) out[1] = x[1];
	//else out[0] = x[0];
}

__global__ void axpy_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int OFFX, int INCX, real_t_device *Y, int OFFY, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
}

__global__ void pow_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX, real_t_device *Y, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY] = pow_real(X[i * INCX], ALPHA);
}

__global__ void const_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = ALPHA;
}

__global__ void constrain_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
}

__global__ void supp_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) {
		if ((X[i * INCX] * X[i * INCX]) < (ALPHA * ALPHA))
			X[i * INCX] = 0;
	}
}

__global__ void add_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] += ALPHA;
}

__global__ void scal_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] *= ALPHA;
}

__global__ void fill_kernel(int N, real_t_device ALPHA, real_t_device *X,
		int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = ALPHA;
}

__global__ void copy_kernel(int N, real_t_device *X, int OFFX, int INCX,
		real_t_device *Y, int OFFY, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY + OFFY] = X[i * INCX + OFFX];
}

__global__ void mul_kernel(int N, real_t_device *X, int INCX, real_t_device *Y,
		int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY] *= X[i * INCX];
}

//extern "C"
void normalize_gpu(real_t_device *x, real_t_device *mean,
		real_t_device *variance, int batch, int filters, int spatial) {
	size_t N = batch * filters * spatial;
	normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch,
			filters, spatial);
	check_error(hipPeekAtLastError());
}

__global__ void l2norm_kernel(int N, real_t_device *x, real_t_device *dx,
		int batch, int filters, int spatial) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int b = index / spatial;
	int i = index % spatial;
	int f;
	real_t_device sum = 0;
	for (f = 0; f < filters; ++f) {
		int index = b * filters * spatial + f * spatial + i;
		sum += powf(x[index], 2);
	}
	sum = sqrt_real(sum);
	if (sum == real_t_device(0))
		sum = 1;
	//printf("%f\n", sum);
	for (f = 0; f < filters; ++f) {
		int index = b * filters * spatial + f * spatial + i;
		x[index] /= sum;
		dx[index] = (real_t_device(1) - x[index]) / sum;
	}
}

//extern "C"
void l2normalize_gpu(real_t_device *x, real_t_device *dx, int batch,
		int filters, int spatial) {
	size_t N = batch * spatial;
	l2norm_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, dx, batch, filters,
			spatial);
	check_error(hipPeekAtLastError());
}

__global__ void fast_mean_kernel(real_t_device *x, int batch, int filters,
		int spatial, real_t_device *mean) {
	const int threads = BLOCK;
	__shared__ real_t_device local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;
			local[id] += (i + id < spatial) ? x[index] : real_t_device(0);
		}
	}

	__syncthreads();

	if (id == 0) {
		mean[filter] = 0;
		for (i = 0; i < threads; ++i) {
			mean[filter] += local[i];
		}
		mean[filter] /= spatial * batch;
	}
}

__global__ void fast_variance_kernel(real_t_device *x, real_t_device *mean,
		int batch, int filters, int spatial, real_t_device *variance) {
	const int threads = BLOCK;
	__shared__ real_t_device local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;

			local[id] +=
					(i + id < spatial) ?
							pow_real((x[index] - mean[filter]), 2) :
							real_t_device(0);
		}
	}

	__syncthreads();

	if (id == 0) {
		variance[filter] = 0;
		for (i = 0; i < threads; ++i) {
			variance[filter] += local[i];
		}
		variance[filter] /= (spatial * batch - 1);
	}
}

//extern "C"
void fast_mean_gpu(real_t_device *x, int batch, int filters, int spatial,
		real_t_device *mean) {
	fast_mean_kernel<<<filters, BLOCK>>>(x, batch, filters, spatial, mean);
	check_error(hipPeekAtLastError());
}

//extern "C"
void fast_variance_gpu(real_t_device *x, real_t_device *mean, int batch,
		int filters, int spatial, real_t_device *variance) {
	fast_variance_kernel<<<filters, BLOCK>>>(x, mean, batch, filters, spatial,
			variance);
	check_error(hipPeekAtLastError());
}

//extern "C"
void mean_gpu(real_t_device *x, int batch, int filters, int spatial,
		real_t_device *mean) {
	mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, batch, filters, spatial,
			mean);
	check_error(hipPeekAtLastError());
}

//extern "C"
void variance_gpu(real_t_device *x, real_t_device *mean, int batch, int filters,
		int spatial, real_t_device *variance) {
	variance_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, mean, batch, filters,
			spatial, variance);
	check_error(hipPeekAtLastError());
}

//extern "C"
void axpy_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX,
		real_t_device * Y, int INCY) {
	axpy_gpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

//extern "C"
void pow_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX,
		real_t_device * Y, int INCY) {
	pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

//extern "C"
void axpy_gpu_offset(int N, real_t_device ALPHA, real_t_device * X, int OFFX,
		int INCX, real_t_device * Y, int OFFY, int INCY) {
	axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY,
			INCY);
	check_error(hipPeekAtLastError());
}

//extern "C"
void copy_gpu(int N, real_t_device * X, int INCX, real_t_device * Y, int INCY) {
	copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

//extern "C"
void mul_gpu(int N, real_t_device * X, int INCX, real_t_device * Y, int INCY) {
	mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

//extern "C"
void copy_gpu_offset(int N, real_t_device * X, int OFFX, int INCX,
		real_t_device * Y, int OFFY, int INCY) {
	copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
	check_error(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, real_t_device *x, int spatial, int layers,
		int batch, int forward, real_t_device *out) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int in_s = i % spatial;
	i = i / spatial;
	int in_c = i % layers;
	i = i / layers;
	int b = i;

	int i1 = b * layers * spatial + in_c * spatial + in_s;
	int i2 = b * layers * spatial + in_s * layers + in_c;

	if (forward)
		out[i2] = x[i1];
	else
		out[i1] = x[i2];
}

//extern "C"
void flatten_gpu(real_t_device *x, int spatial, int layers, int batch,
		int forward, real_t_device *out) {
	int size = spatial * batch * layers;
	flatten_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, spatial, layers,
			batch, forward, out);
	check_error(hipPeekAtLastError());
}

//extern "C"
void reorg_gpu(real_t_device *x, int w, int h, int c, int batch, int stride,
		int forward, real_t_device *out) {
	int size = w * h * c * batch;
	reorg_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, w, h, c, batch,
			stride, forward, out);
	check_error(hipPeekAtLastError());
}

__global__ void mask_kernel(int n, real_t_device *x, real_t_device mask_num,
		real_t_device *mask, real_t_device val) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num)
		x[i] = val;
}

//extern "C"
void mask_gpu(int N, real_t_device * X, real_t_device mask_num,
		real_t_device * mask, real_t_device val) {
	mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, val);
	check_error(hipPeekAtLastError());
}

__global__ void scale_mask_kernel(int n, real_t_device *x,
		real_t_device mask_num, real_t_device *mask, real_t_device scale) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num)
		x[i] *= scale;
}

//extern "C"
void scale_mask_gpu(int N, real_t_device * X, real_t_device mask_num,
		real_t_device * mask, real_t_device scale) {
	scale_mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, scale);
	check_error(hipPeekAtLastError());
}

//extern "C"
void const_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

//extern "C"
void constrain_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	constrain_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

//extern "C"
void add_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	add_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

//extern "C"
void scal_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

//extern "C"
void supp_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	supp_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

//extern "C"
void fill_gpu(int N, real_t_device ALPHA, real_t_device * X, int INCX) {
	fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc,
		int stride, int sample, int batch, int w1, int h1, int c1,
		real_t_device *add, int w2, int h2, int c2, real_t_device s1,
		real_t_device s2, real_t_device *out) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= size)
		return;
	int i = id % minw;
	id /= minw;
	int j = id % minh;
	id /= minh;
	int k = id % minc;
	id /= minc;
	int b = id % batch;

	int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
	int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
	out[out_index] = s1 * out[out_index] + s2 * add[add_index];
	//out[out_index] += add[add_index];
}

//extern "C"
void shortcut_gpu(int batch, int w1, int h1, int c1, real_t_device *add, int w2,
		int h2, int c2, real_t_device s1, real_t_device s2,
		real_t_device *out) {
	int minw = (w1 < w2) ? w1 : w2;
	int minh = (h1 < h2) ? h1 : h2;
	int minc = (c1 < c2) ? c1 : c2;

	int stride = w1 / w2;
	int sample = w2 / w1;
	assert(stride == h1 / h2);
	assert(sample == h2 / h1);
	if (stride < 1)
		stride = 1;
	if (sample < 1)
		sample = 1;

	int size = batch * minw * minh * minc;
	shortcut_kernel<<<cuda_gridsize(size), BLOCK>>>(size, minw, minh, minc,
			stride, sample, batch, w1, h1, c1, add, w2, h2, c2, s1, s2, out);
	check_error(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, real_t_device *pred,
		real_t_device *truth, real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t_device diff = truth[i] - pred[i];
		real_t_device abs_val = fabs_real(diff);
		if (abs_val < real_t_device(1)) {
			error[i] = diff * diff;
			delta[i] = diff;
		} else {
			error[i] = real_t_device(2) * abs_val - real_t_device(1);
			delta[i] =
					(diff > real_t_device(0)) ?
							real_t_device(1) : real_t_device(-1);
		}
	}
}

//extern "C"
void smooth_l1_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	smooth_l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, real_t_device *pred,
		real_t_device *truth, real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t_device t = truth[i];
		real_t_device p = pred[i];
		error[i] = (t) ? -log_real(p) : real_t_device(0);
		delta[i] = t - p;
	}
}

//extern "C"
void softmax_x_ent_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	softmax_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta,
			error);
	check_error(hipPeekAtLastError());
}

__global__ void logistic_x_ent_kernel(int n, real_t_device *pred,
		real_t_device *truth, real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t_device t = truth[i];
		real_t_device p = pred[i];
		error[i] = -t * log_real(p + real_t_device(.0000001))
				- (real_t_device(1) - t)
						* log_real(
								real_t_device(1) - p + real_t_device(.0000001));
		delta[i] = t - p;
	}
}

//extern "C"
void logistic_x_ent_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	logistic_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta,
			error);
	check_error(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t_device diff = truth[i] - pred[i];
		error[i] = diff * diff; //I know this is technically wrong, deal with it.
		delta[i] = diff;
	}
}

//extern "C"
void l2_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	l2_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void l1_kernel(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t_device diff = truth[i] - pred[i];
		error[i] = fabs_real(diff);
		delta[i] =
				(diff > real_t_device(0)) ?
						real_t_device(1) : real_t_device(-1);
	}
}

//extern "C"
void l1_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void wgan_kernel(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		error[i] = truth[i] ? -pred[i] : pred[i];
		delta[i] =
				(truth[i] > real_t_device(0)) ?
						real_t_device(1) : real_t_device(-1);
	}
}

//extern "C"
void wgan_gpu(int n, real_t_device *pred, real_t_device *truth,
		real_t_device *delta, real_t_device *error) {
	wgan_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void weighted_sum_kernel(int n, real_t_device *a, real_t_device *b,
		real_t_device *s, real_t_device *c) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] = s[i] * a[i]
				+ (real_t_device(1) - s[i]) * (b ? b[i] : real_t_device(0));
	}
}

__global__ void deinter_kernel(int NX, real_t_device *X, int NY,
		real_t_device *Y, int B, real_t_device *OUT) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < (NX + NY) * B) {
		int b = i / (NX + NY);
		int j = i % (NX + NY);
		if (j < NX) {
			if (X)
				X[b * NX + j] += OUT[i];
		} else {
			if (Y)
				Y[b * NY + j - NX] += OUT[i];
		}
	}
}

//extern "C"
void deinter_gpu(int NX, real_t_device *X, int NY, real_t_device *Y, int B,
		real_t_device *OUT) {
	deinter_kernel<<<cuda_gridsize((NX + NY) * B), BLOCK>>>(NX, X, NY, Y, B,
			OUT);
	check_error(hipPeekAtLastError());
}

__global__ void inter_kernel(int NX, real_t_device *X, int NY, real_t_device *Y,
		int B, real_t_device *OUT) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < (NX + NY) * B) {
		int b = i / (NX + NY);
		int j = i % (NX + NY);
		if (j < NX) {
			OUT[i] = X[b * NX + j];
		} else {
			OUT[i] = Y[b * NY + j - NX];
		}
	}
}

//extern "C"
void inter_gpu(int NX, real_t_device *X, int NY, real_t_device *Y, int B,
		real_t_device *OUT) {
	inter_kernel<<<cuda_gridsize((NX + NY) * B), BLOCK>>>(NX, X, NY, Y, B, OUT);
	check_error(hipPeekAtLastError());
}

//extern "C"
void weighted_sum_gpu(real_t_device *a, real_t_device *b, real_t_device *s,
		int num, real_t_device *c) {
	weighted_sum_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, c);
	check_error(hipPeekAtLastError());
}

__global__ void weighted_delta_kernel(int n, real_t_device *a, real_t_device *b,
		real_t_device *s, real_t_device *da, real_t_device *db,
		real_t_device *ds, real_t_device *dc) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		if (da)
			da[i] += dc[i] * s[i];
		if (db)
			db[i] += dc[i] * (real_t_device(1) - s[i]);
		ds[i] += dc[i] * (a[i] - b[i]);
	}
}

//extern "C"
void weighted_delta_gpu(real_t_device *a, real_t_device *b, real_t_device *s,
		real_t_device *da, real_t_device *db, real_t_device *ds, int num,
		real_t_device *dc) {
	weighted_delta_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, da, db,
			ds, dc);
	check_error(hipPeekAtLastError());
}

__global__ void mult_add_into_kernel(int n, real_t_device *a, real_t_device *b,
		real_t_device *c) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] += a[i] * b[i];
	}
}

//extern "C"
void mult_add_into_gpu(int num, real_t_device *a, real_t_device *b,
		real_t_device *c) {
	mult_add_into_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, c);
	check_error(hipPeekAtLastError());
}

__device__ void softmax_device(real_t_device *input, int n, real_t_device temp,
		int stride, real_t_device *output) {
	int i;
	real_t_device sum = 0;
	real_t_device largest = -REAL_INFINITY;
	for (i = 0; i < n; ++i) {
		int val = input[i * stride];
		largest = (real_t_device(val) > largest) ? real_t_device(val) : largest;
	}
	for (i = 0; i < n; ++i) {
		real_t_device e = exp_real(input[i * stride] / temp - largest / temp);
		sum += e;
		output[i * stride] = e;
	}
	for (i = 0; i < n; ++i) {
		output[i * stride] /= sum;
	}
}

__global__ void softmax_tree_kernel(real_t_device *input, int spatial,
		int batch, int stride, real_t_device temp, real_t_device *output,
		int groups, int *group_size, int *group_offset) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= spatial * batch * groups)
		return;
	int s = id % spatial;
	id = id / spatial;
	int g = id % groups;
	int b = id / groups;
	int goff = group_offset[g] * spatial;
	int boff = b * stride;
	softmax_device(input + goff + boff + s, group_size[g], temp, spatial,
			output + goff + boff + s);
}

//extern "C"
void softmax_tree(real_t_device *input, int spatial, int batch, int stride,
		real_t_device temp, real_t_device *output, tree hier) {
	int *tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	int *tree_groups_offset = cuda_make_int_array(hier.group_offset,
			hier.groups);
	/*
	 static int *tree_groups_size = 0;
	 static int *tree_groups_offset = 0;
	 if(!tree_groups_size){
	 tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	 tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	 }
	 */
	int num = spatial * batch * hier.groups;
	softmax_tree_kernel<<<cuda_gridsize(num), BLOCK>>>(input, spatial, batch,
			stride, temp, output, hier.groups, tree_groups_size,
			tree_groups_offset);
	check_error(hipPeekAtLastError());
	cuda_free((real_t_device *) tree_groups_size);
	cuda_free((real_t_device *) tree_groups_offset);
}

__global__ void softmax_kernel(real_t_device *input, int n, int batch,
		int batch_offset, int groups, int group_offset, int stride,
		real_t_device temp, real_t_device *output) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= batch * groups)
		return;
	int b = id / groups;
	int g = id % groups;
	softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride,
			output + b * batch_offset + g * group_offset);
}

//extern "C"
void softmax_gpu(real_t_device *input, int n, int batch, int batch_offset,
		int groups, int group_offset, int stride, real_t_device temp,
		real_t_device *output) {
	softmax_kernel<<<cuda_gridsize(batch * groups), BLOCK>>>(input, n, batch,
			batch_offset, groups, group_offset, stride, temp, output);
	check_error(hipPeekAtLastError());
}

__global__ void upsample_kernel(size_t N, real_t_device *x, int w, int h, int c,
		int batch, int stride, int forward, real_t_device scale,
		real_t_device *out) {
	size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int out_index = i;
	int out_w = i % (w * stride);
	i = i / (w * stride);
	int out_h = i % (h * stride);
	i = i / (h * stride);
	int out_c = i % c;
	i = i / c;
	int b = i % batch;

	int in_w = out_w / stride;
	int in_h = out_h / stride;
	int in_c = out_c;

	int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

	if (forward)
		out[out_index] += scale * x[in_index];
	else {
		//TODO
		// FIX it when Titan V is availiable
		atomic_add_real(x + in_index, scale * out[out_index]);
	}
}

//extern "C"
void upsample_gpu(real_t_device *in, int w, int h, int c, int batch, int stride,
		int forward, real_t_device scale, real_t_device *out) {
	size_t size = w * h * c * batch * stride * stride;
	upsample_kernel<<<cuda_gridsize(size), BLOCK>>>(size, in, w, h, c, batch,
			stride, forward, scale, out);
	check_error(hipPeekAtLastError());
}
