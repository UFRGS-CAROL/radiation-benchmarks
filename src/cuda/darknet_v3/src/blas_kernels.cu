#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <assert.h>

extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void scale_bias_kernel(real_t *output, real_t *biases, int n,
		int size) {
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int filter = blockIdx.y;
	int batch = blockIdx.z;

	if (offset < size)
		output[(batch * n + filter) * size + offset] *= biases[filter];
}

void scale_bias_gpu(real_t *output, real_t *biases, int batch, int n,
		int size) {
	dim3 dimGrid((size - 1) / BLOCK + 1, n, batch);
	dim3 dimBlock(BLOCK, 1, 1);

	scale_bias_kernel<<<dimGrid, dimBlock>>>(output, biases, n, size);
	check_error(hipPeekAtLastError());
}

__global__ void backward_scale_kernel(real_t *x_norm, real_t *delta, int batch,
		int n, int size, real_t *scale_updates) {
	__shared__ real_t part[BLOCK];
	int i, b;
	int filter = blockIdx.x;
	int p = threadIdx.x;
	real_t sum = 0;
	for (b = 0; b < batch; ++b) {
		for (i = 0; i < size; i += BLOCK) {
			int index = p + i + size * (filter + n * b);
			sum += (p + i < size) ? delta[index] * x_norm[index] : 0;
		}
	}
	part[p] = sum;
	__syncthreads();
	if (p == 0) {
		for (i = 0; i < BLOCK; ++i)
			scale_updates[filter] += part[i];
	}
}

void backward_scale_gpu(real_t *x_norm, real_t *delta, int batch, int n,
		int size, real_t *scale_updates) {
	backward_scale_kernel<<<n, BLOCK>>>(x_norm, delta, batch, n, size,
			scale_updates);
	check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(real_t *output, real_t *biases, int batch,
		int n, int size) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= n * size * batch)
		return;
	int i = index % size;
	index /= size;
	int j = index % n;
	index /= n;
	int k = index;

	output[(k * n + j) * size + i] += biases[j];
}

void add_bias_gpu(real_t *output, real_t *biases, int batch, int n, int size) {
	int num = n * size * batch;

	add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n,
			size);
	check_error(hipPeekAtLastError());
}

__global__ void backward_bias_conn_kernel(real_t *bias_updates, real_t *delta,
		int batch, int n) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= n)
		return;
	int b;
	real_t sum = 0;
	for (b = 0; b < batch; ++b) {
		int i = b * n + index;
		sum += delta[i];
	}
	bias_updates[index] += sum;
}

__global__ void backward_bias_kernel(real_t *bias_updates, real_t *delta,
		int batch, int n, int size) {
	__shared__ real_t part[BLOCK];
	int i, b;
	int filter = blockIdx.x;
	int p = threadIdx.x;
	real_t sum = 0;
	for (b = 0; b < batch; ++b) {
		for (i = 0; i < size; i += BLOCK) {
			int index = p + i + size * (filter + n * b);
			sum += (p + i < size) ? delta[index] : 0;
		}
	}
	part[p] = sum;
	__syncthreads();
	if (p == 0) {
		for (i = 0; i < BLOCK; ++i)
			bias_updates[filter] += part[i];
	}
}

void backward_bias_gpu(real_t *bias_updates, real_t *delta, int batch, int n,
		int size) {
	if (size == 1) {
		backward_bias_conn_kernel<<<cuda_gridsize(n), BLOCK>>>(bias_updates,
				delta, batch, n);
	} else {
		backward_bias_kernel<<<n, BLOCK>>>(bias_updates, delta, batch, n, size);
	}
	check_error(hipPeekAtLastError());
}

/*
 __global__ void dot_kernel(real_t *output, real_t scale, int batch, int n, int size, real_t *delta)
 {
 int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
 int f1 = index / n;
 int f2 = index % n;
 if (f2 <= f1) return;

 real_t sum = 0;
 real_t norm1 = 0;
 real_t norm2 = 0;
 int b, i;
 for(b = 0; b <  batch; ++b){
 for(i = 0; i < size; ++i){
 int i1 = b * size * n + f1 * size + i;
 int i2 = b * size * n + f2 * size + i;
 sum += output[i1] * output[i2];
 norm1 += output[i1] * output[i1];
 norm2 += output[i2] * output[i2];
 }
 }
 norm1 = sqrt(norm1);
 norm2 = sqrt(norm2);
 real_t norm = norm1 * norm2;
 sum = sum / norm;
 for(b = 0; b <  batch; ++b){
 for(i = 0; i < size; ++i){
 int i1 = b * size * n + f1 * size + i;
 int i2 = b * size * n + f2 * size + i;
 delta[i1] += - scale * sum * output[i2] / norm;
 delta[i2] += - scale * sum * output[i1] / norm;
 }
 }
 }

 void dot_error_gpu(layer l)
 {
 dot_kernel<<<cuda_gridsize(l.n*l.n), BLOCK>>>(l.output_gpu, l.dot, l.batch, l.n, l.out_w * l.out_h, l.delta_gpu);
 check_error(hipPeekAtLastError());
 }
 */

__global__ void adam_kernel(int N, real_t *x, real_t *m, real_t *v, real_t B1,
		real_t B2, real_t rate, real_t eps, int t) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;

	real_t mhat = m[index] / (1.f - powf(B1, t));
	real_t vhat = v[index] / (1.f - powf(B2, t));

	x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}

extern "C" void adam_gpu(int n, real_t *x, real_t *m, real_t *v, real_t B1,
		real_t B2, real_t rate, real_t eps, int t) {
	adam_kernel<<<cuda_gridsize(n), BLOCK>>>(n, x, m, v, B1, B2, rate, eps, t);
	check_error(hipPeekAtLastError());
}

extern "C" void adam_update_gpu(real_t *w, real_t *d, real_t *m, real_t *v,
		real_t B1, real_t B2, real_t eps, real_t decay, real_t rate, int n,
		int batch, int t) {
	scal_gpu(n, B1, m, 1);
	scal_gpu(n, B2, v, 1);
	axpy_gpu(n, -decay * batch, w, 1, d, 1);

	axpy_gpu(n, (1 - B1), d, 1, m, 1);
	mul_gpu(n, d, 1, d, 1);
	axpy_gpu(n, (1 - B2), d, 1, v, 1);

	adam_gpu(n, w, m, v, B1, B2, rate, eps, t);
	fill_gpu(n, 0, d, 1);
}

__global__ void normalize_kernel(int N, real_t *x, real_t *mean,
		real_t *variance, int batch, int filters, int spatial) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int f = (index / spatial) % filters;

	x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
}

__global__ void normalize_delta_kernel(int N, real_t *x, real_t *mean,
		real_t *variance, real_t *mean_delta, real_t *variance_delta, int batch,
		int filters, int spatial, real_t *delta) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int f = (index / spatial) % filters;

	delta[index] = delta[index] * 1.f / (sqrtf(variance[f] + .00001f))
			+ variance_delta[f] * 2.f * (x[index] - mean[f]) / (spatial * batch)
			+ mean_delta[f] / (spatial * batch);
}

extern "C" void normalize_delta_gpu(real_t *x, real_t *mean, real_t *variance,
		real_t *mean_delta, real_t *variance_delta, int batch, int filters,
		int spatial, real_t *delta) {
	size_t N = batch * filters * spatial;
	normalize_delta_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance,
			mean_delta, variance_delta, batch, filters, spatial, delta);
	check_error(hipPeekAtLastError());
}

__global__ void variance_delta_kernel(real_t *x, real_t *delta, real_t *mean,
		real_t *variance, int batch, int filters, int spatial,
		real_t *variance_delta) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	variance_delta[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			variance_delta[i] += delta[index] * (x[index] - mean[i]);
		}
	}
	variance_delta[i] *= -.5f
			* powf(variance[i] + .00001f, (real_t)(-3.f / 2.f));
}

__global__ void accumulate_kernel(real_t *x, int n, int groups, real_t *sum) {
	int k;
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= groups)
		return;
	sum[i] = 0;
	for (k = 0; k < n; ++k) {
		sum[i] += x[k * groups + i];
	}
}

__global__ void fast_mean_delta_kernel(real_t *delta, real_t *variance,
		int batch, int filters, int spatial, real_t *mean_delta) {
	const int threads = BLOCK;
	__shared__ real_t local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;
			local[id] += (i + id < spatial) ? delta[index] : 0;
		}
	}

	__syncthreads();

	if (id == 0) {
		mean_delta[filter] = 0;
		for (i = 0; i < threads; ++i) {
			mean_delta[filter] += local[i];
		}
		mean_delta[filter] *= (-1.f / sqrtf(variance[filter] + .00001f));
	}
}

__global__ void fast_variance_delta_kernel(real_t *x, real_t *delta,
		real_t *mean, real_t *variance, int batch, int filters, int spatial,
		real_t *variance_delta) {
	const int threads = BLOCK;
	__shared__ real_t local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;

			local[id] +=
					(i + id < spatial) ?
							delta[index] * (x[index] - mean[filter]) : 0;
		}
	}

	__syncthreads();

	if (id == 0) {
		variance_delta[filter] = 0;
		for (i = 0; i < threads; ++i) {
			variance_delta[filter] += local[i];
		}
		variance_delta[filter] *= -.5f
				* powf(variance[filter] + .00001f, (real_t)(-3.f / 2.f));
	}
}

__global__ void mean_delta_kernel(real_t *delta, real_t *variance, int batch,
		int filters, int spatial, real_t *mean_delta) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	mean_delta[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			mean_delta[i] += delta[index];
		}
	}
	mean_delta[i] *= (-1.f / sqrtf(variance[i] + .00001f));
}

extern "C" void mean_delta_gpu(real_t *delta, real_t *variance, int batch,
		int filters, int spatial, real_t *mean_delta) {
	mean_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(delta, variance, batch,
			filters, spatial, mean_delta);
	check_error(hipPeekAtLastError());
}

extern "C" void fast_mean_delta_gpu(real_t *delta, real_t *variance, int batch,
		int filters, int spatial, real_t *mean_delta) {
	fast_mean_delta_kernel<<<filters, BLOCK>>>(delta, variance, batch, filters,
			spatial, mean_delta);
	check_error(hipPeekAtLastError());
}

extern "C" void fast_variance_delta_gpu(real_t *x, real_t *delta, real_t *mean,
		real_t *variance, int batch, int filters, int spatial,
		real_t *variance_delta) {
	fast_variance_delta_kernel<<<filters, BLOCK>>>(x, delta, mean, variance,
			batch, filters, spatial, variance_delta);
	check_error(hipPeekAtLastError());
}

__global__ void mean_kernel(real_t *x, int batch, int filters, int spatial,
		real_t *mean) {
	real_t scale = 1.f / (batch * spatial);
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	int j, k;
	mean[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			mean[i] += x[index];
		}
	}
	mean[i] *= scale;
}

__global__ void variance_kernel(real_t *x, real_t *mean, int batch, int filters,
		int spatial, real_t *variance) {
	real_t scale = 1.f / (batch * spatial - 1);
	int j, k;
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= filters)
		return;
	variance[i] = 0;
	for (j = 0; j < batch; ++j) {
		for (k = 0; k < spatial; ++k) {
			int index = j * filters * spatial + i * spatial + k;
			variance[i] += powf((x[index] - mean[i]), 2);
		}
	}
	variance[i] *= scale;
}

__global__ void reorg_kernel(int N, real_t *x, int w, int h, int c, int batch,
		int stride, int forward, real_t *out) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int in_index = i;
	int in_w = i % w;
	i = i / w;
	int in_h = i % h;
	i = i / h;
	int in_c = i % c;
	i = i / c;
	int b = i % batch;

	int out_c = c / (stride * stride);

	int c2 = in_c % out_c;
	int offset = in_c / out_c;
	int w2 = in_w * stride + offset % stride;
	int h2 = in_h * stride + offset / stride;
	//printf("%d\n", offset);
	int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

	// printf("%d %d %d\n", w2, h2, c2);
	//printf("%d %d\n", in_index, out_index);
	//if(out_index >= N || out_index < 0) printf("bad bad bad \n");

	if (forward)
		out[out_index] = x[in_index];
	else
		out[in_index] = x[out_index];
	//if(forward) out[1] = x[1];
	//else out[0] = x[0];
}

__global__ void axpy_kernel(int N, real_t ALPHA, real_t *X, int OFFX, int INCX,
		real_t *Y, int OFFY, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
}

__global__ void pow_kernel(int N, real_t ALPHA, real_t *X, int INCX, real_t *Y,
		int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY] = pow(X[i * INCX], ALPHA);
}

__global__ void const_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = ALPHA;
}

__global__ void constrain_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
}

__global__ void supp_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N) {
		if ((X[i * INCX] * X[i * INCX]) < (ALPHA * ALPHA))
			X[i * INCX] = 0;
	}
}

__global__ void add_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] += ALPHA;
}

__global__ void scal_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] *= ALPHA;
}

__global__ void fill_kernel(int N, real_t ALPHA, real_t *X, int INCX) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		X[i * INCX] = ALPHA;
}

__global__ void copy_kernel(int N, real_t *X, int OFFX, int INCX, real_t *Y,
		int OFFY, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY + OFFY] = X[i * INCX + OFFX];
}

__global__ void mul_kernel(int N, real_t *X, int INCX, real_t *Y, int INCY) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < N)
		Y[i * INCY] *= X[i * INCX];
}

extern "C" void normalize_gpu(real_t *x, real_t *mean, real_t *variance,
		int batch, int filters, int spatial) {
	size_t N = batch * filters * spatial;
	normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch,
			filters, spatial);
	check_error(hipPeekAtLastError());
}

__global__ void l2norm_kernel(int N, real_t *x, real_t *dx, int batch,
		int filters, int spatial) {
	int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x
			+ threadIdx.x;
	if (index >= N)
		return;
	int b = index / spatial;
	int i = index % spatial;
	int f;
	real_t sum = 0;
	for (f = 0; f < filters; ++f) {
		int index = b * filters * spatial + f * spatial + i;
		sum += powf(x[index], 2);
	}
	sum = sqrtf(sum);
	if (sum == 0)
		sum = 1;
	//printf("%f\n", sum);
	for (f = 0; f < filters; ++f) {
		int index = b * filters * spatial + f * spatial + i;
		x[index] /= sum;
		dx[index] = (1 - x[index]) / sum;
	}
}

extern "C" void l2normalize_gpu(real_t *x, real_t *dx, int batch, int filters,
		int spatial) {
	size_t N = batch * spatial;
	l2norm_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, dx, batch, filters,
			spatial);
	check_error(hipPeekAtLastError());
}

__global__ void fast_mean_kernel(real_t *x, int batch, int filters, int spatial,
		real_t *mean) {
	const int threads = BLOCK;
	__shared__ real_t local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;
			local[id] += (i + id < spatial) ? x[index] : 0;
		}
	}

	__syncthreads();

	if (id == 0) {
		mean[filter] = 0;
		for (i = 0; i < threads; ++i) {
			mean[filter] += local[i];
		}
		mean[filter] /= spatial * batch;
	}
}

__global__ void fast_variance_kernel(real_t *x, real_t *mean, int batch,
		int filters, int spatial, real_t *variance) {
	const int threads = BLOCK;
	__shared__ real_t local[threads];

	int id = threadIdx.x;
	local[id] = 0;

	int filter = blockIdx.x;

	int i, j;
	for (j = 0; j < batch; ++j) {
		for (i = 0; i < spatial; i += threads) {
			int index = j * spatial * filters + filter * spatial + i + id;

			local[id] +=
					(i + id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
		}
	}

	__syncthreads();

	if (id == 0) {
		variance[filter] = 0;
		for (i = 0; i < threads; ++i) {
			variance[filter] += local[i];
		}
		variance[filter] /= (spatial * batch - 1);
	}
}

extern "C" void fast_mean_gpu(real_t *x, int batch, int filters, int spatial,
		real_t *mean) {
	fast_mean_kernel<<<filters, BLOCK>>>(x, batch, filters, spatial, mean);
	check_error(hipPeekAtLastError());
}

extern "C" void fast_variance_gpu(real_t *x, real_t *mean, int batch,
		int filters, int spatial, real_t *variance) {
	fast_variance_kernel<<<filters, BLOCK>>>(x, mean, batch, filters, spatial,
			variance);
	check_error(hipPeekAtLastError());
}

extern "C" void mean_gpu(real_t *x, int batch, int filters, int spatial,
		real_t *mean) {
	mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, batch, filters, spatial,
			mean);
	check_error(hipPeekAtLastError());
}

extern "C" void variance_gpu(real_t *x, real_t *mean, int batch, int filters,
		int spatial, real_t *variance) {
	variance_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, mean, batch, filters,
			spatial, variance);
	check_error(hipPeekAtLastError());
}

extern "C" void axpy_gpu(int N, real_t ALPHA, real_t * X, int INCX, real_t * Y,
		int INCY) {
	axpy_gpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_gpu(int N, real_t ALPHA, real_t * X, int INCX, real_t * Y,
		int INCY) {
	pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

extern "C" void axpy_gpu_offset(int N, real_t ALPHA, real_t * X, int OFFX,
		int INCX, real_t * Y, int OFFY, int INCY) {
	axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY,
			INCY);
	check_error(hipPeekAtLastError());
}

extern "C" void copy_gpu(int N, real_t * X, int INCX, real_t * Y, int INCY) {
	copy_gpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void mul_gpu(int N, real_t * X, int INCX, real_t * Y, int INCY) {
	mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
	check_error(hipPeekAtLastError());
}

extern "C" void copy_gpu_offset(int N, real_t * X, int OFFX, int INCX,
		real_t * Y, int OFFY, int INCY) {
	copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
	check_error(hipPeekAtLastError());
}

__global__ void flatten_kernel(int N, real_t *x, int spatial, int layers,
		int batch, int forward, real_t *out) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int in_s = i % spatial;
	i = i / spatial;
	int in_c = i % layers;
	i = i / layers;
	int b = i;

	int i1 = b * layers * spatial + in_c * spatial + in_s;
	int i2 = b * layers * spatial + in_s * layers + in_c;

	if (forward)
		out[i2] = x[i1];
	else
		out[i1] = x[i2];
}

extern "C" void flatten_gpu(real_t *x, int spatial, int layers, int batch,
		int forward, real_t *out) {
	int size = spatial * batch * layers;
	flatten_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, spatial, layers,
			batch, forward, out);
	check_error(hipPeekAtLastError());
}

extern "C" void reorg_gpu(real_t *x, int w, int h, int c, int batch, int stride,
		int forward, real_t *out) {
	int size = w * h * c * batch;
	reorg_kernel<<<cuda_gridsize(size), BLOCK>>>(size, x, w, h, c, batch,
			stride, forward, out);
	check_error(hipPeekAtLastError());
}

__global__ void mask_kernel(int n, real_t *x, real_t mask_num, real_t *mask,
		real_t val) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num)
		x[i] = val;
}

extern "C" void mask_gpu(int N, real_t * X, real_t mask_num, real_t * mask,
		real_t val) {
	mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, val);
	check_error(hipPeekAtLastError());
}

__global__ void scale_mask_kernel(int n, real_t *x, real_t mask_num,
		real_t *mask, real_t scale) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n && mask[i] == mask_num)
		x[i] *= scale;
}

extern "C" void scale_mask_gpu(int N, real_t * X, real_t mask_num,
		real_t * mask, real_t scale) {
	scale_mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask, scale);
	check_error(hipPeekAtLastError());
}

extern "C" void const_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

extern "C" void constrain_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	constrain_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

extern "C" void add_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	add_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

extern "C" void scal_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

extern "C" void supp_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	supp_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

extern "C" void fill_gpu(int N, real_t ALPHA, real_t * X, int INCX) {
	fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
	check_error(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc,
		int stride, int sample, int batch, int w1, int h1, int c1, real_t *add,
		int w2, int h2, int c2, real_t s1, real_t s2, real_t *out) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= size)
		return;
	int i = id % minw;
	id /= minw;
	int j = id % minh;
	id /= minh;
	int k = id % minc;
	id /= minc;
	int b = id % batch;

	int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
	int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
	out[out_index] = s1 * out[out_index] + s2 * add[add_index];
	//out[out_index] += add[add_index];
}

extern "C" void shortcut_gpu(int batch, int w1, int h1, int c1, real_t *add,
		int w2, int h2, int c2, real_t s1, real_t s2, real_t *out) {
	int minw = (w1 < w2) ? w1 : w2;
	int minh = (h1 < h2) ? h1 : h2;
	int minc = (c1 < c2) ? c1 : c2;

	int stride = w1 / w2;
	int sample = w2 / w1;
	assert(stride == h1 / h2);
	assert(sample == h2 / h1);
	if (stride < 1)
		stride = 1;
	if (sample < 1)
		sample = 1;

	int size = batch * minw * minh * minc;
	shortcut_kernel<<<cuda_gridsize(size), BLOCK>>>(size, minw, minh, minc,
			stride, sample, batch, w1, h1, c1, add, w2, h2, c2, s1, s2, out);
	check_error(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, real_t *pred, real_t *truth,
		real_t *delta, real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t diff = truth[i] - pred[i];
		real_t abs_val = fabsf(diff);
		if (abs_val < 1) {
			error[i] = diff * diff;
			delta[i] = diff;
		} else {
			error[i] = 2 * abs_val - 1;
			delta[i] = (diff > 0) ? 1 : -1;
		}
	}
}

extern "C" void smooth_l1_gpu(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	smooth_l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void softmax_x_ent_kernel(int n, real_t *pred, real_t *truth,
		real_t *delta, real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t t = truth[i];
		real_t p = pred[i];
		error[i] = (t) ? -log(p) : 0;
		delta[i] = t - p;
	}
}

extern "C" void softmax_x_ent_gpu(int n, real_t *pred, real_t *truth,
		real_t *delta, real_t *error) {
	softmax_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta,
			error);
	check_error(hipPeekAtLastError());
}

__global__ void logistic_x_ent_kernel(int n, real_t *pred, real_t *truth,
		real_t *delta, real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t t = truth[i];
		real_t p = pred[i];
		error[i] = -t * log(p + .0000001) - (1 - t) * log(1 - p + .0000001);
		delta[i] = t - p;
	}
}

extern "C" void logistic_x_ent_gpu(int n, real_t *pred, real_t *truth,
		real_t *delta, real_t *error) {
	logistic_x_ent_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta,
			error);
	check_error(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t diff = truth[i] - pred[i];
		error[i] = diff * diff; //I know this is technically wrong, deal with it.
		delta[i] = diff;
	}
}

extern "C" void l2_gpu(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	l2_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void l1_kernel(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		real_t diff = truth[i] - pred[i];
		error[i] = abs(diff);
		delta[i] = (diff > 0) ? 1 : -1;
	}
}

extern "C" void l1_gpu(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void wgan_kernel(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		error[i] = truth[i] ? -pred[i] : pred[i];
		delta[i] = (truth[i] > 0) ? 1 : -1;
	}
}

extern "C" void wgan_gpu(int n, real_t *pred, real_t *truth, real_t *delta,
		real_t *error) {
	wgan_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
	check_error(hipPeekAtLastError());
}

__global__ void weighted_sum_kernel(int n, real_t *a, real_t *b, real_t *s,
		real_t *c) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
	}
}

__global__ void deinter_kernel(int NX, real_t *X, int NY, real_t *Y, int B,
		real_t *OUT) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < (NX + NY) * B) {
		int b = i / (NX + NY);
		int j = i % (NX + NY);
		if (j < NX) {
			if (X)
				X[b * NX + j] += OUT[i];
		} else {
			if (Y)
				Y[b * NY + j - NX] += OUT[i];
		}
	}
}

extern "C" void deinter_gpu(int NX, real_t *X, int NY, real_t *Y, int B,
		real_t *OUT) {
	deinter_kernel<<<cuda_gridsize((NX + NY) * B), BLOCK>>>(NX, X, NY, Y, B,
			OUT);
	check_error(hipPeekAtLastError());
}

__global__ void inter_kernel(int NX, real_t *X, int NY, real_t *Y, int B,
		real_t *OUT) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < (NX + NY) * B) {
		int b = i / (NX + NY);
		int j = i % (NX + NY);
		if (j < NX) {
			OUT[i] = X[b * NX + j];
		} else {
			OUT[i] = Y[b * NY + j - NX];
		}
	}
}

extern "C" void inter_gpu(int NX, real_t *X, int NY, real_t *Y, int B,
		real_t *OUT) {
	inter_kernel<<<cuda_gridsize((NX + NY) * B), BLOCK>>>(NX, X, NY, Y, B, OUT);
	check_error(hipPeekAtLastError());
}

extern "C" void weighted_sum_gpu(real_t *a, real_t *b, real_t *s, int num,
		real_t *c) {
	weighted_sum_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, c);
	check_error(hipPeekAtLastError());
}

__global__ void weighted_delta_kernel(int n, real_t *a, real_t *b, real_t *s,
		real_t *da, real_t *db, real_t *ds, real_t *dc) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		if (da)
			da[i] += dc[i] * s[i];
		if (db)
			db[i] += dc[i] * (1 - s[i]);
		ds[i] += dc[i] * (a[i] - b[i]);
	}
}

extern "C" void weighted_delta_gpu(real_t *a, real_t *b, real_t *s, real_t *da,
		real_t *db, real_t *ds, int num, real_t *dc) {
	weighted_delta_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, s, da, db,
			ds, dc);
	check_error(hipPeekAtLastError());
}

__global__ void mult_add_into_kernel(int n, real_t *a, real_t *b, real_t *c) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < n) {
		c[i] += a[i] * b[i];
	}
}

extern "C" void mult_add_into_gpu(int num, real_t *a, real_t *b, real_t *c) {
	mult_add_into_kernel<<<cuda_gridsize(num), BLOCK>>>(num, a, b, c);
	check_error(hipPeekAtLastError());
}

__device__ void softmax_device(real_t *input, int n, real_t temp, int stride,
		real_t *output) {
	int i;
	real_t sum = 0;
	real_t largest = -INFINITY;
	for (i = 0; i < n; ++i) {
		int val = input[i * stride];
		largest = (val > largest) ? val : largest;
	}
	for (i = 0; i < n; ++i) {
		real_t e = expf(input[i * stride] / temp - largest / temp);
		sum += e;
		output[i * stride] = e;
	}
	for (i = 0; i < n; ++i) {
		output[i * stride] /= sum;
	}
}

__global__ void softmax_tree_kernel(real_t *input, int spatial, int batch,
		int stride, real_t temp, real_t *output, int groups, int *group_size,
		int *group_offset) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= spatial * batch * groups)
		return;
	int s = id % spatial;
	id = id / spatial;
	int g = id % groups;
	int b = id / groups;
	int goff = group_offset[g] * spatial;
	int boff = b * stride;
	softmax_device(input + goff + boff + s, group_size[g], temp, spatial,
			output + goff + boff + s);
}

extern "C" void softmax_tree(real_t *input, int spatial, int batch, int stride,
		real_t temp, real_t *output, tree hier) {
	int *tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	int *tree_groups_offset = cuda_make_int_array(hier.group_offset,
			hier.groups);
	/*
	 static int *tree_groups_size = 0;
	 static int *tree_groups_offset = 0;
	 if(!tree_groups_size){
	 tree_groups_size = cuda_make_int_array(hier.group_size, hier.groups);
	 tree_groups_offset = cuda_make_int_array(hier.group_offset, hier.groups);
	 }
	 */
	int num = spatial * batch * hier.groups;
	softmax_tree_kernel<<<cuda_gridsize(num), BLOCK>>>(input, spatial, batch,
			stride, temp, output, hier.groups, tree_groups_size,
			tree_groups_offset);
	check_error(hipPeekAtLastError());
	cuda_free((real_t *) tree_groups_size);
	cuda_free((real_t *) tree_groups_offset);
}

__global__ void softmax_kernel(real_t *input, int n, int batch,
		int batch_offset, int groups, int group_offset, int stride, real_t temp,
		real_t *output) {
	int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= batch * groups)
		return;
	int b = id / groups;
	int g = id % groups;
	softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride,
			output + b * batch_offset + g * group_offset);
}

extern "C" void softmax_gpu(real_t *input, int n, int batch, int batch_offset,
		int groups, int group_offset, int stride, real_t temp, real_t *output) {
	softmax_kernel<<<cuda_gridsize(batch * groups), BLOCK>>>(input, n, batch,
			batch_offset, groups, group_offset, stride, temp, output);
	check_error(hipPeekAtLastError());
}

__global__ void upsample_kernel(size_t N, real_t *x, int w, int h, int c,
		int batch, int stride, int forward, real_t scale, real_t *out) {
	size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= N)
		return;
	int out_index = i;
	int out_w = i % (w * stride);
	i = i / (w * stride);
	int out_h = i % (h * stride);
	i = i / (h * stride);
	int out_c = i % c;
	i = i / c;
	int b = i % batch;

	int in_w = out_w / stride;
	int in_h = out_h / stride;
	int in_c = out_c;

	int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

	if (forward)
		out[out_index] += scale * x[in_index];
	else
		atomicAdd(x + in_index, scale * out[out_index]);
}
extern "C" void upsample_gpu(real_t *in, int w, int h, int c, int batch,
		int stride, int forward, real_t scale, real_t *out) {
	size_t size = w * h * c * batch * stride * stride;
	upsample_kernel<<<cuda_gridsize(size), BLOCK>>>(size, in, w, h, c, batch,
			stride, forward, scale, out);
	check_error(hipPeekAtLastError());
}
