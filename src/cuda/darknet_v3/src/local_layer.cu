#include "hip/hip_runtime.h"
#include "local_layer.h"
#include "utils.h"
#include "im2col.h"
#include "col2im.h"
#include "blas.h"
#include "gemm.h"
#include <stdio.h>
#include <time.h>

int local_out_height(local_layer l) {
	int h = l.h;
	if (!l.pad)
		h -= l.size;
	else
		h -= 1;
	return h / l.stride + 1;
}

int local_out_width(local_layer l) {
	int w = l.w;
	if (!l.pad)
		w -= l.size;
	else
		w -= 1;
	return w / l.stride + 1;
}

local_layer make_local_layer(int batch, int h, int w, int c, int n, int size,
		int stride, int pad, ACTIVATION activation) {
	int i;
	local_layer l; // = { 0 };
	l.type = LOCAL;

	l.h = h;
	l.w = w;
	l.c = c;
	l.n = n;
	l.batch = batch;
	l.stride = stride;
	l.size = size;
	l.pad = pad;

	int out_h = local_out_height(l);
	int out_w = local_out_width(l);
	int locations = out_h * out_w;
	l.out_h = out_h;
	l.out_w = out_w;
	l.out_c = n;
	l.outputs = l.out_h * l.out_w * l.out_c;
	l.inputs = l.w * l.h * l.c;

	l.weights = (real_t*) calloc(c * n * size * size * locations,
			sizeof(real_t));
	l.weight_updates = (real_t*) calloc(c * n * size * size * locations,
			sizeof(real_t));

	l.biases = (real_t*) calloc(l.outputs, sizeof(real_t));
	l.bias_updates = (real_t*) calloc(l.outputs, sizeof(real_t));

	// real_t scale = 1./sqrt(size*size*c);
	real_t scale = real_t(sqrt(2. / (size * size * c)));
	for (i = 0; i < c * n * size * size; ++i)
		l.weights[i] = scale * rand_uniform(real_t(-1), real_t(1));

	l.output = (real_t*) calloc(l.batch * out_h * out_w * n, sizeof(real_t));
	l.delta = (real_t*) calloc(l.batch * out_h * out_w * n, sizeof(real_t));

	l.workspace_size = out_h * out_w * size * size * c;

	l.forward = forward_local_layer;
	l.backward = backward_local_layer;
	l.update = update_local_layer;

#ifdef GPU
	l.forward_gpu = forward_local_layer_gpu;
	l.backward_gpu = backward_local_layer_gpu;
	l.update_gpu = update_local_layer_gpu;

	l.weights_gpu = cuda_make_array(l.weights, c * n * size * size * locations);
	l.weight_updates_gpu = cuda_make_array(l.weight_updates,
			c * n * size * size * locations);

	l.biases_gpu = cuda_make_array(l.biases, l.outputs);
	l.bias_updates_gpu = cuda_make_array(l.bias_updates, l.outputs);

	l.delta_gpu = cuda_make_array(l.delta, l.batch * out_h * out_w * n);
	l.output_gpu = cuda_make_array(l.output, l.batch * out_h * out_w * n);

#endif
	l.activation = activation;

	fprintf(stderr,
			"Local Layer: %d x %d x %d image, %d filters -> %d x %d x %d image\n",
			h, w, c, n, out_h, out_w, n);

	return l;
}

void forward_local_layer(const local_layer l, network net) {
	int out_h = local_out_height(l);
	int out_w = local_out_width(l);
	int i, j;
	int locations = out_h * out_w;

	for (i = 0; i < l.batch; ++i) {
		copy_cpu(l.outputs, l.biases, 1, l.output + i * l.outputs, 1);
	}

	for (i = 0; i < l.batch; ++i) {
		real_t *input = net.input + i * l.w * l.h * l.c;
#ifndef GPU
		im2col_cpu(input, l.c, l.h, l.w, l.size, l.stride, l.pad,
				net.workspace);
		real_t *output = l.output + i * l.outputs;
		for (j = 0; j < locations; ++j) {
			real_t *a = l.weights + j * l.size * l.size * l.c * l.n;
			real_t *b = net.workspace + j;
			real_t *c = output + j;

			int m = l.n;
			int n = 1;
			int k = l.size * l.size * l.c;

			gemm(0, 0, m, n, k, 1, a, k, b, locations, 1, c, locations);
		}
#endif
	}
	activate_array(l.output, l.outputs * l.batch, l.activation);
}

void backward_local_layer(local_layer l, network net) {
	int i, j;
	int locations = l.out_w * l.out_h;

	gradient_array(l.output, l.outputs * l.batch, l.activation, l.delta);

	for (i = 0; i < l.batch; ++i) {
		axpy_cpu(l.outputs, real_t(1), l.delta + i * l.outputs, 1, l.bias_updates, 1);
	}

	for (i = 0; i < l.batch; ++i) {
		real_t *input = net.input + i * l.w * l.h * l.c;
#ifndef GPU
		im2col_cpu(input, l.c, l.h, l.w, l.size, l.stride, l.pad,
				net.workspace);

		for (j = 0; j < locations; ++j) {
			real_t *a = l.delta + i * l.outputs + j;
			real_t *b = net.workspace + j;
			real_t *c = l.weight_updates + j * l.size * l.size * l.c * l.n;
			int m = l.n;
			int n = l.size * l.size * l.c;
			int k = 1;

			gemm(0, 1, m, n, k, 1, a, locations, b, locations, 1, c, n);
		}

		if (net.delta) {
			for (j = 0; j < locations; ++j) {
				real_t *a = l.weights + j * l.size * l.size * l.c * l.n;
				real_t *b = l.delta + i * l.outputs + j;
				real_t *c = net.workspace + j;

				int m = l.size * l.size * l.c;
				int n = 1;
				int k = l.n;

				gemm(1, 0, m, n, k, 1, a, m, b, locations, 0, c, locations);
			}

			col2im_cpu(net.workspace, l.c, l.h, l.w, l.size, l.stride, l.pad,
					net.delta + i * l.c * l.h * l.w);
		}
#endif
	}
}

void update_local_layer(local_layer l, update_args a) {
	real_t learning_rate = a.learning_rate * l.learning_rate_scale;
	real_t momentum = a.momentum;
	real_t decay = a.decay;
	int batch = a.batch;

	int locations = l.out_w * l.out_h;
	int size = l.size * l.size * l.c * l.n * locations;
	axpy_cpu(l.outputs, real_t(learning_rate / batch), l.bias_updates, 1, l.biases, 1);
	scal_cpu(l.outputs, momentum, l.bias_updates, 1);

	axpy_cpu(size, real_t(-decay * batch), l.weights, 1, l.weight_updates, 1);
	axpy_cpu(size, real_t(learning_rate / batch), l.weight_updates, 1, l.weights, 1);
	scal_cpu(size, momentum, l.weight_updates, 1);
}

#ifdef GPU

void forward_local_layer_gpu(const local_layer l, network net) {
	int out_h = local_out_height(l);
	int out_w = local_out_width(l);
	int i, j;
	int locations = out_h * out_w;

	for (i = 0; i < l.batch; ++i) {
		copy_gpu(l.outputs, l.biases_gpu, 1, l.output_gpu + i * l.outputs, 1);
	}

	for (i = 0; i < l.batch; ++i) {
		real_t_device *input = net.input_gpu + i * l.w * l.h * l.c;
		im2col_gpu(input, l.c, l.h, l.w, l.size, l.stride, l.pad,
				net.workspace);
		real_t_device *output = l.output_gpu + i * l.outputs;
		for (j = 0; j < locations; ++j) {
			real_t_device *a = l.weights_gpu + j * l.size * l.size * l.c * l.n;
			real_t_device *b = net.workspace + j;
			real_t_device *c = output + j;

			int m = l.n;
			int n = 1;
			int k = l.size * l.size * l.c;

			gemm_gpu(0, 0, m, n, k, (1), a, k, b, locations, (1), c, locations);
		}
	}
	activate_array_gpu(l.output_gpu, l.outputs * l.batch, l.activation);
}

void backward_local_layer_gpu(local_layer l, network net) {
	int i, j;
	int locations = l.out_w * l.out_h;

	gradient_array_gpu(l.output_gpu, l.outputs * l.batch, l.activation,
			l.delta_gpu);
	for (i = 0; i < l.batch; ++i) {
		axpy_gpu(l.outputs, (1), l.delta_gpu + i * l.outputs, 1,
				l.bias_updates_gpu, 1);
	}

	for (i = 0; i < l.batch; ++i) {
		real_t_device *input = net.input_gpu + i * l.w * l.h * l.c;
		im2col_gpu(input, l.c, l.h, l.w, l.size, l.stride, l.pad,
				net.workspace);

		for (j = 0; j < locations; ++j) {
			real_t_device *a = l.delta_gpu + i * l.outputs + j;
			real_t_device *b = net.workspace + j;
			real_t_device *c = l.weight_updates_gpu + j * l.size * l.size * l.c * l.n;
			int m = l.n;
			int n = l.size * l.size * l.c;
			int k = 1;

			gemm_gpu(0, 1, m, n, k, (1), a, locations, b, locations, (1), c, n);
		}

		if (net.delta_gpu) {
			for (j = 0; j < locations; ++j) {
				real_t_device *a = l.weights_gpu + j * l.size * l.size * l.c * l.n;
				real_t_device *b = l.delta_gpu + i * l.outputs + j;
				real_t_device *c = net.workspace + j;

				int m = l.size * l.size * l.c;
				int n = 1;
				int k = l.n;

				gemm_gpu(1, 0, m, n, k, (1), a, m, b, locations, (0), c, locations);
			}

			col2im_gpu(net.workspace, l.c, l.h, l.w, l.size, l.stride, l.pad,
					net.delta_gpu + i * l.c * l.h * l.w);
		}
	}
}

void update_local_layer_gpu(local_layer l, update_args a) {
	real_t learning_rate = a.learning_rate * l.learning_rate_scale;
	real_t momentum = a.momentum;
	real_t decay = a.decay;
	int batch = a.batch;

	int locations = l.out_w * l.out_h;
	int size = l.size * l.size * l.c * l.n * locations;
	axpy_gpu(l.outputs, (learning_rate / batch), l.bias_updates_gpu, 1,
			l.biases_gpu, 1);
	scal_gpu(l.outputs, CAST(momentum), l.bias_updates_gpu, 1);

	axpy_gpu(size, (-decay * batch), l.weights_gpu, 1, l.weight_updates_gpu, 1);
	axpy_gpu(size, (learning_rate / batch), l.weight_updates_gpu, 1,
			l.weights_gpu, 1);
	scal_gpu(size, CAST(momentum), l.weight_updates_gpu, 1);
}

void pull_local_layer(local_layer l) {
	int locations = l.out_w * l.out_h;
	int size = l.size * l.size * l.c * l.n * locations;
	cuda_pull_array(l.weights_gpu, l.weights, size);
	cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
}

void push_local_layer(local_layer l) {
	int locations = l.out_w * l.out_h;
	int size = l.size * l.size * l.c * l.n * locations;
	cuda_push_array(l.weights_gpu, l.weights, size);
	cuda_push_array(l.biases_gpu, l.biases, l.outputs);
}
#endif
