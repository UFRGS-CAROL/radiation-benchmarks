#include "hip/hip_runtime.h"
#include "Micro.h"
#include "hip/device_functions.h"

template<uint32_t UNROLL_MAX, bool USEFASTMATH, typename real_t>
__global__ void micro_kernel_fma(real_t *d_R0, real_t input_a, real_t input_b,
		real_t output_r, const uint32_t ops) {
	real_t acc = output_r;
	real_t input_a_neg = -input_a;
	real_t input_b_neg = -input_b;

#pragma unroll UNROLL_MAX
	for (uint32_t count = 0; count < ops; count++) {
		acc = fma_inline<USEFASTMATH>(input_a, input_b, acc);
		acc = fma_inline<USEFASTMATH>(input_a_neg, input_b, acc);
		acc = fma_inline<USEFASTMATH>(input_a, input_b_neg, acc);
		acc = fma_inline<USEFASTMATH>(input_a_neg, input_b_neg, acc);
	}

	d_R0[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

template<uint32_t UNROLL_MAX, bool USEFASTMATH, typename real_t>
__global__ void micro_kernel_add(real_t *d_R0, real_t input_a, real_t input_b,
		real_t output_r, const uint32_t ops) {
	real_t acc = output_r;
	real_t input_a_neg = -output_r;

#pragma unroll UNROLL_MAX
	for (uint32_t count = 0; count < ops; count++) {
		acc = add_inline<USEFASTMATH>(acc, input_a);
		acc = add_inline<USEFASTMATH>(acc, input_a_neg);
		acc = add_inline<USEFASTMATH>(acc, input_a_neg);
		acc = add_inline<USEFASTMATH>(acc, input_a);
	}

	d_R0[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

template<uint32_t UNROLL_MAX, bool USEFASTMATH, typename real_t>
__global__ void micro_kernel_mul(real_t *d_R0, real_t input_a, real_t input_b,
		real_t output_r, const uint32_t ops) {
	real_t acc = output_r;
	real_t input_a_inv = real_t(1.0) / input_a;

#pragma unroll UNROLL_MAX
	for (uint32_t count = 0; count < ops; count++) {
		acc = mul_inline<USEFASTMATH>(acc, input_a);
		acc = mul_inline<USEFASTMATH>(acc, input_a_inv);
		acc = mul_inline<USEFASTMATH>(acc, input_a_inv);
		acc = mul_inline<USEFASTMATH>(acc, input_a);
	}

	d_R0[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

template<uint32_t UNROLL_MAX, bool USEFASTMATH, typename real_t>
__global__ void micro_kernel_pythagorean(real_t *d_R0, real_t input_a,
		real_t input_b, real_t output_r, const uint32_t ops) {
	real_t acc = output_r;

#pragma unroll UNROLL_MAX
	for (uint32_t count = 0; count < ops; count++) {
		acc += pythagorean_identity<USEFASTMATH>(input_a, input_a);
	}

	d_R0[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

template<uint32_t UNROLL_MAX, bool USEFASTMATH, typename real_t>
__global__ void micro_kernel_euler(real_t *d_R0, real_t input_a, real_t input_b,
		real_t output_r, const uint32_t ops) {
	real_t acc = 0;

#pragma unroll UNROLL_MAX
	for (uint32_t count = 0; count < ops; count++) {
		acc += euler<USEFASTMATH>(input_a);
	}

	d_R0[blockIdx.x * blockDim.x + threadIdx.x] = acc;
}

template<bool USEFASTMATH, typename real_t>
void execute_kernel(MICROINSTRUCTION& micro, real_t* output, real_t input_a,
		real_t input_b, real_t output_acc, size_t grid_size, size_t block_size,
		uint32_t operation_num) {

	void (*kernel)(real_t*, real_t, real_t, real_t, uint32_t);
	switch (micro) {
	case ADD:
//		kernel = micro_kernel_add<LOOPING_UNROLL, USEFASTMATH>;
		break;
	case MUL:
//		kernel = micro_kernel_mul<LOOPING_UNROLL, USEFASTMATH>;
		break;
	case FMA:
//		kernel = micro_kernel_fma<LOOPING_UNROLL, USEFASTMATH>;
		break;
		throw_line("Not implemented yet")
		;
		break;
	case PYTHAGOREAN:
		kernel = micro_kernel_pythagorean<LOOPING_UNROLL, USEFASTMATH>;
		break;
	case EULER:
		kernel = micro_kernel_euler<LOOPING_UNROLL, USEFASTMATH>;
		break;
	}
	kernel<<<grid_size, block_size>>>(output, input_a, input_b, output_acc,
			operation_num);
}

template<>
void Micro<float>::execute_micro() {
	if (this->parameters.fast_math) {
		execute_kernel<true>(this->parameters.micro, this->output_device.data(),
				this->input_kernel.INPUT_A, this->input_kernel.INPUT_B,
				this->input_kernel.OUTPUT_R, this->parameters.grid_size,
				this->parameters.block_size, this->parameters.operation_num);
	} else {
		execute_kernel<false>(this->parameters.micro,
				this->output_device.data(), this->input_kernel.INPUT_A,
				this->input_kernel.INPUT_B, this->input_kernel.OUTPUT_R,
				this->parameters.grid_size, this->parameters.block_size,
				this->parameters.operation_num);
	}
}
