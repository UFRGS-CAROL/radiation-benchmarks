#include "hip/hip_runtime.h"
#include <iostream>

#include "include/device_vector.h"

#include "Parameters.h"
#include "common_template_functions.h"
#include "GemmCallerMMA.h"

template<typename half_t, typename real_t>
void setup_execute(Parameters& parameters,
		TensorCoresCaller<half_t, real_t>& mult_env, const uint32_t threshold =
				0) {
	/*double elapsed_time = 0;

	 std::vector<half_t> a_vector_host(
	 parameters.size_matrices * parameters.size_matrices);
	 std::vector<half_t> b_vector_host(
	 parameters.size_matrices * parameters.size_matrices);
	 std::vector<real_t> c_vector_host(
	 parameters.size_matrices * parameters.size_matrices);
	 std::vector<real_t> gold_host(
	 parameters.size_matrices * parameters.size_matrices);

	 //Output host vectors are set after computation
	 std::vector<real_t> d_vector_host_real_t;
	 std::vector<real_t> d_vector_host_half_t;

	 if (parameters.generate) {
	 std::cout << "Generating input matrices\n";
	 auto read_abc_files_on_generate = (parameters.check_input_existence
	 && exists(parameters.a_input_path)
	 && exists(parameters.b_input_path)
	 && exists(parameters.c_input_path));

	 get_input_matrices(parameters.size_matrices, a_vector_host,
	 b_vector_host, c_vector_host, parameters.a_input_path,
	 parameters.b_input_path, parameters.c_input_path,
	 read_abc_files_on_generate);
	 } else {
	 std::cout << "Reading input matrices\n";
	 read_abc_files(parameters.a_input_path, a_vector_host,
	 parameters.b_input_path, b_vector_host, parameters.c_input_path,
	 c_vector_host);

	 read_gold(parameters.gold_inout_path, gold_host);
	 }

	 //Alloc only after reading the inputs
	 rad::DeviceVector<half_t> a_vector_device = a_vector_host;
	 rad::DeviceVector<half_t> b_vector_device = b_vector_host;
	 rad::DeviceVector<real_t> c_vector_device = c_vector_host;

	 rad::DeviceVector<real_t> d_vector_device(
	 parameters.size_matrices * parameters.size_matrices);
	 rad::DeviceVector<real_t> d_vector_half_t_device(
	 parameters.size_matrices * parameters.size_matrices);

	 std::cout << "Starting the setup process...\n";
	 std::cout << std::setprecision(5) << std::fixed;
	 for (int it = 0; it < parameters.iterations; it++) {
	 auto computation_time = rad::mysecond();

	 parameters.start_iteration();

	 mult_env.gemm(a_vector_device, b_vector_device, c_vector_device,
	 d_vector_device, d_vector_half_t_device, parameters.alpha,
	 parameters.beta, parameters.size_matrices,
	 parameters.size_matrices, threshold);
	 rad::checkFrameworkErrors(hipDeviceSynchronize());
	 ;
	 rad::checkFrameworkErrors(hipPeekAtLastError());

	 //end iteration
	 parameters.end_iteration();
	 computation_time = rad::mysecond() - computation_time;
	 elapsed_time += computation_time;

	 double copy_time = rad::mysecond();
	 mult_env.memcpy_half_t_mem(d_vector_host_half_t,  d_vector_half_t_device);
	 d_vector_device.to_vector(d_vector_host_real_t);
	 copy_time = rad::mysecond() - copy_time;

	 if (!parameters.generate) {

	 auto comparing_time = rad::mysecond();
	 auto errors =  check_output_errors_dmr(gold_host, d_vector_host_real_t,
	 d_vector_host_half_t, parameters, threshold,
	 mult_env.duplicated);

	 comparing_time = rad::mysecond() - comparing_time;

	 std::cout << "Iteration: " << it << " DMR errors " << errors.first
	 << ". " << "Radiation errors: " << errors.second << ". "
	 << "Time spent on computation: " << computation_time
	 << "s. " << "Time spent on comparing: " << comparing_time
	 << "s. " << "Time spent on copying: " << copy_time << "s. "
	 << std::endl;

	 //If errors != 0 reload matrices to gpu
	 if (errors.first != 0 || errors.second != 0) {
	 read_gold(a_vector_host, b_vector_host, c_vector_host,
	 gold_host, parameters.a_input_path,
	 parameters.b_input_path, parameters.c_input_path,
	 parameters.gold_inout_path);

	 a_vector_device.resize(0);
	 b_vector_device.resize(0);
	 c_vector_device.resize(0);
	 d_vector_device.resize(0);
	 d_vector_half_t_device.resize(0);

	 a_vector_device = a_vector_host;
	 b_vector_device = b_vector_host;
	 c_vector_device = c_vector_host;
	 d_vector_device = d_vector_host_real_t;
	 d_vector_half_t_device = d_vector_host_half_t;

	 }

	 }

	 }

	 std::cout << "Elapsed time: " << (elapsed_time / parameters.iterations)
	 << " s\n";
	 if (parameters.generate) {
	 auto zero_count = 0ul;
	 auto nans_count = 0ul;
	 for (auto s : d_vector_host) {
	 zero_count += (float(s) == 0.0f);
	 nans_count += (std::isnan(float(s)));
	 }
	 std::cout << "Zero values on gold: " << zero_count << std::endl;
	 std::cout << "Nans values on gold: " << nans_count << std::endl;

	 write_gold(parameters.gold_inout_path, d_vector_host);
	 }
	 */
}

void setup_gemm_tensor_cores_unhardened(Parameters& parameters) {
#if __CUDA_ARCH__ >= 600
	if (parameters.precision == "half") {
		UnhardenedTensorCoresCaller<half, half> gemm_obj(parameters.size_matrices,
				parameters.size_matrices);
		setup_execute(parameters, gemm_obj);

	}

#endif
	if (parameters.precision == "float" || parameters.precision == "single"
			|| parameters.precision == "double") {
		throw_line(
				parameters.precision + " using tensorcores not ready yet!!!");
	}

}
void setup_gemm_tensor_cores_dmr(Parameters& parameters) {
#if __CUDA_ARCH__ >= 600
	if (parameters.precision == "half") {
		DMRTensorCoresCaller<half> gemm_obj(parameters.size_matrices,
				parameters.size_matrices);
		setup_execute(parameters, gemm_obj);

	}
#endif

	if (parameters.precision == "float" || parameters.precision == "single"
			|| parameters.precision == "double") {
		throw_line(
				parameters.precision + " using tensorcores not ready yet!!!");
	}
}

