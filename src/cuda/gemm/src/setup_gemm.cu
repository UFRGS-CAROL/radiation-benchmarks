#include "hip/hip_runtime.h"
#include "Parameters.h"
#include "include/device_vector.h"
#include "include/multi_compiler_analysis.h"
#include "common_template_functions.h"

#include "GemmCaller.h"

extern void show_iteration_status(int it, bool verbose, double copy_time, double comparing_time,
		double computation_time, std::pair<int, int> errors);

std::string get_multi_compiler_header() {
	std::string test_info = " nvcc_version:" + rad::get_cuda_cc_version();
	test_info += " nvcc_optimization_flags:" + rad::extract_nvcc_opt_flags_str();
	return test_info;
}

template<const uint32_t COUNT, typename half_t, typename real_t>
void setup_execute(Parameters &parameters, GemmCaller<COUNT, half_t, real_t> &mult_env,
		const uint32_t threshold = 0) {
	double elapsed_time = 0;

	std::vector<real_t> a_vector_host(parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> b_vector_host(parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> c_vector_host(parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> d_vector_host(parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> gold_host(parameters.size_matrices * parameters.size_matrices);

	//Output host vectors are set after computation
	std::vector<half_t> c_vector_host_half_t(parameters.size_matrices * parameters.size_matrices);

	if (parameters.generate) {
		std::cout << "Generating input matrices\n";
		auto read_abc_files_on_generate = (parameters.check_input_existence
				&& exists(parameters.a_input_path) && exists(parameters.b_input_path)
				&& exists(parameters.c_input_path));

		get_input_matrices(parameters.size_matrices, a_vector_host, b_vector_host, c_vector_host,
				parameters.a_input_path, parameters.b_input_path, parameters.c_input_path,
				read_abc_files_on_generate);
	} else {
		std::cout << "Reading input matrices\n";
		read_abc_files(parameters.a_input_path, a_vector_host, parameters.b_input_path,
				b_vector_host, parameters.c_input_path, c_vector_host);

		read_gold(parameters.gold_inout_path, gold_host);
	}

	//Alloc only after reading the inputs
	rad::DeviceVector<real_t> a_vector_device = a_vector_host;
	rad::DeviceVector<real_t> b_vector_device = b_vector_host;
	rad::DeviceVector<real_t> c_vector_device = c_vector_host;
	rad::DeviceVector<half_t> c_vector_device_half_t = c_vector_host_half_t;

	std::cout << "Starting the setup process...\n";
	std::cout << std::setprecision(5) << std::fixed;
	for (int it = 0; it < parameters.iterations; it++) {
		c_vector_device = c_vector_host;

		auto computation_time = rad::mysecond();

		parameters.start_iteration();

		mult_env.gemm(a_vector_device, b_vector_device, c_vector_device, c_vector_device_half_t,
				real_t(parameters.alpha), real_t(parameters.beta), parameters.size_matrices,
				parameters.size_matrices, threshold);
//		rad::checkFrameworkErrors(hipDeviceSynchronize());
//		;
//		rad::checkFrameworkErrors(hipPeekAtLastError());
//		;
		//new DUE setup
		bool is_memory_reload_needed = rad::checkFrameworkErrorsAndResetGPU(hipDeviceSynchronize());

		parameters.end_iteration();
		computation_time = rad::mysecond() - computation_time;
		elapsed_time += computation_time;

		double copy_time = rad::mysecond();

		mult_env.memcpy_half_t_mem(c_vector_host_half_t, c_vector_device_half_t);

		c_vector_device.to_vector(d_vector_host);
		copy_time = rad::mysecond() - copy_time;

		if (!parameters.generate) {

			auto comparing_time = rad::mysecond();
			auto errors = check_output_errors_dmr(gold_host, d_vector_host, c_vector_host_half_t,
					parameters, threshold, mult_env.duplicated);

			comparing_time = rad::mysecond() - comparing_time;

			show_iteration_status(it, parameters.verbose, copy_time, comparing_time,
					computation_time, errors);

			//If errors != 0 reload matrices to gpu
			if (errors.first != 0 || errors.second != 0 || is_memory_reload_needed == true) {
				read_abc_files(parameters.a_input_path, a_vector_host, parameters.b_input_path,
						b_vector_host, parameters.c_input_path, c_vector_host);
				read_gold(parameters.gold_inout_path, gold_host);

				a_vector_device.resize(0);
				b_vector_device.resize(0);
				c_vector_device.resize(0);
				c_vector_device_half_t.resize(0);

				a_vector_device = a_vector_host;
				b_vector_device = b_vector_host;
				c_vector_device = c_vector_host;
				c_vector_device_half_t = c_vector_host_half_t;

			}

		}

	}
	if (parameters.verbose) {

		std::cout << "Elapsed time: " << (elapsed_time / parameters.iterations) << " s\n";
	} else {
		std::cout << "done.\n";
	}

	if (parameters.generate) {
		auto zero_count = 0ul;
		auto nans_count = 0ul;
		for (auto s : d_vector_host) {
			zero_count += (float(s) == 0.0f);
			nans_count += (std::isnan(float(s)));
		}
		std::cout << "Zero values on gold: " << zero_count << std::endl;
		std::cout << "Nans values on gold: " << nans_count << std::endl;

		write_gold(parameters.gold_inout_path, d_vector_host);
	}
}

void setup_gemm_unhardened(Parameters &parameters) {
	if (parameters.precision == "half") {
		UnhardenedGemmCaller<half> gemm_obj(parameters.size_matrices, parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
	}
//
	if (parameters.precision == "float" || parameters.precision == "single") {
		UnhardenedGemmCaller<float> gemm_obj(parameters.size_matrices, parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
	}

	if (parameters.precision == "double") {
		UnhardenedGemmCaller<double> gemm_obj(parameters.size_matrices, parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
	}
}

void setup_gemm_cublas(Parameters &parameters) {
	if (parameters.precision == "half") {
		CUBLASGemmCaller<half> gemm_obj(parameters.size_matrices, parameters.size_matrices,
				parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);

	}

	if (parameters.precision == "float" || parameters.precision == "single") {
		CUBLASGemmCaller<float> gemm_obj(parameters.size_matrices, parameters.size_matrices,
				parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}

	if (parameters.precision == "double") {
		CUBLASGemmCaller<double> gemm_obj(parameters.size_matrices, parameters.size_matrices,
				parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}
}

void setup_gemm_cutlass(Parameters &parameters) {
	throw_line("CUTLASS GEMM not ready yet");
}

void setup_gemm_dmr(Parameters &parameters) {
	if (parameters.precision == "float" || parameters.precision == "single") {
		throw_line("Not ready yet");
	}

	if (parameters.precision == "double") {

		if (parameters.dmr == "mixed") {
			switch (parameters.check_block) {
			case ONE_OP_CHECK: {
				DMRMixedGemmCaller<ONE_OP_CHECK, float, double> gemm_obj(parameters.size_matrices,
						parameters.size_matrices);
				setup_execute(parameters, gemm_obj, THRESHOLD_1);
				break;

			}
			default: {
				//The counter will never be 32, so it will check only at the end
				DMRMixedGemmCaller<AT_END_OP_CHECK, float, double> gemm_obj(
						parameters.size_matrices, parameters.size_matrices);
				setup_execute(parameters, gemm_obj, THRESHOLD_AT_END);
				break;
			}
			}

		} else if (parameters.dmr == "full") {
			DMRGemmCaller<double> gemm_obj(parameters.size_matrices, parameters.size_matrices);
			setup_execute(parameters, gemm_obj);
		}
	}
}

