#include "hip/hip_runtime.h"
#include "setup.h"
#include "Parameters.h"
#include "include/device_vector.h"
#include "include/multi_compiler_analysis.h"
#include "common_template_functions.h"
#include "no_tensor_kernels.h"
#include "GemmCaller.h"

std::string get_multi_compiler_header() {
	std::string test_info = " nvcc_version:" + rad::get_cuda_cc_version();
	test_info += " nvcc_optimization_flags:"
			+ rad::extract_nvcc_opt_flags_str();
	return test_info;
}

void show_iteration_status(int it, bool verbose, double copy_time,
		double comparing_time, double computation_time,
		std::pair<int, int> errors) {
	if (verbose) {
		auto wasted_time = copy_time + comparing_time;
		auto full_time = wasted_time + computation_time;
		std::cout << "Iteration: " << it << " DMR errors " << errors.first
				<< ". " << "Radiation errors: " << errors.second << ". "
				<< "Time spent on computation: " << computation_time << "s. "
				<< "Time spent on comparing: " << comparing_time << "s. "
				<< "Time spent on copying: " << copy_time << "s. " << std::endl;
		std::cout << "Wasted time " << wasted_time << " ("
				<< int((wasted_time / full_time) * 100.0f) << "%)" << std::endl;
	} else {
//				std::cout << "Iteration: " << it << " DMR errors "
//						<< errors.first << ". " << "Radiation errors: "
//						<< errors.second << ". " << std::endl;
	}

}

template<const uint32_t COUNT, typename half_t, typename real_t>
void setup_execute(Parameters& parameters,
		GemmCaller<COUNT, half_t, real_t>& mult_env, const uint32_t threshold =
				0) {
	double elapsed_time = 0;

	std::vector<real_t> a_vector_host(
			parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> b_vector_host(
			parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> c_vector_host(
			parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> d_vector_host(
			parameters.size_matrices * parameters.size_matrices);
	std::vector<real_t> gold_host(
			parameters.size_matrices * parameters.size_matrices);

	//Output host vectors are set after computation
	std::vector<half_t> c_vector_host_half_t(
			parameters.size_matrices * parameters.size_matrices);
	;

	if (parameters.generate) {
		std::cout << "Generating input matrices\n";
		auto read_abc_files_on_generate = (parameters.check_input_existence
				&& exists(parameters.a_input_path)
				&& exists(parameters.b_input_path)
				&& exists(parameters.c_input_path));

		get_input_matrices(parameters.size_matrices, a_vector_host,
				b_vector_host, c_vector_host, parameters.a_input_path,
				parameters.b_input_path, parameters.c_input_path,
				read_abc_files_on_generate);
	} else {
		std::cout << "Reading input matrices\n";
		read_abc_files(parameters.a_input_path, a_vector_host,
				parameters.b_input_path, b_vector_host, parameters.c_input_path,
				c_vector_host);

		read_gold(parameters.gold_inout_path, gold_host);
	}

	//Alloc only after reading the inputs
	rad::DeviceVector<real_t> a_vector_device = a_vector_host;
	rad::DeviceVector<real_t> b_vector_device = b_vector_host;
	rad::DeviceVector<real_t> c_vector_device = c_vector_host;
	rad::DeviceVector<half_t> c_vector_device_half_t = c_vector_host_half_t;
	;

	std::cout << "Starting the setup process...\n";
	std::cout << std::setprecision(5) << std::fixed;
	for (int it = 0; it < parameters.iterations; it++) {
		c_vector_device = c_vector_host;

		auto computation_time = rad::mysecond();

		parameters.start_iteration();

		mult_env.gemm(a_vector_device, b_vector_device, c_vector_device,
				c_vector_device_half_t, real_t(parameters.alpha),
				real_t(parameters.beta), parameters.size_matrices,
				parameters.size_matrices, threshold);
		rad::checkFrameworkErrors(hipDeviceSynchronize());
		;
		rad::checkFrameworkErrors(hipPeekAtLastError());
		;

		parameters.end_iteration();
		computation_time = rad::mysecond() - computation_time;
		elapsed_time += computation_time;

		double copy_time = rad::mysecond();

		mult_env.memcpy_half_t_mem(c_vector_host_half_t,
				c_vector_device_half_t);

		c_vector_device.to_vector(d_vector_host);
		copy_time = rad::mysecond() - copy_time;

		if (!parameters.generate) {

			auto comparing_time = rad::mysecond();
			auto errors = check_output_errors_dmr(gold_host, d_vector_host,
					c_vector_host_half_t, parameters, threshold,
					mult_env.duplicated);

			comparing_time = rad::mysecond() - comparing_time;

			show_iteration_status(it, parameters.verbose, copy_time, comparing_time,
					computation_time, errors);
			//If errors != 0 reload matrices to gpu
			if (errors.first != 0 || errors.second != 0) {
				read_abc_files(parameters.a_input_path, a_vector_host,
						parameters.b_input_path, b_vector_host,
						parameters.c_input_path, c_vector_host);
				read_gold(parameters.gold_inout_path, gold_host);

				a_vector_device.resize(0);
				b_vector_device.resize(0);
				c_vector_device.resize(0);
				c_vector_device_half_t.resize(0);

				a_vector_device = a_vector_host;
				b_vector_device = b_vector_host;
				c_vector_device = c_vector_host;
				c_vector_device_half_t = c_vector_host_half_t;

			}

		}

	}
	if (parameters.verbose) {

		std::cout << "Elapsed time: " << (elapsed_time / parameters.iterations)
				<< " s\n";
	} else {
		std::cout << "done.\n";
	}

	if (parameters.generate) {
		auto zero_count = 0ul;
		auto nans_count = 0ul;
		for (auto s : d_vector_host) {
			zero_count += (float(s) == 0.0f);
			nans_count += (std::isnan(float(s)));
		}
		std::cout << "Zero values on gold: " << zero_count << std::endl;
		std::cout << "Nans values on gold: " << nans_count << std::endl;

		write_gold(parameters.gold_inout_path, d_vector_host);
	}
}

void setup_gemm_unhardened(Parameters& parameters) {
	if (parameters.precision == "half") {
#if __CUDA_ARCH__ >= 550
		UnhardenedGemmCaller<half> gemm_obj(parameters.size_matrices,
				parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
#else
		throw_line("Half MxM is not available for CUDA_ARCH<6.0");
#endif
	}
//
	if (parameters.precision == "float" || parameters.precision == "single") {
		UnhardenedGemmCaller<float> gemm_obj(parameters.size_matrices,
				parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
	}

	if (parameters.precision == "double") {
		UnhardenedGemmCaller<double> gemm_obj(parameters.size_matrices,
				parameters.size_matrices);
		setup_execute(parameters, gemm_obj);
	}
}

void setup_gemm_cublas(Parameters& parameters) {
	if (parameters.precision == "half") {
		CUBLASGemmCaller<half> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);

	}

	if (parameters.precision == "float" || parameters.precision == "single") {
		CUBLASGemmCaller<float> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}

	if (parameters.precision == "double") {
		CUBLASGemmCaller<double> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}
}

void setup_gemm_cutlass(Parameters& parameters) {
	if (parameters.precision == "half") {
		CUBLASGemmCaller<half> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);

	}

	if (parameters.precision == "float" || parameters.precision == "single") {
		CUBLASGemmCaller<float> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}

	if (parameters.precision == "double") {
		CUBLASGemmCaller<double> gemm_obj(parameters.size_matrices,
				parameters.size_matrices, parameters.use_tensor_cores);
		setup_execute(parameters, gemm_obj);
	}
}

void setup_gemm_dmr(Parameters& parameters) {
	if (parameters.precision == "float" || parameters.precision == "single") {
		throw_line("Not ready yet");
	}

	if (parameters.precision == "double") {

		if (parameters.dmr == "mixed") {
			switch (parameters.check_block) {
			case ONE_OP_CHECK: {
				DMRMixedGemmCaller<ONE_OP_CHECK, float, double> gemm_obj(
						parameters.size_matrices, parameters.size_matrices);
				setup_execute(parameters, gemm_obj, THRESHOLD_1);
				break;

			}
			default: {
				//The counter will never be 32, so it will check only at the end
				DMRMixedGemmCaller<AT_END_OP_CHECK, float, double> gemm_obj(
						parameters.size_matrices, parameters.size_matrices);
				setup_execute(parameters, gemm_obj, THRESHOLD_AT_END);
				break;
			}
			}

		} else if (parameters.dmr == "full") {
			DMRGemmCaller<double> gemm_obj(parameters.size_matrices,
					parameters.size_matrices);
			setup_execute(parameters, gemm_obj);
		}
	}
}

