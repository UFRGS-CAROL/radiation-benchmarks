#include "hip/hip_runtime.h"
/*********************************************************************************
 Implementing Breadth first search on CUDA using algorithm given in HiPC'07
 paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

 Copyright (c) 2008 International Institute of Information Technology - Hyderabad.
 All rights reserved.

 Permission to use, copy, modify and distribute this software and its documentation for
 educational purpose is hereby granted without fee, provided that the above copyright
 notice and this permission notice appear in all copies of this software and that you do
 not sell the software.

 THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR
 OTHERWISE.

 The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
 **********************************************************************************/

#include "kernel.h"


__global__ void Kernel(Node* g_graph_nodes, int* g_graph_edges,
		bool_t* g_graph_mask, bool_t* g_updating_graph_mask, bool_t *g_graph_visited,
		int* g_cost, int no_of_nodes) {
	int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < no_of_nodes && g_graph_mask[tid]) {
		g_graph_mask[tid] = FALSE;
		for (int i = g_graph_nodes[tid].starting;
				i
						< (g_graph_nodes[tid].no_of_edges
								+ g_graph_nodes[tid].starting); i++) {
			int id = g_graph_edges[i];
			if (!g_graph_visited[id]) {
				g_cost[id] = g_cost[tid] + 1;
				g_updating_graph_mask[id] = TRUE;
			}
		}
	}
}

__global__ void Kernel2(bool_t* g_graph_mask, bool_t *g_updating_graph_mask,
		bool_t* g_graph_visited, bool_t *g_over, int no_of_nodes) {
	int tid = blockIdx.x * MAX_THREADS_PER_BLOCK + threadIdx.x;
	if (tid < no_of_nodes && g_updating_graph_mask[tid]) {

		g_graph_mask[tid] = TRUE;
		g_graph_visited[tid] = TRUE;
		*g_over = TRUE;
		g_updating_graph_mask[tid] = FALSE;
	}
}

