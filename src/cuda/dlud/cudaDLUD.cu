#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

 // CAROL-RADIATION radiation benchmark implementation - <caio.b.lunardi at gmail.com> - 2018

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <sys/time.h>
#include <omp.h>

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#ifdef RD_WG_SIZE_0_0
#define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
#define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
#define BLOCK_SIZE RD_WG_SIZE
#else
#define BLOCK_SIZE 16
#endif

#ifdef LOGS
#include "log_helper.h"
#endif

#include "lud_kernel.cu"

#define DEFAULT_INPUT_SIZE 8192

int verbose = 0;
int fault_injection = 0;

int k=0; // k x k matrix size
int matrixSize=0; // = k * k matrix size
int iterations=100000000; // global loop iteracion
bool generate=false;

//================== Input paths
char *gold_matrix_path, *input_matrix_path;

FILE* f_INPUT;
FILE* f_B;
FILE* f_GOLD;
//====================================

//================== Host and device matrix ptr's
double *INPUT;
double *B;
double *GOLD;

double *d_INPUT;
double *d_OUTPUT;
//====================================

void GetDevice(){
//================== Retrieve and set the default CUDA device
    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);

    hipSetDevice(0);
       hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void allocCudaMemory()
{
//================== CUDA error handlers
	hipError_t malloc;
	const char *erro;
//====================================
	malloc = hipMalloc( ( void** ) &d_INPUT, matrixSize * sizeof( double ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		if (!generate) log_error_detail("error input"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc( ( void** ) &d_OUTPUT, matrixSize * sizeof( double ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		if (!generate) log_error_detail("error output"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void copyCudaMemory()
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================
	mcpy = hipMemset(d_OUTPUT, 0, matrixSize * sizeof (double));
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		if (!generate) log_error_detail("error gpu output load memset"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure

	mcpy = hipMemcpy( d_INPUT, INPUT, matrixSize * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		if (!generate) log_error_detail("error gpu load input"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void generateInputMatrix(double *m) {
	#pragma omp parallel for
	for (int i = 0; i < DEFAULT_INPUT_SIZE; i++)
		for (int j = 0; j < DEFAULT_INPUT_SIZE; j++)
			m[i * k + j] = (double) rand() / 32768.0;

	if (!(f_INPUT = fopen(input_matrix_path, "wb"))) {
		printf("Error: Could not open input file in wb mode. %s\n", input_matrix_path);
		exit(EXIT_FAILURE);
	} else {
		size_t ret_value = 0;
		for (int i = 0; i < DEFAULT_INPUT_SIZE; i++) {
			ret_value = fwrite(&(m[i * DEFAULT_INPUT_SIZE]), DEFAULT_INPUT_SIZE * sizeof(double), 1, f_INPUT);
			if (ret_value != 1) {
				printf("Failure writing to input: %d\n", ret_value);
				exit(EXIT_FAILURE);
			}
		}
		fclose(f_INPUT);
	}
}

void writeGoldToFile(double *m) {
	if (!(f_GOLD = fopen(gold_matrix_path, "wb"))) {
		printf("Error: Could not open gold file in wb mode. %s\n", gold_matrix_path);
		exit(EXIT_FAILURE);
	} else {
		size_t ret_value = 0;
		for (int i = 0; i < k; i++) {
			ret_value = fwrite(&(m[i * k]), k * sizeof(double), 1, f_GOLD);
			if (ret_value != 1) {
				printf("Failure writing to gold: %d\n", ret_value);
				exit(EXIT_FAILURE);
			}
		}
		fclose(f_GOLD);
	}
}

void ReadMatrixFromFile(){
//================== Read inputs to HOST memory
	int i;
	if (verbose) printf("Reading matrices... ");
	double time = mysecond();
	f_INPUT = fopen(input_matrix_path,"rb");
	if (f_INPUT) {
		// open input successful
    	size_t ret_value;
		for(i=0; i<k; i++)
		{
			ret_value = fread (&(INPUT[ k * i ]), sizeof(double)*k, 1, f_INPUT);
			if (ret_value != 1) {
				printf("Bad input formatting: %lu .\n", ret_value);
				#ifdef LOGS
					log_error_detail("Bad input formatting."); end_log_file();
				#endif
				exit(EXIT_FAILURE);
			}
		}
		fclose(f_INPUT);
	} else if (generate) {
		generateInputMatrix(INPUT);
	} else {
		printf ("Cant open matrices and -generate is false.\n");
		if (generate) {
			generateInputMatrix(INPUT);
		} else {
#ifdef LOGS
			log_error_detail("Cant open matrices"); end_log_file();
#endif
			exit(EXIT_FAILURE);
		}
	}

	if (!generate) {
    	size_t ret_value;
		f_GOLD = fopen(gold_matrix_path,"rb");
		for(i=0; i<k; i++)
		{
			ret_value = fread (&(GOLD[ k * i ]), sizeof(double)*k, 1, f_GOLD);
			if (ret_value != 1) {
				printf("Bad gold formatting: %lu .\n", ret_value);
				#ifdef LOGS
					log_error_detail("Bad gold formatting."); end_log_file();
				#endif
				exit(EXIT_FAILURE);
			}
		}
		fclose(f_GOLD);
	}
	if (verbose) printf("Done reading matrices in %.2fs\n", mysecond() - time);

	if (fault_injection)
	{
		INPUT[3] = (double)6.5;
		printf("!! Injected 6.5 on position INPUT[3]\n");
	}
}

bool badass_memcmp(double *gold, double *found, unsigned long n){
	double result = 0.0;
	int i;
	unsigned long  chunk = ceil(float(n) / float(omp_get_max_threads()));
	// printf("size %d max threads %d chunk %d\n", n, omp_get_max_threads(), chunk);
	double time = mysecond();
#pragma omp parallel for default(shared) private(i) schedule(static,chunk) reduction(+:result)
   for (i=0; i < n; i++)
     result = result + (gold[i] - found[i]);

    //  printf("comparing took %lf seconds, diff %lf\n", mysecond() - time, result);
	if (fabs(result) > 0.0000000001)
		return true;
	return false;
}

void usage() {
    printf("Usage: dlud -size=N [-generate] [-input=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup]\n");
}

int main( int argc, char* argv[] )
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================

//================== Test vars
	int i, j, loop2;
	// int kernel_errors=0;
	// int zero = 0;
	double time;
	double kernel_time, global_time;
    double total_kernel_time, min_kernel_time, max_kernel_time;
	int device_warmup = 1;
    // int gpu_check = 1;
//====================================

//================== Read test parameters
	if (argc<2) {
		usage();
		exit (-1);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        k = getCmdLineArgumentInt(argc, (const char **)argv, "size");

        if ((k <= 0)||(k % 16 != 0))
        {
            printf("Invalid input size given on the command-line: %d\n", k);
            exit(EXIT_FAILURE);
		}
		matrixSize = k * k;
    }
	else
	{
		usage();
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "input"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_a", &input_matrix_path);
    }
    else
    {
        input_matrix_path = new char[100];
        snprintf(input_matrix_path, 100, "dlud_input_%i.matrix", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input path: %s\n", input_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "gold"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold", &gold_matrix_path);
    }
    else
    {
        gold_matrix_path = new char[100];
        snprintf(gold_matrix_path, 100, "dlud_gold_%i.matrix", (signed int)k);
        printf("Using default gold path: %s\n", gold_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "iterations"))
    {
        iterations = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        verbose = 1;
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "debug"))
    {
		fault_injection = 1;
        printf("!! Will be injected an input error\n");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "no-warmup"))
    {
		device_warmup = 0;
        printf("!! The first iteration may not reflect real timing information\n");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "generate"))
    {
		generate = 1;
		device_warmup = 0;
		iterations = 1;
		printf("Will generate input if needed and GOLD.\nIterations setted to 1. no-warmup setted to false.\n");
    } else {
		generate = 0;
	}

	// if (checkCmdLineFlag(argc, (const char **)argv, "no-gpu-gold-check"))
    // {
	// 	gpu_check = 0;
    // } else {
    //     printf("!! The gold check will happen on the GPU and fall back to CPU in case of errors\n");
    // }
//====================================

//================== Init logs
#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "size:%d type:double-precision", k);
	if (!generate) start_log_file("cudaDLUD", test_info);
#endif
//====================================

//================== Alloc HOST memory
	INPUT = ( double* ) malloc( matrixSize * sizeof( double ) );

	GOLD = ( double* ) malloc( matrixSize * sizeof( double ) );

	if (!(INPUT && GOLD)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
    total_kernel_time = 0;
    min_kernel_time = UINT_MAX;
    max_kernel_time = 0;
	GetDevice();
	ReadMatrixFromFile();
	printf( "cudaDLUD\n" );
	fflush(stdout);
//====================================

//================== Init DEVICE memory
	allocCudaMemory();
	copyCudaMemory();
//====================================


	for(loop2=0; loop2<iterations; loop2++)
	{//================== Global test loop

		if (!loop2 && device_warmup) printf("First iteration: device warmup. Please wait...\n");

		// Timer...
		global_time = mysecond();

		hipMemset(d_OUTPUT, 0, matrixSize * sizeof (double));

		if (verbose) printf(",");

		kernel_time = mysecond();
		#ifdef LOGS
		if (loop2 || !device_warmup)
			if (!generate) start_iteration();
		#endif
		//================== Device computation, HMxM
		lud_cuda(d_INPUT, k);

		checkCudaErrors( hipPeekAtLastError() );
		
		checkCudaErrors( hipDeviceSynchronize() );
		checkCudaErrors( hipPeekAtLastError() );
		//====================================
		#ifdef LOGS
		if (loop2 || !device_warmup)
			if (!generate) end_iteration();
		#endif
		kernel_time = mysecond() - kernel_time;
      
		if (loop2 || !device_warmup) {
		  total_kernel_time += kernel_time;
		  min_kernel_time = min(min_kernel_time, kernel_time);
		  max_kernel_time = max(max_kernel_time, kernel_time);
		}

		if (loop2 || !device_warmup)
			if (verbose) printf("Device kernel time for iteration %d: %.3fs\n", loop2, kernel_time);

    	if (verbose) printf(",");

        // Timer...
        time = mysecond();

        //if (kernel_errors != 0) {
        checkCudaErrors( hipMemcpy(INPUT, d_OUTPUT, matrixSize * sizeof( double ), hipMemcpyDeviceToHost) );
		if (generate) {
			writeGoldToFile(INPUT);
		} else if (loop2 || !device_warmup) {
            //~ if (memcmp(A, GOLD, sizeof(double) * k*k)) {
            if (badass_memcmp(GOLD, INPUT, matrixSize)) {
    			char error_detail[150];
    			int host_errors = 0;

                printf("!");

    			#pragma omp parallel for
    			for(i=0; (i<k); i++)
    			{
    				for(j=0; (j<k); j++)
    				{
    					if (INPUT[i + k * j] != GOLD[i + k * j])
    					#pragma omp critical
    					{
    						snprintf(error_detail, 150, "p: [%d, %d], r: %1.16e, e: %1.16e", i, j, (float)(INPUT[i + k * j]), (float)(GOLD[i + k * j]));
    						if (verbose && (host_errors < 10)) printf("%s\n", error_detail);
    						#ifdef LOGS
								if (!generate) log_error_detail(error_detail);
    						#endif
    						host_errors++;
    						//ea++;
    						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + k * j], GOLD[i + k * j], t_ea);

    					}
    				}
    			}

                // printf("numErrors:%d", host_errors);

    			#ifdef LOGS
					if (!generate) log_error_count(host_errors);
    			#endif
    			//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
    			hipFree( d_INPUT );
    			hipFree( d_OUTPUT );
    			//====================================
    			ReadMatrixFromFile();
    			//================== Init DEVICE memory
    			allocCudaMemory();
    			copyCudaMemory();
    			//====================================
    		}
        }

		//====================================

		//================== Console hearthbeat
		/*if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("test number: %d\n", loop2);
			printf(" kernel time: %f\n", kernel_time);
		}
		else
		{*/
			printf(".");
			fflush(stdout);
		//}
		//====================================

		if (loop2 || !device_warmup)
			if (verbose) printf("Gold check time for iteration %d: %.3fs\n", loop2, mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose)
			{
				/////////// PERF
				double outputpersec = (double)matrixSize/kernel_time;
				printf("SIZE:%d OUTPUT/S:%f\n",k, outputpersec);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose) printf("Iteration #%d time: %.3fs\n\n\n", loop2, mysecond() - global_time);
		fflush(stdout);
	}

    double averageKernelTime = total_kernel_time / (iterations - (device_warmup ? 1 : 0));
    printf("\n-- END --\n"
    "Total kernel time: %.3fs\n"
    "Iterations: %d\n"
    "Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n", 
    total_kernel_time, 
    iterations, 
    averageKernelTime, min_kernel_time, max_kernel_time);

	//================== Release device memory
	hipFree( d_INPUT );
	hipFree( d_OUTPUT );
	//====================================

	free( INPUT );
	free( GOLD );
	#ifdef LOGS
		if (!generate) end_log_file();
	#endif

	return 0;
}