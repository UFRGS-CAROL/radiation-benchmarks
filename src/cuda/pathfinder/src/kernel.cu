#include "hip/hip_runtime.h"

#include "common.h"

__global__ void dynproc_kernel(int iteration, int *gpuWall, int *gpuSrc,
		int *gpuResults, int cols, int rows, int startStep, int border) {

	__shared__ int prev[BLOCK_SIZE];
	__shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx = threadIdx.x;

	// each block finally computes result for a small block
	// after N iterations.
	// it is the non-overlapping small blocks that cover
	// all the input data

	// calculate the small block size
	int small_block_cols = BLOCK_SIZE - iteration * HALO * 2;

	// calculate the boundary for the block according to
	// the boundary of its small block
	int blkX = small_block_cols * bx - border;
	int blkXmax = blkX + BLOCK_SIZE - 1;

	// calculate the global thread coordination
	int xidx = blkX + tx;

	// effective range within this block that falls within
	// the valid range of the input data
	// used to rule out computation outside the boundary.
	int validXmin = (blkX < 0) ? -blkX : 0;
	int validXmax = (blkXmax > cols - 1) ?
	BLOCK_SIZE - 1 - (blkXmax - cols + 1) :
											BLOCK_SIZE - 1;

	int W = tx - 1;
	int E = tx + 1;

	W = (W < validXmin) ? validXmin : W;
	E = (E > validXmax) ? validXmax : E;

	bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if (IN_RANGE(xidx, 0, cols - 1)) {
		prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
	bool computed;
	for (int i = 0; i < iteration; i++) {
		computed = false;
		if ( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) && isValid) {
			computed = true;
			int left = prev[W];
			int up = prev[tx];
			int right = prev[E];
			int shortest = MIN(left, up);
			shortest = MIN(shortest, right);
			int index = cols * (startStep + i) + xidx;
			result[tx] = shortest + gpuWall[index];

		}
		__syncthreads();
		if (i == iteration - 1)
			break;
		if (computed)	 //Assign the computation range
			prev[tx] = result[tx];
		__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
	}

	// update the global memory
	// after the last iteration, only threads coordinated within the
	// small block perform the calculation and switch on ``computed''
	if (computed) {
		gpuResults[xidx] = result[tx];
	}
}

/*
 compute N time steps
 */
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols,
		int pyramid_height, int blockCols, int borderCols) {
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(blockCols);

	int src = 1, dst = 0;
	for (int t = 0; t < rows - 1; t += pyramid_height) {
		int temp = src;
		src = dst;
		dst = temp;
		dynproc_kernel<<<dimGrid, dimBlock>>>(MIN(pyramid_height, rows - t - 1),
				gpuWall, gpuResult[src], gpuResult[dst], cols, rows, t,
				borderCols);
	}
	return dst;
}
