#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#include <vector>

#include "common.h"
#include "Parameters.h"
#include "device_vector.h"

template<typename T>
using matrix = std::vector<std::vector<T>>;

template<typename T>
using vector = std::vector<T>;

extern int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols,
		int pyramid_height, int blockCols, int borderCols);

void init(vector<int*>& wall, vector<int>& data, vector<int>& result,
		int pyramid_height, int rows, int cols) {

//	data = new int[rows * cols];
	data.resize(rows * cols);
	wall.resize(rows);

	for (int n = 0; n < rows; n++) {
		wall[n] = data.data() + cols * n;
	}

//	result = new int[cols];
	result.resize(cols);

	int seed = M_SEED;
	srand(seed);
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			wall[i][j] = rand() % 10;
		}
	}

#ifdef BENCH_PRINT

	for (int i = 0; i < rows; i++) {

		for (int j = 0; j < cols; j++) {
			printf("%d ", wall[i][j]);
		}
		printf("\n");
	}

#endif
}

void run(int argc, char** argv) {
	int pyramid_height, cols, rows;
	if (argc == 4) {
		cols = atoi(argv[1]);
		rows = atoi(argv[2]);
		pyramid_height = atoi(argv[3]);
	} else {
		throw_line("Usage: dynproc row_len col_len pyramid_height\n");
	}

	vector<int*> wall;
	vector<int> data;
	vector<int> result;

	init(wall, data, result, pyramid_height, rows, cols);

	/* --------------- pyramid parameters --------------- */
	int borderCols = (pyramid_height) * HALO;
	int smallBlockCol = BLOCK_SIZE - (pyramid_height) * HALO * 2;
	int blockCols = cols / smallBlockCol
			+ ((cols % smallBlockCol == 0) ? 0 : 1);

	printf(
			"pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",
			pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols,
			smallBlockCol);

	//int *gpuWall, *gpuResult[2];
	rad::DeviceVector<int> gpuWall;
	rad::DeviceVector<int> gpuResult[2];

	int size = rows * cols;
//	hipMalloc((void**) &gpuResult[0], sizeof(int) * cols);
//	hipMalloc((void**) &gpuResult[1], sizeof(int) * cols);
	gpuResult[0].resize(cols);
	gpuResult[1].resize(cols);

//	hipMemcpy(gpuResult[0], data, sizeof(int) * cols, hipMemcpyHostToDevice);
	gpuResult[0].fill_n(data.begin(), cols);

//	hipMalloc((void**) &gpuWall, sizeof(int) * (size - cols));
	gpuWall.resize(size - cols);
//	hipMemcpy(gpuWall, data + cols, sizeof(int) * (size - cols),
//			hipMemcpyHostToDevice);

	gpuWall.fill_n(data.begin() + cols, (size - cols));

	int *gpuResult_ptr[2] = { gpuResult[0].data(), gpuResult[1].data() };

	int final_ret = calc_path(gpuWall.data(), gpuResult_ptr, rows, cols,
			pyramid_height, blockCols, borderCols);

//	hipMemcpy(result, gpuResult[final_ret], sizeof(int) * cols,
//			hipMemcpyDeviceToHost);
	gpuResult[final_ret].to_vector(result);

#ifdef BENCH_PRINT

	for (int i = 0; i < cols; i++)

		printf("%d ", data[i]);

	printf("\n");

	for (int i = 0; i < cols; i++)

		printf("%d ", result[i]);

	printf("\n");

#endif

//	hipFree(gpuWall);
//	hipFree(gpuResult[0]);
//	hipFree(gpuResult[1]);
//
//	delete[] data;
//	delete[] wall;
//	delete[] result;

}

