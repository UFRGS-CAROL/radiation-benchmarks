#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#include <vector>

#include "common.h"
#include "Parameters.h"
#include "device_vector.h"

template<typename T>
using matrix = std::vector<std::vector<T>>;

template<typename T>
using vector = std::vector<T>;

extern int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols,
		int pyramid_height, int blockCols, int borderCols);

void init(vector<int*>& wall, vector<int>& data, vector<int>& result,
		int pyramid_height, int rows, int cols) {

//	data = new int[rows * cols];
	data.resize(rows * cols);
	wall.resize(rows);

	for (int n = 0; n < rows; n++) {
		wall[n] = data.data() + cols * n;
	}

//	result = new int[cols];
	result.resize(cols);

	int seed = M_SEED;
	srand(seed);
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			wall[i][j] = rand() % 10;
		}
	}

#ifdef BENCH_PRINT

	for (int i = 0; i < rows; i++) {

		for (int j = 0; j < cols; j++) {
			printf("%d ", wall[i][j]);
		}
		printf("\n");
	}

#endif
}

void run(int argc, char** argv) {
	Parameters parameters(argc, argv);

	vector<int*> wall;
	vector<int> data;
	vector<int> result;

	init(wall, data, result, parameters.pyramid_height, parameters.rows, parameters.cols);

	/* --------------- pyramid parameters --------------- */
	int borderCols = (parameters.pyramid_height) * HALO;
	int smallBlockCol = BLOCK_SIZE - (parameters.pyramid_height) * HALO * 2;
	int blockCols = parameters.cols / smallBlockCol
			+ ((parameters.cols % smallBlockCol == 0) ? 0 : 1);

	printf(
			"pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",
			parameters.pyramid_height, parameters.cols, borderCols, BLOCK_SIZE, blockCols,
			smallBlockCol);

	//int *gpuWall, *gpuResult[2];
	rad::DeviceVector<int> gpuWall;
	rad::DeviceVector<int> gpuResult[2];

	int size = parameters.rows * parameters.cols;
//	hipMalloc((void**) &gpuResult[0], sizeof(int) * cols);
//	hipMalloc((void**) &gpuResult[1], sizeof(int) * cols);
	gpuResult[0].resize(parameters.cols);
	gpuResult[1].resize(parameters.cols);

//	hipMemcpy(gpuResult[0], data, sizeof(int) * cols, hipMemcpyHostToDevice);
	gpuResult[0].fill_n(data.begin(), parameters.cols);

//	hipMalloc((void**) &gpuWall, sizeof(int) * (size - cols));
	gpuWall.resize(size - parameters.cols);
//	hipMemcpy(gpuWall, data + cols, sizeof(int) * (size - cols),
//			hipMemcpyHostToDevice);

	gpuWall.fill_n(data.begin() + parameters.cols, (size - parameters.cols));

	int *gpuResult_ptr[2] = { gpuResult[0].data(), gpuResult[1].data() };

	int final_ret = calc_path(gpuWall.data(), gpuResult_ptr, parameters.rows, parameters.cols,
			parameters.pyramid_height, blockCols, borderCols);

//	hipMemcpy(result, gpuResult[final_ret], sizeof(int) * cols,
//			hipMemcpyDeviceToHost);
	gpuResult[final_ret].to_vector(result);

#ifdef BENCH_PRINT

	for (int i = 0; i < parameters.cols; i++)

		printf("%d ", data[i]);

	printf("\n");

	for (int i = 0; i < parameters.cols; i++)

		printf("%d ", result[i]);

	printf("\n");

#endif

//	hipFree(gpuWall);
//	hipFree(gpuResult[0]);
//	hipFree(gpuResult[1]);
//
//	delete[] data;
//	delete[] wall;
//	delete[] result;

}

