#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>
#include <omp.h>
#include <random>
#include <hip/hip_fp16.h>

#ifdef LOGS
#include "log_helper.h"
#endif

#include "include/persistent_lib.h"
// The timestamp is updated on every log_helper function call.

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#include "half.hpp"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

#define DEFAULT_INPUT_SIZE 8192

//=========== DEFINE TESTED TYPE
#if defined(test_precision_double)

#define GENERATOR_MAXABSVALUE 4.1e+16
#define GENERATOR_MINABSVALUE 0
const char test_precision_description[] = "double";
typedef double tested_type;
typedef double tested_type_host;

#elif defined(test_precision_single)

#define GENERATOR_MAXABSVALUE 4.1e+2
#define GENERATOR_MINABSVALUE 0
const char test_precision_description[] = "single";
typedef float tested_type;
typedef float tested_type_host;

#elif defined(test_precision_half)

#define GENERATOR_MAXABSVALUE 2.0
#define GENERATOR_MINABSVALUE 0
const char test_precision_description[] = "half";
typedef half tested_type;
typedef half_float::half tested_type_host;

#else 
#error TEST TYPE NOT DEFINED OR INCORRECT. USE TYPE=<double|single|half>.
#endif

//====================== benchmark+setup configuration
int generate = 0;
int verbose = 0;
int fault_injection = 0;

//unsigned long long int host_is_memory_bad = 0;

int k = 0; // k x k matrix size
int matrixSize = 0; // = k * k matrix size
int iterations = 100000000; // global loop iteration
//=========================

//======== generator configuration
int generate_safechecks = 0;
bool generate_inputmatricesready = false;
bool host_check = false;
bool generator_debug = false;
//=========================

//================== Input paths
char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

FILE* f_A;
FILE* f_B;
FILE* f_GOLD;
//====================================

//================== Host and device matrix ptr's
tested_type_host *A;
tested_type_host *B;
tested_type_host *C0; //, *C1, *C2;
tested_type_host *GOLD;

tested_type *d_A0; //, *d_A1, *d_A2;
tested_type *d_B0; //, *d_B1, *d_B2;
tested_type *d_C0; //, *d_C1, *d_C2;
//====================================

void GetDevice() {
//================== Retrieve and set the default CUDA device
	hipDeviceProp_t prop;
	int count = 0;
	printf("Get device:");
	rad::checkFrameworkErrors(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		rad::checkFrameworkErrors(hipGetDeviceProperties(&prop, i));
		printf("Name: %s\n", prop.name);
	}
	int *ndevice;
	int dev = 0;
	ndevice = &dev;
	rad::checkFrameworkErrors(hipGetDevice(ndevice));

	rad::checkFrameworkErrors(hipSetDevice(0));
	rad::checkFrameworkErrors(hipGetDeviceProperties(&prop, 0));
	printf("\ndevice: %d %s\n", *ndevice, prop.name);
}

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void* safe_cudaMalloc(size_t size) {
	void* devicePtr;
	void* goldPtr;
	void* outputPtr;

	// First, alloc DEVICE proposed memory and HOST memory for device memory checking
	rad::checkFrameworkErrors(hipMalloc(&devicePtr, size));
	outputPtr = malloc(size);
	goldPtr = malloc(size);
	if ((outputPtr == NULL) || (goldPtr == NULL)) {
		log_error_detail((char *) "error host malloc");
		end_log_file();
		printf("error host malloc\n");
		exit(EXIT_FAILURE);
	}

	// ===> FIRST PHASE: CHECK SETTING BITS TO 10101010
	rad::checkFrameworkErrors(hipMemset(devicePtr, 0xAA, size));
	memset(goldPtr, 0xAA, size);

	rad::checkFrameworkErrors(
			hipMemcpy(outputPtr, devicePtr, size, hipMemcpyDeviceToHost));
	if (memcmp(outputPtr, goldPtr, size)) {
		// Failed
		free(outputPtr);
		free(goldPtr);
		void* newDevicePtr = safe_cudaMalloc(size);
		rad::checkFrameworkErrors(hipFree(devicePtr));
		return newDevicePtr;
	}
	// ===> END FIRST PHASE

	// ===> SECOND PHASE: CHECK SETTING BITS TO 01010101
	rad::checkFrameworkErrors(hipMemset(devicePtr, 0x55, size));
	memset(goldPtr, 0x55, size);

	rad::checkFrameworkErrors(
			hipMemcpy(outputPtr, devicePtr, size, hipMemcpyDeviceToHost));
	if (memcmp(outputPtr, goldPtr, size)) {
		// Failed
		free(outputPtr);
		free(goldPtr);
		void* newDevicePtr = safe_cudaMalloc(size);
		rad::checkFrameworkErrors(hipFree(devicePtr));
		return newDevicePtr;
	}
	// ===> END SECOND PHASE

	free(outputPtr);
	free(goldPtr);
	return devicePtr;
}

void allocCudaMemory() {

#ifdef SAFE_MALLOC
	d_A0 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_A1 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_A2 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));

	d_B0 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_B1 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_B2 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));

	d_C0 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_C1 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
//	d_C2 = (tested_type*) safe_cudaMalloc(matrixSize * sizeof(tested_type));
#else
	rad::checkFrameworkErrors(hipMalloc(&d_A0, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_A1, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_A2, matrixSize * sizeof(tested_type)));

	rad::checkFrameworkErrors(hipMalloc(&d_B0, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_B1, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_B2, matrixSize * sizeof(tested_type)));

	rad::checkFrameworkErrors(hipMalloc(&d_C0, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_C1, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(hipMalloc(&d_C2, matrixSize * sizeof(tested_type)));
#endif

}

void freeCudaMemory() {
	rad::checkFrameworkErrors(hipFree(d_A0));
//	rad::checkFrameworkErrors(hipFree(d_A1));
//	rad::checkFrameworkErrors(hipFree(d_A2));

	rad::checkFrameworkErrors(hipFree(d_B0));
//	rad::checkFrameworkErrors(hipFree(d_B1));
//	rad::checkFrameworkErrors(hipFree(d_B2));

	rad::checkFrameworkErrors(hipFree(d_C0));
//	rad::checkFrameworkErrors(hipFree(d_C1));
//	rad::checkFrameworkErrors(hipFree(d_C2));
}

void copyCudaMemory(rad::HostPersistentControler& pt_control) {
	rad::checkFrameworkErrors(
			hipMemsetAsync(d_C0, 0x00, matrixSize * sizeof(tested_type),
					pt_control.st));
//	rad::checkFrameworkErrors(
//			hipMemset(d_C1, 0x00, matrixSize * sizeof(tested_type)));
//	rad::checkFrameworkErrors(
//			hipMemset(d_C2, 0x00, matrixSize * sizeof(tested_type)));
	pt_control.sync_stream();

	rad::checkFrameworkErrors(
			hipMemcpyAsync(d_A0, A, matrixSize * sizeof(tested_type),
					hipMemcpyHostToDevice, pt_control.st)); // PUSH A
//	rad::checkFrameworkErrors(
//			hipMemcpy(d_A1, A, matrixSize * sizeof(tested_type),
//					hipMemcpyHostToDevice)); // PUSH A
//	rad::checkFrameworkErrors(
//			hipMemcpy(d_A2, A, matrixSize * sizeof(tested_type),
//					hipMemcpyHostToDevice)); // PUSH A
	pt_control.sync_stream();

	rad::checkFrameworkErrors(
			hipMemcpyAsync(d_B0, B, matrixSize * sizeof(tested_type),
					hipMemcpyHostToDevice, pt_control.st)); // PUSH B
//	rad::checkFrameworkErrors(
//			hipMemcpy(d_B1, B, matrixSize * sizeof(tested_type),
//					hipMemcpyHostToDevice)); // PUSH B
//	rad::checkFrameworkErrors(
//			hipMemcpy(d_B2, B, matrixSize * sizeof(tested_type),
//					hipMemcpyHostToDevice)); // PUSH B
	pt_control.sync_stream();
}

void readMatricesFromFile(bool gold = true) {
	int i;
	f_A = fopen(a_matrix_path, "rb");
	f_B = fopen(b_matrix_path, "rb");
	if (!(f_A && f_B)) {
		printf("Cant open input  matrices.\n");
#ifdef LOGS
		if (!generate)
		log_error_detail((char *)"Cant open input matrices"); end_log_file();
#endif
		exit(-3);
	}
	if (gold) {
		if (!(f_GOLD = fopen(gold_matrix_path, "rb"))) {
			printf("Cant open gold matrice.\n");
#ifdef LOGS
			if (!generate)
			log_error_detail((char *)"Cant open gold matrice"); end_log_file();
#endif
			exit(-3);
		}
	}

	size_t ret_value[3];
	for (i = 0; i < k; i++) {
		ret_value[0] = fread(&(A[k * i]), sizeof(tested_type) * k, 1, f_A);
		ret_value[1] = fread(&(B[k * i]), sizeof(tested_type) * k, 1, f_B);
		if (gold) {
			ret_value[2] = fread(&(GOLD[k * i]), sizeof(tested_type) * k, 1,
					f_GOLD);
		}
		if ((ret_value[0] != 1) || (ret_value[1] != 1)
				|| (gold && (ret_value[2] != 1))) {
			printf("Bad input/gold formatting: %lu ; %lu ; %lu .\n",
					ret_value[0], ret_value[1], ret_value[2]);
#ifdef LOGS
			if (!generate)
			log_error_detail((char *)"Bad input/gold formatting."); end_log_file();
#endif
			exit(-3);
		}
	}

	fclose(f_A);
	fclose(f_B);
	if (gold)
		fclose(f_GOLD);
}

void generateInputMatrices() {
	FILE * f_A, *f_B;
	tested_type_host *h_A, *h_B;

	if (k == DEFAULT_INPUT_SIZE) {
		h_A = A;
		h_B = B;
	} else {
		h_A = (tested_type_host*) malloc(
		DEFAULT_INPUT_SIZE * DEFAULT_INPUT_SIZE * sizeof(tested_type));
		h_B = (tested_type_host*) malloc(
		DEFAULT_INPUT_SIZE * DEFAULT_INPUT_SIZE * sizeof(tested_type));
		if (!(h_A && h_B)) {
			printf("Could not alloc h_A or h_B");
			exit(EXIT_FAILURE);
		}
	}

	std::random_device rd; //Will be used to obtain a seed for the random number engine
	std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
	std::uniform_real_distribution<double> dis(-GENERATOR_MAXABSVALUE,
			GENERATOR_MAXABSVALUE);

	if (!generator_debug) {
		for (int i = 0; i < DEFAULT_INPUT_SIZE; i++) {
			for (int j = 0; j < DEFAULT_INPUT_SIZE; j++) {
				h_A[i * DEFAULT_INPUT_SIZE + j] = (tested_type_host) dis(gen);
				h_B[i * DEFAULT_INPUT_SIZE + j] = (tested_type_host) dis(gen);
			}
		}
	} else {
		for (int i = 0; i < DEFAULT_INPUT_SIZE; i++) {
			for (int j = 0; j < DEFAULT_INPUT_SIZE; j++) {
				h_A[i * DEFAULT_INPUT_SIZE + j] = (tested_type_host) 2.0;
				h_B[i * DEFAULT_INPUT_SIZE + j] = (tested_type_host) 2.0;
			}
		}
	}

	if (h_A != A) {
		memcpy(A, h_A, matrixSize * sizeof(tested_type));
		memcpy(B, h_B, matrixSize * sizeof(tested_type));
	}

	int numZeros;
	int numNans;
	int numInfs;
// printf("Write\n");
	f_A = fopen(a_matrix_path, "wb");
	f_B = fopen(b_matrix_path, "wb");
	if (!(f_A && f_B)) {
		printf("Could not open f_A or f_B\n");
		exit(EXIT_FAILURE);
	}

	tested_type_host val;

	numZeros = 0;
	numNans = 0;
	numInfs = 0;
	for (int i = 0; i < DEFAULT_INPUT_SIZE * DEFAULT_INPUT_SIZE; i++) {
		val = h_A[i];
		if (val == 0)
			numZeros++;
		if (isnan(val))
			numNans++;
		if (isinf(val))
			numInfs++;
	}
	printf("Number of zeros/NaNs/INFs on matrix A: %d/%d/%d\n", numZeros,
			numNans, numInfs);

	numZeros = 0;
	numNans = 0;
	numInfs = 0;
	for (int i = 0; i < DEFAULT_INPUT_SIZE * DEFAULT_INPUT_SIZE; i++) {
		val = h_B[i];
		if (val == 0)
			numZeros++;
		if (isnan(val))
			numNans++;
		if (isinf(val))
			numInfs++;
	}
	printf("Number of zeros/NaNs/INFs on matrix B: %d/%d/%d\n", numZeros,
			numNans, numInfs);

	for (int i = 0; i < DEFAULT_INPUT_SIZE; i++) {
		fwrite(&(h_A[i * DEFAULT_INPUT_SIZE]),
				sizeof(tested_type) * DEFAULT_INPUT_SIZE, 1, f_A);
	}

	printf("Element 32 of matrix A: %f\n", (double) A[32]);

	printf("Element 50 of matrix B: %f\n", (double) B[50]);

	for (int i = 0; i < DEFAULT_INPUT_SIZE; i++) {
		fwrite(&(h_B[i * DEFAULT_INPUT_SIZE]),
				sizeof(tested_type_host) * DEFAULT_INPUT_SIZE, 1, f_B);
	}
	printf("Done\n");

	fclose(f_A);
	fclose(f_B);
	if (h_A != A) {
		free(h_A);
		free(h_B);
	}
	return;
}

void retrieveInputMatrices() {
//================== Read inputs to HOST memory
	double time = mysecond();

	if (verbose)
		printf("Preparing input matrices... ");

	FILE *f_A = fopen(a_matrix_path, "rb");
	FILE *f_B = fopen(b_matrix_path, "rb");
	if (generate && (!f_A || !f_B)) {
		if (f_A)
			fclose(f_A);
		if (f_B)
			fclose(f_B);
		generateInputMatrices();
	} else {
		if (f_A)
			fclose(f_A);
		if (f_B)
			fclose(f_B);
		readMatricesFromFile(!generate);
	}

	if ((generate) && (generator_debug) && (k <= 16)) {
		printf("\nMatrix A: \n");
		for (int i = 0; i < k * k; i++) {
			printf(" %.2e", (float) A[i]);
			if ((i + 1) % k == 0)
				printf("\n");
		}
		printf("\nMatrix B: \n");
		for (int i = 0; i < k * k; i++) {
			printf(" %.2e", (float) B[i]);
			if ((i + 1) % k == 0)
				printf("\n");
		}
	}

	if (fault_injection) {
		A[3] = (tested_type_host) 1.666;
		printf("!! Injected 1.666 on position A[3]\n");
	}

	if (verbose)
		printf("Done reading matrices in %.2fs\n", mysecond() - time);
}

void writeGoldtoFile() {
	int i;
	f_GOLD = fopen(gold_matrix_path, "wb");
	if (!f_GOLD) {
		printf("Could not open f_GOLD\n");
		exit(EXIT_FAILURE);
	}

	for (i = 0; i < k; i++) {
		fwrite(&(GOLD[i * k]), sizeof(tested_type) * k, 1, f_GOLD);
	}

	fclose(f_GOLD);
}

template<typename real>
__device__ void process_data(int wA, int wB, real* A, real* B, real* C) {
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;
	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;
	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;
	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;
	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;
	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	real Csub = 0;
	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ real As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ real Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}
	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template<typename real> __global__ void MatrixMulKernel(real *C, real *A,
		real *B, int wA, int wB) {
	rad::PersistentKernel pk;
	while (pk.keep_working()) {
		pk.wait_for_work();
		if (pk.is_able_to_process()) {
			process_data(wA, wB, A, B, C);
			pk.iteration_finished();
		}
	}
}

//__global__ void MatrixMulKernel(half *d_A0, half *d_B0, half *d_C0, int wA,
//		int wB) {
//	register int tx = (blockIdx.x * BLOCK_SIZE) / 2.0 + threadIdx.x;
//	register int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
//	register int k;
//
//	register half2 acc = __float2half2_rn(0.0);
//	for (k = 0; k < wA; k++) {
//
//		acc = __hfma2(__half2half2(d_A0[ty * wA + k]),
//				__half2half2(d_B0[k * (wA / 2) + tx]), acc);
//		// n/2 is needed because we changed how we iterate d_B
//	}
//
//	((half2*) d_C0)[ty * (wA / 2) + tx] = acc;
//}

void usage(int argc, char* argv[]) {
	printf(
			"Usage: %s -size=N [-generate] [-input_a=<path>] [-input_b=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup]\n",
			argv[0]);
}

// Returns true if no errors are found. False if otherwise.
// Set votedOutput pointer to retrieve the voted matrix
bool checkOutputErrors(tested_type_host* votedOutput = NULL,
		bool check = true) {
	int host_errors = 0;
//	int memory_errors = 0;

//	if (host_is_memory_bad != 0) {
//		char info_detail[150];
//		snprintf(info_detail, 150, "b: is_memory_bad: %llu",
//				host_is_memory_bad);
//		if (verbose)
//			printf("%s\n", info_detail);
//
//#ifdef LOGS
//		if (!generate)
//		log_info_detail(info_detail);
//#endif
//		memory_errors++;
//	}

#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < matrixSize; i++) {
		register bool checkFlag = true;
		register tested_type_host valGold = GOLD[i];
		register tested_type_host valOutput = C0[i];

//		register tested_type_host valOutput0 = C0[i];
//		register tested_type_host valOutput1 = C0[i];
//		register tested_type_host valOutput2 = C0[i];
//		if ((valOutput0 != valOutput1) || (valOutput0 != valOutput2)) {
//#pragma omp critical
//			{
//				char info_detail[150];
//				snprintf(info_detail, 150,
//						"m: [%d, %d], r0: %1.20e, r1: %1.20e, r2: %1.20e",
//						(int) floor(i / k), i % k, (double) valOutput0,
//						(double) valOutput1, (double) valOutput2);
//				if (verbose && (memory_errors < 10))
//					printf("%s\n", info_detail);
//
//#ifdef LOGS
//				if (!generate)
//				log_info_detail(info_detail);
//#endif
//				memory_errors++;
//			}
//			if ((valOutput0 != valOutput1) && (valOutput1 != valOutput2)
//					&& (valOutput0 != valOutput2)) {
//				// All 3 values diverge
//				if (valOutput0 == valGold) {
//					valOutput = valOutput0;
//				} else if (valOutput1 == valGold) {
//					valOutput = valOutput1;
//				} else if (valOutput2 == valGold) {
//					valOutput = valOutput2;
//				} else {
//					// NO VALUE MATCHES THE GOLD AND ALL 3 DIVERGE!
//					checkFlag = false;
//#pragma omp critical
//					{
//						char info_detail[150];
//						snprintf(info_detail, 150,
//								"t: [%d, %d], r0: %1.20e, r1: %1.20e, r2: %1.20e, e: %1.20e",
//								(int) floor(i / k), i % k, (double) valOutput0,
//								(double) valOutput1, (double) valOutput2,
//								(double) valGold);
//						if (verbose && (memory_errors < 10))
//							printf("%s\n", info_detail);
//
//#ifdef LOGS
//						if (!generate)
//						log_info_detail(info_detail);
//#endif
//						memory_errors++;
//					}
//				}
//			} else if (valOutput1 == valOutput2) {
//				// Only value 0 diverge
//				valOutput = valOutput1;
//			} else if (valOutput0 == valOutput2) {
//				// Only value 1 diverge
//				valOutput = valOutput0;
//			} else if (valOutput0 == valOutput1) {
//				// Only value 2 diverge
//				valOutput = valOutput0;
//			}
//		}

		if (votedOutput != NULL)
			votedOutput[i] = valOutput;
		// if ((fabs((tested_type_host)(valOutput-valGold)/valGold) > 1e-10)||(fabs((tested_type_host)(valOutput-valGold)/valGold) > 1e-10)) {
		if (check) {
			if (valGold != valOutput) {
				if (checkFlag) {
#pragma omp critical
					{
						char error_detail[150];
						snprintf(error_detail, 150,
								"p: [%d, %d], r: %1.20e, e: %1.20e",
								(int) floor(i / k), i % k, (double) valOutput,
								(double) valGold);
						if (verbose && (host_errors < 10))
							printf("%s\n", error_detail);
#ifdef LOGS
						if (!generate)
						log_error_detail(error_detail);
#endif
						host_errors++;
					}
				}
			}
		}
	}

	// printf("numErrors:%d", host_errors);

#ifdef LOGS
	if (!generate) {
//		log_info_count(memory_errors);
		log_error_count(host_errors);
	}
#endif

	if (host_errors != 0)
		printf("#");

	return (host_errors == 0);
}

void launch_kernel(dim3 dimGrid, dim3 dimBlock) {
	//Starting persistent kernel
	MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A0, d_B0, d_C0, k, k);
	rad::checkFrameworkErrors(hipPeekAtLastError());
	printf("Kernel LAUCHED\n");
}

int main(int argc, char* argv[]) {
//================== Test vars
	int loop2;
	// int kernel_errors=0;
	// int zero = 0;
	double time;
	double kernel_time, global_time;
	double total_kernel_time, min_kernel_time, max_kernel_time;
	int device_warmup = 1;
	// int gpu_check = 1;
//====================================

//================== Read test parameters
	if (argc < 2) {
		usage(argc, argv);
		exit(-1);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "size")) {
		k = getCmdLineArgumentInt(argc, (const char **) argv, "size");

		if ((k <= 0) || (k % 16 != 0)) {
			printf("Invalid input size given on the command-line: %d\n", k);
			exit(EXIT_FAILURE);
		}
		matrixSize = k * k;
	} else {
		usage(argc, argv);
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_a")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_a",
				&a_matrix_path);
	} else {
		a_matrix_path = new char[100];
		snprintf(a_matrix_path, 100, "mxm_a_%s_%i.matrix",
				test_precision_description, (signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", a_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_b")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_b",
				&b_matrix_path);
	} else {
		b_matrix_path = new char[100];
		snprintf(b_matrix_path, 100, "mxm_b_%s_%i.matrix",
				test_precision_description, (signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", b_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold")) {
		getCmdLineArgumentString(argc, (const char **) argv, "gold",
				&gold_matrix_path);
	} else {
		gold_matrix_path = new char[100];
		snprintf(gold_matrix_path, 100, "mxm_gold_%s_%i.matrix",
				test_precision_description, (signed int) k);
		printf("Using default gold path: %s\n", gold_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "iterations")) {
		iterations = getCmdLineArgumentInt(argc, (const char **) argv,
				"iterations");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "verbose")) {
		verbose = 1;
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "debug")) {
		fault_injection = 1;
		printf("!! Will be injected an input error\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "no-warmup")) {
		device_warmup = 0;
		printf(
				"!! The first iteration may not reflect real timing information\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "generate")) {
		generate = 1;
		device_warmup = 0;
		fault_injection = 0;
		iterations = 20;
		generate_safechecks = 5;
		printf(
				"!! Generate !! Disabling device_warmup, fault_injection and iterations limiting.\n");
		printf("!! Generate parameters: generate_safechecks: %d / \n",
				generate_safechecks);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "generator_debug")) {
		if (generate) {
			generator_debug = true;
		} else {
			printf(
					"!! generator_debug ignored: generate is not activated. active with -generate.\n");
		}
	}
//====================================

//================== Set block and grid size for MxM kernel
#if defined(test_precision_double) or defined(test_precision_single)
	int gridsize = k / BLOCK_SIZE < 1 ? 1 : k / BLOCK_SIZE;
	int blocksize = k / BLOCK_SIZE < 1 ? k : BLOCK_SIZE;
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(gridsize, gridsize);
#elif defined(test_precision_half)
	int gridsize = k / BLOCK_SIZE < 1 ? 1 : k / BLOCK_SIZE;
	int blocksize = k / BLOCK_SIZE < 1 ? k : BLOCK_SIZE;
	dim3 dimBlock(blocksize / 2.0, blocksize);
	dim3 dimGrid(gridsize, gridsize);
#endif
//====================================

//================== Init logs
#ifdef LOGS
	if (!generate) {
		char test_info[90];
		char test_name[90];
		snprintf(test_info, 90, "size:%d type:%s-precision", k, test_precision_description);
		snprintf(test_name, 90, "cuda_%s_mxm", test_precision_description);
		start_log_file(test_name, test_info);
	}
#endif
//====================================

//================== Alloc HOST memory
	A = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));
	B = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));
	C0 = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));
//	C1 = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));
//	C2 = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));

	GOLD = (tested_type_host*) malloc(matrixSize * sizeof(tested_type));

	if (!(A && B && C0 && GOLD)) { //&& C1 && C2
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
	total_kernel_time = 0;
	min_kernel_time = UINT_MAX;
	max_kernel_time = 0;
	GetDevice();
	retrieveInputMatrices();
	printf("cuda_%s_mxm\n", test_precision_description);
	fflush(stdout);
//====================================

//================== Init generator if enabled
	int generate_safechecks_count = 0;
//====================================
//================== Init Persistent threads controler
	rad::HostPersistentControler pt_control(dimGrid);

//====================================
//================== Init DEVICE memory
	allocCudaMemory();
	copyCudaMemory(pt_control);
//====================================
	//Starting persistent kernel
	launch_kernel(dimGrid, dimBlock);

	for (loop2 = 0; loop2 < iterations; loop2++) {
		//================== Global test loop

//		host_is_memory_bad = 0;

		if (!loop2 && device_warmup)
			printf("First iteration: device warmup. Please wait...\n");

		global_time = mysecond();

		rad::checkFrameworkErrors(
				hipMemsetAsync(d_C0, 0, matrixSize * sizeof(tested_type),
						pt_control.st));
		pt_control.sync_stream();
		printf("CUDAMEMSET ISSUED\n");
//		rad::checkFrameworkErrors(
//				hipMemset(d_C1, 0, matrixSize * sizeof(tested_type)));
//		rad::checkFrameworkErrors(
//				hipMemset(d_C2, 0, matrixSize * sizeof(tested_type)));

//		rad::checkFrameworkErrors(
//				hipMemcpyToSymbol(HIP_SYMBOL(is_memory_bad), &host_is_memory_bad,
//						sizeof(unsigned long long int), 0,
//						hipMemcpyHostToDevice));

		if (verbose)
			printf(",");

		kernel_time = mysecond();
#ifdef LOGS
		if (!generate)
		if (loop2 || !device_warmup)
		start_iteration();
#endif
		//================== Device computation, MxM

		pt_control.process_data_on_kernel();

//		rad::checkFrameworkErrors(hipDeviceSynchronize());
		rad::checkFrameworkErrors(hipPeekAtLastError());
		//====================================
#ifdef LOGS
		if (!generate)
		if (loop2 || !device_warmup)
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		if (loop2 || !device_warmup) {
			total_kernel_time += kernel_time;
			min_kernel_time = min(min_kernel_time, kernel_time);
			max_kernel_time = max(max_kernel_time, kernel_time);
		}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Device kernel time for iteration %d: %.3fs\n", loop2,
						kernel_time);

		//================== Gold check
		if (verbose)
			printf(",");

		time = mysecond();

		if (loop2 || !device_warmup) {
			rad::checkFrameworkErrors(
					hipMemcpyAsync(C0, d_C0, matrixSize * sizeof(tested_type),
							hipMemcpyDeviceToHost, pt_control.st));
			pt_control.sync_stream();
			if ((generate) && (k <= 16)) {
				printf("\nMatrix C (0): \n");
				for (int i = 0; i < k * k; i++) {
					printf(" %.2e", (float) C0[i]);
					if ((i + 1) % k == 0)
						printf("\n");
				}
				printf("\n");
			}

//			rad::checkFrameworkErrors(
//					hipMemcpy(C1, d_C1, matrixSize * sizeof(tested_type),
//							hipMemcpyDeviceToHost));
//			if ((generate) && (k <= 16)) {
//				printf("\nMatrix C (1): \n");
//				for (int i = 0; i < k * k; i++) {
//					printf(" %.2e", (float) C1[i]);
//					if ((i + 1) % k == 0)
//						printf("\n");
//				}
//				printf("\n");
//			}
//
//			rad::checkFrameworkErrors(
//					hipMemcpy(C2, d_C2, matrixSize * sizeof(tested_type),
//							hipMemcpyDeviceToHost));
//			if ((generate) && (k <= 16)) {
//				printf("\nMatrix C (2): \n");
//				for (int i = 0; i < k * k; i++) {
//					printf(" %.2e", (float) C2[i]);
//					if ((i + 1) % k == 0)
//						printf("\n");
//				}
//				printf("\n");
//			}
//			rad::checkFrameworkErrors(
//					hipMemcpyFromSymbol(&host_is_memory_bad, HIP_SYMBOL(is_memory_bad),
//							sizeof(unsigned long long int), 0,
//							hipMemcpyDeviceToHost));
//			if (verbose) {
//				printf("is_memory_bad: %llu\n", host_is_memory_bad);
//			}

			if (generate) {
				if (generate_safechecks_count == 0) {
					printf(
							"Generate: First generation. Step %d/%d of max. %d \n",
							generate_safechecks_count, generate_safechecks,
							iterations);
					checkOutputErrors(GOLD, false); // This will copy the voted matrix to gold
					generate_safechecks_count++;
					if ((generate) && (k <= 16)) {
						printf("\nMatrix GOLD (VOTED): \n");
						for (int i = 0; i < k * k; i++) {
							printf(" %.2e", (float) GOLD[i]);
							if ((i + 1) % k == 0)
								printf("\n");
						}
						printf("\n");
					}
				} else {
					if (!checkOutputErrors()) {
						printf(
								"Generate: Failed on compare. Step %d/%d of max. %d \n",
								generate_safechecks_count, generate_safechecks,
								iterations);
						generate_safechecks_count = 0;
					} else {
						printf(
								"Generate: Success on compare. Step %d/%d of max. %d\n",
								generate_safechecks_count, generate_safechecks,
								iterations);
						generate_safechecks_count++;
						if (generate_safechecks_count >= generate_safechecks) {
							writeGoldtoFile();
							loop2 = iterations; // This will make the loop end
							//stops only after generating
							pt_control.end_kernel();

						}
					}
				}
			} else {
				bool executed_ok = checkOutputErrors();
				if (executed_ok == false) { // (memory_errors != 0)
					pt_control.end_kernel();

					//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
					freeCudaMemory();
					//====================================
					retrieveInputMatrices();
					//================== Init DEVICE memory
					allocCudaMemory();
					copyCudaMemory(pt_control);
					//====================================
					// Re-launch the kernel
					pt_control.start_kernel();
					launch_kernel(dimGrid, dimBlock);
				}
			}
		}
		//====================================

		//================== Console hearthbeat
		printf(".");
		fflush(stdout);
		//====================================

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Gold check time for iteration %d: %.3fs\n", loop2,
						mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose) {
				/////////// PERF
				double flops = 2.0 * (double) k * k * k;
				double gflops = flops / kernel_time;
				double outputpersec = (double) matrixSize / kernel_time;
				printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", k,
						outputpersec, gflops, gflops / 1000000000);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Iteration #%d time: %.3fs\n\n\n", loop2,
						mysecond() - global_time);
		fflush(stdout);
	}

	double gflops = 2.0 * (double) k * k * k / 1000000000; // Bilion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time
			/ (iterations - (device_warmup ? 1 : 0));
	printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

	pt_control.end_kernel();

	//================== Release device memory
	freeCudaMemory();
	//====================================

	free(A);
	free(B);
	free(C0);
//	free (C1);
//	free (C2);
	free(GOLD);
#ifdef LOGS
	if (!generate)
	end_log_file();
#endif

	return 0;
}
