#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>
#include <omp.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

#include <hipblas.h>

#include <hip/hip_fp16.h>
#include "half.hpp"

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define BLOCK_SIZE 32

#define DEFAULT_INPUT_SIZE 8192

int verbose = 0;
int fault_injection = 0;

int k=0; // k x k matrix size
int iterations=100000000; // global loop iteracion

//================== Input paths
char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

FILE* f_A;
FILE* f_B;
FILE* f_GOLD;
//====================================

//================== Host and device matrix ptr's
half *A;
half *B;
half *C;
half *GOLD;

half *d_A;
half *d_B;
half *d_C;
//====================================

//================== cublas GEMM parameters
const half_float::half oneValue(1.0);
const half alpha = *((half*)&oneValue);
const half beta = *((half*)&oneValue);
hipblasOperation_t transa = HIPBLAS_OP_T;
hipblasOperation_t transb = HIPBLAS_OP_T;
int sizea, sizeb, sizec;

void GetDevice(){
//================== Retrieve and set the default CUDA device
    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);

    hipSetDevice(0);
       hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void allocCudaMemory()
{
//================== CUDA error handlers
	hipError_t malloc;
	const char *erro;
//====================================
	malloc = hipMalloc( ( void** ) &d_A, sizea * sizeof( half ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error a"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc( ( void** ) &d_B, sizea * sizeof( half ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error b"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc( ( void** ) &d_C, sizea * sizeof( half ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error c"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void copyCudaMemory()
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================
	mcpy = hipMemset(d_C, 0, sizea * sizeof (half));
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load c"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure

	mcpy = hipMemcpy( d_A, A, sizeb * sizeof( half ), hipMemcpyHostToDevice ); // PUSH A
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load b"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure

	mcpy = hipMemcpy( d_B, B, sizeb * sizeof( half ), hipMemcpyHostToDevice ); // PUSH B
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load b"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void ReadMatrixFromFile(){
//================== Read inputs to HOST memory
	int i;
	double time = mysecond();
	f_A = fopen(a_matrix_path,"rb");
	f_B = fopen(b_matrix_path,"rb");
	f_GOLD = fopen(gold_matrix_path,"rb");
	if (!(f_A&&f_B&&f_GOLD))
	{
		printf ("Cant open matrices.\n");
#ifdef LOGS
		log_error_detail("Cant open matrices"); end_log_file();
#endif
		exit(-3);
	}
    size_t ret_value[3];
    for(i=0; i<k; i++)
    {
      ret_value[0] = fread (&A[ k * i ], sizeof(half)*k, 1, f_A);
      ret_value[1] = fread (&B[ k * i ], sizeof(half)*k, 1, f_B);
      ret_value[2] = fread (&GOLD[ k * i ], sizeof(half)*k, 1, f_GOLD);
      if (ret_value[0] != 1 || ret_value[1] != 1 || ret_value[2] != 1) {
         printf("Bad input/gold formatting: %lu ; %lu ; %lu .\n", ret_value[0], ret_value[1], ret_value[2]);
         #ifdef LOGS
    		log_error_detail("Bad input/gold formatting."); end_log_file();
         #endif
    		exit(-3);
      }
    }
	if (verbose) printf("Done reading matrices in %.2fs\n", mysecond() - time);

	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);

	if (fault_injection)
	{
        half_float::half tempValue(6.5);
		A[3] = *((half*)&tempValue);
		printf("!! Injected 6.5 on position A[3]\n");
	}
}

bool badass_memcmp(half *gold, half *found, unsigned long n){
	double result = 0.0;
	int i;
	unsigned long  chunk = ceil(float(n) / float(omp_get_max_threads()));
	printf("size %d max threads %d chunk %d\n", n, omp_get_max_threads(), chunk);
	double time = mysecond();
#pragma omp parallel for default(shared) private(i) schedule(static,chunk) reduction(+:result)
   for (i=0; i < n; i++)
     result = result + (gold[i].x - found[i].x);

    //  printf("comparing took %lf seconds, diff %lf\n", mysecond() - time, result);
	if (fabs(result) > 0.0000000001)
		return false;
	return true;
}

// __device__ int kerrors;
//
// __global__ void GoldChkKernel (half *gk, half *ck, int n)//, int *kerrors)
// {
// //================== HW Accelerated output validation
// 	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
// 	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
// 	//if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
// 	if (gk[ty*n + tx].x != ck[ty*n + tx].x)
// 		atomicAdd(&kerrors, 1);
//
// }

void usage() {
    printf("Usage: cudaGemm -size=N [-input_a=<path>] [-input_b=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup]\n");
}

int main( int argc, char* argv[] )
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================

//================== Test vars
	int i, j, loop2;
	// int kernel_errors=0;
	// int zero = 0;
	double time;
	double kernel_time, global_time;
	int device_warmup = 1;
    // int gpu_check = 1;
//====================================

//================== Read test parameters
	if (argc<2) {
		usage();
		exit (-1);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        k = getCmdLineArgumentInt(argc, (const char **)argv, "size");

        if ((k <= 0)||(k % 16 != 0))
        {
            printf("Invalid input size given on the command-line: %d\n", k);
            exit(EXIT_FAILURE);
        }
    }
	else
	{
		usage();
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "input_a"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_a", &a_matrix_path);
    }
    else
    {
        a_matrix_path = new char[100];
        snprintf(a_matrix_path, 100, "hgemm_a_%i", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", a_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "input_b"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_b", &b_matrix_path);
    }
    else
    {
        b_matrix_path = new char[100];
        snprintf(b_matrix_path, 100, "hgemm_b_%i", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", b_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "gold"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold", &gold_matrix_path);
    }
    else
    {
        gold_matrix_path = new char[100];
        snprintf(gold_matrix_path, 100, "hgemm_gold_%i", (signed int)k);
        printf("Using default gold path: %s\n", gold_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "iterations"))
    {
        iterations = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        verbose = 1;
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "debug"))
    {
		fault_injection = 1;
        printf("!! Will be injected an input error\n");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "no-warmup"))
    {
		device_warmup = 0;
        printf("!! The first iteration may not reflect real timing information\n");
    }

	// if (checkCmdLineFlag(argc, (const char **)argv, "no-gpu-gold-check"))
    // {
	// 	gpu_check = 0;
    // } else {
    //     printf("!! The gold check will happen on the GPU and fall back to CPU in case of errors\n");
    // }
//====================================

//================== Set block and grid size for GoldChk kernel
	int gridsize = k/BLOCK_SIZE < 1 ? 1 : k/BLOCK_SIZE;
	int blocksize = k/BLOCK_SIZE < 1 ? k : BLOCK_SIZE;
	dim3 dimBlock(blocksize,blocksize);
	dim3 dimGrid(gridsize,gridsize);
//====================================

//================== Init logs
#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "size:%d type:half-precision", k);
	start_log_file("cudaHalfGEMM", test_info);
#endif
//====================================

//================== cublas GEMM parameters
	sizea = k * k;
	sizeb = k * k;
	sizec = k * k;
//====================================

//================== Alloc HOST memory
	A = ( half* ) malloc( sizea * sizeof( half ) );
	B = ( half* ) malloc( sizeb * sizeof( half ) );
	C = ( half* ) malloc( sizeb * sizeof( half ) );

	GOLD = ( half* ) malloc( sizec * sizeof( half ) );

	if (!(A && B && C && GOLD)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
	GetDevice();
	ReadMatrixFromFile();
	hipblasHandle_t cublasHandle;
	checkCudaErrors( hipblasCreate(&cublasHandle) );
	printf( "cublasHGEMM\n" );
	fflush(stdout);
//====================================

//================== Init DEVICE memory
	allocCudaMemory();
	copyCudaMemory();
//====================================


	for(loop2=0; loop2<iterations; loop2++)
	{//================== Global test loop

		if (!loop2 && device_warmup) printf("First iteration: device warmup. Please wait...\n");

		// Timer...
		global_time = mysecond();

		hipMemset(d_C, 0, sizea * sizeof (half));

		if (verbose) printf(",");

		kernel_time = mysecond();
		#ifdef LOGS
		if (loop2 || !device_warmup)
			start_iteration();
		#endif
		//================== Device computation, GEMM
		checkCudaErrors( hipblasHgemm(cublasHandle, transa, transb,
			   k, k, k,
			   &alpha,
			   d_A, k,
			   d_B, k,
			   &beta,
			   d_C, k ) );
		checkCudaErrors( hipDeviceSynchronize() );
		//====================================
		#ifdef LOGS
		if (loop2 || !device_warmup)
			end_iteration();
		#endif
		kernel_time = mysecond() - kernel_time;

		if (loop2 || !device_warmup)
			if (verbose) printf("Device kernel time for iteration %d: %.3fs\n", loop2, kernel_time);

    	if (verbose) printf(",");

        // Timer...
        time = mysecond();

		// if (gpu_check) {
        //
    	// 	//================== Send GOLD to device, to perform HW output validation
    	// 	mcpy = hipMemcpy(d_A, GOLD, sizea * sizeof( half ), hipMemcpyHostToDevice );
    	// 	erro = hipGetErrorString(mcpy);
    	// 	if(strcmp(erro, "no error") != 0) {
    	// 		printf("error mem load gold\n");
    	// 		#ifdef LOGS
    	// 		log_error_detail("error mem load gold"); end_log_file();
    	// 		#endif
    	// 		return 1;
    	// 	} //mem allocate failure
    	// 	hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));
    	// 	//====================================
        //
    	// 	//================== Device computation, output validation
    	// 	GoldChkKernel<<<dimGrid,dimBlock>>>(d_A, d_C, k);
    	// 	hipDeviceSynchronize();
    	// 	//====================================
        //
    	// 	//================== Retrieve output mismatchs
    	// 	kernel_errors=0;
    	// 	hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors), sizeof(unsigned int));
    	// 	//====================================
        //
        //     if (kernel_errors != 0) {
        //         printf(" kernel error: %d\n", kernel_errors);
        //
		// 		mcpy = hipMemcpy(A, d_C, sizec * sizeof( half ), hipMemcpyDeviceToHost);
		// 		erro = hipGetErrorString(mcpy);
		// 		if(strcmp(erro, "no error") != 0) {
		// 			#ifdef LOGS
		// 			log_error_detail("error mem down c"); end_log_file();
		// 			#endif
		// 			return 1;
		// 		} //mem allocate failure
        //     }
        // }

		//================== If there are errors, check on host (increased reliability)

        // if (gpu_check == 0) {
        //     kernel_errors = 0;
        //     if (memcmp(A, GOLD, sizeof(half) * k*k)) {
        //         kernel_errors = 1;
        //     }
        // }

        //if (kernel_errors != 0) {
        if (loop2 || !device_warmup) {
            if (badass_memcmp(GOLD, A, k * k)){
                mcpy = hipMemcpy(A, d_C, sizec * sizeof( half ), hipMemcpyDeviceToHost );
                erro = hipGetErrorString(mcpy);
                if(strcmp(erro, "no error") != 0) {
                    printf("error mem load gold to host\n");
                    #ifdef LOGS
                        log_error_detail("error mem load gold to host"); end_log_file();
                    #endif
                    return 1;
                } //mem allocate failure
                //~ if (memcmp(A, GOLD, sizeof(double) * k*k)) {
    			char error_detail[150];
    			int host_errors = 0;

                printf("!");

    			#pragma omp parallel for
    			for(i=0; (i<k); i++)
    			{
    				for(j=0; (j<k); j++)
    				{
    					if (A[i + k * j].x != GOLD[i + k * j].x)
    					//if ((fabs((A[i+k*j]-GOLD[i+k*j])/A[i+k*j]) > 0.0000000001)||(fabs((A[i+k*j]-GOLD[i+k*j])/GOLD[i+k*j]) > 0.0000000001))
    					#pragma omp critical
    					{

    						snprintf(error_detail, 150, "p: [%d, %d], r: %hd, e: %hd", i, j, A[i + k * j].x, GOLD[i + k * j].x);
    						//printf("%s\n", error_detail);
    						#ifdef LOGS
    						log_error_detail(error_detail);
    						#endif
    						host_errors++;
    						//ea++;
    						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + k * j], GOLD[i + k * j], t_ea);

    					}
    				}
    			}

                printf("numErrors:%d", host_errors);

    			#ifdef LOGS
    				log_error_count(host_errors);
    			#endif
    			//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
    			hipFree( d_A );
    			hipFree( d_B );
    			hipFree( d_C );
    			//====================================
    			ReadMatrixFromFile();
    			//================== Init DEVICE memory
    			allocCudaMemory();
    			copyCudaMemory();
    			//====================================
    		}
        }

		//====================================

		//================== Console hearthbeat
		/*if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("test number: %d\n", loop2);
			printf(" kernel time: %f\n", kernel_time);
		}
		else
		{*/
			printf(".");
			fflush(stdout);
		//}
		//====================================

// 		if (gpu_check) {
// //================== Send A back to the device
//         	mcpy = hipMemcpy(d_A, A, sizea * sizeof( half ), hipMemcpyHostToDevice );
//         	erro = hipGetErrorString(mcpy);
//         	if(strcmp(erro, "no error") != 0) {
//         		printf("error mem load A\n");
//         		#ifdef LOGS
//         		log_error_detail("error mem load A"); end_log_file();
//         		#endif
//         		return 1;
//         	} //mem allocate failure
//===================================
        // }

		if (loop2 || !device_warmup)
			if (verbose) printf("Gold check time for iteration %d: %.3fs\n", loop2, mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose)
			{
				/////////// PERF
				double flops = 2.0*(double)k*k*k;
				double gflops = flops / kernel_time;
				double outputpersec = (double)k*k/kernel_time;
				printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n",k, outputpersec, gflops, gflops/1000000000);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose) printf("Iteration #%d time: %.3fs\n\n\n", loop2, mysecond() - global_time);
		fflush(stdout);
	}

	//================== Release device memory
	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );
	//====================================

	free( A );
	free( B );
	#ifdef LOGS
	end_log_file();
	#endif

	return 0;
}
