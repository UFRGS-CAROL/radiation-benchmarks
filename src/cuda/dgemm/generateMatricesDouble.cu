#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include <string>
#include <sys/time.h>
// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#include<hipblas.h>

#define MATRIX_PATH "./Double_"
#define DEFAULT_INPUT_SIZE 8192

int k=0;
int sizea, sizeb, sizec;
double *A, *B, *GOLD;

bool host_check = false;

char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

void usage() {
    printf("Usage: generateMatrices -size=N [-host_check] [-input_a=<path>] [-input_b=<path>] [-gold=<path>]\n");
}

void generateInputMatrices()
{
	double temp;
	int i, j;
	FILE *f_A, *f_B;

	f_A = fopen(a_matrix_path, "wb");
	f_B = fopen(b_matrix_path, "wb");


	srand ( time(NULL) );

    int numZerosA = 0;
    int numZerosB = 0;
    #pragma omp parallel for
	for(i=0; i<DEFAULT_INPUT_SIZE; i++)
	{
		for(j=0; j<DEFAULT_INPUT_SIZE+16; j++){
			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.0004e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_A );
            if (temp == 0 || isnan(temp) || isinf(temp)) {
                numZerosA++;
            }

			temp = (rand()/((double)(RAND_MAX)+1)*(-4.06e16-4.4e16))+4.1e16;
			fwrite( &temp, sizeof(double), 1, f_B );
            if (temp == 0 || isnan(temp) || isinf(temp)) {
                numZerosB++;
            }


		}
	}
	printf("Number of zeros/NaNs/INFs on A: %d\n", numZerosA);
	printf("Number of zeros/NaNs/INFs on B: %d\n", numZerosB);

	fclose(f_A);
	fclose(f_B);

	return;
}

void ReadMatrixFromFile(){

	int i;
	FILE *f_A, *f_B;
    printf("Each matrix size: %.4fGB\n", (float)sizeof(double) * DEFAULT_INPUT_SIZE*DEFAULT_INPUT_SIZE / (1024*1024*1024));

	f_A = fopen(a_matrix_path,"rb");
	f_B = fopen(b_matrix_path,"rb");
	if (!(f_A&&f_B))
	{
		printf("Error opening matrices A, B.\n");
		exit(-1);
	}
	for(i=0; i<k; i++)
	{
		fread (&A[ k * i ], sizeof(double)*k, 1, f_A);
		fread (&B[ k * i ], sizeof(double)*k, 1, f_B);
	}
printf("Done reading matrices\n");

	fclose(f_A);
	fclose(f_B);
}

void GetDevice(){

    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("Get Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);

    hipSetDevice(0);
        hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

double* openmpMul(double* a, double* b, size_t size) {
	double time = mysecond();

	double* bT = (double*) malloc(sizeof(double)*size*size);
	double* c = (double*) calloc(size*size, sizeof(double));

	if (c == NULL || bT == NULL) {
		printf("could not alloc hostGold matrix.");
		return NULL;
	}

	#pragma omp parallel for
	for (int i=0;i<size;i++)
		for (int j=0;j<size;j++)
			bT[j*size+i] = b[i*size+j];
	
	#pragma omp parallel for
	for (int i=0;i<size;i++)
		for (int j=0;j<size;j++)
			for (int k=0; k<size;k++) 
				c[i*size+j] += a[j*size+k] * bT[i*size+k];

	printf("host mmul time: %.2f seconds\n", mysecond()-time);

	return c;
}

void generateGoldMatrix()
{
	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	char transa = 't', transb = 't';
	////////////////////////////////////////////////////

	////////////////////////////////////////////////////
	//////////DEVICE VARS///////////////////////////////
	hipError_t cumalloc_err;
	const char *cumalloc_err_str;

	double *d_A;
	double *d_B;
	double *d_C;
	////////////////////////////////////////////////////

	A = ( double* ) malloc( sizea * sizeof( double ) );
	B = ( double* ) malloc( sizeb * sizeof( double ) );
	GOLD = ( double* ) malloc( sizec * sizeof( double ) );

	GetDevice();

	ReadMatrixFromFile();

	cumalloc_err = hipMalloc( ( void** ) &d_A, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);} //mem allocate failure

	cumalloc_err = hipMalloc( ( void** ) &d_B, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMalloc( ( void** ) &d_C, sizea * sizeof( double ) );
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}


	cumalloc_err = hipMemset( d_C, 0, sizeb * sizeof( double )); // ZERA C
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_A, A, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	cumalloc_err = hipMemcpy( d_B, B, sizeb * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	printf("hipblasDgemm... k=%d transa=%c transb=%c\n", k, transa, transb);
	double time = mysecond();


	hipblasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
			   k, k, k,
			   alpha,
			   d_A, k,
			   d_B, k,
			   beta,
			   d_C, k );
	hipDeviceSynchronize();

	time=mysecond()-time;

	/////////// PERF
    double flops = 2.0*(double)k*k*k;
    double gflops = flops / time;
    double outputpersec = (double)k*k/time;
    printf("kernel time: %lf\n",time);
    printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n",k, outputpersec, gflops, gflops/1000000000);
	///////////

	cumalloc_err = hipMemcpy(GOLD, d_C, sizec * sizeof( double ), hipMemcpyDeviceToHost);
	cumalloc_err_str = hipGetErrorString(cumalloc_err);
	if(strcmp(cumalloc_err_str, "no error") != 0) {exit(-3);}

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

	int i, j;
	FILE *f_GOLD;

	f_GOLD = fopen(gold_matrix_path, "wb");

	//printf("-------------------------\n%.10f\n%.10f\n%.10f\n", GOLD[0], GOLD[1], GOLD[2]);

	if (host_check) {
		printf("Calculating mMul using OpenMP on Host...\n");
		double *hostGold = openmpMul(A, B, k);
		printf("Comparing GPU result with Host result...\n");
		double maxDiff = 0.0;
		#pragma omp parallel for
		for (i=0; i<k; i++) {
			for (j=0; j<k; j++) {
				register double diff = fabs((hostGold[i*k+j]-GOLD[i*k+j])/hostGold[i*k+j]);
				if (diff > maxDiff) {
					#pragma omp critical
					maxDiff = max(diff, maxDiff);
				}
				if (diff > 0.1) {
					printf("Fail! hostGold!=gpuGold %f != %f (diff: %e)\n", hostGold[i*k+j], GOLD[i*k+j], fabs((hostGold[i*k+j]-GOLD[i*k+j])/hostGold[i*k+j]));
					fflush(stdout);
					exit(-1);
				}
			}
		}
		printf("CPU and GPU match by an error of up to %e element difference. Writing to file...\n", maxDiff);
	}

    int numZeros = 0;
	for(i=0; i<k; i++)
	{
		fwrite( &GOLD[i * k], sizeof(double)*k, 1, f_GOLD );
        for(j=0; j<k; j++) {
            if (isnan(GOLD[i*k + j]) || isinf(GOLD[i*k + j]) || GOLD[i*k + j]==0) {
                numZeros++;
            }
        }
	}
	printf("Number of zeros/NaNs/INFs on GOLD: %d\n", numZeros);

	fclose(f_GOLD);

	return;
}

int main (int argc, char** argv)
{
//====================================
//================== Read parameters
	if (argc<2) {
		usage();
		exit (-1);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        k = getCmdLineArgumentInt(argc, (const char **)argv, "size");

        if ((k <= 0)||(k % 16 != 0))
        {
            printf("Invalid input size given on the command-line: %d\n", k);
            exit(EXIT_FAILURE);
        }
    }
	else
	{
		usage();
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "input_a"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_a", &a_matrix_path);
    }
    else
    {
        a_matrix_path = new char[100];
        snprintf(a_matrix_path, 100, "dgemm_a_%i", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", a_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "input_b"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_b", &b_matrix_path);
    }
    else
    {
        b_matrix_path = new char[100];
        snprintf(b_matrix_path, 100, "dgemm_b_%i", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", b_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "gold"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold", &gold_matrix_path);
    }
    else
    {
        gold_matrix_path = new char[100];
        snprintf(gold_matrix_path, 100, "dgemm_gold_%i", (signed int)k);
        printf("Using default gold path: %s\n", gold_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "host_check"))
    {
		host_check = true;
	}
//====================================

	sizea = k * k;
	sizeb = k * k;
	sizec = k * k;


    printf("Each matrix size: %.4fGB\n", (float)sizeof(double) * DEFAULT_INPUT_SIZE*DEFAULT_INPUT_SIZE / (1024*1024*1024));

	FILE *test_file;
	test_file=fopen(a_matrix_path, "rb");
	if (!test_file)
	{
		printf("Generating input matrices...\n");
		generateInputMatrices();
	}
	else
	{	printf("Input matrices already exist...\n");	}

	generateGoldMatrix();

	return 0;
}
