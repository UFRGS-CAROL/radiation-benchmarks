#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>

#ifdef USE_OMP
#include <omp.h>
#endif

#include <hipblas.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define DEFAULT_INPUT_SIZE 8192
#define ERROR_THRESHOLD  1.0e-10

#define GOLDCHK_BLOCK_SIZE 32

__device__ int kerrors;

typedef char byte;

////====================================

void get_device() {
//================== Retrieve and set the default CUDA device
	hipDeviceProp_t prop;
	hipError_t teste;
	int count = 0;
	teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Name: %s\n", prop.name);
	}
	int *ndevice;
	int dev = 0;
	ndevice = &dev;
	hipGetDevice(ndevice);

	hipSetDevice(0);
	hipGetDeviceProperties(&prop, 0);
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void alloc_cuda_memory(double *d_a, double *d_b, double *d_c, int matrix_size) {
//================== CUDA error handlers
	hipError_t malloc;
	const char *erro;
//====================================
	malloc = hipMalloc((void**) &d_a, matrix_size * sizeof(double));
	erro = hipGetErrorString(malloc);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error a"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc((void**) &d_b, matrix_size * sizeof(double));
	erro = hipGetErrorString(malloc);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error b"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc((void**) &d_c, matrix_size * sizeof(double));
	erro = hipGetErrorString(malloc);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error c"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure
}

void copy_cuda_memory(double *d_a, double *d_b, double *d_c, double *a,
		double *b, int matrix_size) {
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================
	mcpy = hipMemset(d_c, 0, matrix_size * sizeof(double));
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load c"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	mcpy = hipMemcpy(d_a, a, matrix_size * sizeof(double),
			hipMemcpyHostToDevice); // PUSH A
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load a"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	mcpy = hipMemcpy(d_b, b, matrix_size * sizeof(double),
			hipMemcpyHostToDevice); // PUSH B
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load b"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure
}

void read_matrix_from_file(FILE *f_A, FILE *f_B, FILE *f_GOLD, double *a,
		double *b, double *gold, char *a_matrix_path, char *b_matrix_path,
		char *gold_matrix_path, bool verbose, bool fault_injection, int k) {
//================== Read inputs to HOST memory
	if (verbose)
		printf("Reading matrices... ");
	double time = mysecond();
	f_A = fopen(a_matrix_path, "rb");
	f_B = fopen(b_matrix_path, "rb");
	f_GOLD = fopen(gold_matrix_path, "rb");
	if (!(f_A && f_B && f_GOLD)) {
		printf("Cant open matrices.\n");
#ifdef LOGS
		log_error_detail("Cant open matrices"); end_log_file();
#endif
		exit(-3);
	}
	size_t ret_value[3];
	for (int i = 0; i < k; i++) {
		ret_value[0] = fread(&(a[k * i]), sizeof(double) * k, 1, f_A);
		ret_value[1] = fread(&(b[k * i]), sizeof(double) * k, 1, f_B);
		ret_value[2] = fread(&(gold[k * i]), sizeof(double) * k, 1, f_GOLD);
		if ((ret_value[0] != 1) || (ret_value[1] != 1) || (ret_value[2] != 1)) {
			printf("Bad input/gold formatting: %lu ; %lu ; %lu .\n",
					ret_value[0], ret_value[1], ret_value[2]);
#ifdef LOGS
			log_error_detail("Bad input/gold formatting."); end_log_file();
#endif
			exit(-3);
		}
	}
	if (verbose)
		printf("Done reading matrices in %.2fs\n", mysecond() - time);

	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);

	if (fault_injection) {
		a[3] = (double) 6.5;
		printf("!! Injected 6.5 on position A[3]\n");
	}
}

__global__ void gold_check_kernel(double *gk, double *ck, int n) {
//================== HW Accelerated output validation
	int tx = blockIdx.x * GOLDCHK_BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * GOLDCHK_BLOCK_SIZE + threadIdx.y;
	//if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
	if (gk[ty * n + tx] != ck[ty * n + tx])
		atomicAdd(&kerrors, 1);

}

void usage() {
	printf(
			"Usage: dgemm -size=N [-input_a=<path>] [-input_b=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup] [-gold_gpu_check]\n");
}

void check_output_errors(double *GOLD, double *d_A, double *d_B, double *d_C,
		double *C, int k, bool verbose) {
	char error_detail[150];
	int host_errors = 0;
#ifdef USE_OMP
#pragma omp parallel for
#endif
	for (int i = 0; (i < k); i++) {
		for (int j = 0; (j < k); j++) {
			double valGold = GOLD[i + k * j];
			double valOutput = C[i + k * j];
			// if ((fabs((double)(valOutput-valGold)/valGold) > 1e-10)||(fabs((double)(valOutput-valGold)/valGold) > 1e-10)) {
			if (valGold != valOutput) {
#ifdef USE_OMP
#pragma omp critical
#endif
				{
					snprintf(error_detail, 150,
							"p: [%d, %d], r: %1.20e, e: %1.20e", i, j,
							valOutput, valGold);
					if (verbose && (host_errors < 10))
						printf("%s\n", error_detail);
#ifdef LOGS
					log_error_detail(error_detail);
#endif
					host_errors++;
					//ea++;
					//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + k * j], GOLD[i + k * j], t_ea);
				}
			}
		}
	}

	// printf("numErrors:%d", host_errors);

	if (host_errors != 0) {
		printf("#");
#ifdef LOGS
		log_error_count(host_errors);
#endif
		//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		//====================================
		read_matrix_from_file();
		//================== Init DEVICE memory
		alloc_cuda_memory();
		copy_cuda_memory();
		//====================================
	}
}

int main(int argc, char* argv[]) {
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================

	int verbose = 0;
	int fault_injection = 0;
	int gold_gpu_check = 0;

	int k = 0; // k x k matrix size
	int matrixSize = 0; // = k * k matrix size
	int iterations = 100000000; // global loop iteracion

	//================== Input paths
	char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

	FILE* f_A;
	FILE* f_B;
	FILE* f_GOLD;
	//====================================

	//================== Host and device matrix ptr's
	double *A;
	double *B;
	double *C;
	double *GOLD;

	double *d_A;
	double *d_B;
	double *d_C;
	//====================================

//================== Test vars
	int loop2;
	// int kernel_errors=0;
	int zero = 0;
	double time;
	double kernel_time, global_time;
	double total_kernel_time, min_kernel_time, max_kernel_time;
	int device_warmup = 1;
//====================================

//================== Read test parameters
	if (argc < 2) {
		usage();
		exit(-1);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "size")) {
		k = getCmdLineArgumentInt(argc, (const char **) argv, "size");

		if ((k <= 0) || (k % 16 != 0)) {
			printf("Invalid input size given on the command-line: %d\n", k);
			exit(EXIT_FAILURE);
		}
		matrixSize = k * k;
	} else {
		usage();
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_a")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_a",
				&a_matrix_path);
	} else {
		a_matrix_path = new char[100];
		snprintf(a_matrix_path, 100, "dgemm_a_%i.matrix",
				(signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", a_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_b")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_b",
				&b_matrix_path);
	} else {
		b_matrix_path = new char[100];
		snprintf(b_matrix_path, 100, "dgemm_b_%i.matrix",
				(signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", b_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold")) {
		getCmdLineArgumentString(argc, (const char **) argv, "gold",
				&gold_matrix_path);
	} else {
		gold_matrix_path = new char[100];
		snprintf(gold_matrix_path, 100, "dgemm_gold_%i.matrix", (signed int) k);
		printf("Using default gold path: %s\n", gold_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "iterations")) {
		iterations = getCmdLineArgumentInt(argc, (const char **) argv,
				"iterations");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "verbose")) {
		verbose = 1;
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "debug")) {
		fault_injection = 1;
		printf("!! Will be injected an input error\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "no-warmup")) {
		device_warmup = 0;
		printf(
				"!! The first iteration may not reflect real timing information\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold_gpu_check")) {
		gold_gpu_check = 1;
	}
//====================================

	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	char transa = 't', transb = 't';
	////////////////////////////////////////////////////

//================== Init logs
#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "size:%d gpu-gold-check:%d type:double-precision", k, gold_gpu_check);
	start_log_file("cudaDGEMM", test_info);
#endif
//====================================

//================== Alloc HOST memory
	A = (double*) malloc(matrixSize * sizeof(double));
	B = (double*) malloc(matrixSize * sizeof(double));
	C = (double*) malloc(matrixSize * sizeof(double));

	GOLD = (double*) malloc(matrixSize * sizeof(double));

	if (!(A && B && C && GOLD)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
	total_kernel_time = 0;
	min_kernel_time = UINT_MAX;
	max_kernel_time = 0;
	get_device();
	read_matrix_from_file();
	printf("cudaDGEMM\n");
	fflush(stdout);
//====================================

//================== Init DEVICE memory
	alloc_cuda_memory();
	copy_cuda_memory();
//====================================

	for (loop2 = 0; loop2 < iterations; loop2++) {//================== Global test loop

		if (!loop2 && device_warmup)
			printf("First iteration: device warmup. Please wait...\n");

		// Timer...
		global_time = mysecond();

		hipMemset(d_C, 0, matrixSize * sizeof(double));
		checkCudaErrors(hipPeekAtLastError());
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipPeekAtLastError());

		if (verbose)
			printf(",");

		kernel_time = mysecond();
#ifdef LOGS
		if (loop2 || !device_warmup)
		start_iteration();
#endif
		//================== Device computation, HMxM

		hipblasDgemm((hipblasOperation_t) transa, (hipblasOperation_t) transb, k,
				k, k, alpha, d_A, k, d_B, k, beta, d_C, k);

		checkCudaErrors(hipPeekAtLastError());
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipPeekAtLastError());
		//====================================
#ifdef LOGS
		if (loop2 || !device_warmup)
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		if (loop2 || !device_warmup) {
			total_kernel_time += kernel_time;
			min_kernel_time = min(min_kernel_time, kernel_time);
			max_kernel_time = max(max_kernel_time, kernel_time);
		}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Device kernel time for iteration %d: %.3fs\n", loop2,
						kernel_time);

		if (verbose)
			printf(",");

		// Timer...
		time = mysecond();

		//if (kernel_errors != 0) {
		if (loop2 || !device_warmup) {
			bool checkHost = true;
			if (gold_gpu_check == 1) {
				checkHost = false;

				//================== Set block and grid size for GoldChk kernel
				int gridsize =
						k / GOLDCHK_BLOCK_SIZE < 1 ? 1 : k / GOLDCHK_BLOCK_SIZE;
				int blocksize =
						k / GOLDCHK_BLOCK_SIZE < 1 ? k : GOLDCHK_BLOCK_SIZE;
				dim3 dimBlock(blocksize, blocksize);
				dim3 dimGrid(gridsize, gridsize);
				//====================================

				//================== Send GOLD to device, to perform HW output validation
				mcpy = hipMemcpy(d_A, GOLD, matrixSize * sizeof(double),
						hipMemcpyHostToDevice);
				erro = hipGetErrorString(mcpy);
				if (strcmp(erro, "no error") != 0) {
					printf("error mem load gold\n");
#ifdef LOGS
					log_error_detail("error mem load gold"); end_log_file();
#endif
					return 1;
				} //mem allocate failure
				hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));
				//====================================

				//================== Device computation, output validation
				gold_check_kernel<<<dimGrid, dimBlock>>>(d_A, d_C, k);
				checkCudaErrors(hipPeekAtLastError());

				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipPeekAtLastError());
				//====================================

				//================== Retrieve output mismatchs
				unsigned int kernel_errors = 0;
				hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors),
						sizeof(unsigned int));
				if (kernel_errors != 0)
					checkHost = true;
				//====================================

				//================== Send A back to the device
				mcpy = hipMemcpy(d_A, A, matrixSize * sizeof(double),
						hipMemcpyHostToDevice);
				erro = hipGetErrorString(mcpy);
				if (strcmp(erro, "no error") != 0) {
					printf("error mem load A\n");
#ifdef LOGS
					log_error_detail("error mem load A"); end_log_file();
#endif
					return 1;
				} //mem allocate failure
				  //====================================
			}
			if (checkHost) {
				checkCudaErrors(
						hipMemcpy(C, d_C, matrixSize * sizeof(double),
								hipMemcpyDeviceToHost));
				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipPeekAtLastError());
				//~ if (memcmp(A, GOLD, sizeof(double) * k*k)) {
				check_output_errors(GOLD, d_A, d_B, d_C, C, k, verbose);
			}
		}

		//====================================

		//================== Console hearthbeat
		/*if(kernel_errors > 0 || (loop2 % 10 == 0))
		 {
		 printf("test number: %d\n", loop2);
		 printf(" kernel time: %f\n", kernel_time);
		 }
		 else
		 {*/
		printf(".");
		fflush(stdout);
		//}
		//====================================

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Gold check time for iteration %d: %.3fs\n", loop2,
						mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose) {
				/////////// PERF
				double flops = 2.0 * (double) k * k * k;
				double gflops = flops / kernel_time;
				double outputpersec = (double) matrixSize / kernel_time;
				printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", k,
						outputpersec, gflops, gflops / 1000000000);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Iteration #%d time: %.3fs\n\n\n", loop2,
						mysecond() - global_time);
		fflush(stdout);
	}

	double gflops = 2.0 * (double) k * k * k / 1000000000; // Bilion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time
			/ (iterations - (device_warmup ? 1 : 0));
	printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

	//================== Release device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//====================================

	free(A);
	free(B);
	free(C);
	free(GOLD);
#ifdef LOGS
	end_log_file();
#endif

	return 0;
}
