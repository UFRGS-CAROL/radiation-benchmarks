#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>
#include <omp.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#ifdef SAFE_MALLOC
#include "safe_memory.h"
#endif

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp isupdated on every log_helper function call.

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define DEFAULT_INPUT_SIZE 8192

int verbose = 0;
int fault_injection = 0;
int gold_gpu_check = 0;

int k = 0; // k x k matrix size
int matrixSize = 0; // = k * k matrix size
int iterations = 100000000; // global loop iteracion

//================== Input paths
char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

FILE* f_A;
FILE* f_B;
FILE* f_GOLD;
//====================================

//================== Host and device matrix ptr's
double *A;
double *B;
double *C;
double *GOLD;

double *d_A;
double *d_B;
double *d_C;
//====================================

typedef char byte;

void GetDevice() {
//================== Retrieve and set the default CUDA device
	hipDeviceProp_t prop;
	hipError_t teste;
	int count = 0;
	teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Name: %s\n", prop.name);
	}
	int *ndevice;
	int dev = 0;
	ndevice = &dev;
	hipGetDevice(ndevice);

	hipSetDevice(0);
	hipGetDeviceProperties(&prop, 0);
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void allocCudaMemory() {
//================== CUDA error handlers
	hipError_t malloc;
	const char *erro = 0;
//====================================
#ifdef SAFE_MALLOC
	d_A = (double*)safe_malloc(matrixSize * sizeof(double));
#else
	malloc = hipMalloc((void**) &d_A, matrixSize * sizeof(double));
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error a"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure
#endif

#ifdef SAFE_MALLOC
	d_B = (double*)safe_malloc(matrixSize * sizeof(double));
#else
	malloc = hipMalloc((void**) &d_B, matrixSize * sizeof(double));
	erro = hipGetErrorString(malloc);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error b"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure
#endif

#ifdef SAFE_MALLOC
	d_C = (double*)safe_malloc(matrixSize * sizeof(double));
#else
	malloc = hipMalloc((void**) &d_C, matrixSize * sizeof(double));
	erro = hipGetErrorString(malloc);

	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error c"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure
#endif
}

void copyCudaMemory() {
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================
	mcpy = hipMemset(d_C, 0, matrixSize * sizeof(double));
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error gpu load c"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure

	mcpy = hipMemcpy(d_A, A, matrixSize * sizeof(double),
			hipMemcpyHostToDevice); // PUSH A
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error gpu load a"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure

	mcpy = hipMemcpy(d_B, B, matrixSize * sizeof(double),
			hipMemcpyHostToDevice); // PUSH B
	erro = hipGetErrorString(mcpy);
	if (strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail((char *)"error gpu load b"); end_log_file();
#endif
		exit (EXIT_FAILURE);
	} //mem allocate failure
}

void ReadMatrixFromFile() {
//================== Read inputs to HOST memory
	if (verbose)
		printf("Reading matrices... ");
	double time = mysecond();
	f_A = fopen(a_matrix_path, "rb");
	f_B = fopen(b_matrix_path, "rb");
	f_GOLD = fopen(gold_matrix_path, "rb");
	if (!(f_A && f_B && f_GOLD)) {
		printf("Cant open matrices.\n");
#ifdef LOGS
		log_error_detail((char *)"Cant open matrices"); end_log_file();
#endif
		exit(-3);
	}
	size_t ret_value[3];
	for (int i = 0; i < k; i++) {
		ret_value[0] = fread(&(A[k * i]), sizeof(double) * k, 1, f_A);
		ret_value[1] = fread(&(B[k * i]), sizeof(double) * k, 1, f_B);
		ret_value[2] = fread(&(GOLD[k * i]), sizeof(double) * k, 1, f_GOLD);
		if ((ret_value[0] != 1) || (ret_value[1] != 1) || (ret_value[2] != 1)) {
			printf("Bad input/gold formatting: %lu ; %lu ; %lu .\n",
					ret_value[0], ret_value[1], ret_value[2]);
#ifdef LOGS
			log_error_detail((char *)"Bad input/gold formatting."); end_log_file();
#endif
			exit(-3);
		}
	}
	if (verbose)
		printf("Done reading matrices in %.2fs\n", mysecond() - time);

	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);

	if (fault_injection) {
		A[3] = (double) 6.5;
		printf("!! Injected 6.5 on position A[3]\n");
	}
}

// bool badass_memcmp(double *gold, double *found, unsigned long n){
// 	double result = 0.0;
// 	int i;
// 	unsigned long  chunk = ceil(double(n) / double(omp_get_max_threads()));
// 	// printf("size %d max threads %d chunk %d\n", n, omp_get_max_threads(), chunk);
// 	double time = mysecond();
// #pragma omp parallel for default(shared) private(i) schedule(static,chunk) reduction(+:result)
//    for (i=0; i < n; i++)
//      result = result + (gold[i] - found[i]);

//     //  printf("comparing took %lf seconds, diff %lf\n", mysecond() - time, result);
// 	if (fabs(result) > 0.0000000001)
// 		return true;
// 	return false;
// }

// bool badass_memcmp(byte *gold, byte *found, unsigned long n){
// 	bool flag = false;
// 	//#pragma omp parallel for private(gold,found) shared(flag)
// 	for (int i=0; i < n; i++) {
// 		if (gold[i] != found[i]) {
// 			//printf("memcmp found an error at position [%d]: gold: 0x%hhX | output: 0x%hhX\n", i, gold[i], found[i]);
// 			flag = true;
// 		}
// 	}

// 	return flag;
// }

// bool badass_memcmp_double(double *gold, double *found, unsigned long n){
// 	bool flag = false;
//     double t = mysecond();
//     double min = 1.0e-10;
// 	#pragma omp parallel for shared(flag)    
// 	for (unsigned long i=0; i < n; i++) {
//         // double valGold = GOLD[i];
// 		// double valOutput = C[i];
// 		if (GOLD[i] != C[i]) {
// 		//if (fabs((valOutput-valGold)/valGold > min) || fabs((valOutput-valGold)/valGold) > min){
// 			//printf("memcmp found an error at position [%d]: gold: 0x%hhX | output: 0x%hhX\n", i, gold[i], found[i]);
// 			flag = true;
// 		}
// 	}

//     double final_time = mysecond() - t;
//     if (verbose) printf("Time comparing %lf\n", final_time);
// 	return flag;
// }

#define GOLDCHK_BLOCK_SIZE 32

__device__ int kerrors;

__global__ void GoldChkKernel(double *gk, double *ck, int n) //, int *kerrors)
		{
//================== HW Accelerated output validation
	int tx = blockIdx.x * GOLDCHK_BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * GOLDCHK_BLOCK_SIZE + threadIdx.y;
	//if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
	if (gk[ty * n + tx] != ck[ty * n + tx])
		atomicAdd(&kerrors, 1);

}

void usage() {
	printf(
			"Usage: dgemm -size=N [-input_a=<path>] [-input_b=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup] [-gold_gpu_check]\n");
}

void checkOutputErrors() {
	int host_errors = 0;

#pragma omp parallel for shared(host_errors)
	for (int i = 0; (i < k * k); i++) {
		register double valGold = GOLD[i];
		register double valOutput = C[i];
		// if ((fabs((double)(valOutput-valGold)/valGold) > 1e-10)||(fabs((double)(valOutput-valGold)/valGold) > 1e-10)) {
		if (valGold != valOutput) {
#pragma omp critical
			{
				char error_detail[150];
				snprintf(error_detail, 150, "p: [%d, %d], r: %1.20e, e: %1.20e",
						(int) floor(i / k), i % k, valOutput, valGold);
				if (verbose && (host_errors < 10))
					printf("%s\n", error_detail);

#ifdef LOGS
				log_error_detail(error_detail);
#endif
				host_errors++;
			}
		}
	}

	// printf("numErrors:%d", host_errors);

	if (host_errors != 0) {
		printf("#");
#ifdef LOGS
		log_error_count(host_errors);
#endif
		//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		//====================================
		ReadMatrixFromFile();
		//================== Init DEVICE memory
		allocCudaMemory();
		copyCudaMemory();
		//====================================
	}
}

int main(int argc, char* argv[]) {
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================

//================== Test vars
	int loop2;
	// int kernel_errors=0;
	int zero = 0;
	double time;
	double kernel_time, global_time;
	double total_kernel_time, min_kernel_time, max_kernel_time;
	int device_warmup = 1;
	unsigned char use_tensor_cores = 0;
//====================================

//================== Read test parameters
	if (argc < 2) {
		usage();
		exit(-1);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "size")) {
		k = getCmdLineArgumentInt(argc, (const char **) argv, "size");

		if ((k <= 0) || (k % 16 != 0)) {
			printf("Invalid input size given on the command-line: %d\n", k);
			exit (EXIT_FAILURE);
		}
		matrixSize = k * k;
	} else {
		usage();
		exit (EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_a")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_a",
				&a_matrix_path);
	} else {
		a_matrix_path = new char[100];
		snprintf(a_matrix_path, 100, "dgemm_a_%i.matrix",
				(signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", a_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_b")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_b",
				&b_matrix_path);
	} else {
		b_matrix_path = new char[100];
		snprintf(b_matrix_path, 100, "dgemm_b_%i.matrix",
				(signed int) DEFAULT_INPUT_SIZE);
		printf("Using default input_a path: %s\n", b_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold")) {
		getCmdLineArgumentString(argc, (const char **) argv, "gold",
				&gold_matrix_path);
	} else {
		gold_matrix_path = new char[100];
		snprintf(gold_matrix_path, 100, "dgemm_gold_%i.matrix", (signed int) k);
		printf("Using default gold path: %s\n", gold_matrix_path);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "iterations")) {
		iterations = getCmdLineArgumentInt(argc, (const char **) argv,
				"iterations");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "verbose")) {
		verbose = 1;
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "debug")) {
		fault_injection = 1;
		printf("!! Will be injected an input error\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "no-warmup")) {
		device_warmup = 0;
		printf(
				"!! The first iteration may not reflect real timing information\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold_gpu_check")) {
		gold_gpu_check = 1;
	}

	//flag for tensor cores
	if (checkCmdLineFlag(argc, (const char **) argv, "use_tensors") && getCmdLineArgumentInt(argc, (const char **) argv,
			"use_tensors") == 1) {
		printf(
				"Use tensor cores is not available for Double precision, using normal cores instead\n");
	}
//====================================

	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	hipblasOperation_t transa = HIPBLAS_OP_T, transb = HIPBLAS_OP_T;
	////////////////////////////////////////////////////

//================== Init logs
#ifdef LOGS
	char test_info[200];
	snprintf(test_info, 200, "size:%d gpu-gold-check:%d type:double-precision use_tensor_cores:%d", k, gold_gpu_check, use_tensor_cores);
	start_log_file((char *)"cudaDGEMM", test_info);
#endif
//====================================

//================== Alloc HOST memory
	A = (double*) malloc(matrixSize * sizeof(double));
	B = (double*) malloc(matrixSize * sizeof(double));
	C = (double*) malloc(matrixSize * sizeof(double));

	GOLD = (double*) malloc(matrixSize * sizeof(double));

	if (!(A && B && C && GOLD)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

	// Alloc blas handle
	hipblasHandle_t blas_handle;

	checkCudaErrors(hipblasCreate(&blas_handle));

	printf("Tensor cores %d, is handle available\n", use_tensor_cores, (blas_handle && true));
	if (use_tensor_cores == 0) {
		hipblasSetMathMode(blas_handle, HIPBLAS_DEFAULT_MATH);
	} else if (use_tensor_cores == 1) {
		//TODO: implement when double precision is available
	}

//================== Init test environment
	// kernel_errors=0;
	total_kernel_time = 0;
	min_kernel_time = UINT_MAX;
	max_kernel_time = 0;
	GetDevice();
	ReadMatrixFromFile();
	printf("cudaDGEMM\n");
	fflush (stdout);
//====================================

//================== Init DEVICE memory
	allocCudaMemory();
	copyCudaMemory();
//====================================

	for (loop2 = 0; loop2 < iterations; loop2++) {//================== Global test loop

		if (!loop2 && device_warmup)
			printf("First iteration: device warmup. Please wait...\n");

		// Timer...
		global_time = mysecond();

		hipMemset(d_C, 0, matrixSize * sizeof(double));
		checkCudaErrors(hipPeekAtLastError());
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipPeekAtLastError());

		if (verbose)
			printf(",");

		kernel_time = mysecond();
#ifdef LOGS
		if (loop2 || !device_warmup)
		start_iteration();
#endif
		//================== Device computation, HMxM
//		hipblasStatus_t hipblasDgemm(hipblasHandle_t handle,
//		                           hipblasOperation_t transa, hipblasOperation_t transb,
//		                           int m, int n, int k,
//		                           const double          *alpha,
//		                           const double          *A, int lda,
//		                           const double          *B, int ldb,
//		                           const double          *beta,
//		                           double          *C, int ldc)

		hipblasDgemm(blas_handle, 			//hipblasHandle_t handle,
				transa, //hipblasOperation_t transa
				transb, //hipblasOperation_t transb
				k,							//int m,
				k, 							//int n
				k, 							//int k
				&alpha, 						//const double          *alpha
				d_A, 		//const double          *A
				k, 							// int lda
				d_B, 		//const double          *B
				k, 							// int ldb,
				&beta, 						// const double          *beta
				d_C, 						//double          *C
				k);							//int ldc

		checkCudaErrors(hipPeekAtLastError());
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipPeekAtLastError());
		//====================================
#ifdef LOGS
		if (loop2 || !device_warmup)
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		if (loop2 || !device_warmup) {
			total_kernel_time += kernel_time;
			min_kernel_time = min(min_kernel_time, kernel_time);
			max_kernel_time = max(max_kernel_time, kernel_time);
		}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Device kernel time for iteration %d: %.3fs\n", loop2,
						kernel_time);

		if (verbose)
			printf(",");

		// Timer...
		time = mysecond();

		//if (kernel_errors != 0) {
		if (loop2 || !device_warmup) {
			bool checkHost = true;
			if (gold_gpu_check == 1) {
				checkHost = false;

				//================== Set block and grid size for GoldChk kernel
				int gridsize =
						k / GOLDCHK_BLOCK_SIZE < 1 ? 1 : k / GOLDCHK_BLOCK_SIZE;
				int blocksize =
						k / GOLDCHK_BLOCK_SIZE < 1 ? k : GOLDCHK_BLOCK_SIZE;
				dim3 dimBlock(blocksize, blocksize);
				dim3 dimGrid(gridsize, gridsize);
				//====================================

				//================== Send GOLD to device, to perform HW output validation
				mcpy = hipMemcpy(d_A, GOLD, matrixSize * sizeof(double),
						hipMemcpyHostToDevice);
				erro = hipGetErrorString(mcpy);
				if (strcmp(erro, "no error") != 0) {
					printf("error mem load gold\n");
#ifdef LOGS
					log_error_detail((char *)"error mem load gold"); end_log_file();
#endif
					return 1;
				} //mem allocate failure
				hipMemcpyToSymbol(HIP_SYMBOL(kerrors), &zero, sizeof(int));
				//====================================

				//================== Device computation, output validation
				GoldChkKernel<<<dimGrid, dimBlock>>>(d_A, d_C, k);
				checkCudaErrors(hipPeekAtLastError());

				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipPeekAtLastError());
				//====================================

				//================== Retrieve output mismatchs
				unsigned int kernel_errors = 0;
				hipMemcpyFromSymbol(&kernel_errors, HIP_SYMBOL(kerrors),
						sizeof(unsigned int));
				if (kernel_errors != 0)
					checkHost = true;
				//====================================

				//================== Send A back to the device
				mcpy = hipMemcpy(d_A, A, matrixSize * sizeof(double),
						hipMemcpyHostToDevice);
				erro = hipGetErrorString(mcpy);
				if (strcmp(erro, "no error") != 0) {
					printf("error mem load A\n");
#ifdef LOGS
					log_error_detail((char *)"error mem load A"); end_log_file();
#endif
					return 1;
				} //mem allocate failure
				  //====================================
			}
			if (checkHost) {
				checkCudaErrors(
						hipMemcpy(C, d_C, matrixSize * sizeof(double),
								hipMemcpyDeviceToHost));
				checkCudaErrors(hipDeviceSynchronize());
				checkCudaErrors(hipPeekAtLastError());
				//~ if (memcmp(A, GOLD, sizeof(double) * k*k)) {
				//if (badass_memcmp_double(GOLD, C, matrixSize)){ //badass_memcmp((byte*)GOLD, (byte*)C, matrixSize * sizeof( double ) )) {
				//	printf("!");
				checkOutputErrors();
				//}
			}
		}

		//====================================

		//================== Console hearthbeat
		/*if(kernel_errors > 0 || (loop2 % 10 == 0))
		 {
		 printf("test number: %d\n", loop2);
		 printf(" kernel time: %f\n", kernel_time);
		 }
		 else
		 {*/
		printf(".");
		fflush(stdout);
		//}
		//====================================

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Gold check time for iteration %d: %.3fs\n", loop2,
						mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose) {
				/////////// PERF
				double flops = 2.0 * (double) k * k * k;
				double gflops = flops / kernel_time;
				double outputpersec = (double) matrixSize / kernel_time;
				printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", k,
						outputpersec, gflops, gflops / 1000000000);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose)
				printf("Iteration #%d time: %.3fs\n\n\n", loop2,
						mysecond() - global_time);
		fflush(stdout);
	}

	double gflops = 2.0 * (double) k * k * k / 1000000000; // Bilion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time
			/ (iterations - (device_warmup ? 1 : 0));
	printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

	//================== Release device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	//====================================
	//Release the handle
	hipblasDestroy(blas_handle);

	free(A);
	free(B);
	free(C);
	free(GOLD);
#ifdef LOGS
	end_log_file();
#endif

	return 0;
}
