#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>
#include <omp.h>

#include <hipblas.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define DEFAULT_INPUT_SIZE 8192

int verbose = 0;
int fault_injection = 0;

int k=0; // k x k matrix size
int matrixSize=0; // = k * k matrix size
int iterations=100000000; // global loop iteracion

//================== Input paths
char *gold_matrix_path, *a_matrix_path, *b_matrix_path;

FILE* f_A;
FILE* f_B;
FILE* f_GOLD;
//====================================

//================== Host and device matrix ptr's
double *A;
double *B;
double *GOLD;

double *d_A;
double *d_B;
double *d_C;
//====================================

void GetDevice(){
//================== Retrieve and set the default CUDA device
    hipDeviceProp_t prop;
    hipError_t teste;
    int count=0;
    teste = hipGetDeviceCount(&count);
	printf("\nGet Device Test: %s\n", hipGetErrorString(teste));
    for (int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "Name: %s\n", prop.name );
    }
    int *ndevice; int dev = 0;
    ndevice = &dev;
    hipGetDevice(ndevice);

    hipSetDevice(0);
       hipGetDeviceProperties( &prop, 0 );
	printf("\ndevice: %d %s\n", *ndevice, prop.name);

}

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void allocCudaMemory()
{
//================== CUDA error handlers
	hipError_t malloc;
	const char *erro;
//====================================
	malloc = hipMalloc( ( void** ) &d_A, matrixSize * sizeof( double ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error a"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc( ( void** ) &d_B, matrixSize * sizeof( double ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error b"); end_log_file();
#endif
		exit(EXIT_FAILURE);
	} //mem allocate failure

	malloc = hipMalloc( ( void** ) &d_C, matrixSize * sizeof( double ) );
	erro = hipGetErrorString(malloc);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error c"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void copyCudaMemory()
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================
	mcpy = hipMemset(d_C, 0, matrixSize * sizeof (double));
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load c"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure

	mcpy = hipMemcpy( d_A, A, matrixSize * sizeof( double ), hipMemcpyHostToDevice ); // PUSH A
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load a"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure

	mcpy = hipMemcpy( d_B, B, matrixSize * sizeof( double ), hipMemcpyHostToDevice ); // PUSH B
	erro = hipGetErrorString(mcpy);
	if(strcmp(erro, "no error") != 0) {
#ifdef LOGS
		log_error_detail("error gpu load b"); end_log_file();
#endif
		exit(EXIT_FAILURE);} //mem allocate failure
}

void ReadMatrixFromFile(){
//================== Read inputs to HOST memory
	int i, j;
	if (verbose) printf("Reading matrices... ");
	double time = mysecond();
	f_A = fopen(a_matrix_path,"rb");
	f_B = fopen(b_matrix_path,"rb");
	f_GOLD = fopen(gold_matrix_path,"rb");
	if (!(f_A&&f_B&&f_GOLD))
	{
		printf ("Cant open matrices.\n");
#ifdef LOGS
		log_error_detail("Cant open matrices"); end_log_file();
#endif
		exit(-3);
	}
    size_t ret_value[3];
    for(i=0; i<k; i++)
    {
      ret_value[0] = fread (&(A[ k * i ]), sizeof(double)*k, 1, f_A);
      ret_value[1] = fread (&(B[ k * i ]), sizeof(double)*k, 1, f_B);
      ret_value[2] = fread (&(GOLD[ k * i ]), sizeof(double)*k, 1, f_GOLD);
      if ((ret_value[0] != 1) || (ret_value[1] != 1) || (ret_value[2] != 1)) {
         printf("Bad input/gold formatting: %lu ; %lu ; %lu .\n", ret_value[0], ret_value[1], ret_value[2]);
         #ifdef LOGS
    		log_error_detail("Bad input/gold formatting."); end_log_file();
         #endif
    		exit(-3);
      }
    }
	if (verbose) printf("Done reading matrices in %.2fs\n", mysecond() - time);

	fclose(f_A);
	fclose(f_B);
	fclose(f_GOLD);

	if (fault_injection)
	{
		A[3] = (double)6.5;
		printf("!! Injected 6.5 on position A[3]\n");
	}
}

bool badass_memcmp(double *gold, double *found, unsigned long n){
	double result = 0.0;
	int i;
	unsigned long  chunk = ceil(double(n) / double(omp_get_max_threads()));
	// printf("size %d max threads %d chunk %d\n", n, omp_get_max_threads(), chunk);
	double time = mysecond();
#pragma omp parallel for default(shared) private(i) schedule(static,chunk) reduction(+:result)
   for (i=0; i < n; i++)
     result = result + (gold[i] - found[i]);

    //  printf("comparing took %lf seconds, diff %lf\n", mysecond() - time, result);
	if (fabs(result) > 0.0000000001)
		return true;
	return false;
}

// __device__ int kerrors;
//
// __global__ void GoldChkKernel (double *gk, double *ck, int n)//, int *kerrors)
// {
// //================== HW Accelerated output validation
// 	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
// 	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
// 	//if ((fabs((gk[ty*n+tx]-ck[ty*n+tx])/gk[ty*n+tx]) > 0.0000000001)||(fabs((gk[ty*n+tx]-ck[ty*n+tx])/ck[ty*n+tx]) > 0.0000000001))
// 	if (gk[ty*n + tx].x != ck[ty*n + tx].x)
// 		atomicAdd(&kerrors, 1);
//
// }

void usage() {
    printf("Usage: dgemm -size=N [-input_a=<path>] [-input_b=<path>] [-gold=<path>] [-iterations=N] [-verbose] [-no-warmup]\n");
}

int main( int argc, char* argv[] )
{
//================== CUDA error handlers
	hipError_t mcpy;
	const char *erro;
//====================================

//================== Test vars
	int i, j, loop2;
	// int kernel_errors=0;
	// int zero = 0;
	double time;
	double kernel_time, global_time;
    double total_kernel_time, min_kernel_time, max_kernel_time;
	int device_warmup = 1;
    // int gpu_check = 1;
//====================================

//================== Read test parameters
	if (argc<2) {
		usage();
		exit (-1);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        k = getCmdLineArgumentInt(argc, (const char **)argv, "size");

        if ((k <= 0)||(k % 16 != 0))
        {
            printf("Invalid input size given on the command-line: %d\n", k);
            exit(EXIT_FAILURE);
		}
		matrixSize = k * k;
    }
	else
	{
		usage();
		exit(EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **)argv, "input_a"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_a", &a_matrix_path);
    }
    else
    {
        a_matrix_path = new char[100];
        snprintf(a_matrix_path, 100, "dgemm_a_%i.matrix", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", a_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "input_b"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input_b", &b_matrix_path);
    }
    else
    {
        b_matrix_path = new char[100];
        snprintf(b_matrix_path, 100, "dgemm_b_%i.matrix", (signed int)DEFAULT_INPUT_SIZE);
        printf("Using default input_a path: %s\n", b_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "gold"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold", &gold_matrix_path);
    }
    else
    {
        gold_matrix_path = new char[100];
        snprintf(gold_matrix_path, 100, "dgemm_gold_%i.matrix", (signed int)k);
        printf("Using default gold path: %s\n", gold_matrix_path);
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "iterations"))
    {
        iterations = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        verbose = 1;
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "debug"))
    {
		fault_injection = 1;
        printf("!! Will be injected an input error\n");
    }

	if (checkCmdLineFlag(argc, (const char **)argv, "no-warmup"))
    {
		device_warmup = 0;
        printf("!! The first iteration may not reflect real timing information\n");
    }

	// if (checkCmdLineFlag(argc, (const char **)argv, "no-gpu-gold-check"))
    // {
	// 	gpu_check = 0;
    // } else {
    //     printf("!! The gold check will happen on the GPU and fall back to CPU in case of errors\n");
    // }
//====================================

	////////////////////////////////////////////////////
	/////////////CUBLAS GEMM VARS///////////////////////
	const double alpha = 1.0;
	const double beta = 1.0;
	char transa = 't', transb = 't';
	////////////////////////////////////////////////////

//================== Init logs
#ifdef LOGS
	char test_info[90];
	snprintf(test_info, 90, "size:%d type:double-precision", k);
	start_log_file("cudaDGEMM", test_info);
#endif
//====================================

//================== Alloc HOST memory
	A = ( double* ) malloc( matrixSize * sizeof( double ) );
	B = ( double* ) malloc( matrixSize * sizeof( double ) );

	GOLD = ( double* ) malloc( matrixSize * sizeof( double ) );

	if (!(A && B && GOLD)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
    total_kernel_time = 0;
    min_kernel_time = UINT_MAX;
    max_kernel_time = 0;
	GetDevice();
	ReadMatrixFromFile();
	printf( "cudaDGEMM\n" );
	fflush(stdout);
//====================================

//================== Init DEVICE memory
	allocCudaMemory();
	copyCudaMemory();
//====================================


	for(loop2=0; loop2<iterations; loop2++)
	{//================== Global test loop

		if (!loop2 && device_warmup) printf("First iteration: device warmup. Please wait...\n");

		// Timer...
		global_time = mysecond();

		hipMemset(d_C, 0, matrixSize * sizeof (double));

		if (verbose) printf(",");

		kernel_time = mysecond();
		#ifdef LOGS
		if (loop2 || !device_warmup)
			start_iteration();
		#endif
		//================== Device computation, HMxM
	
        hipblasDgemm( (hipblasOperation_t)transa, (hipblasOperation_t)transb,
            k, k, k,
            alpha,
            d_A, k,
            d_B, k,
            beta,
            d_C, k );

        checkCudaErrors( hipPeekAtLastError() );

        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors( hipPeekAtLastError() );
		//====================================
		#ifdef LOGS
		if (loop2 || !device_warmup)
			end_iteration();
		#endif
		kernel_time = mysecond() - kernel_time;
      
		if (loop2 || !device_warmup) {
		  total_kernel_time += kernel_time;
		  min_kernel_time = min(min_kernel_time, kernel_time);
		  max_kernel_time = max(max_kernel_time, kernel_time);
		}

		if (loop2 || !device_warmup)
			if (verbose) printf("Device kernel time for iteration %d: %.3fs\n", loop2, kernel_time);

    	if (verbose) printf(",");

        // Timer...
        time = mysecond();

        //if (kernel_errors != 0) {
        if (loop2 || !device_warmup) {
            checkCudaErrors( hipMemcpy(A, d_C, matrixSize * sizeof( double ), hipMemcpyDeviceToHost) );
            //~ if (memcmp(A, GOLD, sizeof(double) * k*k)) {
            if (badass_memcmp(GOLD, A, matrixSize)) {
    			char error_detail[150];
    			int host_errors = 0;

                printf("!");

    			#pragma omp parallel for
    			for(i=0; (i<k); i++)
    			{
    				for(j=0; (j<k); j++)
    				{
    					if (A[i + k * j] != GOLD[i + k * j])
    					//if ((fabs((A[i+k*j]-GOLD[i+k*j])/A[i+k*j]) > 0.0000000001)||(fabs((A[i+k*j]-GOLD[i+k*j])/GOLD[i+k*j]) > 0.0000000001))
    					#pragma omp critical
    					{

    						snprintf(error_detail, 150, "p: [%d, %d], r: %1.16e, e: %1.16e", i, j, (double)(A[i + k * j]), (double)(GOLD[i + k * j]));
    						if (verbose && (host_errors < 10)) printf("%s\n", error_detail);
    						#ifdef LOGS
    						log_error_detail(error_detail);
    						#endif
    						host_errors++;
    						//ea++;
    						//fprintf(file, "\n p: [%d, %d], r: %1.16e, e: %1.16e, error: %d\n", i, j, A[i + k * j], GOLD[i + k * j], t_ea);

    					}
    				}
    			}

                // printf("numErrors:%d", host_errors);

    			#ifdef LOGS
    				log_error_count(host_errors);
    			#endif
    			//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
    			hipFree( d_A );
    			hipFree( d_B );
    			hipFree( d_C );
    			//====================================
    			ReadMatrixFromFile();
    			//================== Init DEVICE memory
    			allocCudaMemory();
    			copyCudaMemory();
    			//====================================
    		}
        }

		//====================================

		//================== Console hearthbeat
		/*if(kernel_errors > 0 || (loop2 % 10 == 0))
		{
			printf("test number: %d\n", loop2);
			printf(" kernel time: %f\n", kernel_time);
		}
		else
		{*/
			printf(".");
			fflush(stdout);
		//}
		//====================================

		if (loop2 || !device_warmup)
			if (verbose) printf("Gold check time for iteration %d: %.3fs\n", loop2, mysecond() - time);

		if (loop2 || !device_warmup)
			if (verbose)
			{
				/////////// PERF
				double flops = 2.0*(double)k*k*k;
				double gflops = flops / kernel_time;
				double outputpersec = (double)matrixSize/kernel_time;
				printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n",k, outputpersec, gflops, gflops/1000000000);
				///////////
			}

		if (loop2 || !device_warmup)
			if (verbose) printf("Iteration #%d time: %.3fs\n\n\n", loop2, mysecond() - global_time);
		fflush(stdout);
	}

    double gflops = 2.0*(double)k*k*k / 1000000000; // Bilion FLoating-point OPerationS
    double averageKernelTime = total_kernel_time / (iterations - (device_warmup ? 1 : 0));
    printf("\n-- END --\n"
    "Total kernel time: %.3fs\n"
    "Iterations: %d\n"
    "Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
    "Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n", 
    total_kernel_time, 
    iterations, 
    averageKernelTime, min_kernel_time, max_kernel_time,
    gflops / averageKernelTime, gflops / min_kernel_time, gflops / max_kernel_time);

	//================== Release device memory
	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );
	//====================================

	free( A );
	free( B );
	free( GOLD );
	#ifdef LOGS
	end_log_file();
	#endif

	return 0;
}