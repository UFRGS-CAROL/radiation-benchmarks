#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <string>
#include <omp.h>
#include <random>
#include <hip/hip_fp16.h>
#include <vector>

#include "dmr_kernels.h"
#include "none_kernels.h"
#include "device_vector.h"
#include "cuda_utils.h"
#include "Parameters.h"

// helper functions
#include "hip/hip_runtime_api.h"

#define HALF_ROUND_STYLE 1
#define HALF_ROUND_TIES_TO_EVEN 1
#include "half.hpp"

#define BLOCK_SIZE 32

#define DEFAULT_INPUT_SIZE 8192

void usage(int argc, char* argv[]) {
	printf("Usage: %s [-iterations=N] [-verbose]\n", argv[0]);
}

// Returns true if no errors are found. False if otherwise.
// Set votedOutput pointer to retrieve the voted matrix
template<typename T>
int check_output_errors(std::vector<T> &R, T OUTPUT_R, bool verbose) {
	int host_errors = 0;
#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < R.size(); i++) {
		register bool checkFlag = true;
		register T valGold = (OUTPUT_R);
		register T valOutput = R[i];
		if (valGold != valOutput) {
			if (checkFlag) {
#pragma omp critical
				{
					char error_detail[150];
					snprintf(error_detail, 150, "p: [%d], r: %1.20e, e: %1.20e",
							i, (double) valOutput, (double) valGold);
					if (verbose && (host_errors < 10))
						printf("%s\n", error_detail);
#ifdef LOGS
					log_error_detail(error_detail);
#endif
					host_errors++;
				}
			}
		}
	}

	if (host_errors != 0) {
		printf("#");
#ifdef LOGS
		log_error_count(host_errors);
#endif
	}
	return host_errors;
}

template<typename full, typename incomplete = void>
void test_radiation(const incomplete OUTPUT_R, const incomplete INPUT_A,
		const incomplete INPUT_B, Parameters& parameters) {
	//================== Init test environment
	// kernel_errors=0;
	double total_kernel_time = 0;
	double min_kernel_time = UINT_MAX;
	double max_kernel_time = 0;

	parameters.print_details();
	//====================================
	// FULL PRECIISON
	std::vector<full> host_vector_full(parameters.r_size, 0);
	DeviceVector<full> device_vector_full(parameters.r_size);

	//====================================
	// SECOND PRECISION ONLY IF IT IS DEFINED
	DeviceVector<incomplete> device_vector_inc;
	std::vector<incomplete> host_vector_inc;
	if (std::is_void<incomplete>::value != true) {
		host_vector_inc = std::vector<incomplete>(parameters.r_size, 0);
		device_vector_inc = DeviceVector<incomplete>(parameters.r_size);
	}

	for (int iteration = 0; iteration < parameters.iterations; iteration++) {
		//================== Global test loop
		double kernel_time = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//================== Device computation
		if (std::is_void<incomplete>::value) {
			switch (parameters.micro) {
			case ADD:
				MicroBenchmarkKernel_ADD<full> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_full.data,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			case MUL:
				MicroBenchmarkKernel_MUL<full> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_full.data,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			case FMA:
				MicroBenchmarkKernel_FMA<full> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_full.data,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			}
		} else {
			switch (parameters.micro) {
			case ADD:
				MicroBenchmarkKernel_ADD<incomplete, full> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data, device_vector_full.data, 0.1,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			case MUL:
				MicroBenchmarkKernel_MUL<incomplete, full> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data, device_vector_full.data, 0.1,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			case FMA:
				MicroBenchmarkKernel_FMA<incomplete, full> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data, device_vector_full.data, 0.1,
						OUTPUT_R, INPUT_A, INPUT_B);
				break;
			}
		}

		checkFrameworkErrors(hipPeekAtLastError());
		checkFrameworkErrors(hipDeviceSynchronize());
		checkFrameworkErrors(hipPeekAtLastError());

		kernel_time = mysecond() - kernel_time;

		//====================================
#ifdef LOGS
		end_iteration();
#endif

		total_kernel_time += kernel_time;
		min_kernel_time = std::min(min_kernel_time, kernel_time);
		max_kernel_time = std::max(max_kernel_time, kernel_time);

		std::cout << ".";
		if (parameters.verbose) {
			//check output
			host_vector_full = device_vector_full.to_vector();
			int errors = check_output_errors<full>(host_vector_full, OUTPUT_R,
					parameters.verbose);
			unsigned long long relative_errors = copy_errors();

			/////////// PERF
			double outputpersec = double(parameters.r_size) / kernel_time;
			std::cout << "SIZE:" << parameters.r_size;
			std::cout << " OUTPUT/S:" << outputpersec;
			std::cout << " ITERATION " << iteration << " time: " << kernel_time
					<< std::endl;

		}
	}

	double gflops = parameters.r_size * OPS / 1e9; // Billion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time / parameters.iterations;
	std::printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, parameters.iterations, averageKernelTime,
			min_kernel_time, max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

}

void dmr(Parameters& parameters) {
	switch (parameters.redundancy) {
	//NONE REDUNDANCY ----------------------------------------------------------
	case NONE:
		switch (parameters.precision) {
		case HALF:
			test_radiation<half>(OUTPUT_R_HALF, INPUT_A_HALF, INPUT_B_HALF,
					parameters);
			break;
		case SINGLE:
			test_radiation<float>(OUTPUT_R_SINGLE, INPUT_A_SINGLE,
			INPUT_B_SINGLE, parameters);
			break;

		case DOUBLE:
			test_radiation<double>(OUTPUT_R_DOUBLE, INPUT_A_DOUBLE,
			INPUT_B_DOUBLE, parameters);
			break;
		}
		break;

		//DMR MIXED REDUNDANCY -------------------------------------------------------
	case DMRMIXED:
		switch (parameters.precision) {
		case DOUBLE:
			test_radiation<double, float>(OUTPUT_R_SINGLE, INPUT_A_SINGLE,
			INPUT_B_SINGLE, parameters);
			break;
		case SINGLE:
			test_radiation<float, half>(OUTPUT_R_HALF, INPUT_A_HALF,
			INPUT_B_HALF, parameters);
			break;
		}
		break;

//		//DMR REDUNDANCY -------------------------------------------------------
	case DMR:
		switch (parameters.precision) {
		case DOUBLE:
			test_radiation<double, double>(OUTPUT_R_DOUBLE, INPUT_A_DOUBLE,
			INPUT_B_DOUBLE, parameters);
			break;
		case SINGLE:
			test_radiation<float, float>(OUTPUT_R_SINGLE, INPUT_A_SINGLE,
			INPUT_B_SINGLE, parameters);
			break;
		case HALF:
			test_radiation<half, half>(OUTPUT_R_HALF, INPUT_A_HALF,
			INPUT_B_HALF, parameters);
			break;
		}
		break;

	}

}

int main(int argc, char* argv[]) {

//================== Set block and grid size for MxM kernel
	hipDeviceProp_t prop = GetDevice();
	Parameters parameters(argc, argv, prop.multiProcessorCount, 256);
//================== Init logs
#ifdef LOGS
	std::string test_info = std::string("ops:") + std::to_string(OPS)
	+ " gridsize:" + std::to_string(parameters.grid_size)
	+ " blocksize:" + std::to_string(parameters.block_size) + " type:"
	+ parameters.instruction_str + "-" + parameters.precision_str
	+ "-precision hard:" + parameters.hardening_str;

	std::string test_name = std::string("cuda_") + parameters.precision_str
	+ "_micro-" + parameters.instruction_str;
	start_log_file(const_cast<char*>(test_name.c_str()),
			const_cast<char*>(test_info.c_str()));
#endif

	dmr(parameters);

#ifdef LOGS
	end_log_file();
#endif
	return 0;
}
