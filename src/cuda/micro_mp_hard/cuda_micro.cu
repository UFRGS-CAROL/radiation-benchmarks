#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <string>
#include <omp.h>
#include <random>
#include <hip/hip_fp16.h>
#include <vector>

#include "dmr_kernels.h"
#include "none_kernels.h"
#include "device_vector.h"
#include "cuda_utils.h"
#include "Parameters.h"

// helper functions
#include "hip/hip_runtime_api.h"

#define HALF_ROUND_STYLE 1
#define HALF_ROUND_TIES_TO_EVEN 1
#include "half.hpp"

#define BLOCK_SIZE 32

#define DEFAULT_INPUT_SIZE 8192

//===================================== DEFINE TESTED PRECISION
//FOR DMR APPROACH I NEED to use the smallest precision
//as a limit, since it is not possible to store the bigger precisions
//on smaller precisions

//If double it means that DMR will be double and float
//so the limits are the float ones

#define INPUT_A_DOUBLE 1.1945305291614955E+103 // 0x5555555555555555
#define INPUT_B_DOUBLE 3.7206620809969885E-103 // 0x2AAAAAAAAAAAAAAA
#define OUTPUT_R_DOUBLE 4.444444444444444 //0x4011C71C71C71C71

#define INPUT_A_SINGLE 1.4660155E+13 // 0x55555555
#define INPUT_B_SINGLE 3.0316488E-13 // 0x2AAAAAAA
#define OUTPUT_R_SINGLE 4.444444 //0x408E38E3

#define INPUT_A_HALF 1.066E+2 // 0x56AA
#define INPUT_B_HALF 4.166E-2 // 0x2955
#define OUTPUT_R_HALF 4.44 // 0x4471

#define OPS_PER_THREAD_OPERATION 1

void usage(int argc, char* argv[]) {
	printf("Usage: %s [-iterations=N] [-verbose]\n", argv[0]);
}

// Returns true if no errors are found. False if otherwise.
// Set votedOutput pointer to retrieve the voted matrix
template<typename T, int OUTPUT_R>
bool checkOutputErrors(std::vector<T> &R, bool verbose) {
	int host_errors = 0;
#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < R.size(); i++) {
		register bool checkFlag = true;
		register T valGold = (OUTPUT_R);
		register T valOutput = R[i];
		if (valGold != valOutput) {
			if (checkFlag) {
#pragma omp critical
				{
					char error_detail[150];
					snprintf(error_detail, 150, "p: [%d], r: %1.20e, e: %1.20e",
							i, (double) valOutput, (double) valGold);
					if (verbose && (host_errors < 10))
						printf("%s\n", error_detail);
#ifdef LOGS
					log_error_detail(error_detail);
#endif
					host_errors++;
				}
			}
		}
	}

	if (host_errors != 0) {
		printf("#");
#ifdef LOGS
		log_error_count(host_errors);
#endif
	}
	return host_errors == 0;
}

template<typename incomplete, typename full>
void test_radiation(int iterations, bool verbose, int r_size, int gridsize,
		int blocksize, const incomplete OUTPUT_R, const incomplete INPUT_A,
		const incomplete INPUT_B, Parameters& parameters) {
	//================== Init test environment
	// kernel_errors=0;
	double total_kernel_time = 0;
	double min_kernel_time = UINT_MAX;
	double max_kernel_time = 0;
	double global_time;

	parameters.print_details();
	//====================================
	std::vector<incomplete> host_vector_inc(r_size, 0);
	std::vector<full> host_vector_ful(r_size, 0);

	DeviceVector<incomplete> device_vector_inc;
	DeviceVector<full> device_vector_ful;

	device_vector_ful = host_vector_ful;
	device_vector_inc = host_vector_inc;

	for (int loop2 = 0; loop2 < iterations; loop2++) {
		//================== Global test loop

		global_time = mysecond();
		double kernel_time = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//================== Device computation
		switch (parameters.micro) {
		case ADD:
			MicroBenchmarkKernel_ADD<incomplete, full> <<<gridsize, blocksize>>>(
					device_vector_inc.data, device_vector_ful.data, 0.1,
					OUTPUT_R, INPUT_A, INPUT_B);
			break;
		case MUL:
			MicroBenchmarkKernel_MUL<incomplete, full> <<<gridsize, blocksize>>>(
					device_vector_inc.data, device_vector_ful.data, 0.1,
					OUTPUT_R, INPUT_A, INPUT_B);
			break;
		case FMA:
			MicroBenchmarkKernel_FMA<incomplete, full> <<<gridsize, blocksize>>>(
					device_vector_inc.data, device_vector_ful.data, 0.1,
					OUTPUT_R, INPUT_A, INPUT_B);
			break;
		}

		checkFrameworkErrors(hipPeekAtLastError());
		checkFrameworkErrors(hipDeviceSynchronize());
		checkFrameworkErrors(hipPeekAtLastError());

		std::printf("ERRORS %lld\n", copy_errors());

		//====================================
#ifdef LOGS
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		total_kernel_time += kernel_time;
		min_kernel_time = std::min(min_kernel_time, kernel_time);
		max_kernel_time = std::max(max_kernel_time, kernel_time);

		if (verbose)
			std::printf("Device kernel time for iteration %d: %.3fs\n", loop2,
					kernel_time);

		double gold_check_time = mysecond();

		std::printf(".");

		if (verbose)
			std::printf("Gold check time for iteration %d: %.3fs\n", loop2,
					mysecond() - gold_check_time);

		if (verbose) {
			/////////// PERF
			double flops = r_size * OPS * OPS_PER_THREAD_OPERATION;
			double gflops = flops / kernel_time;
			double outputpersec = (double) r_size / kernel_time;
			std::printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", r_size,
					outputpersec, gflops, gflops / 1000000000);
			///////////
		}

		if (verbose)
			std::printf("Iteration #%d time: %.3fs\n\n\n", loop2,
					mysecond() - global_time);
	}

	double gflops = r_size * OPS * OPS_PER_THREAD_OPERATION / 1000000000; // Billion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time / iterations;
	std::printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

}

template<typename full>
void test_radiation(int iterations, bool verbose, int r_size, int gridsize,
		int blocksize, const full OUTPUT_R, const full INPUT_A,
		const full INPUT_B, Parameters& parameters) {
	//================== Init test environment
	// kernel_errors=0;
	double total_kernel_time = 0;
	double min_kernel_time = UINT_MAX;
	double max_kernel_time = 0;
	double global_time;

	parameters.print_details();
	//====================================
	std::vector<full> host_vector_ful(r_size, 0);
	DeviceVector<full> device_vector_ful;

	device_vector_ful = host_vector_ful;

	for (int loop2 = 0; loop2 < iterations; loop2++) {
		//================== Global test loop

		global_time = mysecond();
		double kernel_time = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//================== Device computation
		switch (parameters.micro) {
		case ADD:
			MicroBenchmarkKernel_ADD<full> <<<gridsize, blocksize>>>(
					device_vector_ful.data, OUTPUT_R, INPUT_A, INPUT_B);
			break;
		case MUL:
			MicroBenchmarkKernel_MUL<full> <<<gridsize, blocksize>>>(
					device_vector_ful.data, OUTPUT_R, INPUT_A, INPUT_B);
			break;
		case FMA:
			MicroBenchmarkKernel_FMA<full> <<<gridsize, blocksize>>>(
					device_vector_ful.data, OUTPUT_R, INPUT_A, INPUT_B);
			break;
		}

		checkFrameworkErrors(hipPeekAtLastError());
		checkFrameworkErrors(hipDeviceSynchronize());
		checkFrameworkErrors(hipPeekAtLastError());

		std::printf("ERRORS %lld\n", copy_errors());

		//====================================
#ifdef LOGS
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		total_kernel_time += kernel_time;
		min_kernel_time = std::min(min_kernel_time, kernel_time);
		max_kernel_time = std::max(max_kernel_time, kernel_time);

		if (verbose)
			std::printf("Device kernel time for iteration %d: %.3fs\n", loop2,
					kernel_time);

		double gold_check_time = mysecond();

		std::printf(".");

		if (verbose)
			std::printf("Gold check time for iteration %d: %.3fs\n", loop2,
					mysecond() - gold_check_time);

		if (verbose) {
			/////////// PERF
			double flops = r_size * OPS * OPS_PER_THREAD_OPERATION;
			double gflops = flops / kernel_time;
			double outputpersec = (double) r_size / kernel_time;
			std::printf("SIZE:%d OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", r_size,
					outputpersec, gflops, gflops / 1000000000);
			///////////
		}

		if (verbose)
			std::printf("Iteration #%d time: %.3fs\n\n\n", loop2,
					mysecond() - global_time);
	}

	double gflops = r_size * OPS * OPS_PER_THREAD_OPERATION / 1000000000; // Billion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time / iterations;
	std::printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

}

void dmr(Parameters& parameters) {
	switch (parameters.redundancy) {
	//NONE REDUNDANCY ----------------------------------------------------------
	case NONE:
		switch (parameters.precision) {
		case DOUBLE:
			test_radiation<half>(parameters.iterations, parameters.verbose, r_size, gridsize,
					blocksize, OUTPUT_R_HALF, INPUT_A_HALF, INPUT_B_HALF);
			break;
		case SINGLE:
			test_radiation<float>(parameters.iterations, parameters.verbose, r_size, gridsize,
					blocksize,
					OUTPUT_R_SINGLE, INPUT_A_SINGLE, INPUT_B_SINGLE);
			break;

		case DOUBLE:
			test_radiation<double>(parameters.iterations, parameters.verbose, r_size, gridsize,
					blocksize,
					OUTPUT_R_DOUBLE, INPUT_A_DOUBLE, INPUT_B_DOUBLE);
			break;
		}
		break;

		//DMR MIXED REDUNDANCY -------------------------------------------------------
	case DMRMIXED:
		switch (parameters.precision) {
		case DOUBLE:
			test_radiation<float, double>(parameters.iterations, parameters.verbose, r_size, gridsize,
					blocksize, OUTPUT_R_SINGLE, INPUT_A_SINGLE, INPUT_B_SINGLE);
			break;
		case SINGLE:
			test_radiation<half, float>(iterations, verbose, r_size, gridsize,
					blocksize, OUTPUT_R_HALF, INPUT_A_HALF, INPUT_B_HALF);
			break;
		}
		break;

		//DMR REDUNDANCY -------------------------------------------------------
	case DMR:
		switch (parameters.precision) {
		case DOUBLE:
			test_radiation<double, double>(iterations, verbose, r_size,
					gridsize, blocksize, OUTPUT_R_DOUBLE, INPUT_A_DOUBLE,
					INPUT_B_DOUBLE);
			break;
		case SINGLE:
			test_radiation<float, float>(iterations, verbose, r_size, gridsize,
					blocksize, OUTPUT_R_SINGLE, INPUT_A_SINGLE, INPUT_B_SINGLE);
			break;
		case HALF:
			test_radiation<half, half>(iterations, verbose, r_size, gridsize,
					blocksize,
					OUTPUT_R_HALF, INPUT_A_HALF, INPUT_B_HALF);
			break;
		}
		break;

	}

}

int main(int argc, char* argv[]) {

//================== Set block and grid size for MxM kernel
	hipDeviceProp_t prop = GetDevice();
	Parameters parameters(argc, argv);

	parameters.gridsize = prop.multiProcessorCount;
	parameters.blocksize = 256;
	parameters.r_size = gridsize * blocksize * OPS_PER_THREAD_OPERATION;
	parameters.print_details();

//====================================

//================== Init logs
#ifdef LOGS
	std::string test_info = std::string("ops:") + std::to_string(OPS) + " gridsize:" + std::to_string(gridsize) +
		" blocksize:" + std::to_string(blocksize) + " type:" + parameters.test_type_description +
		"-" + parameters.test_precision_description + "-precision hard:" + parameters.hardening;

	std::string test_name = std::string("cuda_") +  parameters.test_precision_description + "_micro-" +
	parameters.test_type_description;
	start_log_file(const_cast<char*>(test_name.c_str()), const_cast<char*>(test_info.c_str()));
#endif

	dmr(parameters);

#ifdef LOGS
	end_log_file();
#endif
	return 0;
}
