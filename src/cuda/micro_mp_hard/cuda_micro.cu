#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <string>
#include <omp.h>
#include <random>
#include <hip/hip_fp16.h>

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

// helper functions
#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#define HALF_ROUND_STYLE 1
#define HALF_ROUND_TIES_TO_EVEN 1
#include "half.hpp"

#undef min
#define min( x, y ) ( (x) < (y) ? (x) : (y) )
#undef max
#define max( x, y ) ( (x) > (y) ? (x) : (y) )

#define BLOCK_SIZE 32

#define DEFAULT_INPUT_SIZE 8192

//#define OPS 1000000000

//===================================== DEFINE TESTED PRECISION
//FOR DMR APPROACH I NEED to use the smallest precision
//as a limit, since it is not possible to store the bigger precisions
//on smaller precisions

//If double it means that DMR will be double and float
//so the limits are the float ones
#if defined(test_precision_double)

//#define OPS_PER_THREAD_OPERATION 1
//#define INPUT_A 1.1945305291614955E+103 // 0x5555555555555555
//#define INPUT_B 3.7206620809969885E-103 // 0x2AAAAAAAAAAAAAAA
//#define OUTPUT_R 4.444444444444444 //0x4011C71C71C71C71
//const char test_precision_description[] = "double";
//typedef double tested_type;
//typedef double tested_type_host;

//#elif defined(test_precision_single)

#define OPS_PER_THREAD_OPERATION 1
#define INPUT_A 1.4660155E+13 // 0x55555555
#define INPUT_B 3.0316488E-13 // 0x2AAAAAAA
#define OUTPUT_R 4.444444 //0x408E38E3
const char test_precision_description[] = "single_and_double";
typedef float tested_type;
typedef float tested_type_host;


//If single it means that DMR will be single and half
//so the limits are the half ones
#elif defined(test_precision_single)

#define OPS_PER_THREAD_OPERATION 1
#define INPUT_A 1.066E+2 // 0x56AA
#define INPUT_B 4.166E-2 // 0x2955
#define OUTPUT_R 4.44 // 0x4471
const char test_precision_description[] = "half_and_single";
typedef half tested_type;
typedef half_float::half tested_type_host;

#endif
//=====================================================

#if defined(test_type_fma) 
const char test_type_description[] = "fma_dmr";
#elif defined(test_type_add) 
const char test_type_description[] = "add_dmr";
#elif defined(test_type_mul)
const char test_type_description[] = "mul_dmr";
#endif

//====================== benchmark+setup configuration
int verbose = 0;

size_t r_size = 0;

int iterations = 100000000; // global loop iteration
//=========================

//================== Host and device matrix ptr's
tested_type_host *R;

tested_type *d_R;
//====================================

#define checkFrameworkErrors(error) __checkFrameworkErrors(error, __LINE__, __FILE__)

void __checkFrameworkErrors(hipError_t error, int line, const char* file) {
	if (error == hipSuccess) {
		return;
	}
	char errorDescription[250];
	snprintf(errorDescription, 250, "CUDA Framework error: %s. Bailing.",
			hipGetErrorString(error));
#ifdef LOGS
	log_error_detail((char *)errorDescription); end_log_file();
#endif
	printf("%s - Line: %d at %s\n", errorDescription, line, file);
	exit(EXIT_FAILURE);
}

hipDeviceProp_t GetDevice() {
//================== Retrieve and set the default CUDA device
	hipDeviceProp_t prop;
	int count = 0;
	printf("Get device:");
	checkFrameworkErrors(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		checkFrameworkErrors(hipGetDeviceProperties(&prop, i));
		printf("Name: %s\n", prop.name);
	}
	int *ndevice;
	int dev = 0;
	ndevice = &dev;
	checkFrameworkErrors(hipGetDevice(ndevice));

	checkFrameworkErrors(hipSetDevice(0));
	checkFrameworkErrors(hipGetDeviceProperties(&prop, 0));
	printf("\ndevice: %d %s\n", *ndevice, prop.name);
	return prop;
}

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void* safe_cudaMalloc(size_t size) {
	void* devicePtr;
	void* goldPtr;
	void* outputPtr;

	// First, alloc DEVICE proposed memory and HOST memory for device memory checking
	checkFrameworkErrors(hipMalloc(&devicePtr, size));
	outputPtr = malloc(size);
	goldPtr = malloc(size);
	if ((outputPtr == NULL) || (goldPtr == NULL)) {
		log_error_detail((char *) "error host malloc");
		end_log_file();
		printf("error host malloc\n");
		exit(EXIT_FAILURE);
	}

	// ===> FIRST PHASE: CHECK SETTING BITS TO 10101010
	checkFrameworkErrors(hipMemset(devicePtr, 0xAA, size));
	memset(goldPtr, 0xAA, size);

	checkFrameworkErrors(
			hipMemcpy(outputPtr, devicePtr, size, hipMemcpyDeviceToHost));
	if (memcmp(outputPtr, goldPtr, size)) {
		// Failed
		free(outputPtr);
		free(goldPtr);
		void* newDevicePtr = safe_cudaMalloc(size);
		checkFrameworkErrors(hipFree(devicePtr));
		return newDevicePtr;
	}
	// ===> END FIRST PHASE

	// ===> SECOND PHASE: CHECK SETTING BITS TO 01010101
	checkFrameworkErrors(hipMemset(devicePtr, 0x55, size));
	memset(goldPtr, 0x55, size);

	checkFrameworkErrors(
			hipMemcpy(outputPtr, devicePtr, size, hipMemcpyDeviceToHost));
	if (memcmp(outputPtr, goldPtr, size)) {
		// Failed
		free(outputPtr);
		free(goldPtr);
		void* newDevicePtr = safe_cudaMalloc(size);
		checkFrameworkErrors(hipFree(devicePtr));
		return newDevicePtr;
	}
	// ===> END SECOND PHASE

	free(outputPtr);
	free(goldPtr);
	return devicePtr;
}

void allocCudaMemory() {
	d_R = (tested_type*) safe_cudaMalloc(r_size * sizeof(tested_type));
//	d_R[1] = (tested_type*) safe_cudaMalloc(r_size * sizeof(tested_type));
//	d_R[2] = (tested_type*) safe_cudaMalloc(r_size * sizeof(tested_type));
}

void freeCudaMemory() {
	checkFrameworkErrors(hipFree(d_R));
//	checkFrameworkErrors(hipFree(d_R[1]));
//	checkFrameworkErrors(hipFree(d_R[2]));
}

void setCudaMemory() {
	checkFrameworkErrors(hipMemset(d_R, 0x00, r_size * sizeof(tested_type)));
//	checkFrameworkErrors(hipMemset(d_R[1], 0x00, r_size * sizeof(tested_type)));
//	checkFrameworkErrors(hipMemset(d_R[2], 0x00, r_size * sizeof(tested_type)));
}


void usage(int argc, char* argv[]) {
	printf("Usage: %s [-iterations=N] [-verbose]\n", argv[0]);
}

// Returns true if no errors are found. False if otherwise.
// Set votedOutput pointer to retrieve the voted matrix
bool checkOutputErrors() {
	int host_errors = 0;
//	int memory_errors = 0;

#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < r_size; i++) {
		register bool checkFlag = true;
		register tested_type_host valGold = tested_type_host(OUTPUT_R);
		register tested_type_host valOutput = R[i];
		if (valGold != valOutput) {
			if (checkFlag) {
#pragma omp critical
				{
					char error_detail[150];
					snprintf(error_detail, 150, "p: [%d], r: %1.20e, e: %1.20e",
							i, (double) valOutput, (double) valGold);
					if (verbose && (host_errors < 10))
						printf("%s\n", error_detail);
#ifdef LOGS
					log_error_detail(error_detail);
#endif
					host_errors++;
				}
			}
		}
	}

	// printf("numErrors:%d", host_errors);

	if (host_errors != 0) {
		printf("#");
#ifdef LOGS
		log_error_count(host_errors);
#endif
		//================== Release device memory to ensure there is no corrupted data on the inputs of the next iteration
		freeCudaMemory();
		//================== Init DEVICE memory
		allocCudaMemory();
		setCudaMemory();
		//====================================
	}
	return host_errors == 0;
}

int main(int argc, char* argv[]) {
//================== Test vars
	int loop2;
	double time;
	double kernel_time, global_time;
	double total_kernel_time, min_kernel_time, max_kernel_time;
//====================================

//================== Read test parameters
	if (checkCmdLineFlag(argc, (const char **) argv, "iterations")) {
		iterations = getCmdLineArgumentInt(argc, (const char **) argv,
				"iterations");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "verbose")) {
		verbose = 1;
	}
//====================================

//================== Set block and grid size for MxM kernel
	hipDeviceProp_t prop = GetDevice();
	int gridsize = prop.multiProcessorCount;
	int blocksize = 256;

	printf("grid size = %d ; block size = %d\n", gridsize, blocksize);

	r_size = gridsize * blocksize * OPS_PER_THREAD_OPERATION;
//====================================

//================== Init logs
#ifdef LOGS
	char test_info[250];
	char test_name[250];
	snprintf(test_info, 250, "ops:%d gridsize:%d blocksize:%d type:%s-%s-precision", OPS, gridsize, blocksize, test_type_description, test_precision_description);
	snprintf(test_name, 250, "cuda_%s_micro-%s", test_precision_description, test_type_description);
	start_log_file(test_name, test_info);
#endif
//====================================

//================== Alloc HOST memory
	R = (tested_type_host*) malloc(r_size * sizeof(tested_type));
//	R[1] = (tested_type_host*) malloc(r_size * sizeof(tested_type));
//	R[2] = (tested_type_host*) malloc(r_size * sizeof(tested_type));

	if (!(R)) {
		printf("Failed on host malloc.\n");
		exit(-3);
	}
//====================================

//================== Init test environment
	// kernel_errors=0;
	total_kernel_time = 0;
	min_kernel_time = UINT_MAX;
	max_kernel_time = 0;
	printf("cuda_micro-%s_%s\n", test_type_description,
			test_precision_description);
	fflush(stdout);
//====================================

//================== Init DEVICE memory
	allocCudaMemory();
	setCudaMemory();
//====================================

	for (loop2 = 0; loop2 < iterations; loop2++) {
		//================== Global test loop

		global_time = mysecond();

		setCudaMemory();

		if (verbose)
			printf(",");

		kernel_time = mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//================== Device computation
#if test_type_fma
		MicroBenchmarkKernel_FMA<<<gridsize, blocksize>>>(d_R);
#elif test_type_add
		MicroBenchmarkKernel_ADD<<<gridsize, blocksize>>>(d_R);
#elif test_type_mul
		MicroBenchmarkKernel_MUL<<<gridsize, blocksize>>>(d_R);
#endif

		checkFrameworkErrors(hipPeekAtLastError());

		checkFrameworkErrors(hipDeviceSynchronize());
		checkFrameworkErrors(hipPeekAtLastError());
		//====================================
#ifdef LOGS
		end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		total_kernel_time += kernel_time;
		min_kernel_time = min(min_kernel_time, kernel_time);
		max_kernel_time = max(max_kernel_time, kernel_time);

		if (verbose)
			printf("Device kernel time for iteration %d: %.3fs\n", loop2,
					kernel_time);

		//================== Gold check
		if (verbose)
			printf(",");

		time = mysecond();

		checkFrameworkErrors(
				hipMemcpy(R, d_R, r_size * sizeof(tested_type),
						hipMemcpyDeviceToHost));

//		checkFrameworkErrors(
//				hipMemcpy(R[1], d_R[1], r_size * sizeof(tested_type),
//						hipMemcpyDeviceToHost));
//
//		checkFrameworkErrors(
//				hipMemcpy(R[2], d_R[2], r_size * sizeof(tested_type),
//						hipMemcpyDeviceToHost));

		checkOutputErrors();
		//====================================

		//================== Console hearthbeat
		printf(".");
		fflush(stdout);
		//====================================

		if (verbose)
			printf("Gold check time for iteration %d: %.3fs\n", loop2,
					mysecond() - time);

		if (verbose) {
			/////////// PERF
			double flops = r_size * OPS * OPS_PER_THREAD_OPERATION;
			double gflops = flops / kernel_time;
			double outputpersec = (double) r_size / kernel_time;
			printf("SIZE:%ld OUTPUT/S:%f FLOPS:%f (GFLOPS:%.2f)\n", r_size,
					outputpersec, gflops, gflops / 1000000000);
			///////////
		}

		if (verbose)
			printf("Iteration #%d time: %.3fs\n\n\n", loop2,
					mysecond() - global_time);
		fflush(stdout);
	}

	double gflops = r_size * OPS * OPS_PER_THREAD_OPERATION / 1000000000; // Bilion FLoating-point OPerationS
	double averageKernelTime = total_kernel_time / iterations;
	printf("\n-- END --\n"
			"Total kernel time: %.3fs\n"
			"Iterations: %d\n"
			"Average kernel time: %.3fs (best: %.3fs ; worst: %.3fs)\n"
			"Average GFLOPs: %.2f (best: %.2f ; worst: %.2f)\n",
			total_kernel_time, iterations, averageKernelTime, min_kernel_time,
			max_kernel_time, gflops / averageKernelTime,
			gflops / min_kernel_time, gflops / max_kernel_time);

	//================== Release device memory
	freeCudaMemory();
	//====================================

	free(R);
//	free(R[1]);
//	free(R[2]);
#ifdef LOGS
	end_log_file();
#endif

	return 0;
}
