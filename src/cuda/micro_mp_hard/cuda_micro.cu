#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <string>
#include <omp.h>
#include <random>
#include <hip/hip_fp16.h>
#include <vector>
#include <sstream>

//#include "include/cuda_utils.h"
#include "include/device_vector.h"

#include "utils.h"

#include "dmr_kernels.h"
#include "none_kernels.h"
#include "Parameters.h"
#include "nonconstant_setup.h"

hipDeviceProp_t get_device() {
//================== Retrieve and set the default CUDA device
	hipDeviceProp_t prop;
	int count = 0;

	rad::checkFrameworkErrors(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		rad::checkFrameworkErrors(hipGetDeviceProperties(&prop, i));
	}
	int *ndevice;
	int dev = 0;
	ndevice = &dev;
	rad::checkFrameworkErrors(hipGetDevice(ndevice));

	rad::checkFrameworkErrors(hipSetDevice(0));
	rad::checkFrameworkErrors(hipGetDeviceProperties(&prop, 0));

	return prop;
}

std::string get_double_representation(double val) {
	std::string output = "";
	if (sizeof(double) == 8) {

		uint64_t int_val;

		memcpy(&int_val, &val, sizeof(double));
		for (uint64_t i = uint64_t(1) << 63; i > 0; i = i / 2) {
			if (int_val & i) {
				output += "1";
			} else {
				output += "0";
			}
		}
	} else {
		std::cerr << "USING more than 64 bits double" << std::endl;
	}
	return output;
}

// Returns the number of errors found
// if no errors were found it returns 0
template<typename half_t, typename real_t, typename output_type = half_t>
int check_output_errors(std::vector<half_t> &R_half_t, std::vector<real_t> &R,
		output_type OUTPUT_R, bool verbose, unsigned long long dmr_errors) {
	int host_errors = 0;
	double gold = double(OUTPUT_R);
	double threshold = -3;
#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < R.size(); i++) {
		double output = double(R[i]);
		double output_inc = double(R_half_t[i]);
		threshold = max(threshold, fabs(output - output_inc));
		if (!cmp(gold, output, 0.000000001)
				|| !cmp(output, output_inc, ZERO_FLOAT)) {
#pragma omp critical
			{
				std::stringstream error_detail;
				error_detail.precision(20);
				error_detail << "p: [" << i << "], r: " << std::scientific
						<< output << ", e: " << gold << " smaller_precision: "
						<< output_inc;

				if (verbose && (host_errors < 10))
					std::cout << error_detail.str() << std::endl;
#ifdef LOGS
				log_error_detail(const_cast<char*>(error_detail.str().c_str()));
#endif
				host_errors++;
			}
		}
	}

	if (dmr_errors != 0) {
		std::stringstream error_detail;
		error_detail << "detected_dmr_errors: " << dmr_errors;
		;
#ifdef LOGS
		log_error_detail(const_cast<char*>(error_detail.str().c_str()));
#endif
	}

	if (host_errors != 0) {
		std::cout << "#";
#ifdef LOGS
		log_error_count(host_errors);
#endif
	}
	return host_errors;
}

template<typename half_t, typename real_t, typename ... TypeArgs>
void test_radiation(Type<TypeArgs...>& type_, Parameters& parameters) {
	std::cout << "Input values " << type_ << std::endl;
#ifdef CHECKBLOCK
	std::cout << "Instruction block checking size " << CHECKBLOCK << std::endl;
#endif
	// Init test environment
	// kernel_errors=0;
	double total_kernel_time = 0;
	double min_kernel_time = UINT_MAX;
	double max_kernel_time = 0;
	//====================================

	// real_t PRECIISON
	std::vector<real_t> host_vector_real_t(parameters.r_size, 0);
	rad::DeviceVector<real_t> device_vector_real_t(parameters.r_size);

	//====================================

	// SECOND PRECISION ONLY IF IT IS DEFINED
	std::vector<half_t> host_vector_inc(parameters.r_size, 0);
	rad::DeviceVector<half_t> device_vector_inc(parameters.r_size);
	//====================================
	// Verbose in csv format
	if (parameters.verbose == false) {
		std::cout << "output/s,iteration,time,output errors,relative errors"
				<< std::endl;
	}

	auto gold = type_.output_r;
	for (int iteration = 0; iteration < parameters.iterations; iteration++) {
		//================== Global test loop
		double kernel_time = rad::mysecond();
#ifdef LOGS
		start_iteration();
#endif
		//================== Device computation
		if (parameters.redundancy == NONE) {
			switch (parameters.micro) {
			case ADD:
				MicroBenchmarkKernel_ADD<real_t> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_real_t.data(),
						type_.output_r, type_.input_a);
				break;
			case MUL:
				MicroBenchmarkKernel_MUL<real_t> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_real_t.data(),
						type_.output_r, type_.input_a);
				break;
			case FMA:
				MicroBenchmarkKernel_FMA<real_t> <<<parameters.grid_size,
						parameters.block_size>>>(device_vector_real_t.data(),
						type_.output_r, type_.input_a, type_.input_b);
				break;
			}

		} else {
			switch (parameters.micro) {
			case ADD: {
				MicroBenchmarkKernel_ADD<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data(),
						type_.output_r, type_.input_a);
				break;
			}
			case MUL: {
				MicroBenchmarkKernel_MUL<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data(),
						type_.output_r, type_.input_a);
				break;
			}
			case FMA: {
				MicroBenchmarkKernel_FMA<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data(),
						type_.output_r, type_.input_a, type_.input_b);
				break;
			}
			case ADDNOTBIASED: {
				MicroBenchmarkKernel_ADDNOTBIASAED<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data(),
						type_.output_r);
				break;
			}
			case MULNOTBIASED: {
				MicroBenchmarkKernel_MULNOTBIASAED<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data());
				gold = 1.10517102313140469505;
				break;
			}
			case FMANOTBIASED: {
				MicroBenchmarkKernel_FMANOTBIASAED<half_t, real_t> <<<
						parameters.grid_size, parameters.block_size>>>(
						device_vector_inc.data(), device_vector_real_t.data());
				gold = 2.50000000001979527653e-01;
				break;
			}
			}
		}

		rad::checkFrameworkErrors(hipPeekAtLastError());
		rad::checkFrameworkErrors(hipDeviceSynchronize());
		rad::checkFrameworkErrors(hipPeekAtLastError());

		kernel_time = rad::mysecond() - kernel_time;

		//====================================
#ifdef LOGS
		end_iteration();
#endif

		total_kernel_time += kernel_time;
		min_kernel_time = std::min(min_kernel_time, kernel_time);
		max_kernel_time = std::max(max_kernel_time, kernel_time);

		//check output
		host_vector_real_t = device_vector_real_t.to_vector();
		host_vector_inc = device_vector_inc.to_vector();
		unsigned long long relative_errors = copy_errors();

		int errors = check_output_errors(host_vector_inc, host_vector_real_t,
				gold, parameters.verbose, relative_errors);

		double outputpersec = double(parameters.r_size) / kernel_time;
		if (parameters.verbose) {
			/////////// PERF
			std::cout << "SIZE:" << parameters.r_size;
			std::cout << " OUTPUT/S:" << outputpersec;
			std::cout << " ITERATION " << iteration;
			std::cout << " time: " << kernel_time;
			std::cout << " output errors: " << errors;
			std::cout << " relative errors: " << relative_errors << std::endl;

		} else {
			// CSV format
			std::cout << outputpersec << ",";
			std::cout << iteration << ",";
			std::cout << kernel_time << ",";
			std::cout << errors << ",";
			std::cout << relative_errors << std::endl;

		}
	}

	if (parameters.verbose) {
		double averageKernelTime = total_kernel_time / parameters.iterations;
		std::cout << std::endl << "-- END --" << std::endl;
		std::cout << "Total kernel time: " << total_kernel_time << std::endl;
		std::cout << "Iterations: " << parameters.iterations << std::endl;
		std::cout << "Average kernel time: " << averageKernelTime << std::endl;
		std::cout << "Best: " << min_kernel_time << std::endl;
		std::cout << "Worst: " << max_kernel_time << std::endl;
	}
}

void dmr(Parameters& parameters) {
	/* DMRMIXED REDUNDANCY -------------------------------------------------- */
	if (parameters.redundancy == DMRMIXED) {

		if (parameters.precision == DOUBLE) {
			Type<float, double> type_;
//			Type<float> type_;
			test_radiation<float, double, float, double>(type_, parameters);
//			test_radiation<float, double>(type_, parameters);

		}

		if (parameters.precision == SINGLE) {
//			Type<half, float> type_;
			Type<half> type_;
//			test_radiation<half, float, half, float>(type_, parameters);
			test_radiation<half, float>(type_, parameters);

		}
	}

	/* DMR REDUNDANCY ------------------------------------------------------- */
	/* NONE REDUNDANCY ------------------------------------------------------ */
	if (parameters.redundancy == NONE || parameters.redundancy == DMR) {
		if (parameters.precision == HALF) {
			Type<half> type_;
			test_radiation<half, half>(type_, parameters);

		}

		if (parameters.precision == SINGLE) {
			Type<float> type_;
			test_radiation<float, float>(type_, parameters);
		}

		if (parameters.precision == DOUBLE) {
			Type<double> type_;
			test_radiation<double, double>(type_, parameters);
		}

	}

}

int main(int argc, char* argv[]) {

//================== Set block and grid size for MxM kernel
	hipDeviceProp_t prop = get_device();
	Parameters parameters(argc, argv, prop.multiProcessorCount, 256);
	if (parameters.verbose) {
		std::cout << "Get device Name: " << prop.name << std::endl;
	}
//================== Init logs
#ifdef LOGS
	std::string test_info = std::string("ops:") + std::to_string(OPS)
	+ " gridsize:" + std::to_string(parameters.grid_size)
	+ " blocksize:" + std::to_string(parameters.block_size) + " type:"
	+ parameters.instruction_str + "-" + parameters.precision_str
	+ "-precision hard:" + parameters.hardening_str;
	test_info += " checkblock:";
#ifdef CHECKBLOCK
	test_info += std::to_string(CHECKBLOCK);
#else
	test_info += std::to_string(OPS);
#endif

	test_info += " nonconst:" + std::to_string(parameters.nonconstant);
	test_info += " numop:" + std::to_string(parameters.operation_num);

	std::string test_name = std::string("cuda_") + parameters.precision_str
	+ "_micro-" + parameters.instruction_str;
	start_log_file(const_cast<char*>(test_name.c_str()),
			const_cast<char*>(test_info.c_str()));

	std::cout << "LOGFILENAME:" << get_log_file_name() << std::endl;

#endif

	std::cout << parameters << std::endl;

	if (parameters.nonconstant) {
		dmr_nonconstant(parameters);
	} else {
		dmr(parameters);
	}
#ifdef LOGS
	end_log_file();
#endif
	return 0;
}
