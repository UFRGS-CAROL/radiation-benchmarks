#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>

#ifdef USE_OMP
#include <omp.h>
#endif

#ifdef PRECISION_HALF
#include <hip/hip_fp16.h>
#include "half.hpp"
#endif

int generate;

// Helper functions
#include "hip/hip_runtime_api.h"
#include "helper_string.h"

#ifdef LOGS
#include "log_helper.h"
#endif
// The timestamp is updated on every log_helper function call.

#ifdef SAFE_MALLOC
#include "safe_memory/safe_memory.h"
#endif

#ifndef DEFAULT_SIM_TIME
#define DEFAULT_SIM_TIME 10000
#endif

//=========== DEFINE TESTED TYPE
#if defined(PRECISION_DOUBLE)
	const char test_precision_description[] = "double";
	typedef double tested_type;
	typedef double tested_type_host;
#elif defined(PRECISION_SINGLE)
	const char test_precision_description[] = "single";
	typedef float tested_type;
	typedef float tested_type_host;
#elif defined(PRECISION_HALF)
	const char test_precision_description[] = "half";
	typedef half tested_type;
	typedef half_float::half tested_type_host;
#else 
	#error TEST TYPE NOT DEFINED OR INCORRECT. USE PRECISION=<double|single|half>.
#endif

#define BLOCK_SIZE 16

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
tested_type_host t_chip(0.0005);
tested_type_host chip_height(0.016);
tested_type_host chip_width(0.016);
/* ambient temperature, assuming no package at all	*/
tested_type_host amb_temp(80.0);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

typedef struct parameters_t {
	int grid_cols, grid_rows;
	tested_type_host *FilesavingTemp1, *FilesavingPower1, *MatrixOut1, *GoldMatrix1;
	tested_type_host *FilesavingTemp2, *FilesavingPower2, *MatrixOut2;
	tested_type_host *FilesavingTemp3, *FilesavingPower3, *MatrixOut3;

	char *tfile, *pfile, *ofile;
	int nstreams;
	int sim_time;
	int pyramid_height;
	int setup_loops;
	int verbose;
	int fault_injection;
	int generate;
} parameters;

void run(int argc, char** argv);
int check_output_errors(parameters *setup_parameters, int streamIdx);

double mysecond() {
	struct timeval tp;
	struct timezone tzp;
	int i = gettimeofday(&tp, &tzp);
	return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

void fatal(const char *s) {
	fprintf(stderr, "error: %s\n", s);
#ifdef LOGS
	if (!generate) {end_log_file();}
#endif
	exit(1);
}

void readInput(parameters *params) {
	// =================== Read all files
	int i, j;
	FILE *ftemp, *fpower, *fgold;
	char str[STR_SIZE];
	float val;
	int num_zeros = 0;
	int num_nans = 0;

	if ((ftemp = fopen(params->tfile, "r")) == 0)
		fatal("The temp file was not opened");
	if ((fpower = fopen(params->pfile, "r")) == 0)
		fatal("The power file was not opened");

	if (!(params->generate))
		if ((fgold = fopen(params->ofile, "rb")) == 0)
			fatal("The gold was not opened");

	for (i = 0; i <= (params->grid_rows) - 1; i++) {
		for (j = 0; j <= (params->grid_cols) - 1; j++) {
			if (!fgets(str, STR_SIZE, ftemp)) {
				fatal("not enough lines in temp file");
			}
			if (feof(ftemp)) {
				printf("[%d,%d] size: %d ", i, j, params->grid_rows);
				fatal("not enough lines in temp file");
			}
			if ((sscanf(str, "%f", &val) != 1))
				fatal("invalid temp file format");

			// =======================
			//HARDENING AGAINST BAD BOARDS
			//-----------------------------------------------------------------------------------

			params->FilesavingTemp1[i * (params->grid_cols) + j] = tested_type_host(val);
			params->FilesavingTemp2[i * (params->grid_cols) + j] = tested_type_host(val);
			params->FilesavingTemp3[i * (params->grid_cols) + j] = tested_type_host(val);

			//-----------------------------------------------------------------------------------

			if (tested_type_host(val) == 0)
				num_zeros++;
			if (isnan(tested_type_host(val)))
				num_nans++;

			if (!fgets(str, STR_SIZE, fpower)) {
				fatal("not enough lines in power file");
			}
			if (feof(fpower))
				fatal("not enough lines in power file");
			if ((sscanf(str, "%f", &val) != 1))
				fatal("invalid power file format");
			// =======================
			//HARDENING AGAINST BAD BOARDS
			//-----------------------------------------------------------------------------------

			params->FilesavingPower1[i * (params->grid_cols) + j] = tested_type_host(val);
			params->FilesavingPower2[i * (params->grid_cols) + j] = tested_type_host(val);
			params->FilesavingPower3[i * (params->grid_cols) + j] = tested_type_host(val);

			//-----------------------------------------------------------------------------------

			if (tested_type_host(val) == 0)
				num_zeros++;
			if (isnan(tested_type_host(val)))
				num_nans++;

			if (!(params->generate)) {
				// fgets(str, STR_SIZE, fgold);
				// if (feof(fgold))
				// 	fatal("not enough lines in gold file");
				// if ((sscanf(str, "%f", &val) != 1))
				// 	fatal("invalid gold file format");
				assert( fread(&(params->GoldMatrix1[i * (params->grid_cols) + j]), sizeof(tested_type), 1, fgold) == 1 );

				// =======================
				//HARDENING AGAINST BAD BOARDS
				//-----------------------------------------------------------------------------------
				// params->GoldMatrix1[i * (params->grid_cols) + j] = val;
				//-----------------------------------------------------------------------------------

			}
		}
	}

	printf("Zeros in the input: %d\n", num_zeros);
	printf("NaNs in the input: %d\n", num_nans);

	// =================== FAULT INJECTION
	if (params->fault_injection) {
		params->FilesavingTemp1[32] = 6.231235;
		params->FilesavingTemp2[32] = 6.231235;
		params->FilesavingTemp3[32] = 6.231235;
		printf("!!!!!!!!! Injected error: FilesavingTemp1[32] = %f\n",
				(double)params->FilesavingTemp1[32]);
		printf("!!!!!!!!! Injected error: FilesavingTemp2[32] = %f\n",
				(double)params->FilesavingTemp2[32]);
		printf("!!!!!!!!! Injected error: FilesavingTemp3[32] = %f\n",
				(double)params->FilesavingTemp3[32]);
	}
	// ==================================

	fclose(ftemp);
	fclose(fpower);
	if (!(params->generate))
		fclose(fgold);
}

void writeOutput(parameters *params) {
	// =================== Write output to gold file
	int i, j;
	FILE *fgold;
	// char str[STR_SIZE];
	int num_zeros = 0;
	int num_nans = 0;

	if ((fgold = fopen(params->ofile, "wb")) == 0)
		fatal("The gold was not opened");

	for (i = 0; i <= (params->grid_rows) - 1; i++) {
		for (j = 0; j <= (params->grid_cols) - 1; j++) {
			// =======================
			//HARDENING AGAINST BAD BOARDS
			//-----------------------------------------------------------------------------------

			if (params->MatrixOut1[i * (params->grid_cols) + j] == 0)
				num_zeros++;

			if (isnan(params->MatrixOut1[i * (params->grid_cols) + j]))
				num_nans++;

			//-----------------------------------------------------------------------------------
			fwrite(&(params->MatrixOut1[i * (params->grid_cols) + j]), sizeof(tested_type), 1, fgold);
			// sprintf(str, "%f\n",
			// 		params->MatrixOut1[i * (params->grid_cols) + j]);
			// fputs(str, fgold);
		}
	}
	fclose(fgold);
	printf("Zeros in the output: %d\n", num_zeros);
	printf("NaNs in the output: %d\n", num_nans);
}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__device__ unsigned long long int is_memory_bad = 0;

__device__ tested_type inline read_voter(tested_type *v1, tested_type *v2, tested_type *v3,
		int offset) {

	register tested_type in1 = v1[offset];
	register tested_type in2 = v2[offset];
	register tested_type in3 = v3[offset];

	if (in1 == in2 || in1 == in3) {
		return in1;
	}

	if (in2 == in3) {
		return in2;
	}

	if (in1 != in2 && in2 != in3 && in1 != in3) {
		atomicAdd(&is_memory_bad, 1);
	}

	return in1;
}

__device__ tested_type inline read_voter_2d(tested_type v1[][BLOCK_SIZE],
	tested_type v2[][BLOCK_SIZE], tested_type v3[][BLOCK_SIZE], int x, int y) {
	register tested_type in1 = v1[x][y];
	register tested_type in2 = v2[x][y];
	register tested_type in3 = v3[x][y];

	if (in1 == in2 || in1 == in3) {
		return in1;
	}

	if (in2 == in3) {
		return in2;
	}

	if (in1 != in2 && in2 != in3 && in1 != in3) {
		atomicAdd(&is_memory_bad, 1);
	}

	return in1;
}

__global__ void calculate_temp(int iteration,  //number of iteration
		//Hardening against bad boards
		tested_type* power1,   //power input
		tested_type* temp_src1,    //temperature input/output
		tested_type* temp_dst1,    //temperature input/output
		//---------------------------------------------------
		tested_type* power2,   //power input
		tested_type* temp_src2,    //temperature input/output
		tested_type* temp_dst2,    //temperature input/output
		//---------------------------------------------------
		tested_type* power3,   //power input
		tested_type* temp_src3,    //temperature input/output
		tested_type* temp_dst3,    //temperature input/output
		//---------------------------------------------------

		int grid_cols,  //Col of grid
		int grid_rows,  //Row of grid
		int border_cols,  // border offset
		int border_rows,  // border offset
		float Cap,      //Capacitance
		float Rx, float Ry, float Rz, float step, float time_elapsed) {

	//----------------------------------------------------
	__shared__ tested_type temp_on_cuda_1[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ tested_type power_on_cuda_1[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ tested_type temp_t_1[BLOCK_SIZE][BLOCK_SIZE]; // saving temporary temperature result

	// //----------------------------------------------------
	// __shared__ tested_type temp_on_cuda_2[BLOCK_SIZE][BLOCK_SIZE];
	// __shared__ tested_type power_on_cuda_2[BLOCK_SIZE][BLOCK_SIZE];
	// __shared__ tested_type temp_t_2[BLOCK_SIZE][BLOCK_SIZE]; // saving temporary temperature result

	// //----------------------------------------------------
	// __shared__ tested_type temp_on_cuda_3[BLOCK_SIZE][BLOCK_SIZE];
	// __shared__ tested_type power_on_cuda_3[BLOCK_SIZE][BLOCK_SIZE];
	// __shared__ tested_type temp_t_3[BLOCK_SIZE][BLOCK_SIZE]; // saving temporary temperature result
	// //---------------------------------------------------

	tested_type amb_temp = 80.0;
	tested_type step_div_Cap;
	tested_type Rx_1, Ry_1, Rz_1;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	step_div_Cap = step / Cap;

	Rx_1 = 1 / Rx;
	Ry_1 = 1 / Ry;
	Rz_1 = 1 / Rz;

	// each block finally computes result for a small block
	// after N iterations.
	// it is the non-overlapping small blocks that cover
	// all the input data

	// calculate the small block size
	int small_block_rows = BLOCK_SIZE - iteration * 2;    //EXPAND_RATE
	int small_block_cols = BLOCK_SIZE - iteration * 2;    //EXPAND_RATE

	// calculate the boundary for the block according to
	// the boundary of its small block
	int blkY = small_block_rows * by - border_rows;
	int blkX = small_block_cols * bx - border_cols;
	int blkYmax = blkY + BLOCK_SIZE - 1;
	int blkXmax = blkX + BLOCK_SIZE - 1;

	// calculate the global thread coordination
	int yidx = blkY + ty;
	int xidx = blkX + tx;

	// load data if it is within the valid input range
	int loadYidx = yidx, loadXidx = xidx;
	int index = grid_cols * loadYidx + loadXidx;

	if (IN_RANGE(loadYidx, 0,
			grid_rows - 1) && IN_RANGE(loadXidx, 0, grid_cols - 1)) {

		//v1
		temp_on_cuda_1[ty][tx] = read_voter(temp_src1, temp_src2, temp_src3,
				index); // Load the temperature data from global memory to shared memory
		power_on_cuda_1[ty][tx] = read_voter(power1, power2, power3, index); // Load the power data from global memory to shared memory

		//v2
		// temp_on_cuda_2[ty][tx] = read_voter(temp_src1, temp_src2, temp_src3,
		// 		index); // Load the temperature data from global memory to shared memory
		// power_on_cuda_2[ty][tx] = read_voter(power1, power2, power3, index); // Load the power data from global memory to shared memory

		// //v3
		// temp_on_cuda_3[ty][tx] = read_voter(temp_src1, temp_src2, temp_src3,
		// 		index); // Load the temperature data from global memory to shared memory
		// power_on_cuda_3[ty][tx] = read_voter(power1, power2, power3, index); // Load the power data from global memory to shared memory

	}
	__syncthreads();

	// effective range within this block that falls within
	// the valid range of the input data
	// used to rule out computation outside the boundary.
	int validYmin = (blkY < 0) ? -blkY : 0;
	int validYmax = (blkYmax > grid_rows - 1) ?
	BLOCK_SIZE - 1 - (blkYmax - grid_rows + 1) :
												BLOCK_SIZE - 1;
	int validXmin = (blkX < 0) ? -blkX : 0;
	int validXmax = (blkXmax > grid_cols - 1) ?
	BLOCK_SIZE - 1 - (blkXmax - grid_cols + 1) :
												BLOCK_SIZE - 1;

	int N = ty - 1;
	int S = ty + 1;
	int W = tx - 1;
	int E = tx + 1;

	N = (N < validYmin) ? validYmin : N;
	S = (S > validYmax) ? validYmax : S;
	W = (W < validXmin) ? validXmin : W;
	E = (E > validXmax) ? validXmax : E;

	bool computed;
	for (int i = 0; i < iteration; i++) {
		computed = false;
		if ( IN_RANGE(tx, i + 1, BLOCK_SIZE-i-2) &&
		IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&
		IN_RANGE(tx, validXmin, validXmax) &&
		IN_RANGE(ty, validYmin, validYmax)) {
			computed = true;
			register tested_type calculated = temp_on_cuda_1[ty][tx]//read_voter_2d(temp_on_cuda_1,
					//temp_on_cuda_2, temp_on_cuda_3, ty, tx)
					+ step_div_Cap
							* (power_on_cuda_1[ty][tx]//read_voter_2d(power_on_cuda_1, power_on_cuda_2,
									//power_on_cuda_3, ty, tx)
									+ (temp_on_cuda_1[S][tx]//read_voter_2d(temp_on_cuda_1,
											//temp_on_cuda_2, temp_on_cuda_3, S,
											//tx)
											+ temp_on_cuda_1[N][tx]//read_voter_2d(temp_on_cuda_1,
													//temp_on_cuda_2,
													//temp_on_cuda_3, N, tx)

											- tested_type(2.0)
													* temp_on_cuda_1[ty][tx]//read_voter_2d(
															//temp_on_cuda_1,
															//temp_on_cuda_2,
															//temp_on_cuda_3, ty,
															//tx)
														) 
														* Ry_1
									+ (temp_on_cuda_1[ty][E]//read_voter_2d(temp_on_cuda_1,
											//temp_on_cuda_2, temp_on_cuda_3, ty,
											//E)
											+ temp_on_cuda_1[ty][W]//read_voter_2d(temp_on_cuda_1,
													//temp_on_cuda_2,
													//temp_on_cuda_3, ty, W)
											- tested_type(2.0)
													* temp_on_cuda_1[ty][tx]//read_voter_2d(
															//temp_on_cuda_1,
															//temp_on_cuda_2,
															//temp_on_cuda_3, ty,
															//tx)
														) 
														* Rx_1
									+ (amb_temp
											- temp_on_cuda_1[ty][tx]//read_voter_2d(temp_on_cuda_1,
													//temp_on_cuda_2,
													//temp_on_cuda_3, ty, tx)
											)
											* Rz_1);
			temp_t_1[ty][tx] = calculated;

			//--------------------------------------------------------------------------------------------------------------------------
			// temp_t_2[ty][tx] = calculated;

			//--------------------------------------------------------------------------------------------------------------------------
			// temp_t_3[ty][tx] = calculated;

		}
		__syncthreads();
		if (i == iteration - 1)
			break;
		if (computed) {	 //Assign the computation range

			temp_on_cuda_1[ty][tx] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3,
					//ty, tx);
			//temp_on_cuda_2[ty][tx] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3,
					//ty, tx);
			//temp_on_cuda_3[ty][tx] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3,
					//ty, tx);

		}
		__syncthreads();
	}

	// update the global memory
	// after the last iteration, only threads coordinated within the
	// small block perform the calculation and switch on ``computed''
	if (computed) {
		//--------------------------------------------------------------------------------------------------------------------------

		temp_dst1[index] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3, ty, tx);
		//--------------------------------------------------------------------------------------------------------------------------

		temp_dst2[index] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3, ty, tx);
		//--------------------------------------------------------------------------------------------------------------------------

		temp_dst3[index] = temp_t_1[ty][tx];//read_voter_2d(temp_t_1, temp_t_2, temp_t_3, ty, tx);
	}
}

/*
 compute N time steps
 */
long long int flops = 0;

int compute_tran_temp(
		//Memory triplication
		tested_type_host *MatrixPower1, tested_type_host *MatrixPower2, tested_type_host *MatrixPower3,
		tested_type_host *MatrixTemp1[2], tested_type_host *MatrixTemp2[2], tested_type_host *MatrixTemp3[2],
		//-------------------------------------------------------------
		int col, int row, int sim_time, int num_iterations, int blockCols,
		int blockRows, int borderCols, int borderRows, hipStream_t stream) {
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(blockCols, blockRows);

	float grid_height = chip_height / row;
	float grid_width = chip_width / col;

	float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	float Rz = t_chip / (K_SI * grid_height * grid_width);

	float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
	float step = PRECISION / max_slope;
	float t;
	float time_elapsed;
	time_elapsed = 0.001;

	int src = 1, dst = 0;
	for (t = 0; t < sim_time; t += num_iterations) {
		int temp = src;
		src = dst;
		dst = temp;
		//printf("[%d]", omp_get_thread_num());
		calculate_temp<<<dimGrid, dimBlock, 0, stream>>>(
				MIN(num_iterations, sim_time - t),
				//memory hardening --------------------------------
				(tested_type*)MatrixPower1, (tested_type*)MatrixTemp1[src],
				(tested_type*)MatrixTemp1[dst], //default copy
				(tested_type*)MatrixPower2, (tested_type*)MatrixTemp2[src],
				(tested_type*)MatrixTemp2[dst], //second copy
				(tested_type*)MatrixPower3, (tested_type*)MatrixTemp3[src],
				(tested_type*)MatrixTemp3[dst], //third copy

				col, row, borderCols, borderRows, Cap, Rx, Ry, Rz, step,
				time_elapsed);
		flops += col * row * MIN(num_iterations, sim_time - t) * 15;
	}
	hipStreamSynchronize(stream);
	return dst;
}

void usage(int argc, char** argv) {
	printf(
			"Usage: %s [-size=N] [-generate] [-sim_time=N] [-input_temp=<path>] [-input_power=<path>] [-gold=<path>] [-iterations=N] [-streams=N] [-debug] [-verbose]\n",
			argv[0]);
}

void getParams(int argc, char** argv, parameters *params) {
	params->nstreams = 1;
	params->sim_time = DEFAULT_SIM_TIME;
	params->pyramid_height = 1;
	params->setup_loops = 10000000;
	params->verbose = 0;
	params->fault_injection = 0;
	params->generate = 0;
	generate = 0;

	if (argc < 2) {
		usage(argc, argv);
		exit (EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "size")) {
		params->grid_cols = getCmdLineArgumentInt(argc, (const char **) argv,
				"size");
		params->grid_rows = params->grid_cols;

		if ((params->grid_cols <= 0) || (params->grid_cols % 16 != 0)) {
			printf("Invalid input size given on the command-line: %d\n",
					params->grid_cols);
			exit (EXIT_FAILURE);
		}
	} else {
		usage(argc, argv);
		exit (EXIT_FAILURE);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "generate")) {
		params->generate = 1;
		generate = 1;
		printf(
				">> Output will be written to file. Only stream #0 output will be considered.\n");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "sim_time")) {
		params->sim_time = getCmdLineArgumentInt(argc, (const char **) argv,
				"sim_time");

		if (params->sim_time < 1) {
			printf("Invalid sim_time given on the command-line: %d\n",
					params->sim_time);
			exit (EXIT_FAILURE);
		}
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_temp")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_temp",
				&(params->tfile));
	} else {
		params->tfile = new char[100];
		snprintf(params->tfile, 100, "temp_%i", params->grid_rows);
		printf("Using default input_temp path: %s\n", params->tfile);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "input_power")) {
		getCmdLineArgumentString(argc, (const char **) argv, "input_power",
				&(params->pfile));
	} else {
		params->pfile = new char[100];
		snprintf(params->pfile, 100, "power_%i", params->grid_rows);
		printf("Using default input_power path: %s\n", params->pfile);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "gold")) {
		getCmdLineArgumentString(argc, (const char **) argv, "gold",
				&(params->ofile));
	} else {
		params->ofile = new char[100];
		snprintf(params->ofile, 100, "gold_%s_%i_%i", test_precision_description, params->grid_rows,
				params->sim_time);
		printf("Using default gold path: %s\n", params->ofile);
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "iterations")) {
		params->setup_loops = getCmdLineArgumentInt(argc, (const char **) argv,
				"iterations");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "streams")) {
		params->nstreams = getCmdLineArgumentInt(argc, (const char **) argv,
				"streams");
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "verbose")) {
		params->verbose = 1;
	}

	if (checkCmdLineFlag(argc, (const char **) argv, "debug")) {
		params->fault_injection = 1;
		printf("!! Will be injected an input error\n");
	}
}

int main(int argc, char** argv) {
	printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

	run(argc, argv);

	return EXIT_SUCCESS;
}

void run(int argc, char** argv) {
	//int streamIdx;
	double timestamp, globaltime;

	parameters *setupParams = (parameters *) malloc(sizeof(parameters));

	// =============== Get setup parameters from command line
	getParams(argc, argv, setupParams);
	// =======================

	// ===============  pyramid parameters
# define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
	int borderCols = (setupParams->pyramid_height) * EXPAND_RATE / 2;
	int borderRows = (setupParams->pyramid_height) * EXPAND_RATE / 2;
	int smallBlockCol = BLOCK_SIZE - (setupParams->pyramid_height) * EXPAND_RATE;
	int smallBlockRow = BLOCK_SIZE - (setupParams->pyramid_height) * EXPAND_RATE;
	int blockCols = setupParams->grid_cols / smallBlockCol
			+ ((setupParams->grid_cols % smallBlockCol == 0) ? 0 : 1);
	int blockRows = setupParams->grid_rows / smallBlockRow
			+ ((setupParams->grid_rows % smallBlockRow == 0) ? 0 : 1);

	int size = (setupParams->grid_cols) * (setupParams->grid_rows);
	// =======================
	//HARDENING AGAINST BAD BOARDS
	//-----------------------------------------------------------------------------------
	setupParams->FilesavingTemp1 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->FilesavingPower1 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->MatrixOut1 = (tested_type_host *) calloc(size, sizeof(tested_type));
	setupParams->GoldMatrix1 = (tested_type_host *) calloc(size, sizeof(tested_type));

	setupParams->FilesavingTemp2 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->FilesavingPower2 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->MatrixOut2 = (tested_type_host *) calloc(size, sizeof(tested_type));

	setupParams->FilesavingTemp3 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->FilesavingPower3 = (tested_type_host *) malloc(size * sizeof(tested_type));
	setupParams->MatrixOut3 = (tested_type_host *) calloc(size, sizeof(tested_type));

	if (!(setupParams->FilesavingPower1) || !(setupParams->FilesavingTemp1)
			|| !(setupParams->MatrixOut1) || !(setupParams->GoldMatrix1))
		fatal("unable to allocate memory");

	if (!(setupParams->FilesavingPower2) || !(setupParams->FilesavingTemp2)
			|| !(setupParams->MatrixOut2))
		fatal("unable to allocate memory");

	if (!(setupParams->FilesavingPower3) || !(setupParams->FilesavingTemp3)
			|| !(setupParams->MatrixOut3))
		fatal("unable to allocate memory");

	//-----------------------------------------------------------------------------------

	printf("cudaTripHOTSPOT\nstreams:%d size:%d pyramidHeight:%d simTime:%d\n",
			setupParams->nstreams, setupParams->grid_rows,
			setupParams->pyramid_height, setupParams->sim_time);

#ifdef LOGS
	char test_info[150];
	char test_name[90];
	snprintf(test_info, 150, "streams:%d precision:%s size:%d pyramidHeight:%d simTime:%d", setupParams -> nstreams, test_precision_description, setupParams -> grid_rows, setupParams -> pyramid_height, setupParams -> sim_time);
	snprintf(test_name, 90, "cuda_trip_hotspot_%s", test_precision_description);
	if (!(setupParams->generate)) start_log_file(test_name, test_info);
#endif

	timestamp = mysecond();
	readInput(setupParams);
	if (setupParams->verbose)
		printf("readInput time: %.4fs\n", mysecond() - timestamp);
	fflush (stdout);

	hipStream_t *streams = (hipStream_t *) malloc(
			(setupParams->nstreams) * sizeof(hipStream_t));

	// =======================
	//HARDENING AGAINST BAD BOARDS
	//-----------------------------------------------------------------------------------
	tested_type_host *MatrixTemp1[setupParams->nstreams][2],
			*MatrixPower1[setupParams->nstreams];

	tested_type_host *MatrixTemp2[setupParams->nstreams][2],
			*MatrixPower2[setupParams->nstreams];

	tested_type_host *MatrixTemp3[setupParams->nstreams][2],
			*MatrixPower3[setupParams->nstreams];
	//-----------------------------------------------------------------------------------
	for (int streamIdx = 0; streamIdx < (setupParams->nstreams); streamIdx++) {
		checkCudaErrors(
				hipStreamCreateWithFlags(&(streams[streamIdx]),
						hipStreamNonBlocking));

#ifdef SAFE_MALLOC
		// =======================
		//HARDENING AGAINST BAD BOARDS
		//-----------------------------------------------------------------------------------
		for(int z = 0; z < 2; z++) {
			safe_cuda_malloc_cover((void**)&(MatrixTemp1[streamIdx][z]), sizeof(tested_type)*size);
//		safe_cuda_malloc_cover((void**)&(MatrixTemp1[streamIdx][1]), sizeof(float)*size);

			safe_cuda_malloc_cover((void**)&(MatrixTemp2[streamIdx][z]), sizeof(tested_type)*size);
//		safe_cuda_malloc_cover((void**)&(MatrixTemp2[streamIdx][1]), sizeof(float)*size);

			safe_cuda_malloc_cover((void**)&(MatrixTemp3[streamIdx][z]), sizeof(tested_type)*size);
//		safe_cuda_malloc_cover((void**)&(MatrixTemp3[streamIdx][1]), sizeof(float)*size);
		}
		//-----------------------------------------------------------------------------------

#else
		// =======================
		//HARDENING AGAINST BAD BOARDS
		//-----------------------------------------------------------------------------------
		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp1[streamIdx][0]),
						sizeof(tested_type) * size));
		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp1[streamIdx][1]),
						sizeof(tested_type) * size));

		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp2[streamIdx][0]),
						sizeof(tested_type) * size));
		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp2[streamIdx][1]),
						sizeof(tested_type) * size));

		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp3[streamIdx][0]),
						sizeof(tested_type) * size));
		checkCudaErrors(
				hipMalloc((void**) &(MatrixTemp3[streamIdx][1]),
						sizeof(tested_type) * size));

		//-----------------------------------------------------------------------------------

#endif

#ifdef SAFE_MALLOC
		// =======================
		//HARDENING AGAINST BAD BOARDS
		//-----------------------------------------------------------------------------------
		safe_cuda_malloc_cover((void**)&(MatrixPower1[streamIdx]), sizeof(tested_type)*size);
		safe_cuda_malloc_cover((void**)&(MatrixPower2[streamIdx]), sizeof(tested_type)*size);
		safe_cuda_malloc_cover((void**)&(MatrixPower3[streamIdx]), sizeof(tested_type)*size);

		//-----------------------------------------------------------------------------------

#else
		// =======================
		//HARDENING AGAINST BAD BOARDS
		//-----------------------------------------------------------------------------------

		checkCudaErrors(
				hipMalloc((void**) &(MatrixPower1[streamIdx]),
						sizeof(tested_type) * size));

		checkCudaErrors(
				hipMalloc((void**) &(MatrixPower2[streamIdx]),
						sizeof(tested_type) * size));

		checkCudaErrors(
				hipMalloc((void**) &(MatrixPower3[streamIdx]),
						sizeof(tested_type) * size));
		//-----------------------------------------------------------------------------------

#endif

	}
	for (int loop1 = 0; loop1 < (setupParams->setup_loops); loop1++) {
		globaltime = mysecond();

		int ret[setupParams->nstreams];
//		// =======================
//		//HARDENING AGAINST BAD BOARDS
//		//-----------------------------------------------------------------------------------
//		float *MatrixTemp1[setupParams->nstreams][2],
//				*MatrixPower1[setupParams->nstreams];
//
//		float *MatrixTemp2[setupParams->nstreams][2],
//				*MatrixPower2[setupParams->nstreams];
//
//		float *MatrixTemp3[setupParams->nstreams][2],
//				*MatrixPower3[setupParams->nstreams];
//		//-----------------------------------------------------------------------------------

		timestamp = mysecond();
		for (int streamIdx = 0; streamIdx < (setupParams->nstreams);
				streamIdx++) {
//			CHANGED
//			checkCudaErrors(
//					hipStreamCreateWithFlags(&(streams[streamIdx]),
//							hipStreamNonBlocking));
//
//#ifdef SAFE_MALLOC
//			// =======================
//			//HARDENING AGAINST BAD BOARDS
//			//-----------------------------------------------------------------------------------
//
//			safe_cuda_malloc_cover((void**)&(MatrixTemp1[streamIdx][0]), sizeof(float)*size);
//			safe_cuda_malloc_cover((void**)&(MatrixTemp1[streamIdx][1]), sizeof(float)*size);
//
//			safe_cuda_malloc_cover((void**)&(MatrixTemp2[streamIdx][0]), sizeof(float)*size);
//			safe_cuda_malloc_cover((void**)&(MatrixTemp2[streamIdx][1]), sizeof(float)*size);
//
//			safe_cuda_malloc_cover((void**)&(MatrixTemp3[streamIdx][0]), sizeof(float)*size);
//			safe_cuda_malloc_cover((void**)&(MatrixTemp3[streamIdx][1]), sizeof(float)*size);
//			//-----------------------------------------------------------------------------------
//
//#else
//			// =======================
//			//HARDENING AGAINST BAD BOARDS
//			//-----------------------------------------------------------------------------------
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp1[streamIdx][0]),
//							sizeof(float) * size));
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp1[streamIdx][1]),
//							sizeof(float) * size));
//
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp2[streamIdx][0]),
//							sizeof(float) * size));
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp2[streamIdx][1]),
//							sizeof(float) * size));
//
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp3[streamIdx][0]),
//							sizeof(float) * size));
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixTemp3[streamIdx][1]),
//							sizeof(float) * size));
//
//			//-----------------------------------------------------------------------------------
//
//#endif
			// =======================
			//HARDENING AGAINST BAD BOARDS
			//-----------------------------------------------------------------------------------

			hipMemcpy(MatrixTemp1[streamIdx][0], setupParams->FilesavingTemp1,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);
			hipMemset(MatrixTemp2[streamIdx][1], 0.0, sizeof(tested_type) * size);

			hipMemcpy(MatrixTemp2[streamIdx][0], setupParams->FilesavingTemp2,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);
			hipMemset(MatrixTemp2[streamIdx][1], 0.0, sizeof(tested_type) * size);

			hipMemcpy(MatrixTemp3[streamIdx][0], setupParams->FilesavingTemp3,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);
			hipMemset(MatrixTemp3[streamIdx][1], 0.0, sizeof(tested_type) * size);

			//-----------------------------------------------------------------------------------
//
//#ifdef SAFE_MALLOC
//			// =======================
//			//HARDENING AGAINST BAD BOARDS
//			//-----------------------------------------------------------------------------------
//			safe_cuda_malloc_cover((void**)&(MatrixPower1[streamIdx]), sizeof(float)*size);
//			safe_cuda_malloc_cover((void**)&(MatrixPower2[streamIdx]), sizeof(float)*size);
//			safe_cuda_malloc_cover((void**)&(MatrixPower3[streamIdx]), sizeof(float)*size);
//
//			//-----------------------------------------------------------------------------------
//
//#else
//			// =======================
//			//HARDENING AGAINST BAD BOARDS
//			//-----------------------------------------------------------------------------------
//
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixPower1[streamIdx]),
//							sizeof(float) * size));
//
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixPower2[streamIdx]),
//							sizeof(float) * size));
//
//			checkCudaErrors(
//					hipMalloc((void**) &(MatrixPower3[streamIdx]),
//							sizeof(float) * size));
//			//-----------------------------------------------------------------------------------
//
//#endif
			//-----------------------------------------------------------------------------------

			hipMemcpy(MatrixPower1[streamIdx], setupParams->FilesavingPower1,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);

			hipMemcpy(MatrixPower2[streamIdx], setupParams->FilesavingPower2,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);

			hipMemcpy(MatrixPower3[streamIdx], setupParams->FilesavingPower3,
					sizeof(tested_type) * size, hipMemcpyHostToDevice);

			//-----------------------------------------------------------------------------------

		}
		if (setupParams->verbose)
			printf("[Iteration #%i] GPU prepare time: %.4fs\n", loop1,
					mysecond() - timestamp);

		//printf("Start computing the transient temperature\n");
		double kernel_time = mysecond();
#ifdef LOGS
		if (!(setupParams->generate)) start_iteration();
#endif
#pragma omp parallel for
		for (int streamIdx = 0; streamIdx < (setupParams->nstreams);
				streamIdx++) {
			unsigned long long int is_memory_bad_host = 0;

			hipMemcpyToSymbol(HIP_SYMBOL("is_memory_bad"), &is_memory_bad_host,
					sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);

			ret[streamIdx] = compute_tran_temp(
					///compute_tran_temp(
					//float *MatrixPower1, float *MatrixPower2, float *MatrixPower3,
					MatrixPower1[streamIdx], MatrixPower2[streamIdx],
					MatrixPower3[streamIdx],
					//float *MatrixTemp1[2], float *MatrixTemp2[2], float *MatrixTemp3[2],
					MatrixTemp1[streamIdx], MatrixTemp2[streamIdx],
					MatrixTemp3[streamIdx],
					//-----------------------------
					setupParams->grid_cols, setupParams->grid_rows,
					setupParams->sim_time, setupParams->pyramid_height,
					blockCols, blockRows, borderCols, borderRows,
					streams[streamIdx]);
		}
		for (int streamIdx = 0; streamIdx < (setupParams->nstreams);
				streamIdx++) {
			hipStreamSynchronize(streams[streamIdx]);
		}
#ifdef LOGS
		if (!(setupParams->generate)) end_iteration();
#endif
		kernel_time = mysecond() - kernel_time;

		/////////// PERF
		if (setupParams->verbose) {

			double outputpersec = (double) ((setupParams->grid_rows
					* setupParams->grid_rows * setupParams->nstreams)
					/ kernel_time);
			printf("[Iteration #%i] kernel time: %.4lfs\n", loop1, kernel_time);
			printf(
					"[Iteration #%i] SIZE:%d OUTPUT/S:%f FLOPS: %f (GFLOPS: %.2f)\n",
					loop1, setupParams->grid_rows, outputpersec,
					(double) flops / kernel_time,
					(double) flops / (kernel_time * 1000000000));
		}
		flops = 0;

		//printf("Ending simulation\n");
		timestamp = mysecond();
		int kernel_errors = 0;
		if (setupParams->generate) {
			// =======================
			//HARDENING AGAINST BAD BOARDS
			//-----------------------------------------------------------------------------------
			hipMemcpy(setupParams->MatrixOut1, MatrixTemp1[0][ret[0]],
					sizeof(tested_type) * size, hipMemcpyDeviceToHost);
			hipMemcpy(setupParams->MatrixOut2, MatrixTemp2[0][ret[0]],
					sizeof(tested_type) * size, hipMemcpyDeviceToHost);
			hipMemcpy(setupParams->MatrixOut3, MatrixTemp3[0][ret[0]],
					sizeof(tested_type) * size, hipMemcpyDeviceToHost);
			//-----------------------------------------------------------------------------------

			writeOutput(setupParams);
		} else {
			for (int streamIdx = 0; streamIdx < (setupParams->nstreams);
					streamIdx++) {
				hipMemcpy(setupParams->MatrixOut1,
						MatrixTemp1[streamIdx][ret[streamIdx]],
						sizeof(tested_type) * size, hipMemcpyDeviceToHost);
				hipMemcpy(setupParams->MatrixOut2,
						MatrixTemp2[streamIdx][ret[streamIdx]],
						sizeof(tested_type) * size, hipMemcpyDeviceToHost);
				hipMemcpy(setupParams->MatrixOut3,
						MatrixTemp3[streamIdx][ret[streamIdx]],
						sizeof(tested_type) * size, hipMemcpyDeviceToHost);

				check_output_errors(setupParams, streamIdx);
			}
			//			for (streamIdx = 0; streamIdx < setupParams->nstreams;
//					streamIdx++) {
//				memset(setupParams->MatrixOut, 0, sizeof(float) * size);
//				hipMemcpy(setupParams->MatrixOut,
//						MatrixTemp[streamIdx][ret[streamIdx]],
//						sizeof(float) * size, hipMemcpyDeviceToHost);
//				char error_detail[150];
//				if (memcmp(setupParams->GoldMatrix, setupParams->MatrixOut,
//						sizeof(float) * size)) {
//#pragma omp parallel for
//					for (int i = 0; i < (setupParams->grid_rows); i++) {
//						register float *ptrGold = &(setupParams->GoldMatrix[i
//								* (setupParams->grid_rows) + 0]);
//						register float *ptrOut = &(setupParams->MatrixOut[i
//								* (setupParams->grid_rows) + 0]);
//						for (int j = 0; j < (setupParams->grid_cols); j++) {
//							if (ptrGold[j] != ptrOut[j])
//#pragma omp critical
//									{
//								kernel_errors++;
//								snprintf(error_detail, 150,
//										"stream: %d, p: [%d, %d], r: %1.16e, e: %1.16e",
//										streamIdx, i, j,
//										setupParams->GoldMatrix[i
//												* (setupParams->grid_rows) + j],
//										setupParams->MatrixOut[i
//												* (setupParams->grid_rows) + j]);
//								printf(
//										"stream: %d, p: [%d, %d], r: %1.16e, e: %1.16e\n",
//										streamIdx, i, j,
//										setupParams->GoldMatrix[i
//												* (setupParams->grid_rows) + j],
//										setupParams->MatrixOut[i
//												* (setupParams->grid_rows) + j]);
//#ifdef LOGS
//								if (!(setupParams->generate)) log_error_detail(error_detail);
//#endif
//							}
//						}
//					}
//				}
//			}
		}

		if (setupParams->verbose)
			printf("[Iteration #%i] Gold check time: %.4fs\n", loop1,
					mysecond() - timestamp);
		if (kernel_errors != 0)
			printf("ERROR detected.\n");
		else
			printf(".");

		fflush(stdout);

//		for (streamIdx = 0; streamIdx < setupParams->nstreams; streamIdx++) {
//			// =======================
//			//HARDENING AGAINST BAD BOARDS
//			//-----------------------------------------------------------------------------------
//
//			hipFree(MatrixPower1[streamIdx]);
//			hipFree(MatrixTemp1[streamIdx][0]);
//			hipFree(MatrixTemp1[streamIdx][1]);
//
//			hipFree(MatrixPower2[streamIdx]);
//			hipFree(MatrixTemp2[streamIdx][0]);
//			hipFree(MatrixTemp2[streamIdx][1]);
//
//			hipFree(MatrixPower3[streamIdx]);
//			hipFree(MatrixTemp3[streamIdx][0]);
//			hipFree(MatrixTemp3[streamIdx][1]);
//
////			hipStreamDestroy(streams[streamIdx]);
//
//			//-----------------------------------------------------------------------------------
//
//		}
		if (setupParams->verbose)
			printf("[Iteration #%i] elapsed time: %.4fs\n", loop1,
					mysecond() - globaltime);
	}

	for (int streamIdx = 0; streamIdx < setupParams->nstreams; streamIdx++) {
		// =======================
		//HARDENING AGAINST BAD BOARDS
		//-----------------------------------------------------------------------------------
		hipFree(MatrixPower1[streamIdx]);
		hipFree(MatrixTemp1[streamIdx][0]);
		hipFree(MatrixTemp1[streamIdx][1]);

		hipFree(MatrixPower2[streamIdx]);
		hipFree(MatrixTemp2[streamIdx][0]);
		hipFree(MatrixTemp2[streamIdx][1]);

		hipFree(MatrixPower3[streamIdx]);
		hipFree(MatrixTemp3[streamIdx][0]);
		hipFree(MatrixTemp3[streamIdx][1]);
		hipStreamDestroy(streams[streamIdx]);
		//-----------------------------------------------------------------------------------
	}

#ifdef LOGS
	if (!(setupParams->generate)) end_log_file();
#endif
}

// Returns true if no errors are found. False if otherwise.
// Set votedOutput pointer to retrieve the voted matrix
int check_output_errors(parameters *setup_parameters, int streamIdx) {
	int host_errors = 0;
	int memory_errors = 0;

	unsigned long long int is_memory_bad_host = 0;
	hipMemcpyFromSymbol(&is_memory_bad_host, HIP_SYMBOL("is_memory_bad"), sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
	if(is_memory_bad_host != 0) {
		char info_detail[150];
		snprintf(info_detail, 150,
				"b: is_memory_bad: %llu",
				is_memory_bad_host);
		if (setup_parameters->verbose)
			printf("%s\n", info_detail);

#ifdef LOGS
		if (!generate) 
			log_info_detail(info_detail);
#endif
		memory_errors++;
	}

//#pragma omp parallel for shared(host_errors)
	for (int i = 0; i < setup_parameters->grid_rows; i++) {
		for (int j = 0; j < setup_parameters->grid_cols; j++) {
			int index = i * setup_parameters->grid_rows + j;
			register bool checkFlag = true;
			register tested_type_host valGold = setup_parameters->GoldMatrix1[index];
			register tested_type_host valOutput1 = setup_parameters->MatrixOut1[index];
			register tested_type_host valOutput2 = setup_parameters->MatrixOut2[index];
			register tested_type_host valOutput3 = setup_parameters->MatrixOut3[index];
			register tested_type_host valOutput = valOutput1;

			if ((valOutput1 != valOutput2) || (valOutput2 != valOutput3)) {
#pragma omp critical
				{
					char info_detail[150];
					snprintf(info_detail, 150,
							"stream: %d, m: [%d, %d], r0: %1.20e, r1: %1.20e, r2: %1.20e",
							streamIdx, i, j, 
							(double)valOutput1, (double)valOutput2, (double)valOutput3);
					if (setup_parameters->verbose && (memory_errors < 10))
						printf("%s\n", info_detail);

#ifdef LOGS
					if (!generate)
					log_info_detail(info_detail);
#endif
					memory_errors += 1;
				}
				if ((valOutput1 != valOutput2) && (valOutput2 != valOutput3)) {
					// All 3 values diverge
					if (valOutput1 == valGold) {
						valOutput = valOutput1;
					} else if (valOutput2 == valGold) {
						valOutput = valOutput2;
					} else if (valOutput3 == valGold) {
						valOutput = valOutput3;
					} else {
						// NO VALUE MATCHES THE GOLD AND ALL 3 DIVERGE!
						printf("#");
					}
				} else if (valOutput2 == valOutput3) {
					// Only value 1 diverge
					valOutput = valOutput2;
				} else if (valOutput1 == valOutput3) {
					// Only value 2 diverge
					valOutput = valOutput1;
				} else if (valOutput1 == valOutput2) {
					// Only value 3 diverge
					valOutput = valOutput1;
				}
			}
//			if (votedOutput != NULL)
//				votedOutput[i] = valOutput;
			// if ((fabs((tested_type_host)(valOutput-valGold)/valGold) > 1e-10)||(fabs((tested_type_host)(valOutput-valGold)/valGold) > 1e-10)) {
//			if (!(generate && (votedOutput != NULL))) {
			if ((valGold != valOutput) && checkFlag) {
#pragma omp critical
				{
					char error_detail[150];
					snprintf(error_detail, 150,
							"stream: %d, p: [%d, %d], r: %1.20e, e: %1.20e",
							streamIdx, i, j, 
							(double)valOutput, (double)valGold);
					if (setup_parameters->verbose && (host_errors < 10))
						printf("%s\n", error_detail);
#ifdef LOGS
					if (!generate)
					log_error_detail(error_detail);
#endif
					host_errors++;

				}
			}
		}
	}

#ifdef LOGS
	if (!generate) {
		log_info_count(memory_errors);
		log_error_count(host_errors);
	}
#endif
	if (memory_errors != 0) printf("M");
	if (host_errors != 0) printf("#");

	return (host_errors == 0) && (memory_errors == 0);
}
