#include "hip/hip_runtime.h"
/**
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

////////////////////////////////////////////////////////////////////////////////
//
//  QUICKSORT.CU
//
//  Implementation of a parallel quicksort in CUDA. It comes in
//  several parts:
//
//  1. A small-set insertion sort. We do this on any set with <=32 elements
//  2. A partitioning kernel, which - given a pivot - separates an input
//     array into elements <=pivot, and >pivot. Two quicksorts will then
//     be launched to resolve each of these.
//  3. A quicksort co-ordinator, which figures out what kernels to launch
//     and when.
//
////////////////////////////////////////////////////////////////////////////////
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <sys/time.h>
#include <omp.h>
#include "cdpQuicksort.h"

#define INPUTSIZE 100000000

double mysecond()
{
   struct timeval tp;
   struct timezone tzp;
   int i = gettimeofday(&tp,&tzp);
   return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

////////////////////////////////////////////////////////////////////////////////
// Inline PTX call to return index of highest non-zero bit in a word
////////////////////////////////////////////////////////////////////////////////
static __device__ __forceinline__ unsigned int __qsflo(unsigned int word)
{
    unsigned int ret;
    asm volatile("bfind.u32 %0, %1;" : "=r"(ret) : "r"(word));
    return ret;
}

////////////////////////////////////////////////////////////////////////////////
//
//  ringbufAlloc
//
//  Allocates from a ringbuffer. Allows for not failing when we run out
//  of stack for tracking the offset counts for each sort subsection.
//
//  We use the atomicMax trick to allow out-of-order retirement. If we
//  hit the size limit on the ringbuffer, then we spin-wait for people
//  to complete.
//
////////////////////////////////////////////////////////////////////////////////
template< typename T >
static __device__ T *ringbufAlloc(qsortRingbuf *ringbuf)
{
    // Wait for there to be space in the ring buffer. We'll retry only a fixed
    // number of times and then fail, to avoid an out-of-memory deadlock.
    unsigned int loop = 10000;

    while (((ringbuf->head - ringbuf->tail) >= ringbuf->stacksize) && (loop-- > 0));

    if (loop == 0)
        return NULL;

    // Note that the element includes a little index book-keeping, for freeing later.
    unsigned int index = atomicAdd((unsigned int *) &ringbuf->head, 1);
    T *ret = (T *)(ringbuf->stackbase) + (index & (ringbuf->stacksize-1));
    ret->index = index;

    return ret;
}

////////////////////////////////////////////////////////////////////////////////
//
//  ringBufFree
//
//  Releases an element from the ring buffer. If every element is released
//  up to and including this one, we can advance the tail to indicate that
//  space is now available.
//
////////////////////////////////////////////////////////////////////////////////
template< typename T >
static __device__ void ringbufFree(qsortRingbuf *ringbuf, T *data)
{
    unsigned int index = data->index;       // Non-wrapped index to free
    unsigned int count = atomicAdd((unsigned int *)&(ringbuf->count), 1) + 1;
    unsigned int max = atomicMax((unsigned int *)&(ringbuf->max), index + 1);

    // Update the tail if need be. Note we update "max" to be the new value in ringbuf->max
    if (max < (index+1)) max = index+1;

    if (max == count)
        atomicMax((unsigned int *)&(ringbuf->tail), count);
}

////////////////////////////////////////////////////////////////////////////////
//
//  qsort_warp
//
//  Simplest possible implementation, does a per-warp quicksort with no inter-warp
//  communication. This has a high atomic issue rate, but the rest should actually
//  be fairly quick because of low work per thread.
//
//  A warp finds its section of the data, then writes all data <pivot to one
//  buffer and all data >pivot to the other. Atomics are used to get a unique
//  section of the buffer.
//
//  Obvious optimisation: do multiple chunks per warp, to increase in-flight loads
//  and cover the instruction overhead.
//
////////////////////////////////////////////////////////////////////////////////
__global__ void qsort_warp(unsigned *indata,
                           unsigned *outdata,
                           unsigned int offset,
                           unsigned int len,
                           qsortAtomicData *atomicData,
                           qsortRingbuf *atomicDataStack,
                           unsigned int source_is_indata,
                           unsigned int depth)
{
    // Find my data offset, based on warp ID
    unsigned int thread_id = threadIdx.x + (blockIdx.x << QSORT_BLOCKSIZE_SHIFT);
    //unsigned int warp_id = threadIdx.x >> 5;   // Used for debug only
    unsigned int lane_id = threadIdx.x & (warpSize-1);

    // Exit if I'm outside the range of sort to be done
    if (thread_id >= len)
        return;

    //
    // First part of the algorithm. Each warp counts the number of elements that are
    // greater/less than the pivot.
    //
    // When a warp knows its count, it updates an atomic counter.
    //

    // Read in the data and the pivot. Arbitrary pivot selection for now.
    unsigned pivot = indata[offset + len/2];
    unsigned data  = indata[offset + thread_id];

    // Count how many are <= and how many are > pivot.
    // If all are <= pivot then we adjust the comparison
    // because otherwise the sort will move nothing and
    // we'll iterate forever.
    unsigned int greater = (data > pivot);
    unsigned int gt_mask = __ballot(greater);

    if (gt_mask == 0)
    {
        greater = (data >= pivot);
        gt_mask = __ballot(greater);    // Must re-ballot for adjusted comparator
    }

    unsigned int lt_mask = __ballot(!greater);
    unsigned int gt_count = __popc(gt_mask);
    unsigned int lt_count = __popc(lt_mask);

    // Atomically adjust the lt_ and gt_offsets by this amount. Only one thread need do this. Share the result using shfl
    unsigned int lt_offset, gt_offset;

    if (lane_id == 0)
    {
        if (lt_count > 0)
            lt_offset = atomicAdd((unsigned int *) &atomicData->lt_offset, lt_count);

        if (gt_count > 0)
            gt_offset = len - (atomicAdd((unsigned int *) &atomicData->gt_offset, gt_count) + gt_count);
    }

    lt_offset = __shfl((int)lt_offset, 0);   // Everyone pulls the offsets from lane 0
    gt_offset = __shfl((int)gt_offset, 0);

    __syncthreads();

    // Now compute my own personal offset within this. I need to know how many
    // threads with a lane ID less than mine are going to write to the same buffer
    // as me. We can use popc to implement a single-operation warp scan in this case.
    unsigned lane_mask_lt;
    asm("mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt));
    unsigned int my_mask = greater ? gt_mask : lt_mask;
    unsigned int my_offset = __popc(my_mask & lane_mask_lt);

    // Move data.
    my_offset += greater ? gt_offset : lt_offset;
    outdata[offset + my_offset] = data;


    // Count up if we're the last warp in. If so, then Kepler will launch the next
    // set of sorts directly from here.
    if (lane_id == 0)
    {
        // Count "elements written". If I wrote the last one, then trigger the next qsorts
        unsigned int mycount = lt_count + gt_count;

        if (atomicAdd((unsigned int *) &atomicData->sorted_count, mycount) + mycount == len)
        {
            // We're the last warp to do any sorting. Therefore it's up to us to launch the next stage.
            unsigned int lt_len = atomicData->lt_offset;
            unsigned int gt_len = atomicData->gt_offset;

            hipStream_t lstream, rstream;
            hipStreamCreateWithFlags(&lstream, hipStreamNonBlocking);
            hipStreamCreateWithFlags(&rstream, hipStreamNonBlocking);

            // Begin by freeing our atomicData storage. It's better for the ringbuffer algorithm
            // if we free when we're done, rather than re-using (makes for less fragmentation).
            ringbufFree<qsortAtomicData>(atomicDataStack, atomicData);

            // Exceptional case: if "lt_len" is zero, then all values in the batch
            // are equal. We are then done (may need to copy into correct buffer, though)
            if (lt_len == 0)
            {
                if (source_is_indata)
                    hipMemcpyAsync(indata+offset, outdata+offset, gt_len*sizeof(unsigned), hipMemcpyDeviceToDevice, lstream);

                return;
            }

            // Start with lower half first
            if (lt_len > BITONICSORT_LEN)
            {
                // If we've exceeded maximum depth, fall through to backup big_bitonicsort
                if (depth >= QSORT_MAXDEPTH)
                {
                    // The final bitonic stage sorts in-place in "outdata". We therefore
                    // re-use "indata" as the out-of-range tracking buffer. For (2^n)+1
                    // elements we need (2^(n+1)) bytes of oor buffer. The backup qsort
                    // buffer is at least this large when sizeof(QTYPE) >= 2.
                    big_bitonicsort<<< 1, BITONICSORT_LEN, 0, lstream >>>(outdata, source_is_indata ? indata : outdata, indata, offset, lt_len);
                }
                else
                {
                    // Launch another quicksort. We need to allocate more storage for the atomic data.
                    if ((atomicData = ringbufAlloc<qsortAtomicData>(atomicDataStack)) == NULL)
                        printf("Stack-allocation error. Failing left child launch.\n");
                    else
                    {
                        atomicData->lt_offset = atomicData->gt_offset = atomicData->sorted_count = 0;
                        unsigned int numblocks = (unsigned int)(lt_len+(QSORT_BLOCKSIZE-1)) >> QSORT_BLOCKSIZE_SHIFT;
                        qsort_warp<<< numblocks, QSORT_BLOCKSIZE, 0, lstream >>>(outdata, indata, offset, lt_len, atomicData, atomicDataStack, !source_is_indata, depth+1);
                    }
                }
            }
            else if (lt_len > 1)
            {
                // Final stage uses a bitonic sort instead. It's important to
                // make sure the final stage ends up in the correct (original) buffer.
                // We launch the smallest power-of-2 number of threads that we can.
                unsigned int bitonic_len = 1 << (__qsflo(lt_len-1U)+1);
                bitonicsort<<< 1, bitonic_len, 0, lstream >>>(outdata, source_is_indata ? indata : outdata, offset, lt_len);
            }
            // Finally, if we sorted just one single element, we must still make
            // sure that it winds up in the correct place.
            else if (source_is_indata && (lt_len == 1))
                indata[offset] = outdata[offset];

            if (hipPeekAtLastError() != hipSuccess)
                printf("Left-side launch fail: %s\n", hipGetErrorString(hipGetLastError()));


            // Now the upper half.
            if (gt_len > BITONICSORT_LEN)
            {
                // If we've exceeded maximum depth, fall through to backup big_bitonicsort
                if (depth >= QSORT_MAXDEPTH)
                    big_bitonicsort<<< 1, BITONICSORT_LEN, 0, rstream >>>(outdata, source_is_indata ? indata : outdata, indata, offset+lt_len, gt_len);
                else
                {
                    // Allocate new atomic storage for this launch
                    if ((atomicData = ringbufAlloc<qsortAtomicData>(atomicDataStack)) == NULL)
                        printf("Stack allocation error! Failing right-side launch.\n");
                    else
                    {
                        atomicData->lt_offset = atomicData->gt_offset = atomicData->sorted_count = 0;
                        unsigned int numblocks = (unsigned int)(gt_len+(QSORT_BLOCKSIZE-1)) >> QSORT_BLOCKSIZE_SHIFT;
                        qsort_warp<<< numblocks, QSORT_BLOCKSIZE, 0, rstream >>>(outdata, indata, offset+lt_len, gt_len, atomicData, atomicDataStack, !source_is_indata, depth+1);
                    }
                }
            }
            else if (gt_len > 1)
            {
                unsigned int bitonic_len = 1 << (__qsflo(gt_len-1U)+1);
                bitonicsort<<< 1, bitonic_len, 0, rstream >>>(outdata, source_is_indata ? indata : outdata, offset+lt_len, gt_len);
            }
            else if (source_is_indata && (gt_len == 1))
                indata[offset+lt_len] = outdata[offset+lt_len];

            if (hipPeekAtLastError() != hipSuccess)
                printf("Right-side launch fail: %s\n", hipGetErrorString(hipGetLastError()));
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//
//  run_quicksort
//
//  Host-side code to run the Kepler version of quicksort. It's pretty
//  simple, because all launch control is handled on the device via CDP.
//
//  All parallel quicksorts require an equal-sized scratch buffer. This
//  must be passed in ahead of time.
//
//  Returns the time elapsed for the sort.
//
////////////////////////////////////////////////////////////////////////////////
float run_quicksort_cdp(unsigned *gpudata, unsigned *scratchdata, unsigned int count, hipStream_t stream)
{
    unsigned int stacksize = QSORT_STACK_ELEMS;

    // This is the stack, for atomic tracking of each sort's status
    qsortAtomicData *gpustack;
    checkCudaErrors(hipMalloc((void **)&gpustack, stacksize * sizeof(qsortAtomicData)));
    checkCudaErrors(hipMemset(gpustack, 0, sizeof(qsortAtomicData)));     // Only need set first entry to 0

    // Create the memory ringbuffer used for handling the stack.
    // Initialise everything to where it needs to be.
    qsortRingbuf buf;
    qsortRingbuf *ringbuf;
    checkCudaErrors(hipMalloc((void **)&ringbuf, sizeof(qsortRingbuf)));
    buf.head = 1;           // We start with one allocation
    buf.tail = 0;
    buf.count = 0;
    buf.max = 0;
    buf.stacksize = stacksize;
    buf.stackbase = gpustack;
    checkCudaErrors(hipMemcpy(ringbuf, &buf, sizeof(buf), hipMemcpyHostToDevice));


    // Timing events...
    hipEvent_t ev1, ev2;
    checkCudaErrors(hipEventCreate(&ev1));
    checkCudaErrors(hipEventCreate(&ev2));
    checkCudaErrors(hipEventRecord(ev1));

    // Now we trivially launch the qsort kernel
    if (count > BITONICSORT_LEN)
    {
        unsigned int numblocks = (unsigned int)(count+(QSORT_BLOCKSIZE-1)) >> QSORT_BLOCKSIZE_SHIFT;
        qsort_warp<<< numblocks, QSORT_BLOCKSIZE, 0, stream >>>(gpudata, scratchdata, 0U, count, gpustack, ringbuf, true, 0);
    }
    else
    {
        bitonicsort<<< 1, BITONICSORT_LEN >>>(gpudata, gpudata, 0, count);
    }

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(ev2));
    checkCudaErrors(hipDeviceSynchronize());

    float elapse=0.0f;

    if (hipPeekAtLastError() != hipSuccess)
        printf("Launch failure: %s\n", hipGetErrorString(hipGetLastError()));
    else
        checkCudaErrors(hipEventElapsedTime(&elapse, ev1, ev2));

    // Sanity check that the stack allocator is doing the right thing
    checkCudaErrors(hipMemcpy(&buf, ringbuf, sizeof(*ringbuf), hipMemcpyDeviceToHost));

    if (count > BITONICSORT_LEN && buf.head != buf.tail)
    {
        printf("Stack allocation error!\nRingbuf:\n");
        printf("\t head = %u\n", buf.head);
        printf("\t tail = %u\n", buf.tail);
        printf("\tcount = %u\n", buf.count);
        printf("\t  max = %u\n", buf.max);
    }

    // Release our stack data once we're done
    checkCudaErrors(hipFree(ringbuf));
    checkCudaErrors(hipFree(gpustack));

    return elapse;
}

int dataRead(unsigned *data, unsigned *gold, unsigned int size, char *inputName, char *goldName, int verbose)
{
  FILE *finput, *fgold;

  if (!(finput = fopen(inputName, "rb")))
  { // GENERATE INPUT
    unsigned *ndata = new unsigned[INPUTSIZE];
    printf("Input was not detected. Generating, this will take a long time...");
    for (unsigned int i=0; i<INPUTSIZE; i++)
    {
      // Build data 8 bits at a time
      ndata[i] = 0;
      char *ptr = (char *)&(ndata[i]);

      for (unsigned j=0; j<sizeof(unsigned); j++)
      {
          // Easy-to-read data in debug mode
          if (verbose)
          {
              *ptr++ = (char)(rand() % 10);
              break;
          }

          *ptr++ = (char)(rand() & 255);
      }
    }
    if (!(finput = fopen(inputName, "wb")))
    { printf("Warning! Couldn't write the input to file, proceeding anyway...\n"); }
    else
    {
      fwrite(ndata, INPUTSIZE*sizeof(unsigned), 1 , finput);
      fclose(finput);
    }
    memcpy(data, ndata, size*sizeof(unsigned));
    printf("Done.\n");
  }
  else
  { // READ INPUT
    printf("Reading input...");
    double timer = mysecond();
    fread(data, size*sizeof(unsigned), 1 , finput);
    fclose(finput);
    printf("Done in %.2fs\n", mysecond() - timer);
  }

  if (verbose)
  {
    for (unsigned int i=0; i<size; i++)
    {
      if (i && !(i%32))
          printf("\n        ");

      printf("%u ", data[i]);
    }
  }

  if (!(fgold = fopen(goldName, "rb")))
  { // Gold does not exist, propagate this information so that gold should be generated.
    printf("Gold does not exist. yet.\n");
    return 0;
  }
  else
  {
    fread(gold, size*sizeof(unsigned), 1 , fgold);
    fclose(fgold);
  }
  return 1;
}

void goldWrite(unsigned *gold, unsigned int size, char *goldName)
{
  FILE *fgold;
  if (!(fgold = fopen(goldName, "wb")))
  {
    printf("Gold file could not be open in wb mode.\n");
  }
  else
  {
    fwrite(gold, size*sizeof(unsigned), 1, fgold);
    fclose(fgold);
  }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
int run_qsort(unsigned int size, int seed, int loop, int verbose, char *inputName, char *goldName)
{
    if (seed > 0)
        srand(seed);

    // Create and set up our test
    unsigned *gpudata, *scratchdata;
    checkCudaErrors(hipMalloc((void **)&gpudata, size*sizeof(unsigned)));
    checkCudaErrors(hipMalloc((void **)&scratchdata, size*sizeof(unsigned)));

    int goldstatus; // Wether gold is ready=1 or not=0

    // Create CPU data.
    unsigned *data = new unsigned[size];
    unsigned *outdata = new unsigned[size];
    unsigned *gold = new unsigned[size];

    goldstatus = dataRead(data, gold, size, inputName, goldName, verbose);

    unsigned int loop1; // Test loop iterator
    for (loop1 = 0; loop1 < loop; loop1++)
    {

        checkCudaErrors(hipMemcpy(gpudata, data, size*sizeof(unsigned), hipMemcpyHostToDevice));

        // So we're now populated and ready to go! We size our launch as
        // blocks of up to BLOCKSIZE threads, and appropriate grid size.
        // One thread is launched per element.
        float elapse;
        elapse = run_quicksort_cdp(gpudata, scratchdata, size, NULL);

        //run_bitonicsort<SORTTYPE>(gpudata, scratchdata, size, verbose);
        checkCudaErrors(hipDeviceSynchronize());

        // Copy back the data and verify correct sort
        checkCudaErrors(hipMemcpy(outdata, gpudata, size*sizeof(unsigned), hipMemcpyDeviceToHost));

        if (verbose)
        {
            printf("Output: ");

            for (unsigned int i=0; i<size; i++)
            {
                if (i && !(i%32)) printf("\n        ");

                printf("%u ", outdata[i]);
            }

            printf("\n");
        }

        if (!goldstatus)
        {// Write gold to file
          printf("Assuming no errors occurred, i'm writing output to gold file %s...\n", goldName);
          goldWrite(outdata, size, goldName);
          memcpy(gold, outdata, size*sizeof(unsigned));
          goldstatus = 1; // gold ready
          printf("Done.\n");
        }
        double timer;
        unsigned int check;

        printf("Checking for errors (cpu compute)...");
        timer = mysecond();
        #pragma omp parallel for
        for (check=1; check<size; check++)
        {
            if (outdata[check] < outdata[check-1])
            {
                printf("FAILED at element: %d\n", check);
                //break;
            }
        }
        printf("Done in %.2fs.\n", mysecond() - timer);
        printf("Checking for errors (gold comparison)...");
        timer = mysecond();
        #pragma omp parallel for
        for (check=0; check<size; check++)
        {
            if (gold[check] != outdata[check])
            {
                printf("FAILED at element: %d\n", check);
            }
        }
        printf("Done in %.2fs.\n", mysecond() - timer);

      /*  if (check != size)
        {
            printf("    cdpAdvancedQuicksort FAILED\n");
            exit(EXIT_FAILURE);
        }
        else
            printf("    cdpAdvancedQuicksort PASSED\n");*/

        // Display the time between event recordings
        printf("Sorted %u elems in %.3f ms (%.3f Melems/sec)\n", size, elapse, (float)size/(elapse*1000.0f));
        fflush(stdout);
    }

    // Release everything and we're done
    checkCudaErrors(hipFree(scratchdata));
    checkCudaErrors(hipFree(gpudata));
    delete(data);
    return 0;
}

static void usage()
{
    printf("Syntax: qsort [-size=<num>] [-seed=<num>] [-verbose] [-input=<inputfile>] [-gold=<goldfile>]\n");
    printf("If loop_step is non-zero, will run from 1->array_len in steps of loop_step\n");
    printf("The default goldfile file name is quickSortGold[SIZE], the default inputfile is quickSortInput100000000\n");
    printf("If goldfile/inputfile does not exist, it will be generated\n");
}


// Host side entry
int main(int argc, char *argv[])
{
    int size = 5000;     // TODO: make this 1e6
    unsigned int seed = 100;    // TODO: make this 0
    int loop = 1;
    int verbose = 0;
    char *goldName, *inputName;

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "h"))
    {
        usage();
        printf("&&&& cdpAdvancedQuicksort WAIVED\n");
        exit(EXIT_WAIVED);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        size = getCmdLineArgumentInt(argc, (const char **)argv, "size");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "seed"))
    {
        seed = getCmdLineArgumentInt(argc, (const char **)argv, "seed");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "loop"))
    {
        loop = getCmdLineArgumentInt(argc, (const char **)argv, "loop");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        verbose = 1;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "gold"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold", &goldName);
    }
    else
    {
        goldName = new char[100];
        snprintf(goldName, 100, "quickSortGold%i", (signed int)size);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "input"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "input", &inputName);
    }
    else
    {
        inputName = new char[100];
        snprintf(inputName, 100, "quickSortInput%i", (signed int)INPUTSIZE);
    }

    // Get device properties
    int cuda_device = findCudaDevice(argc, (const char **)argv);
    hipDeviceProp_t properties;
    checkCudaErrors(hipGetDeviceProperties(&properties, cuda_device));
    int cdpCapable = (properties.major == 3 && properties.minor >= 5) || properties.major >=4;

    printf("GPU device %s has compute capabilities (SM %d.%d)\n", properties.name, properties.major, properties.minor);

    if (!cdpCapable)
    {
        printf("cdpAdvancedQuicksort requires SM 3.5 or higher to use CUDA Dynamic Parallelism.  Exiting...\n");
        exit(EXIT_WAIVED);
    }

    printf("Running qsort on %d elements with seed %d, on %s\n", size, seed, properties.name);

    run_qsort(size, seed, loop, verbose, inputName, goldName);

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    checkCudaErrors(hipDeviceReset());
    exit(EXIT_SUCCESS);
}
