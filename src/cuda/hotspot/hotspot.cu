#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#include <omp.h>

int generate;

// Helper functions
#include "hip/hip_runtime_api.h"
#include "helper_string.h"

#ifdef LOGS
#include "log_helper.h"
#endif

#define BLOCK_SIZE 16

#define STR_SIZE 256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5

/* chip parameters	*/
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all	*/
float amb_temp = 80.0;

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

typedef struct parameters_t {
    int grid_cols, grid_rows;
    float *FilesavingTemp, *FilesavingPower, *MatrixOut, *GoldMatrix;
    char *tfile, *pfile, *ofile;
    int nstreams;
    int sim_time;
    int pyramid_height;
    int setup_loops;
    int verbose;
    int fault_injection;
    int generate;
} parameters;

double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

void fatal(const char *s)
{
    fprintf(stderr, "error: %s\n", s);
    #ifdef LOGS
    	if (!generate) { end_log_file(); }
    #endif
    exit(1);
}

void readInput(parameters *params)
{
    // =================== Read all files
    int i,j;
    FILE *ftemp, *fpower, *fgold;
    char str[STR_SIZE];
    float val;
	int num_zeros = 0;
	int num_nans = 0;

    if( (ftemp  = fopen(params->tfile, "r" )) ==0 )
        fatal( "The temp file was not opened" );
    if( (fpower  = fopen(params->pfile, "r" )) ==0 )
        fatal( "The power file was not opened" );

    if (!(params->generate))
        if( (fgold  = fopen(params->ofile, "r" )) ==0 )
            fatal( "The gold was not opened" );

    for (i=0; i <= (params->grid_rows)-1; i++) {
        for (j=0; j <= (params->grid_cols)-1; j++) {
            fgets(str, STR_SIZE, ftemp);
            if (feof(ftemp)) { printf("[%d,%d] size: %d ", i, j, params->grid_rows);
                fatal("not enough lines in temp file"); }
            if ((sscanf(str, "%f", &val) != 1))
                fatal("invalid temp file format");
            params->FilesavingTemp[i*(params->grid_cols)+j] = val;
			if (val==0) num_zeros++;
			if (isnan(val)) num_nans++;

            fgets(str, STR_SIZE, fpower);
            if (feof(fpower))
                fatal("not enough lines in power file");
            if ((sscanf(str, "%f", &val) != 1))
                fatal("invalid power file format");
            params->FilesavingPower[i*(params->grid_cols)+j] = val;
			if (val==0) num_zeros++;
			if (isnan(val)) num_nans++;

            if (!(params->generate)) {
                fgets(str, STR_SIZE, fgold);
                if (feof(fgold))
                    fatal("not enough lines in gold file");
                if ((sscanf(str, "%f", &val) != 1))
                    fatal("invalid gold file format");
                params->GoldMatrix[i*(params->grid_cols)+j] = val;
            }
        }
    }

	printf("Zeros in the input: %d\n", num_zeros);
	printf("NaNs in the input: %d\n", num_nans);

    // =================== FAULT INJECTION
    if (params->fault_injection) {
        params->FilesavingTemp[32] = 6.231235;
        printf("!!!!!!!!! Injected error: FilesavingTemp[32] = %f\n", params->FilesavingTemp[32]);
    }
    // ==================================


    fclose(ftemp);
    fclose(fpower);
    if (!(params->generate))
        fclose(fgold);
}

void writeOutput(parameters *params)
{
    // =================== Write output to gold file
    int i,j;
    FILE *fgold;
    char str[STR_SIZE];
	int num_zeros = 0;
	int num_nans = 0;

    if( (fgold  = fopen(params->ofile, "w" )) ==0 )
        fatal( "The gold was not opened" );

    for (i=0; i <= (params->grid_rows)-1; i++) {
        for (j=0; j <= (params->grid_cols)-1; j++) {
			if (params->MatrixOut[i*(params->grid_cols)+j] == 0) num_zeros++;
			if (isnan(params->MatrixOut[i*(params->grid_cols)+j])) num_nans++;
            sprintf(str, "%f\n", params->MatrixOut[i*(params->grid_cols)+j]);
            fputs(str,fgold);
        }
    }
    fclose(fgold);
	printf("Zeros in the output: %d\n", num_zeros);
	printf("NaNs in the output: %d\n", num_nans);
}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void calculate_temp(int iteration,  //number of iteration
float *power,   //power input
float *temp_src,    //temperature input/output
float *temp_dst,    //temperature input/output
int grid_cols,  //Col of grid
int grid_rows,  //Row of grid
int border_cols,  // border offset
int border_rows,  // border offset
float Cap,      //Capacitance
float Rx,
float Ry,
float Rz,
float step,
float time_elapsed)
{

    __shared__ float temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temparary temperature result

    float amb_temp = 80.0;
    float step_div_Cap;
    float Rx_1,Ry_1,Rz_1;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx=threadIdx.x;
    int ty=threadIdx.y;

    step_div_Cap=step/Cap;

    Rx_1=1/Rx;
    Ry_1=1/Ry;
    Rz_1=1/Rz;

    // each block finally computes result for a small block
    // after N iterations.
    // it is the non-overlapping small blocks that cover
    // all the input data

    // calculate the small block size
    int small_block_rows = BLOCK_SIZE-iteration*2;//EXPAND_RATE
    int small_block_cols = BLOCK_SIZE-iteration*2;//EXPAND_RATE

    // calculate the boundary for the block according to
    // the boundary of its small block
    int blkY = small_block_rows*by-border_rows;
    int blkX = small_block_cols*bx-border_cols;
    int blkYmax = blkY+BLOCK_SIZE-1;
    int blkXmax = blkX+BLOCK_SIZE-1;

    // calculate the global thread coordination
    int yidx = blkY+ty;
    int xidx = blkX+tx;

    // load data if it is within the valid input range
    int loadYidx=yidx, loadXidx=xidx;
    int index = grid_cols*loadYidx+loadXidx;

    if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
        temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from global memory to shared memory
        power_on_cuda[ty][tx] = power[index];// Load the power data from global memory to shared memory
    }
    __syncthreads();

    // effective range within this block that falls within
    // the valid range of the input data
    // used to rule out computation outside the boundary.
    int validYmin = (blkY < 0) ? -blkY : 0;
    int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
    int validXmin = (blkX < 0) ? -blkX : 0;
    int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

    int N = ty-1;
    int S = ty+1;
    int W = tx-1;
    int E = tx+1;

    N = (N < validYmin) ? validYmin : N;
    S = (S > validYmax) ? validYmax : S;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    bool computed;
    for (int i=0; i<iteration ; i++){
        computed = false;
        if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
        IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
        IN_RANGE(tx, validXmin, validXmax) && \
        IN_RANGE(ty, validYmin, validYmax) ) {
            computed = true;
            temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] +
            (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 +
            (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 +
            (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);

        }
        __syncthreads();
        if(i==iteration-1)
            break;
        if(computed)	 //Assign the computation range
            temp_on_cuda[ty][tx]= temp_t[ty][tx];
        __syncthreads();
    }

    // update the global memory
    // after the last iteration, only threads coordinated within the
    // small block perform the calculation and switch on ``computed''
    if (computed){
        temp_dst[index]= temp_t[ty][tx];
    }
}

/*
compute N time steps
*/
long long int flops = 0;

int compute_tran_temp(float *MatrixPower,float *MatrixTemp[2], int col, int row, \
int sim_time, int num_iterations, int blockCols, int blockRows, int borderCols, int borderRows, hipStream_t stream)
{
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(blockCols, blockRows);

    float grid_height = chip_height / row;
    float grid_width = chip_width / col;

    float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
    float Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
    float Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
    float Rz = t_chip / (K_SI * grid_height * grid_width);

    float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    float step = PRECISION / max_slope;
    float t;
    float time_elapsed;
    time_elapsed=0.001;

    int src = 1, dst = 0;
    for (t = 0; t < sim_time; t+=num_iterations)
    {
        int temp = src;
        src = dst;
        dst = temp;
        //printf("[%d]", omp_get_thread_num());
        calculate_temp<<<dimGrid, dimBlock, 0, stream>>>(MIN(num_iterations, sim_time-t), MatrixPower,MatrixTemp[src],MatrixTemp[dst],\
        col,row,borderCols, borderRows, Cap,Rx,Ry,Rz,step,time_elapsed);
        flops += col * row * MIN(num_iterations, sim_time-t) * 15;
    }
    hipStreamSynchronize(stream);
    return dst;
}

void usage(int argc, char** argv) {
    printf("Usage: %s -size=N [-generate] [-sim_time=N] [-temp_file=<path>] [-power_file=<path>] [-gold_file=<path>] [-iterations=N] [-streams=N] [-debug] [-verbose]\n", argv[0]);
}

void getParams(int argc, char** argv, parameters *params)
{
    params -> nstreams = 1;
    params -> sim_time = 1000;
    params -> pyramid_height = 1;
    params -> setup_loops = 10000000;
    params -> verbose = 0;
    params -> fault_injection = 0;
    params -> generate = 0;
    generate = 0;

    if (argc<2) {
        usage(argc, argv);
        exit (EXIT_FAILURE);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "size"))
    {
        params -> grid_cols = getCmdLineArgumentInt(argc, (const char **)argv, "size");
        params -> grid_rows = params -> grid_cols;

        if ((params -> grid_cols  <= 0)||(params -> grid_cols  % 16 != 0))
        {
            printf("Invalid input size given on the command-line: %d\n", params -> grid_cols );
            exit(EXIT_FAILURE);
        }
    }
    else
    {
        usage(argc, argv);
        exit(EXIT_FAILURE);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "generate"))
    {
        params -> generate = 1;
	generate = 1;
        printf(">> Output will be written to file. Only stream #0 output will be considered.\n");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "sim_time"))
    {
        params -> sim_time = getCmdLineArgumentInt(argc, (const char **)argv, "sim_time");

        if (params -> sim_time < 1)
        {
            printf("Invalid sim_time given on the command-line: %d\n", params -> sim_time );
            exit(EXIT_FAILURE);
        }
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "temp_file"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "temp_file", &(params -> tfile));
    }
    else
    {
        params -> tfile = new char[100];
        snprintf(params -> tfile, 100, "temp_%i", params -> grid_rows);
        printf("Using default temp_file path: %s\n", params -> tfile);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "power_file"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "power_file", &(params -> pfile));
    }
    else
    {
        params -> pfile = new char[100];
        snprintf(params -> pfile, 100, "power_%i", params -> grid_rows);
        printf("Using default power_file path: %s\n", params -> pfile);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "gold_file"))
    {
        getCmdLineArgumentString(argc, (const char **)argv, "gold_file", &(params -> ofile));
    }
    else
    {
        params -> ofile = new char[100];
        snprintf(params -> ofile, 100, "gold_float_%i_%i", params -> grid_rows, params -> sim_time);
        printf("Using default gold path: %s\n", params -> ofile);
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "iterations"))
    {
        params -> setup_loops = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "streams"))
    {
        params -> nstreams = getCmdLineArgumentInt(argc, (const char **)argv, "streams");
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "verbose"))
    {
        params -> verbose = 1;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "debug"))
    {
        params -> fault_injection = 1;
        printf("!! Will be injected an input error\n");
    }
}

int main(int argc, char** argv)
{
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    int streamIdx;
    double timestamp, globaltime;

    parameters *setupParams = (parameters *)malloc(sizeof(parameters));

    // =============== Get setup parameters from command line
    getParams(argc, argv, setupParams);
    // =======================

    // ===============  pyramid parameters
    # define EXPAND_RATE 2// add one iteration will extend the pyramid base by 2 per each borderline
    int borderCols = (setupParams->pyramid_height)*EXPAND_RATE/2;
    int borderRows = (setupParams->pyramid_height)*EXPAND_RATE/2;
    int smallBlockCol = BLOCK_SIZE-(setupParams->pyramid_height)*EXPAND_RATE;
    int smallBlockRow = BLOCK_SIZE-(setupParams->pyramid_height)*EXPAND_RATE;
    int blockCols = setupParams->grid_cols/smallBlockCol+((setupParams->grid_cols%smallBlockCol==0)?0:1);
    int blockRows = setupParams->grid_rows/smallBlockRow+((setupParams->grid_rows%smallBlockRow==0)?0:1);

    int size = (setupParams -> grid_cols) * (setupParams -> grid_rows);
    // =======================

    setupParams -> FilesavingTemp = (float *) malloc(size *sizeof(float));
    setupParams -> FilesavingPower = (float *) malloc(size *sizeof(float));
    setupParams -> MatrixOut = (float *) calloc (size, sizeof(float));
    setupParams -> GoldMatrix = (float *) calloc (size, sizeof(float));

    if( !(setupParams -> FilesavingPower) || !(setupParams -> FilesavingTemp) || !(setupParams -> MatrixOut) || !(setupParams -> GoldMatrix))
        fatal("unable to allocate memory");

    printf("cudaHOTSPOT\nstreams:%d size:%d pyramidHeight:%d simTime:%d\n", setupParams -> nstreams, setupParams -> grid_rows, setupParams -> pyramid_height, setupParams -> sim_time);

    #ifdef LOGS
        char test_info[90];
        snprintf(test_info, 90, "streams:%d size:%d pyramidHeight:%d simTime:%d", setupParams -> nstreams, setupParams -> grid_rows, setupParams -> pyramid_height, setupParams -> sim_time);
        if (!(setupParams->generate)) start_log_file("cudaHotspot", test_info);
    #endif

    timestamp = mysecond();
    readInput(setupParams);
    if (setupParams->verbose) printf("readInput time: %.4fs\n", mysecond()-timestamp);fflush(stdout);

    hipStream_t *streams = (hipStream_t *) malloc((setupParams -> nstreams) * sizeof(hipStream_t));
    for ( int loop1=0 ; loop1<(setupParams->setup_loops) ; loop1++)
    {
        globaltime = mysecond();

        int ret[setupParams->nstreams];
        float *MatrixTemp[setupParams->nstreams][2], *MatrixPower[setupParams->nstreams];

        timestamp = mysecond();
        for (streamIdx = 0; streamIdx < (setupParams-> nstreams); streamIdx++) {
            checkCudaErrors( hipStreamCreateWithFlags(&(streams[streamIdx]), hipStreamNonBlocking) );

            checkCudaErrors( hipMalloc((void**)&(MatrixTemp[streamIdx][0]), sizeof(float)*size) );
            checkCudaErrors( hipMalloc((void**)&(MatrixTemp[streamIdx][1]), sizeof(float)*size) );
            hipMemcpy(MatrixTemp[streamIdx][0], setupParams->FilesavingTemp, sizeof(float)*size, hipMemcpyHostToDevice);
			hipMemset(MatrixTemp[streamIdx][1], 0.0, sizeof(float)*size);

            checkCudaErrors( hipMalloc((void**)&(MatrixPower[streamIdx]), sizeof(float)*size) );
            hipMemcpy(MatrixPower[streamIdx], setupParams->FilesavingPower, sizeof(float)*size, hipMemcpyHostToDevice);
        }
        if (setupParams->verbose) printf("[Iteration #%i] GPU prepare time: %.4fs\n", loop1, mysecond()-timestamp);

        //printf("Start computing the transient temperature\n");
        double kernel_time = mysecond();
        #ifdef LOGS
            if (!(setupParams->generate)) start_iteration();
        #endif
        #pragma omp parallel for
        for (streamIdx = 0; streamIdx < (setupParams->nstreams); streamIdx++) {
            ret[streamIdx] = compute_tran_temp(MatrixPower[streamIdx],MatrixTemp[streamIdx],setupParams->grid_cols,setupParams->grid_rows, \
            setupParams->sim_time,setupParams->pyramid_height, blockCols, blockRows, borderCols, borderRows, streams[streamIdx]);
        }
        for (streamIdx = 0; streamIdx < (setupParams->nstreams); streamIdx++) {
            hipStreamSynchronize(streams[streamIdx]);
        }
        #ifdef LOGS
            if (!(setupParams->generate)) end_iteration();
        #endif
        kernel_time = mysecond() - kernel_time;

        /////////// PERF
        if (setupParams->verbose)
        {

            double outputpersec = (double)((setupParams->grid_rows*setupParams->grid_rows*setupParams->nstreams)/kernel_time);
            printf("[Iteration #%i] kernel time: %.4lfs\n", loop1,kernel_time);
            printf("[Iteration #%i] SIZE:%d OUTPUT/S:%f FLOPS: %f (GFLOPS: %.2f)\n", loop1,setupParams->grid_rows, outputpersec, (double)flops / kernel_time,(double)flops / (kernel_time *1000000000));
        }
        flops = 0;

        //printf("Ending simulation\n");
        timestamp = mysecond();
        int kernel_errors=0;
        if (setupParams->generate) {
            hipMemcpy(setupParams->MatrixOut, MatrixTemp[0][ret[0]], sizeof(float)*size, hipMemcpyDeviceToHost);
            writeOutput(setupParams);
        } else {
            for (streamIdx = 0; streamIdx < setupParams->nstreams; streamIdx++) {
                memset(setupParams->MatrixOut, 0, sizeof(float)*size);
                hipMemcpy(setupParams->MatrixOut, MatrixTemp[streamIdx][ret[streamIdx]], sizeof(float)*size, hipMemcpyDeviceToHost);
                char error_detail[150];
				if (memcmp(setupParams->GoldMatrix, setupParams->MatrixOut, sizeof(float)*size))
				{
		            #pragma omp parallel for
		            for (int i=0; i<(setupParams->grid_rows); i++)
		            {
						register float *ptrGold = &(setupParams->GoldMatrix[i*(setupParams->grid_rows)+0]);
						register float *ptrOut = &(setupParams->MatrixOut[i*(setupParams->grid_rows)+0]);
		                for (int j=0 ; j<(setupParams->grid_cols); j++)
		                {
		                    if (ptrGold[j]!=ptrOut[j])
		                    #pragma omp critical
		                    {
		                        kernel_errors++;
		                        snprintf(error_detail, 150, "stream: %d, p: [%d, %d], r: %1.16e, e: %1.16e", streamIdx, i, j, setupParams->GoldMatrix[i*(setupParams->grid_rows)+j], setupParams->MatrixOut[i*(setupParams->grid_rows)+j]);
		                        printf("stream: %d, p: [%d, %d], r: %1.16e, e: %1.16e\n", streamIdx, i, j, setupParams->GoldMatrix[i*(setupParams->grid_rows)+j], setupParams->MatrixOut[i*(setupParams->grid_rows)+j]);
		                        #ifdef LOGS
		                            if (!(setupParams->generate)) log_error_detail(error_detail);
		                        #endif
		                    }
		                }
					}
                }
            }
            #ifdef LOGS
                if (!(setupParams->generate)) log_error_count(kernel_errors);
            #endif
        }

        if (setupParams->verbose) printf("[Iteration #%i] Gold check time: %.4fs\n", loop1, mysecond() - timestamp);
        if (kernel_errors!=0)
            printf("ERROR detected.\n");
        else
            printf(".");

        fflush(stdout);

        for (streamIdx = 0; streamIdx < setupParams->nstreams; streamIdx++) {
            hipFree(MatrixPower[streamIdx]);
            hipFree(MatrixTemp[streamIdx][0]);
            hipFree(MatrixTemp[streamIdx][1]);
            hipStreamDestroy(streams[streamIdx]);
        }
        if (setupParams->verbose) printf("[Iteration #%i] elapsed time: %.4fs\n", loop1, mysecond() - globaltime);
    }
    #ifdef LOGS
    	if (!(setupParams->generate)) end_log_file();
    #endif
}
