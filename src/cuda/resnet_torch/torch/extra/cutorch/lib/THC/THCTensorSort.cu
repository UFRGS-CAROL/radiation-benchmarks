#include "THCTensorSort.cuh"

void THCudaLongTensor_fillSliceWithIndex(THCState* state,
                                         THCudaLongTensor* t,
                                         int dim) {
  long dims = THCudaLongTensor_nDimension(state, t);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  ptrdiff_t inElements = THCudaLongTensor_nElement(state, t);
  long sliceSize = THCudaLongTensor_size(state, t, dim);
  ptrdiff_t numSlices = inElements / sliceSize;

  dim3 grid;
  if (!THC_getGridFromTiles(numSlices, grid)) {
    THError("Slice to fill with indices is too large");
  }

  long maxThreads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  long numThreads = sliceSize;
  if (numThreads > maxThreads) {
    numThreads = maxThreads;
  }

  dim3 block(numThreads);

#define FILL_INDEX(T, DIM)                                       \
  fillSliceWithIndex<T, DIM>                                     \
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(      \
      info, numSlices, sliceSize, info.strides[collapseDim])

  if (TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, t)) {
    TensorInfo<long, unsigned int> info =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, t);
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);

    if (info.isContiguous()) {
      FILL_INDEX(unsigned int, -2);
    } else {
      if (info.dims == 1) {
        FILL_INDEX(unsigned int, 1);
      } else if (info.dims == 2) {
        FILL_INDEX(unsigned int, 2);
      } else {
        FILL_INDEX(unsigned int, -1);
      }
    }
  } else {
    TensorInfo<long, unsigned long> info =
      getTensorInfo<THCudaLongTensor, unsigned long>(state, t);
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);

    // catch-all implementation
    FILL_INDEX(unsigned long, -1);
  }

#undef FILL_INDEX

  THCudaCheck(hipGetLastError());
}
