#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"
#include "THCTensorMathReduce.cuh"
#include "THCTensorMathPointwise.cuh"

struct TensorATan2Op {
  __device__ __forceinline__ void operator()(float* out, float* a, float* b) {
    *out = atan2f(*a, *b);
  }
};

void THCudaTensor_atan2(THCState *state, THCudaTensor *self_, THCudaTensor *tx, THCudaTensor *ty)
{
  THCAssertSameGPU(THCudaTensor_checkGPU(state, 3, self_, tx, ty));
  THArgCheck(THCudaTensor_nElement(state, tx) ==
             THCudaTensor_nElement(state, ty), 3, "sizes do not match");
  THCudaTensor_resizeAs(state, self_, tx);

  if (!THC_pointwiseApply3(state, self_, tx, ty, TensorATan2Op())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

