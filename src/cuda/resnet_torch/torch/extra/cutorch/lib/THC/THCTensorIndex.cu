#include "hip/hip_runtime.h"
#include "THC.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCHalf.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"
#include "THCDeviceUtils.cuh"
#include "THCNumerics.cuh"
#include "THCAtomics.cuh"
#include <algorithm> // for std::min

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexCopyLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexCopySmallIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<T, IndexType> src,
                                    TensorInfo<long, IndexType> indices,
                                    int dstCopyDim,
                                    int srcCopyDim,
                                    IndexType innerSize,
                                    long dstCopyDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(srcIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex < dstCopyDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);

      dstOffset += dstIndex * dst.strides[dstCopyDim];

      IndexType srcOffset =
        IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcCopyDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexCopySmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexCopyLargeIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<T, IndexType> src,
                                    TensorInfo<long, IndexType> indices,
                                    int dstCopyDim,
                                    int srcCopyDim,
                                    IndexType innerSize,
                                    long dstCopyDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < innerSize * indices.sizes[0];
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex = linearIndex / innerSize;
    IndexType elementInSlice = linearIndex % innerSize;

    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(srcIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex < dstCopyDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstCopyDim];

    IndexType srcOffset =
      IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcCopyDim];

    dst.data[dstOffset] = src.data[srcOffset];
  }
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexAddLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexAddSmallIndex(TensorInfo<T, IndexType> dst,
                                   TensorInfo<T, IndexType> src,
                                   TensorInfo<long, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType innerSize,
                                   long dstAddDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(srcIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex < dstAddDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstAddDim];

      IndexType srcOffset =
        IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcAddDim];

      atomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexAddSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexAddLargeIndex(TensorInfo<T, IndexType> dst,
                                   TensorInfo<T, IndexType> src,
                                   TensorInfo<long, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType innerSize,
                                   long dstAddDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < innerSize * indices.sizes[0];
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex = linearIndex / innerSize;
    IndexType elementInSlice = linearIndex % innerSize;

    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(srcIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex < dstAddDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstAddDim];

    IndexType srcOffset =
      IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcAddDim];

    atomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
  }
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexFillLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int IdxDim>
__global__ void indexFillSmallIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<long, IndexType> indices,
                                    int dstFillDim,
                                    IndexType innerSize,
                                    long dstFillDimSize,
                                    T val) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    // Lua indices begin at 1
    IndexType dstIndex_ =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(dstIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex < dstFillDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex_ * dst.strides[dstFillDim];

      dst.data[dstOffset] = val;
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexFillSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int IdxDim>
__global__ void indexFillLargeIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<long, IndexType> indices,
                                    int dstFillDim,
                                    IndexType innerSize,
                                    long dstFillDimSize,
                                    T val) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < innerSize * indices.sizes[0];
       linearIndex += gridDim.x * blockDim.x) {
    IndexType dstIndex = linearIndex / innerSize;
    IndexType elementInSlice = linearIndex % innerSize;

    // Lua indices begin at 1
    IndexType dstIndex_ =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(dstIndex, indices)] - TH_INDEX_BASE;
    assert(dstIndex_ < dstFillDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex_ * dst.strides[dstFillDim];

    dst.data[dstOffset] = val;
  }
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexSelectLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexSelectSmallIndex(TensorInfo<T, IndexType> dst,
                                      TensorInfo<T, IndexType> src,
                                      TensorInfo<long, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType innerSize,
                                      long srcSelectDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    // Lua indices begin at 1
    IndexType srcIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(dstIndex, indices)] - TH_INDEX_BASE;
    assert(srcIndex < srcSelectDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstSelectDim];

      IndexType srcOffset =
        IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcSelectDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexSelectSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexSelectLargeIndex(TensorInfo<T, IndexType> dst,
                                      TensorInfo<T, IndexType> src,
                                      TensorInfo<long, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType totalSize,
                                      IndexType innerSize,
                                      long srcSelectDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType dstIndex = linearIndex / innerSize;
    IndexType elementInSlice = linearIndex % innerSize;

    // Lua indices begin at 1
    IndexType srcIndex =
      indices.data[IndexToOffset<long, IndexType, IdxDim>::get(dstIndex, indices)] - TH_INDEX_BASE;
    assert(srcIndex < srcSelectDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstSelectDim];

    IndexType srcOffset =
      IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcSelectDim];

    dst.data[dstOffset] = src.data[srcOffset];
  }
}

template <typename IndexType, unsigned int Dims>
struct LinearIndexCalcData {
  // sizes for Tensor dims (either from the Tensor, or the size of the adv indexer at that dim)
  IndexType sizes[Dims];
  // strides for the Tensor we are indexing into
  IndexType strides[Dims];
  // these are pointers to the buffers containing the index selected at each dimension
  // for all of the indices we want to generate. If a dimension is not under advanced indexing
  // then the pointer is NULL
  long *advIndexTensors[Dims];
};

template <typename IndexType, unsigned int Dims>
__device__ __forceinline__ long calculateOffset(
  IndexType index,
  LinearIndexCalcData<IndexType, Dims> data
)
{
  IndexType offset = 0;

#pragma unroll
  for (int dim = Dims - 1; dim >= 0; --dim) {
    IndexType sizeAtDim, strideAtDim, indexAtDim, nextIndex;

    strideAtDim = data.strides[dim];
    sizeAtDim = data.sizes[dim];

    if (data.advIndexTensors[dim] != NULL) {
      indexAtDim = data.advIndexTensors[dim][index % sizeAtDim];
      // Check if next dimension is also advanced indexing, if so we must keep the index
      // the same and iterate together
      if (dim > 0 && data.advIndexTensors[dim - 1] != NULL) {
        nextIndex = index;
      } else {
        nextIndex = index / sizeAtDim;
      }
    } else {
      nextIndex = index / sizeAtDim;
      indexAtDim = index - nextIndex * sizeAtDim;
    }

    offset += indexAtDim * strideAtDim;
    index = nextIndex;
  }

  return offset;
}

template <typename IndexType, unsigned int Dims>
__global__ void calculateLinearIndices(
  long *output,               // output Tensor for indices
  int elements,               // number of elements in output <-> indices to calculate
  ptrdiff_t baseOffset,       // base offset into the Tensor
  LinearIndexCalcData<IndexType, Dims> data
)
{
  for (long i = blockIdx.x * blockDim.x + threadIdx.x;
         i < elements;
         i += blockDim.x * gridDim.x) {
      output[i] = baseOffset + calculateOffset<IndexType, Dims>(i, data);
   }
}

#include "generic/THCTensorIndex.cu"
#include "THCGenerateAllTypes.h"
