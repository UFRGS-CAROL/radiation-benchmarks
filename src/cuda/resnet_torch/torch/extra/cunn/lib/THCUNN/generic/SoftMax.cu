
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftMax.cu"
#else

#include "../common.h"

void THNN_(SoftMax_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, output, input);
  long batchSize, dim, stride0, stride1 = 1;
  long blocksY = 1, blocksZ = 1;

  if (input->nDimension == 1)
  {
    batchSize = 1;
    dim = input->size[0];
    stride0 = 1;
  }
  else if (input->nDimension == 2)
  {
    batchSize = input->size[0];
    dim = input->size[1];
    stride0 = 1;
  }
  else if (input->nDimension == 3)
  {
    batchSize = 1;
    dim = input->size[0];
    blocksY = input->size[1];
    blocksZ = input->size[2];
    stride0 = blocksY * blocksZ;
    stride1 = blocksZ;
  }
  else if (input->nDimension == 4)
  {
    batchSize = input->size[0];
    dim = input->size[1];
    blocksY = input->size[2];
    blocksZ = input->size[3];
    stride0 = blocksY * blocksZ;
    stride1 = blocksZ;
  }
  else
  {
    THError("1D, 2D, 3D or 4D tensor expected");
  }

  // when possible use only 2d grid of thread blocks to stay compatible with compute capability 2.X devices.
  if (blocksY * blocksZ < 65536)
  {
    blocksY *= blocksZ;
    blocksZ = 1;
    if (input->nDimension == 3 || input->nDimension == 4) {
      stride0 = blocksY * blocksZ;
      stride1 = blocksZ;
    }
  }

  dim3 blocks(batchSize, blocksY, blocksZ);
  dim3 threads(SOFTMAX_THREADS);
  cunn_SoftMax_updateOutput_kernel<real, accreal><<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, output),
    THCTensor_(data)(state, input),
    batchSize, dim, stride0, stride1
  );
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
}

void THNN_(SoftMax_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  output = THCTensor_(newContiguous)(state, output);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, output);
  long batchSize, dim, stride0, stride1 = 1;
  long blocksY = 1, blocksZ = 1;

  if (gradInput->nDimension == 1)
  {
    batchSize = 1;
    dim = gradInput->size[0];
    stride0 = 1;
  }
  else if (gradInput->nDimension == 2)
  {
    batchSize = gradInput->size[0];
    dim = gradInput->size[1];
    stride0 = 1;
  }
  else if (gradInput->nDimension == 3)
  {
    batchSize = 1;
    dim = gradInput->size[0];
    blocksY = gradInput->size[1];
    blocksZ = gradInput->size[2];
    stride0 = blocksY * blocksZ;
    stride1 = blocksZ;
  }
  else if (gradInput->nDimension == 4)
  {
    batchSize = gradInput->size[0];
    dim = gradInput->size[1];
    blocksY = gradInput->size[2];
    blocksZ = gradInput->size[3];
    stride0 = blocksY * blocksZ;
    stride1 = blocksZ;
  }
  else
  {
    THError("1D, 2D, 3D or 4D tensor expected");
  }

  // when possible use only 2d grid of thread blocks to stay compatible with compute capability 2.X devices.
  if (blocksY * blocksZ < 65536)
  {
    blocksY *= blocksZ;
    blocksZ = 1;
    if (input->nDimension == 3 || input->nDimension == 4) {
      stride0 = blocksY * blocksZ;
      stride1 = blocksZ;
    }
  }

  dim3 blocks(batchSize, blocksY, blocksZ);
  dim3 threads(SOFTMAX_THREADS);
  cunn_SoftMax_updateGradInput_kernel<real, accreal><<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
    THCTensor_(data)(state, gradInput),
    THCTensor_(data)(state, output),
    THCTensor_(data)(state, gradOutput),
    batchSize, dim, stride0, stride1
  );
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, gradOutput);
  THCTensor_(free)(state, output);
}

#endif
