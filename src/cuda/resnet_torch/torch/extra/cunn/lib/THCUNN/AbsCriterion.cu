#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

template <typename Dtype, typename Acctype>
struct abs_functor
{
  __host__ __device__ Acctype operator()(const Dtype& x, const Dtype& y) const
  {
    Dtype z = x-y;
    return ScalarConvert<Dtype, Acctype>::to(z >= 0 ? z : -z);
  }
};

template <typename Dtype>
struct abs_updateGradInput_functor
{
  const Dtype norm;

  abs_updateGradInput_functor(Dtype norm_)
    : norm(norm_)
  {}

  __host__ __device__ Dtype operator()(const Dtype& x, const Dtype& y) const
  {
    return (x - y) >= 0 ? norm : -norm;
  }
};

#include "generic/AbsCriterion.cu"
#include "THCGenerateFloatTypes.h"
