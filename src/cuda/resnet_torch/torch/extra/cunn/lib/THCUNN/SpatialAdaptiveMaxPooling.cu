#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function adaptively maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y
 */
 template <typename T>
__global__ void adaptivemaxpool(T *input, T *output, THCIndex_t *indices_x, THCIndex_t *indices_y,
                        int input_n, int input_h, int input_w,
                        int output_h, int output_w,
                        int strideh, int stridew,
                        int strided)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  const int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  const int yy_step = blockDim.y*gridDim.y;
  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*strided;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);
    int y_end   = (int)ceil(float(yy+1) / output_h * input_h);
    int kH = y_end-y_start;

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      int x_start = (int)floor(float(xx) / output_w * input_w);
      int x_end   = (int)ceil(float(xx + 1) / output_w * input_w);

      int kW = x_end-x_start;

      // Compute the mean of the input image...
      T *ptr_input = input + y_start*strideh + x_start*stridew;
      T *ptr_output = output + yy*output_w + xx;
      THCIndex_t *ptr_ind_x = indices_x + yy*output_w + xx;
      THCIndex_t *ptr_ind_y = indices_y + yy*output_w + xx;
      int argmax_x = -1;
      int argmax_y = -1;
      T max = THCNumerics<T>::min();
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++) {
          T val = ptr_input[kx*stridew];
          if (val > max) {
            max = val;
            argmax_x = kx;
            argmax_y = ky;
          }
        }
        ptr_input += strideh; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind_x = argmax_x + TH_INDEX_BASE;
      *ptr_ind_y = argmax_y + TH_INDEX_BASE;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
 template <typename T>
__global__ void adaptivemaxgradinput(T *gradInput, T *gradOutput, THCIndex_t *indices_x, THCIndex_t *indices_y,
                             int input_n, int input_h, int input_w,
                             int output_h, int output_w)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      T *ptr_gradInput = gradInput + y_start*input_w + x_start;
      T *ptr_gradOutput = gradOutput + yy*output_w + xx;
      THCIndex_t *ptr_ind_x = indices_x + yy*output_w + xx;
      THCIndex_t *ptr_ind_y = indices_y + yy*output_w + xx;
      T z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x) - TH_INDEX_BASE;
      int argmax_y = (*ptr_ind_y) - TH_INDEX_BASE;

      ptr_gradInput[argmax_x + argmax_y*input_w] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
 template <typename T>
__global__ void atomicadaptivemaxgradinput(
  T *gradInput, T *gradOutput, THCIndex_t *indices_x, THCIndex_t *indices_y,
  int input_n, int input_h, int input_w, int output_h, int output_w
)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {

    int y_start = (int)floor(float(yy) / output_h * input_h);

    for(xx = xx_start; xx < xx_end; xx+=xx_step) {

      int x_start = (int)floor(float(xx) / output_w * input_w);

      T *ptr_gradInput = gradInput + y_start*input_w + x_start;
      T *ptr_gradOutput = gradOutput + yy*output_w + xx;
      THCIndex_t *ptr_ind_x = indices_x + yy*output_w + xx;
      THCIndex_t *ptr_ind_y = indices_y + yy*output_w + xx;
      T z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x) - TH_INDEX_BASE;
      int argmax_y = (*ptr_ind_y) - TH_INDEX_BASE;

      // atomic add since different threads could update same variable
      atomicAdd(&(ptr_gradInput[argmax_x + argmax_y*input_w]), z);
    }
  }
}

#include "generic/SpatialAdaptiveMaxPooling.cu"
#include "THCGenerateFloatTypes.h"

#undef CUDA_MAX_THREADS
