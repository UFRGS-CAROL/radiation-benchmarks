
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricAveragePooling.cu"
#else

static inline void THNN_(VolumetricAveragePooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         int kT,
                         int kW,
                         int kH,
                         int dT,
                         int dW,
                         int dH) {
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int ndim = input->nDimension;
  int dimN = 0;
  int dimt = 1;
  int dimh = 2;
  int dimw = 3;

  if (input->nDimension == 5)
  {
    dimN++;
    dimt++;
    dimh++;
    dimw++;
  }

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    THArgCheck(input->size[dimw] >= kW && input->size[dimh] >= kH
               && input->size[dimt] >= kT, 2,
               "input image (T: %d H: %d W: %d) smaller than "
               "kernel size (kT: %d kH: %d kW: %d)",
               input->size[dimt], input->size[dimh], input->size[dimw],
               kT, kH, kW);

    /* sizes */
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    THArgCheck(input->size[dimw] >= kW && input->size[dimh] >= kH
               && input->size[dimt] >= kT, 2,
               "input image (T: %d H: %d W: %d) smaller than "
               "kernel size (kT: %d kH: %d kW: %d)",
               input->size[dimt], input->size[dimh], input->size[dimw],
               kT, kH, kW);

    /* sizes */
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }
  else
  {
    THArgCheck(false, 2, "4D or 5D tensor expected, but got: %d", input->nDimension);
  }

  int outputTime   = (inputTime   - kT) / dT + 1;
  int outputHeight = (inputHeight - kH) / dH + 1;
  int outputWidth  = (inputWidth  - kW) / dW + 1;

  if (gradOutput != NULL) {
     THCUNN_check_dim_size(state, gradOutput, ndim, dimN, inputSlices);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimt, outputTime);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
     THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

void THNN_(VolumetricAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int kT, int kW, int kH,
           int dT, int dW, int dH)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int dimt = 1;
  int dimh = 2;
  int dimw = 3;

  if (input->nDimension == 5)
  {
    dimt++;
    dimh++;
    dimw++;
  }

  THNN_(VolumetricAveragePooling_shapeCheck)
       (state, input, NULL, kT, kW, kH, dT, dW, dH);

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  int outputTime   = (inputTime   - kT) / dT + 1;
  int outputHeight = (inputHeight - kH) / dH + 1;
  int outputWidth  = (inputWidth  - kW) / dW + 1;

  if (input->nDimension == 4) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }
  else /* 5D */
  {
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                         outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaInput;
  THCDeviceTensor<real, 4> cudaOutput;
  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaInput  = toDeviceTensor<real, 4>(state, input);
    cudaOutput = toDeviceTensor<real, 4>(state, output);
  }
  else
  {
    cudaInput  = toDeviceTensor<real, 5>(state, input).downcastOuter<4>();
    cudaOutput = toDeviceTensor<real, 5>(state, output).downcastOuter<4>();
  }

  int totalZ = outputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);
  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
              THCCeilDiv(outputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    accreal normFactor = ScalarConvert<int, accreal>::to(1) / static_cast<accreal>(kT * kH * kW);
    switch (kW)
      {
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(1);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(2);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(3);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(4);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(5);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(6);
        LAUNCH_UPDATE_OUTPUT_KERNEL_WIDTH(7);
      default:
        cuda_VolumetricAveragePooling_updateOutput<real, accreal><<<grid, block>>>(
                                                                    cudaInput,
                                                                    cudaOutput,
                                                                    kT, kH, kW,
                                                                    dT, dH, dW,
                                                                    normFactor,
                                                                    offsetZ
                                                                    );
        break;
      }
    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }
  THCTensor_(free)(state, input);
}

void THNN_(VolumetricAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int kT, int kW, int kH,
           int dT, int dW, int dH)
{

  THNN_(VolumetricAveragePooling_shapeCheck)
       (state, input, gradOutput, kT, kW, kH, dT, dW, dH);
  bool kernelsOverlap = (dT < kT) || (dH < kH) || (dW < kW);

  // Resize and initialize result tensor.
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  int outputTime;
  int outputHeight;
  int outputWidth;

  if (THCTensor_(nDimension)(state, input) == 4) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime    = THCTensor_(size)(state, input, 1);
    inputHeight  = THCTensor_(size)(state, input, 2);
    inputWidth   = THCTensor_(size)(state, input, 3);

    outputTime   = THCTensor_(size)(state, gradOutput, 1);
    outputHeight = THCTensor_(size)(state, gradOutput, 2);
    outputWidth  = THCTensor_(size)(state, gradOutput, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime    = THCTensor_(size)(state, input, 2);
    inputHeight  = THCTensor_(size)(state, input, 3);
    inputWidth   = THCTensor_(size)(state, input, 4);

    outputTime   = THCTensor_(size)(state, gradOutput, 2);
    outputHeight = THCTensor_(size)(state, gradOutput, 3);
    outputWidth  = THCTensor_(size)(state, gradOutput, 4);
  }

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaGradInput;
  THCDeviceTensor<real, 4> cudaGradOutput;
  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaGradInput  = toDeviceTensor<real, 4>(state, gradInput);
    cudaGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
  }
  else
  {
    cudaGradInput =
      toDeviceTensor<real, 5>(state, gradInput).downcastOuter<4>();
    cudaGradOutput =
      toDeviceTensor<real, 5>(state, gradOutput).downcastOuter<4>();
  }

  dim3 block(32, 8);

  // Optimizing for stride 1 is probably only of limited value, but this
  // specialization yields 3x speedup over the atomicAdd implementation.
  if (dT == 1 && dH == 1 && dW == 1)
  {
    int totalZ = inputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {
      dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
                THCCeilDiv(inputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      cuda_VolumetricAveragePooling_updateGradInput_Stride1<real, accreal><<<grid, block>>>(
         cudaGradOutput, cudaGradInput, kT, kH, kW, 1.0f/(kT * kH * kW), offsetZ);
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }
  else
  {
    int totalZ = outputTime * inputSlices * batchSize;
    int offsetZ = 0;
    while (totalZ > 0) {

      dim3 grid(THCCeilDiv(outputWidth, static_cast<int>(block.x)),
                THCCeilDiv(outputHeight, static_cast<int>(block.y)),
                totalZ > 65535 ? 65535 : totalZ);
      if (kernelsOverlap)
        {
          cuda_VolumetricAveragePooling_updateGradInput_atomicAdd<real, accreal><<<grid, block>>>(
            cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      else
        {
          cuda_VolumetricAveragePooling_updateGradInput<real, accreal><<<grid, block>>>(
             cudaGradOutput, cudaGradInput, kT, kH, kW, dT, dH, dW, offsetZ);
        }
      THCudaCheck(cudaGetLastError());
      totalZ -= 65535;
      offsetZ += 65535;
    }
  }

  THCTensor_(free)(state, gradOutput);
}

#endif
