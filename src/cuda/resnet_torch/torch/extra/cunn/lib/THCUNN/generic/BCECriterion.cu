
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/BCECriterion.cu"
#else

void THNN_(BCECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 3, input, target, weights);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));

  accreal sum;
  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<real> weights_data(THCTensor_(data)(state, weights));
    sum = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      bce_functor_weights<real, accreal>(),
      (accreal) 0,
      thrust::plus<accreal>()
    );
    THCTensor_(free)(state, weights);
  } else {
    sum = thrust::transform_reduce(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      bce_functor<real, accreal>(),
      (accreal) 0,
      thrust::plus<accreal>()
    );
  }

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(BCECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_nElement(state, input, weights);
  THCUNN_assertSameGPU(state, 4, input, target, gradInput, weights);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  real norm = ScalarConvert<accreal, real>::to(sizeAverage ? accreal(1)/size : accreal(1));

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  if (weights) {
    weights = THCTensor_(newContiguous)(state, weights);
    thrust::device_ptr<real> weights_data(THCTensor_(data)(state, weights));
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data, weights_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size, weights_data+size)),
      gradInput_data,
      bce_updateGradInput_functor_weights<real, accreal>(norm)
    );
    THCTensor_(free)(state, weights);
  } else {
    thrust::transform(
      thrust::make_zip_iterator(thrust::make_tuple(input_data, target_data)),
      thrust::make_zip_iterator(thrust::make_tuple(input_data+size, target_data+size)),
      gradInput_data,
      bce_updateGradInput_functor<real, accreal>(norm)
    );
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
