
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialClassNLLCriterion.cu"
#else

void THNN_(SpatialClassNLLCriterion_shapeCheck)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *weights)
{
  THArgCheck(THCIndexTensor_(nDimension)(state, target) == 3, 1,
             "only batches of spatial targets supported (3D tensors)" \
             " but got targets of dimension: %d",
             THCIndexTensor_(nDimension)(state, target));
  THArgCheck(THCTensor_(nDimension)(state, input) == 4, 2,
             "only batches of spatial inputs supported (4D tensors), "      \
             "but got input of dimension: %d", THCTensor_(nDimension)(state, input));
  if (THCTensor_(size)(state, input, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, input, 2) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, input, 3) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff input_size = THCTensor_(sizeDesc)(state, input);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("input and target batch or spatial sizes don't match: target %s, input %s",
            target_size.str, input_size.str);
  }

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }
}

void THNN_(SpatialClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);

  if (weights)
    THCUNN_assertSameGPU(state, 5, input, target, weights, output, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, output, total_weight);

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  THCTensor_(fill)(state, output, ScalarConvert<int, real>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, real>::to(0));

  cunn_SpatialClassNLLCriterion_updateOutput_kernel<real, accreal>
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      output_data,
      total_weight_data,
      input_data,
      target_data,
      weights_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());
  if (sizeAverage) {
    cunn_SpatialClassNLLCriterion_sizeAverage_kernel<<<1, 1, 0, THCState_getCurrentStream(state)>>>(
      output_data, total_weight_data
    );
    THCudaCheck(cudaGetLastError());
  }

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SpatialClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");

  if (weights)
    THCUNN_assertSameGPU(state, 5, weights, input, target, gradInput, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, gradInput, total_weight);

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
  int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
  blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
  int total_blocks = blocks_per_sample * batch_size;

  cunn_SpatialClassNLLCriterion_updateGradInput_kernel
    <<<total_blocks, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      gradInput_data,
      target_data,
      weights_data,
      total_weight_data,
      sizeAverage,
      THCTensor_(size)(state, input, 0),
      THCTensor_(size)(state, input, 1),
      THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
      blocks_per_sample
  );
  THCudaCheck(cudaGetLastError());

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

#endif
