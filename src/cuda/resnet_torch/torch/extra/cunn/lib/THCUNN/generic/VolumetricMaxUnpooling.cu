
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricMaxUnpooling.cu"
#else

static inline void THNN_(VolumetricMaxUnpooling_shapeCheck)(
                         THCState *state,
                         THCTensor *input,
                         THCTensor *gradOutput,
                         THCIndexTensor *indices,
                         int oT,
                         int oW,
                         int oH,
                         int dT,
                         int dW,
                         int dH,
                         int pT,
                         int pW,
                         int pH) {
  int inputSlices;

  THCUNN_check_shape_indices(state, indices, input);

  THArgCheck(dT > 0 && dW > 0 && dH > 0, 10,
             "stride should be greater than zero, but got dT: %d dH: %d dW: %d",
             dT, dH, dW);

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    inputSlices = THCTensor_(size)(state, input, 0);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    inputSlices = THCTensor_(size)(state, input, 1);
  }
  else
  {
    THArgCheck(false, 2, "4D or 5D tensor expected, got %d",
               THCTensor_(nDimension)(state, input));
  }

  int dimw = 3;
  int dimh = 2;
  int dimt = 1;
  int dimn = 0;
  if (input->nDimension == 5)
  {
    dimt++;
    dimw++;
    dimh++;
    dimn++;
  }

  if (gradOutput != NULL) {
    if (oT != gradOutput->size[dimt] || oW != gradOutput->size[dimw] || oH != gradOutput->size[dimh])
    {
      THError(
        "Inconsistent gradOutput size. oT= %d, oH= %d, oW= %d, gradOutput: %dx%dx%d",
        oT, oH, oW, gradOutput->size[dimt], gradOutput->size[dimh], gradOutput->size[dimw]);
    }

    THCUNN_check_dim_size(state, gradOutput, input->nDimension, dimn, inputSlices);
  }
}

void THNN_(VolumetricMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{
  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  THNN_(VolumetricMaxUnpooling_shapeCheck)(
        state, input, NULL, indices,
        outputTime, outputWidth, outputHeight,
        dT, dW, dH, padT, padW, padH);
  THCUNN_assertSameGPU(state, 3, input, indices, output);

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    /* sizes */
    batchSize   = 1;
    inputSlices = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else if (THCTensor_(nDimension)(state, input) == 5)
  {
    /* sizes */
    batchSize   = THCTensor_(size)(state, input, 0);
    inputSlices = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  if (input->nDimension == 4) /* 4D */
  {
    /* resize output */
    THCTensor_(resize4d)(state, output, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }
  else
  { /* 5D */
    THCTensor_(resize5d)(state, output, batchSize, inputSlices,
                          outputTime, outputHeight, outputWidth);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  THCTensor_(zero)(state, output);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaInput;
  THCDeviceTensor<real, 4> cudaOutput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaInput  = toDeviceTensor<real, 4>(state, input);
    cudaOutput = toDeviceTensor<real, 4>(state, output);
    cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);
  }
  else
  {
    cudaInput  = toDeviceTensor<real, 5>(state, input).downcastOuter<4>();
    cudaOutput = toDeviceTensor<real, 5>(state, output).downcastOuter<4>();
    cudaIndices = toDeviceTensor<THCIndex_t, 5>(state, indices).downcastOuter<4>();
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateOutput<<<grid, block,
          0, THCState_getCurrentStream(state)>>>(
                             cudaInput, cudaIndices, cudaOutput,
                             dT, dH, dW,
                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
}

void THNN_(VolumetricMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int outputTime, int outputWidth, int outputHeight,
           int dT, int dW, int dH,
           int padT, int padW, int padH)
{

  int batchSize;
  int inputSlices;
  int inputTime;
  int inputHeight;
  int inputWidth;

  THNN_(VolumetricMaxUnpooling_shapeCheck)(
        state, input, gradOutput, indices,
        outputTime, outputWidth, outputHeight,
        dT, dW, dH, padT, padW, padH);
  THCUNN_assertSameGPU(state, 4, input, indices, gradOutput, gradInput);

  if (THCTensor_(nDimension)(state, input) == 4) /* 4D */
  {
    batchSize = 1;
    inputSlices  = THCTensor_(size)(state, input, 0);
    inputTime   = THCTensor_(size)(state, input, 1);
    inputHeight = THCTensor_(size)(state, input, 2);
    inputWidth  = THCTensor_(size)(state, input, 3);
  }
  else
  {
    batchSize    = THCTensor_(size)(state, input, 0);
    inputSlices  = THCTensor_(size)(state, input, 1);
    inputTime   = THCTensor_(size)(state, input, 2);
    inputHeight = THCTensor_(size)(state, input, 3);
    inputWidth  = THCTensor_(size)(state, input, 4);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  // Collapse batch and feature dimensions
  THCDeviceTensor<real, 4> cudaGradInput;
  THCDeviceTensor<real, 4> cudaGradOutput;
  THCDeviceTensor<THCIndex_t, 4> cudaIndices;

  if (THCTensor_(nDimension)(state, input) == 4)
  {
    cudaGradInput  = toDeviceTensor<real, 4>(state, gradInput);
    cudaGradOutput = toDeviceTensor<real, 4>(state, gradOutput);
    cudaIndices = toDeviceTensor<THCIndex_t, 4>(state, indices);
  }
  else
  {
    cudaGradInput =
      toDeviceTensor<real, 5>(state, gradInput).downcastOuter<4>();
    cudaGradOutput =
      toDeviceTensor<real, 5>(state, gradOutput).downcastOuter<4>();
    cudaIndices =
      toDeviceTensor<THCIndex_t, 5>(state, indices).downcastOuter<4>();
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(THCCeilDiv(inputWidth, static_cast<int>(block.x)),
              THCCeilDiv(inputHeight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    cuda_VolumetricMaxUnpooling_updateGradInput<<<grid, block,
      0, THCState_getCurrentStream(state)>>>(
                                             cudaGradOutput,
                                             cudaIndices,
                                             cudaGradInput,
                                             dT, dH, dW,
                                             padT, padH, padW, offsetZ);
    THCudaCheck(cudaGetLastError());
    totalZ -= 65535;
    offsetZ += 65535;
  }

  // cleanup
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  THCIndexTensor_(free)(state, indices);
}

#endif
