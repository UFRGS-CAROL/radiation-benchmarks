
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialAdaptiveAveragePooling.cu"
#else

#include "../common.h"

void THNN_(SpatialAdaptiveAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int nOutputCols,
           int nOutputRows)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  real *output_data;
  real *input_data;

  THCUNN_argCheck(state, input->nDimension == 3 || input->nDimension == 4, 2, input,
                  "3D or 4D (batch mode) tensor expected for input, but got: %s");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];

    long istride_d = input->stride[0];
    long istride_h = input->stride[1];
    long istride_w = input->stride[2];

    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);

    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run averagepool kernel
    adaptiveaveragepool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                   istride_h, istride_w, istride_d);
    THCudaCheck(cudaGetLastError());

  } else {
    input = THCTensor_(newContiguous)(state, input);
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];

    long istride_d = input->stride[1];
    long istride_h = input->stride[2];
    long istride_w = input->stride[3];

    input_data = THCTensor_(data)(state, input);

    THCTensor_(resize4d)(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);

    output_data = THCTensor_(data)(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run averagepool kernel
    adaptiveaveragepool <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                   nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                   istride_h, istride_w, istride_d);
    THCudaCheck(cudaGetLastError());
    // clean
    THCTensor_(free)(state, input);
  }
}

void THNN_(SpatialAdaptiveAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput)
{
  bool atomic = true; // suboptimal, but without atomic it doesn't pass the tests

  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  real *gradInput_data;
  real *gradOutput_data;

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];
    long nOutputCols = gradOutput->size[2];
    long nOutputRows = gradOutput->size[1];

    //bool atomic = (nInputCols%nOutputCols != 0) || (nInputRows%nOutputRows != 0);

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);

    gradOutput_data = THCTensor_(data)(state, gradOutput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicadaptiveaveragegradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    else
    {
      // run updateGradInput kernel
      adaptiveaveragegradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    THCudaCheck(cudaGetLastError());
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];
    long nOutputCols = gradOutput->size[3];
    long nOutputRows = gradOutput->size[2];

    //bool atomic = //(nInputCols%nOutputCols != 0) || (nInputRows%nOutputRows != 0);

    THCTensor_(resizeAs)(state, gradInput, input);
    THCTensor_(zero)(state, gradInput);

    gradOutput_data = THCTensor_(data)(state, gradOutput);
    gradInput_data = THCTensor_(data)(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicadaptiveaveragegradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    else
    {
      // run updateGradInput kernel, accumulate gradients atomically
      adaptiveaveragegradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols);
    }
    THCudaCheck(cudaGetLastError());
  }

  // clean
  THCTensor_(free)(state,gradOutput);

}

#endif
