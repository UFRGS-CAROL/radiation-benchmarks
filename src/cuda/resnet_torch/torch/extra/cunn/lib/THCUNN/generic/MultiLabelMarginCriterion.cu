
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiLabelMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           bool sizeaverage)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, input);

  if(input->nDimension == 1)
  {
    int dim = input->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == dim), 3,
        "inconsistent target size");
    THCTensor_(resize1d)(state, output, 1);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal> <<<blocks,threads>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        sizeaverage
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->nDimension == 2)
  {
    int nframe = input->size[0];
    int dim = input->size[1];
    THArgCheck((target->nDimension == 2) && (target->size[0] == nframe)
               && (target->size[1] == dim), 3, "inconsistent target size");
    THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size[0]);

    dim3 blocks(input->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal> <<<blocks,threads>>>(
        THCTensor_(data)(state, output_tmp),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        nframe, dim,
        sizeaverage
        );
    THCudaCheck(cudaGetLastError());
    THCTensor_(resize1d)(state, output, 1);
    THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(THCTensor_(sumall)(state, output_tmp)));
    THCTensor_(free)(state, output_tmp);
  }
  else
    THError("vector or matrix expected");

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradInput,
            THCTensor *istarget,
            bool sizeaverage)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->nDimension == 1)
  {
    int dim = gradInput->size[0];
    THArgCheck((target->nDimension == 1) && (target->size[0] == dim), 3,
               "inconsistent target size");
    THArgCheck((istarget->nDimension == 1) && (istarget->size[0] == dim), 3,
               "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal> <<<blocks,threads>>>(THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, gradInput->size[0],
        sizeaverage);

  }
  else if(gradInput->nDimension == 2)
  {
    int nframe = gradInput->size[0];
    int dim = gradInput->size[1];
    THArgCheck((target->nDimension == 2) && (target->size[0] == nframe)
               && (target->size[1] == dim), 3, "inconsistent target size");
    THArgCheck((istarget->nDimension == 2) && (istarget->size[0] == nframe)
               && (istarget->size[1] == dim), 3, "inconsistent isTarget size");
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal> <<<blocks,threads>>>(THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size[0], gradInput->size[1],
        sizeaverage);
  }
  else
    THError("vector or matrix expected");

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

#endif
