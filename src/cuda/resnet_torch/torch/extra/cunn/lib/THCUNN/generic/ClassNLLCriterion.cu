
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/ClassNLLCriterion.cu"
#else

void THNN_(ClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight,
           long ignore_index) {
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_check_dim_size(state, total_weight, 1, 0, 1);
  ignore_index -= TH_INDEX_BASE;

  if (THCIndexTensor_(nDimension)(state, target) > 1) {
    THError("multi-target not supported");
  }

  int n_dims = THCTensor_(nDimension)(state, input);
  int n_classes = THCTensor_(size)(state, input, n_dims - 1);

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, input, target, weights, output, total_weight
    );
  } else {
    THCUNN_assertSameGPU(
      state, 4, input, target, output, total_weight
    );
  }

  THArgCheck(n_dims <= 2 && n_dims > 0, 2, "vector or matrix expected");

  long batch_size = n_dims == 1 ? 1 : THCTensor_(size)(state, input, 0);
  long num_targets = THCudaLongTensor_size(state, target, 0);
  THArgCheck(batch_size == num_targets,
      2, "mismatch between the batch size of input (%ld) and that of target (%ld)",
      batch_size, num_targets);

  if (weights && THCTensor_(nElement)(state, weights) != n_classes) {
    THCDescBuff s1 = THCTensor_(sizeDesc)(state, weights);
    THError("weight tensor should be defined either for all %d classes or no classes"
            " but got weight tensor of shape: %s", n_classes, s1.str);
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *input_data = THCTensor_(data)(state, input);
  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *output_data = THCTensor_(data)(state, output);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  if (THCTensor_(nDimension)(state, input) == 1) {
    cunn_ClassNLLCriterion_updateOutput_kernel1<real>
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        n_classes,
        ignore_index
    );

  } else if (THCTensor_(nDimension)(state, input) == 2) {
    cunn_ClassNLLCriterion_updateOutput_kernel<real, accreal>
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        n_classes,
        ignore_index
    );
  }
  THCudaCheck(cudaGetLastError());

  if (weights) {
    THCTensor_(free)(state, weights);
  }
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(ClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           THCTensor *weights,
           THCTensor *total_weight,
           long ignore_index) {
  if (THCIndexTensor_(nDimension)(state, target) > 1) {
    THError("multi-target not supported");
  }
  ignore_index -= TH_INDEX_BASE;

  int n_dims = THCTensor_(nDimension)(state, input);
  int n_classes = THCTensor_(size)(state, input, n_dims - 1);

  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4, "gradInput must be contiguous");

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, weights, input, target, gradInput, total_weight
    );
  }
  else {
    THCUNN_assertSameGPU(
      state, 4, input, target, gradInput, total_weight
    );
  }

  THArgCheck(n_dims <= 2 && n_dims > 0, 2, "vector or matrix expected");

  long batch_size = n_dims == 1 ? 1 : THCTensor_(size)(state, input, 0);
  long num_targets = THCudaLongTensor_size(state, target, 0);
  THArgCheck(batch_size == num_targets,
      2, "mismatch between the batch size of input (%ld) and that of target (%ld)",
      batch_size, num_targets);

  if (weights && THCTensor_(nElement)(state, weights) != n_classes) {
    THError("weight tensor should be defined either for all or no classes");
  }

  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  real *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  real *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  real *total_weight_data = THCTensor_(data)(state, total_weight);

  if (THCTensor_(nDimension)(state, input) == 1) {
    cunn_ClassNLLCriterion_updateGradInput_kernel1<real>
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        weights_data,
        target_data,
        total_weight_data,
        sizeAverage,
        n_classes,
        ignore_index
    );
  } else {
    cunn_ClassNLLCriterion_updateGradInput_kernel<real>
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        target_data,
        weights_data,
        total_weight_data,
        sizeAverage,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        n_classes,
        ignore_index
    );
  }
  THCudaCheck(cudaGetLastError());

  if (weights) {
    THCTensor_(free)(state, weights);
  }
  THCIndexTensor_(free)(state, target);
}

#endif
