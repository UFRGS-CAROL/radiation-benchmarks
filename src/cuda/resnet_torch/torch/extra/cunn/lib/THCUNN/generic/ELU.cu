
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/ELU.cu"
#else

#include "../common.h"


void THNN_(ELU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           accreal alpha_,
           bool inplace)
{
  real alpha = ScalarConvert<accreal, real>::to(alpha_);
  THCUNN_assertSameGPU(state, 2, input, output);

  if (inplace)
  {
    THC_pointwiseApply1(state, input, ELUupdateOutputIP_functor<real>(alpha));
    THCTensor_(set)(state, output, input);
  }
  else
  {
    THCTensor_(resizeAs)(state, output, input);
    THC_pointwiseApply2(state, output, input, ELUupdateOutput_functor<real>(alpha));
  }
}


void THNN_(ELU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           accreal alpha_,
           bool inplace)
{
  real alpha = ScalarConvert<accreal, real>::to(alpha_);
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);

  if (inplace)
  {
    THC_pointwiseApply2(state, gradOutput, output, ELUupdateGradInputIP_functor<real>(alpha));
    THCTensor_(set)(state, gradInput, gradOutput);
  }
  else
  {
    THCTensor_(resizeAs)(state, gradInput, output);
    THC_pointwiseApply3(state, gradInput, output, gradOutput, ELUupdateGradInput_functor<real>(alpha));
  }
}

#endif
