#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "common.h"

// kernels borrowed from Caffe
template <typename Dtype, typename AccType>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* top_data,
    long* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    AccType maxval = THCNumerics<AccType>::min();
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        if (ScalarConvert<Dtype, AccType>::to(bottom_data[h * width + w]) > maxval) {
          maxidx = h * width + w;
          maxval = ScalarConvert<Dtype, AccType>::to(bottom_data[maxidx]);
        }
      }
    }
    top_data[index] = ScalarConvert<AccType, Dtype>::to(maxval);
    top_mask[index] = maxidx + TH_INDEX_BASE;
  }
}


template <typename Dtype, typename AccType>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const long* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);

    AccType gradient = AccType(0);
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    top_mask += offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (top_mask[ph * pooled_width + pw] - TH_INDEX_BASE == h * width + w) {
          gradient += ScalarConvert<Dtype, AccType>::to(top_diff[ph * pooled_width + pw]);
        }
      }
    }
    bottom_diff[index] = ScalarConvert<AccType, Dtype>::to(gradient);
  }
}

#include "generic/SpatialDilatedMaxPooling.cu"
#include "THCGenerateFloatTypes.h"
