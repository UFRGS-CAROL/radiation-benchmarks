#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

#include <cfloat>

template <typename Dtype>
__global__ void cuda_VolumetricDilatedMaxPooling_updateOutput(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<THCIndex_t, 4> indices,
  THCDeviceTensor<Dtype, 4> output,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  int dilationT, int dilationH, int dilationW,
  int offsetZ)
{
  int oColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow    = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame  = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice   = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oColumn < output.getSize(3))
  {
    int iColumn = oColumn * dW - padW;
    int iRow    = oRow    * dH - padH;
    int iFrame  = oFrame  * dT - padT;

    int maxColumn = 0;
    int maxRow = 0;
    int maxFrame = 0;

    Dtype max = THCNumerics<Dtype>::min();

    for (int frame = 0; frame < kT; ++frame)
    {
      if (iFrame + frame * dilationT < input.getSize(1) && iFrame + frame * dilationT >= 0)
      {
        for (int row = 0; row < kH; ++row)
        {
          if (iRow + row * dilationH < input.getSize(2) && iRow + row * dilationH >= 0)
          {
            for (int column = 0; column < kW; ++column)
            {
              if (iColumn + column * dilationW < input.getSize(3) && iColumn + column * dilationW >= 0)
              {
                Dtype val = input[slice][iFrame + frame * dilationT][iRow + row * dilationH][iColumn + column * dilationW];

                if (max < val)
                {
                  max = val;
                  maxColumn = column;
                  maxRow    = row;
                  maxFrame  = frame;
                }
              }
            }
          }
        }
      }
    }

    output[slice][oFrame][oRow][oColumn] = max;
    THCIndex_t *idx = &indices[slice][oFrame][oRow][oColumn];
    ((unsigned char*)(idx))[0] = maxFrame;
    ((unsigned char*)(idx))[1] = maxRow;
    ((unsigned char*)(idx))[2] = maxColumn;
    ((unsigned char*)(idx))[3] = 0;
  }
}

template <int KERNEL_WIDTH, typename Dtype>
__global__ void cuda_VolumetricDilatedMaxPooling_updateOutput(
  THCDeviceTensor<Dtype, 4> input, THCDeviceTensor<THCIndex_t, 4> indices,
  THCDeviceTensor<Dtype, 4> output,
  int kT, int kH,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  int dilationT, int dilationH, int dilationW,
  int offsetZ)
{
  int oColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow    = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame  = (blockIdx.z + offsetZ) % output.getSize(1); // output frame/time
  int slice   = (blockIdx.z + offsetZ) / output.getSize(1); // output slice/feature

  if (oRow < output.getSize(2) && oColumn < output.getSize(3))
  {
    int iColumn = oColumn * dW - padW;
    int iRow    = oRow    * dH - padH;
    int iFrame  = oFrame  * dT - padT;

    int maxColumn = 0;
    int maxRow = 0;
    int maxFrame;

    Dtype max = THCNumerics<Dtype>::min();

    for (int frame = 0; frame < kT; ++frame)
    {
      if (iFrame + frame * dilationT < input.getSize(1) && iFrame + frame * dilationT >= 0)
      {
        for (int row = 0; row < kH; ++row)
        {
          if (iRow + row * dilationH < input.getSize(2) && iRow + row * dilationH >= 0)
          {
            for (int column = 0; column < KERNEL_WIDTH; ++column)
            {
              if (iColumn + column * dilationW < input.getSize(3) && iColumn + column * dilationW >= 0)
              {
                Dtype val = input[slice][iFrame + frame * dilationT][iRow + row * dilationH][iColumn + column * dilationW];

                if (max < val)
                {
                  max = val;
                  maxColumn = column;
                  maxRow    = row;
                  maxFrame  = frame;
                }
              }
            }
          }
        }
      }
    }

    output[slice][oFrame][oRow][oColumn] = max;
    THCIndex_t *idx = &indices[slice][oFrame][oRow][oColumn];
    ((unsigned char*)(idx))[0] = maxFrame;
    ((unsigned char*)(idx))[1] = maxRow;
    ((unsigned char*)(idx))[2] = maxColumn;
    ((unsigned char*)(idx))[3] = 0;
  }
}

template <typename Dtype>
__global__ void cuda_VolumetricDilatedMaxPooling_updateGradInput(
  THCDeviceTensor<Dtype, 4> gradOutput,
  THCDeviceTensor<THCIndex_t, 4> indices,
  THCDeviceTensor<Dtype, 4> gradInput,
  int dT, int dH, int dW,
  int padT, int padH, int padW,
  int dilationT, int dilationH, int dilationW,
  int offsetZ)
{
  int oColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow    = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame  = (blockIdx.z + offsetZ) % gradOutput.getSize(1); // output frame/time
  int slice   = (blockIdx.z + offsetZ) / gradOutput.getSize(1); // output slice/feature

  if (oRow < gradOutput.getSize(2) && oColumn < gradOutput.getSize(3))
  {
    THCIndex_t *idx = &indices[slice][oFrame][oRow][oColumn];
    int iFrame  = ((unsigned char*)(idx))[0] * dilationT + oFrame  * dT - padT;
    int iRow    = ((unsigned char*)(idx))[1] * dilationH + oRow    * dH - padH;
    int iColumn = ((unsigned char*)(idx))[2] * dilationW + oColumn * dW - padW;
    atomicAdd(&gradInput[slice][iFrame][iRow][iColumn],
              gradOutput[slice][oFrame][oRow][oColumn]);
  }
}

#include "generic/VolumetricDilatedMaxPooling.cu"
#include "THCGenerateFloatTypes.h"
