#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */

#include <hip/hip_cooperative_groups.h>
#include <stdexcept>
#include "hipblas.h"

#include "matrixMul_kernel.h"
#include "persistent_lib.h"

template<typename real_t>
__device__ void process_mxm_ii(real_t *C, real_t *A, real_t *B, int wA,
		int wB) {
	// Handle to thread block group
	cooperative_groups::thread_block cta =
			cooperative_groups::this_thread_block();
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wA * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wA - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * wB;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	real_t Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ real_t As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ real_t Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = A[a + wA * ty + tx];
		Bs[ty][tx] = B[b + wB * ty + tx];

		// Synchronize to make sure the matrices are loaded
		cooperative_groups::sync(cta);

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll
		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		cooperative_groups::sync(cta);

	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	C[c + wB * ty + tx] = Csub;
}

template<typename real_t>
__global__ void matrixMulCUDANonpersistent(real_t* c, real_t* a, real_t* b,
		int wA, int wB) {
	process_mxm_ii(c, a, b, wA, wB);
}

template<typename real_t>
__global__ void matrixMulCUDAPersistent(real_t* c, real_t* a, real_t* b, int wA,
		int wB, int nStreams) {
	rad::PersistentKernel pk;
	while (pk.keep_working()) {
		pk.wait_for_work();
		if (pk.is_able_to_process()) {
			for (int streamI = 0; streamI < nStreams; streamI++) {
				int ptr_index = streamI * wA * wB;
				real_t* c_i_ptr = c + ptr_index;
				real_t* a_i_ptr = a + ptr_index;
				real_t* b_i_ptr = b + ptr_index;
				process_mxm_ii(c_i_ptr, a_i_ptr, b_i_ptr, wA, wB);

			}
			pk.iteration_finished();
		}
	}
}

void matrixMulCUDA(float *C, float *A, float *B, int wA, int wB,
		const std::vector<CudaStream>& streams, KernelType t, dim3 gridDim,
		dim3 blockDim) {
	auto streamSize = streams.size();
	switch (t) {
	case PERSISTENT: {
		matrixMulCUDAPersistent<<<gridDim, blockDim, 0, streams[0].stream>>>(C,
				A, B, wA, wB, streamSize);
		rad::checkFrameworkErrors (hipPeekAtLastError());

break;	}
	case STATIC: {
		for (int streamI = 0; streamI < streamSize; streamI++) {
			int ptr_index = streamI * wA * wB;
			float* c_i_ptr = C + ptr_index;
			float* a_i_ptr = A + ptr_index;
			float* b_i_ptr = B + ptr_index;
			matrixMulCUDANonpersistent<<<gridDim, blockDim, 0,
			streams[streamI].stream>>>(c_i_ptr, a_i_ptr, b_i_ptr, wA,
					wB);
		}

		for (auto stream : streams) {
			stream.sync();
		}
		break;
	}

	case GEMM: {

		static hipblasHandle_t handle;
		float alpha = 1;
		float beta = 0;
		hipblasStatus_t status = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, wA, wB, wB, &alpha,
				A, wA, B, wB, &beta, C, wB, streamSize);
		break;
	}

}
}

